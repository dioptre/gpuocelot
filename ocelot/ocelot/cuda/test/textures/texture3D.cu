#include "hip/hip_runtime.h"
/*!
	\file texture3D.cu

	\author Andrew Kerr <arkerr@gatech.edu>

	\brief tests implementation of hipBindTextureToArray

	\date 27 Oct 2009
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

// declare texture reference for 2D float texture
texture<float, 3, hipReadModeElementType> Surface;

extern "C" __global__ void kernelMemset(float *out, int width, int height, float value) {
	unsigned int x = threadIdx.x;
	unsigned int y = threadIdx.y;
	unsigned int z = blockIdx.x;
	out[x + y * width + z * width * height] = value;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
	
*/
extern "C" __global__ void kernel(float *out, int width, int height) {

	unsigned int x = threadIdx.x;
	unsigned int y = threadIdx.y;
	unsigned int z = blockIdx.x;

	float sample = tex3D(Surface, x, y, z);

	out[x + y * width + z * width * height] = sample;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **arg) {
	int width = 8, height = 8, depth = 4;

	float *in_data_host, *out_data_host;
	float *out_data_gpu;

	hipError_t result;
	hipArray *arrayPointer;
	size_t bytes = width * height * depth * sizeof(float);
	in_data_host = (float *)malloc(bytes);
	out_data_host = (float *)malloc(bytes);

	// initialize data
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
	hipExtent extent;
	extent.width = width;
	extent.depth = depth;
	extent.height = height;

	result = hipMalloc3DArray(&arrayPointer, &channelDesc, extent, 0);
	if (result != hipSuccess) {
		fprintf(stderr, "Texture3D - failed to malloc 3D array - %s", hipGetErrorString(result));
		return 1;
	}
	
	// memcpy
	for (int k = 0; k <depth; k++) {
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				float c = 0;
				
				// impulse response
				if (i == 4 && j == 4 && k == 1) { c = 1.0f; }
				
				in_data_host[i + j * width + k * width * height] = c;
			}
		}
	}
	
	hipMemcpy3DParms params;

	memset(&params, 0, sizeof(params));
	params.srcPtr.pitch = sizeof(float) * width;
	params.srcPtr.ptr = in_data_host;
	params.srcPtr.xsize = width;
	params.srcPtr.ysize = height;

	params.srcPos.x = 0;
	params.srcPos.y = 0;
	params.srcPos.z = 0;

	params.dstArray = arrayPointer;

	params.dstPos.x = 0;
	params.dstPos.y = 0;
	params.dstPos.z = 0;

	params.extent.width = width;
	params.extent.depth = depth;
	params.extent.height = height;

	params.kind = hipMemcpyHostToDevice;

	result = hipMemcpy3D(&params);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy3D - failed to copy from host buffer to device array - %s\n", hipGetErrorString(result));
		return 1;
	}
	
	// clear it, copy back from array
	for (int k = 0; k <depth; k++) {
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				in_data_host[i + j * width + k * width * height] = -1;
			}
		}
	}
	
	memset(&params, 0, sizeof(params));
	params.dstPtr.pitch = sizeof(float) * width;
	params.dstPtr.ptr = in_data_host;
	params.dstPtr.xsize = width;
	params.dstPtr.ysize = height;

	params.srcPos.x = 0;
	params.srcPos.y = 0;
	params.srcPos.z = 0;

	params.srcArray = arrayPointer;

	params.dstPos.x = 0;
	params.dstPos.y = 0;
	params.dstPos.z = 0;

	params.extent.width = width;
	params.extent.depth = depth;
	params.extent.height = height;

	params.kind = hipMemcpyDeviceToHost;

	result = hipMemcpy3D(&params);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy3D - failed to copy from array to host buffer for verification - %s", hipGetErrorString(result));
		return 1;
	}
	for (int k = 0; k <depth; k++) {
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				float c = in_data_host[i + j * width + k * width * height];
				if (c < 0) {
					fprintf(stderr, "You can't copy to and from a 3D array. What makes you think you can sample from one? (%d, %d, %d)\n",
						i, j, k);
					fprintf(stderr, " = %f\n", c);
					return 1;
				}
			}
		}
	}

	Surface.addressMode[0] = hipAddressModeWrap;
	Surface.addressMode[1] = hipAddressModeWrap;
	Surface.addressMode[2] = hipAddressModeWrap;
	Surface.filterMode = hipFilterModeLinear;
	Surface.normalized = false;
	
	// bind to array
	result = hipBindTextureToArray(Surface, arrayPointer, channelDesc);
	if (result != hipSuccess) {
		fprintf(stderr, "hipBindTextureToArray() - failed to bind texture to array - %s", hipGetErrorString(result));
		return 2;
	}
	
	// allocate output
	if (hipMalloc((void **)&out_data_gpu, bytes) != hipSuccess) {
		printf("hipMalloc(out_data_gpu) - failed to allocate %d bytes: %s\n", (int)bytes,
			hipGetErrorString(hipGetLastError()));
		return -2;
	}
	
	dim3 grid(depth, 1);
	dim3 block(width, height, 1);
	kernelMemset<<< grid, block >>>(out_data_gpu, width, height, 0.0f);
	kernel<<< grid, block >>>(out_data_gpu, width, height);
	hipDeviceSynchronize();
	
	result = hipMemcpy(out_data_host, out_data_gpu, bytes, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy() - failed to copy from device output buffer to host buffer %s", hipGetErrorString(result));
		return 3;
	}
	
	// print result as a set of matrices
	for (int k = 0; k < depth; k++) {
		float *ptr = &out_data_host[k * width * height];
		
		printf("\nA_%d = [\n", k);
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				printf(" %f ", ptr[i + j * width]);
			}
			printf(";\n");
		}
		printf("];\n");
	}
	
	// clean up
	hipFree(out_data_gpu);
	free(out_data_host);
	free(in_data_host);
	hipFreeArray(arrayPointer);
	
	return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////


#include "hip/hip_runtime.h"
/*!
	\file textureArray.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief tests implementation of cube texture mapping
	\date February 10, 2012
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

// declare texture reference for 2D float texture
texture<float, hipTextureTypeCubemap> Surface;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
__global__ void kernel(float *out, int width) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	const float pi = 3.14159265358f;

	float theta = pi * (2.0f - (float)x / (float)width);
	float phi = pi * (2.0f - (float)y / (float)width);

	float cx = cos(theta)*cos(phi);
	float cy = sin(phi);
	float cz = sin(theta)*cos(phi);
	
	float sample = texCubemap( Surface, cx, cy, cz );

	out[x + y * width] = sample;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **arg) {
	int width = 16, layers = 6;

	float *in_data_host, *out_data_host;
	float *out_data_gpu;
	hipArray *in_data_gpu = 0;

	size_t bytes = width * width * sizeof(float);
	in_data_host = (float *)malloc(bytes);
	out_data_host = (float *)malloc(bytes);

	// procedural texture generation
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = (float)((122 + i*3 + j*2) % 128) / 128.0f;
			out_data_host[i*width+j] = 0;
		}
	}

	// construct array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  
  if (hipMalloc3DArray( &in_data_gpu, &channelDesc, 
  	make_hipExtent(width, width, layers), hipArrayCubemap ) != hipSuccess) {
  
		printf("hipMalloc3DArray() - failed: %s\n", 
			hipGetErrorString(hipGetLastError()));
		return -1;
  }
   
  hipMemcpy3DParms parameters = {0};
  parameters.srcPos = make_hipPos(0,0,0); 
  parameters.dstPos = make_hipPos(0,0,0); 
  parameters.srcPtr = make_hipPitchedPtr(in_data_host, width * sizeof(float), width, width); 
  parameters.dstArray = in_data_gpu;
  parameters.extent = make_hipExtent(width, width, layers);
  parameters.kind = hipMemcpyHostToDevice;
  
  if (hipMemcpy3D(&parameters) != hipSuccess) {
		printf("hipMemcpy3D() - failed: %s\n", 
			hipGetErrorString(hipGetLastError()));
		return -1;
  }
  
  // set texture parameters
	Surface.addressMode[0] = hipAddressModeWrap;
	Surface.addressMode[1] = hipAddressModeWrap;
	Surface.filterMode = hipFilterModePoint;
	Surface.normalized = true;  // access with normalized texture coordinates
	
	if (hipBindTextureToArray(Surface, in_data_gpu, channelDesc) != hipSuccess) {
		printf("hipBindTextureToArray() - failed to bind texture: %s\n", 
			hipGetErrorString(hipGetLastError()));
		
		free(in_data_host);
		free(out_data_host);
		hipFreeArray(in_data_gpu);
		return -2;
	}

	if (hipMalloc((void **)&out_data_gpu, bytes) != hipSuccess) {
		
		printf("hipMalloc(out_data_gpu) - failed to allocate %d bytes: %s\n", (int)bytes,
			hipGetErrorString(hipGetLastError()));
		
		free(in_data_host);
		free(out_data_host);
		hipFreeArray(in_data_gpu);
		return -2;
	}

	dim3 grid(width / 16, width / 16), block(16, 16);
	
	kernel<<< grid, block >>>(out_data_gpu, width);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytes, hipMemcpyDeviceToHost);
	hipFreeArray(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	
	// unit test not written
	// ...
	
	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));

	free(in_data_host);
	free(out_data_host);

	return (errors ? -1 : 0);
}

/////////////////////////////////////////////////////////////////////////////////////////////////


#include "hip/hip_runtime.h"
/*!
	\file textureArray.cu

	\author Andrew Kerr <arkerr@gatech.edu>

	\brief tests implementation of cudaBindTextureArray

	\date 27 Oct 2009
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> Surface;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
__global__ void kernel(float *out, int width, int height) {

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(Surface, x, y);

	out[x + y * width] = sample;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **arg) {
	int width = 64, height = 64;

	float *in_data_host, *out_data_host;
	float *out_data_gpu;
	hipArray *in_data_gpu = 0;

	size_t bytes = width * height * sizeof(float);
	in_data_host = (float *)malloc(bytes);
	out_data_host = (float *)malloc(bytes);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = (float)((122 + i*3 + j*2) % 128) / 128.0f;
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	if (hipMallocArray(&in_data_gpu, &channelDesc, width, height) != hipSuccess) {
		printf("failed to malloc array: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		return -2;
	}
	
	if (hipMemcpyToArray(in_data_gpu, 0, 0, in_data_host, bytes, hipMemcpyHostToDevice)
		!= hipSuccess) {
		
		printf("hipMemcpyToArray() - failed to bind texture: %s\n", 
			hipGetErrorString(hipGetLastError()));
		
		free(in_data_host);
		free(out_data_host);
		hipFreeArray(in_data_gpu);
		return -2;		
	}

	Surface.addressMode[0] = hipAddressModeWrap;
	Surface.addressMode[1] = hipAddressModeWrap;
	Surface.filterMode = hipFilterModePoint;
	Surface.normalized = false;
	
	if (hipBindTextureToArray(Surface, in_data_gpu, channelDesc) != hipSuccess) {
		
		printf("hipBindTextureToArray() - failed to bind texture: %s\n", 
			hipGetErrorString(hipGetLastError()));
		
		free(in_data_host);
		free(out_data_host);
		hipFreeArray(in_data_gpu);
		return -2;
	}

	if (hipMalloc((void **)&out_data_gpu, bytes) != hipSuccess) {
		
		printf("hipMalloc(out_data_gpu) - failed to allocate %d bytes: %s\n", (int)bytes,
			hipGetErrorString(hipGetLastError()));
		
		free(in_data_host);
		free(out_data_host);
		hipFreeArray(in_data_gpu);
		return -2;
	}

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernel<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytes, hipMemcpyDeviceToHost);
	hipFreeArray(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			float in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			if (fabs(in - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - in = %f, out = %f %s\n", i, j, in, out, (errors ? "***":""));
			}
		}
	}

	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));

	free(in_data_host);
	free(out_data_host);

	return (errors ? -1 : 0);
}

/////////////////////////////////////////////////////////////////////////////////////////////////


#include "hip/hip_runtime.h"
/*!
	\brief sequence.cu
	\author Andrew Kerr

	\brief simple test of a CUDA implementation's ability to allocate memory on the device, launch
		a kernel, and fetch its results. One kernel requires no syncthreads, another kernel requires
		one synchronization
*/

#include <stdio.h>
#include <dlfcn.h>

#if 1

extern "C" __global__ void simple(int *A) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	A[i] = i;
}

extern "C" __global__ void sequence(int *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N) {
		A[i] = 2*i;
	}
}

extern "C" __global__ void testShareSimple(int *A) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ int Share[32];
	
	int a = A[i];
	Share[threadIdx.x] = a;
	__syncthreads();
	A[i] = Share[threadIdx.x ^ 1];
	A[i] = Share[31 - threadIdx.x];
}

extern "C" __global__ void v4sequence(int4 *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int4 b = make_int4(i, 2*i, 3*i, 4*i);
	A[i-1] = b;
}

#endif

extern "C" __global__ void testShr(int *A) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int b;
	__shared__ int storage[256];
	
	storage[threadIdx.x] = 2*i;
	__syncthreads();
	if (i & 1) {
		b = storage[threadIdx.x ^ 1] * 19;
	}
	else {
		b = storage[threadIdx.x ^ 1] * 13;
	}
	A[i] = b;
}

int main(int argc, char *arg[]) {

	const int BlockSize = 4;
	const int N = 4;
	int *A_host, *A_gpu =0;
	int errors = 0;

	size_t bytes = sizeof(int)*N;
	
	hipError_t result = hipDeviceSynchronize();
	if (result != hipSuccess) {
		printf("hipDeviceSynchronize() = %s\n", hipGetErrorString(result));
		printf("Failed to load CUDA library:\n%s\n", dlerror());
		return 0;
	}

	if (hipMalloc((void **)&A_gpu, bytes) != hipSuccess) {
		printf("hipMalloc() - failed to allocate %d bytes on device\n", (int)bytes);
		return -1;
	}

	A_host = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_host[i] = -1;
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);
	
	dim3 grid((N+BlockSize-1)/BlockSize,1);
	dim3 block(BlockSize, 1);
	
	simple<<< grid, block >>>(A_gpu);
	
	sequence<<< grid, block >>>(A_gpu, N);
	
	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < N && errors < 5; i++) {
		if (A_host[i] != 2*i) {
			
			printf("ERROR 1 [%d] - expected: %d, got: %d\n", i, 2*i, A_host[i]);
			++errors;
		}
	}
#if 0
	grid.x /= 4;
	v4sequence<<< grid, block >>>((int4 *)A_gpu, N/4);
	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	grid.x *= 4;


	if (!errors) {
		sequence<<< grid, block >>>(A_gpu, N);
		
		printf("\n\n\n\n\n\n");
		
		testShareSimple<<< grid, block >>>(A_gpu);
		if (hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost) != hipSuccess) {
			printf("hipMemcpy(A, A) - failed to copy %d bytes from device to host\n", (int)bytes);
			hipFree(A_gpu);
			free(A_host);
		}
	
		for (int i = 0; i < N; i++) {
//			int p = i + 31 - 2 * (i % 32);
			int p = i;
			if (p & 0x01) {
				p --;
			}
			else {
				p ++;
			}
			int expected = p * 2;
			int got = A_host[i];
			if (expected != got) {
				printf("ERROR 2 [%d] - expected: %d, got: %d\n", i, expected, got);
				++errors;
			}
		}
	}
#endif
	
	if (!errors) {

		testShr<<< grid, block >>>(A_gpu);
	
		if (hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost) != hipSuccess) {
			printf("hipMemcpy(A, B) - failed to copy %d bytes from device to host\n", (int)bytes);
			hipFree(A_gpu);
			free(A_host);
		}
	
		for (int i = 0; (errors < 5) && i < N; ++i) {
			int b;
			if (i & 1) {
				b = (i ^ 1) * 2 * 19;
			}
			else {
				b = (i ^ 1) * 2 * 13;
			}
			int got = A_host[i];
			if (b != got) {
				printf("ERROR 3 [%d] - expected: %d, got: %d\n", i, b, got);
				++errors;
			}
		}
	}

	hipFree(A_gpu);
	free(A_host);


	if (errors) {
		printf("Pass/Fail : Fail\n");
	}
	else {
		printf("Pass/Fail : Pass\n");
	}

	return 0;
}


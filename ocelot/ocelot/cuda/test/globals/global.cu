/*!
	\file global.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief verifies a CUDA application's ability to use global symbols
	\date Feburary 12, 2010
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float Pi;

extern "C" __global__ void copyFromGlobal(float *result) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	result[i] = Pi * (float)(i % 128);
}

int main(int argc, char *arg[]) {
	int N = 64;
	bool verbose = false;
	size_t bytes = sizeof(float) * N;
	float *results_gpu = 0;
	float *results_cpu = (float *)malloc(bytes);
	int devices = 0;
	hipGetDeviceCount(&devices);
	
	int errors = 0;
	for (int device = 0; device != devices; ++device) {
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		printf("cudaSetDevice() - %d - %s \n", device, properties.name);
		hipSetDevice(device);
		errors = 0;
		
		if (hipMalloc((void **)&results_gpu, bytes) != hipSuccess) {
			printf("cudaMalloc() failed to allocate %d bytes on device\n", (int)bytes);
			return -1;
		}

		for (int i = 0; i < N; i++) {
			results_cpu[i] = -1;
		}
		if (verbose) {
			printf(" [1]\n");
		}
		hipMemcpy(results_gpu, results_cpu, bytes, hipMemcpyHostToDevice);
	
		if (verbose) {
			printf(" [2]\n");
		}
	
		float pi = 3.14159f;
		if (hipMemcpyToSymbol(HIP_SYMBOL("Pi"), &pi, sizeof(float), 0, 
			hipMemcpyHostToDevice) != hipSuccess) {
		
			printf("cudaMemcpyToSymbol() failed to copy 4 bytes to symbol 'Pi'\n");
		
			hipFree(results_gpu);
			free(results_cpu);
			return -1;
		}
	
	
		if (verbose) {
			printf(" [3]\n");
		}
		float copy_pi = 0;
		if (hipMemcpyFromSymbol(&copy_pi, HIP_SYMBOL("Pi"), sizeof(float), 0,
			hipMemcpyDeviceToHost) != hipSuccess) {
		
			printf("cudaMemcpyFromSymbol() failed to copy 4 bytes from symbol 'Pi'\n");
		
			hipFree(results_gpu);
			free(results_cpu);
			return -1;
		}
	
		if (fabs(copy_pi - 3.14159f) > 0.001f) {
			printf("value copied from symbol (%f) did not match expected 3.14159\n",
				copy_pi);
		
			hipFree(results_gpu);
			free(results_cpu);
			return -1;
		}
	
		dim3 block(64, 1);
		dim3 grid((63 + block.x) / 64, 1);
	
		copyFromGlobal<<< grid, block >>>(results_gpu);
	
		if (verbose) {
			printf(" [4]\n");
		}
		hipMemcpy(results_cpu, results_gpu, bytes, hipMemcpyDeviceToHost);
	
		for (int i = 0; i < N; i++) {
			float expected = 3.14159f * (float)(i % 128);
			float got = results_cpu[i];
			if (fabs(expected - got) > 0.001f) {
				printf("ERROR 0 - [%d] - got: %f, expected: %f\n", i, got, expected);
				if (++errors > 5) { break; }
			}
		}
	
		if (verbose) {
			printf("[5]\n");
		}
		float *pi_gpu = 0;
		if (hipGetSymbolAddress((void **)&pi_gpu, HIP_SYMBOL("Pi")) != hipSuccess) {
			printf("failed to get address of global variable 'Pi'\n");
			hipFree(results_gpu);
			free(results_cpu);
			return -1;
		}
	
		if (verbose) {
			printf(" [6]\n");
		}
		copy_pi = 2.0f * 3.14159f;
		if (hipMemcpy(pi_gpu, &copy_pi, sizeof(float), hipMemcpyHostToDevice) !=
			hipSuccess) {
		
			printf("failed to copy value to symbol 'Pi'\n");
			hipFree(results_gpu);
			free(results_cpu);
			return -1;		
		}

		copyFromGlobal<<< grid, block >>>(results_gpu);
	
		if (verbose) {
			printf(" [7]\n");
		}
		hipMemcpy(results_cpu, results_gpu, bytes, hipMemcpyDeviceToHost);
	
		for (int i = 0; i < N; i++) {
			float expected = 2.0f * 3.14159f * (float)(i % 128);
			float got = results_cpu[i];
			if (fabs(expected - got) > 0.001f) {
				printf("ERROR 1 - [%d] - got: %f, expected: %f\n",
					i, got, expected);
				if (++errors > 5) { break; }
			}
		}
	
		hipFree(results_gpu);

	}
	
	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));

	free(results_cpu);

	return 0;
}


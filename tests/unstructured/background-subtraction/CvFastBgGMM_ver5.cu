
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define CUDAGMM_VERSION 5
 
#if(CUDAGMM_VERSION == 5)

#define SWAP(a, b, t)	t = (a); a = (b); b = (t)


typedef struct CvFastBgGMMData
{
	float4* ucGaussian;
	float* rWeight;
	int* rnUsedModes;

} CvFastBgGMMData;


enum ImageInfo
{
#if(CUDAGMM_VERSION >= 2)
	ImageInfoPixelCount = 0,		//
	ImageInfoPixelsPerThread = 1,	//
	ImageInfoCount = 2
#else
	ImageInfoInpWidth = 0,
	ImageInfoInpHeight = 1,
	ImageInfoInpWidthStep = 2,
	ImageInfoOutWidth = 3,
	ImageInfoOutHeight = 4,
	ImageInfoOutWidthStep = 5,
	ImageInfoPixelCount = 6,		//
	ImageInfoPixelsPerThread = 7,	//
	ImageInfoCount = 8
#endif
};

// algorithm parameters
typedef struct CvFastBgGMMParams
{
	/////////////////////////
	//very important parameters - things you will change
	////////////////////////
	float fAlphaT;
	//alpha - speed of update - if the time interval you want to average over is T
	//set alpha=1/T. It is also useful at start to make T slowly increase
	//from 1 until the desired T
	float fTb;
	//Tb - threshold on the squared Mahalan. dist. to decide if it is well described
	//by the background model or not. Related to Cthr from the paper.
	//This does not influence the update of the background. A typical value could be 4 sigma
	//and that is Tb=4*4=16;

	/////////////////////////
	//less important parameters - things you might change but be carefull
	////////////////////////
	float fTg;
	//Tg - threshold on the squared Mahalan. dist. to decide 
	//when a sample is close to the existing components. If it is not close
	//to any a new component will be generated. I use 3 sigma => Tg=3*3=9.
	//Smaller Tg leads to more generated components and higher Tg might make
	//lead to small number of components but they can grow too large
	float fTB;//1-cf from the paper
	//TB - threshold when the component becomes significant enough to be included into
	//the background model. It is the TB=1-cf from the paper. So I use cf=0.1 => TB=0.
	//For alpha=0.001 it means that the mode should exist for approximately 105 frames before
	//it is considered foreground
	float fSigma;
	//initial standard deviation  for the newly generated components. 
	//It will will influence the speed of adaptation. A good guess should be made. 
	//A simple way is to estimate the typical standard deviation from the images.
	//I used here 10 as a reasonable value
	float fCT;//CT - complexity reduction prior
	//this is related to the number of samples needed to accept that a component
	//actually exists. We use CT=0.05 of all the samples. By setting CT=0 you get
	//the standard Stauffer&Grimson algorithm (maybe not exact but very similar)

	//even less important parameters
	int nM;//max number of modes - const - 4 is usually enough

	//shadow detection parameters
	int bShadowDetection;//do shadow detection
	float fTau;
	// Tau - shadow threshold. The shadow is detected if the pixel is darker
	//version of the background. Tau is a threshold on how much darker the shadow can be.
	//Tau= 0.5 means that if pixel is more than 2 times darker then it is not shadow
	//See: Prati,Mikic,Trivedi,Cucchiarra,"Detecting Moving Shadows...",IEEE PAMI,2003.

	float fPrune;	//=-m_fAlphaT*m_fCT;

	//data
	int nNBands;//only RGB now ==3
	int nWidth;//image size
	int nHeight;
	int nSize;
	int bRemoveForeground;
} CvFastBgGMMParams;


__constant__ CvFastBgGMMParams d_GMMParams;
__constant__ CvFastBgGMMData d_GMMData;
__constant__ int d_arrImageInfo[ImageInfoCount];

/*=======================================================================================*/

__device__ int _cudaUpdateFastBgGMM(int pixel, 
									float red, float green, float blue,
									int* pModesUsed
									)
{
	//calculate distances to the modes (+ sort)
	//here we need to go in descending order!!!

	int pos;
	bool bFitsPDF = 0;
	int bBackground = 0;
	float m_fOneMinAlpha = 1 - d_GMMParams.fAlphaT;
	int nModes = (*pModesUsed);
	float weight, totalWeight = 0.0f;
	float dR, dG, dB;
	float dist, k, sigmanew;

	//go through all modes
	for (int iModes = 0; iModes < nModes; iModes++)
	{
		pos = pixel + iModes*d_arrImageInfo[ImageInfoPixelCount];
		weight = d_GMMData.rWeight[pos];

		//fit not found yet
		if (!bFitsPDF)
		{
			//check if it belongs to some of the modes
			//calculate distance
			float4 cGauss = d_GMMData.ucGaussian[pos];

			dR = cGauss.x - red;
			dG = cGauss.y - green;
			dB = cGauss.z - blue;

			//check if it fits the current mode (Factor * sigma)

			//square distance -slower and less accurate
			//float maxDistance = cvSqrt(m_fTg*var);
			//if ((fabs(dR) <= maxDistance) && (fabs(dG) <= maxDistance) && (fabs(dB) <= maxDistance))
			//circle
			dist = dR*dR + dG*dG + dB*dB;

			//background? - m_fTb
			if ((totalWeight < d_GMMParams.fTB) && (dist < d_GMMParams.fTb * cGauss.w))
				bBackground = 1;

			//check fit
			if (dist < d_GMMParams.fTg * cGauss.w)
			{
				//belongs to the mode
				bFitsPDF = 1;

				//update distribution
				k = d_GMMParams.fAlphaT/weight;
				weight = m_fOneMinAlpha * weight + d_GMMParams.fPrune;
				weight += d_GMMParams.fAlphaT;
				cGauss.x -= k*(dR);
				cGauss.y -= k*(dG);
				cGauss.z -= k*(dB);

				//limit update speed for cov matrice
				//not needed
				sigmanew = cGauss.w + k*(dist - cGauss.w);

				//limit the variance
				cGauss.w = sigmanew < 4 ? 4 : 
					sigmanew > 5 * d_GMMParams.fSigma ? 5 * d_GMMParams.fSigma : sigmanew;

				d_GMMData.ucGaussian[pos] = cGauss;

				//sort
				//all other weights are at the same place and 
				//only the matched (iModes) is higher -> just find the new place for it

				for (int iLocal = iModes; iLocal > 0; iLocal--)
				{
					int posLocal = pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount];
					if (weight < (d_GMMData.rWeight[posLocal-d_arrImageInfo[ImageInfoPixelCount]]))
					{
						break;
					}
					else
					{
						//swap
						float tmpVal;
						float4 tmpuChar;
						SWAP(d_GMMData.ucGaussian[posLocal],
							d_GMMData.ucGaussian[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
							tmpuChar);
						SWAP(d_GMMData.rWeight[posLocal],
							d_GMMData.rWeight[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
							tmpVal);
					}
				}

				//belongs to the mode
			}
			else
			{
				weight = m_fOneMinAlpha * weight + d_GMMParams.fPrune;

				//check prune
				if (weight < -(d_GMMParams.fPrune))
				{
					weight = 0.0f;
					nModes--;
					//	bPrune=1;
					//break;//the components are sorted so we can skip the rest
				}
			}
			//check if it fits the current mode (2.5 sigma)
			///////
		}	//fit not found yet
		else
		{
			weight = m_fOneMinAlpha * weight + d_GMMParams.fPrune;

			if (weight < -(d_GMMParams.fPrune))
			{
				weight=0.0;
				nModes--;
				//bPrune=1;
				//break;//the components are sorted so we can skip the rest
			}
		}
		totalWeight += weight;
		d_GMMData.rWeight[pos] = weight;
	}
	//go through all modes
	//////

	//renormalize weights
	for (int iLocal = 0; iLocal < nModes; iLocal++)
	{
		d_GMMData.rWeight[pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount]] /= totalWeight;
	}

	//make new mode if needed and exit
	if (!bFitsPDF)
	{
		if (nModes == d_GMMParams.nM)
		{
			//replace the weakest
		}
		else
		{
			//add a new one
			//totalWeight+=m_fAlphaT;
			//pos++;
			nModes++;
		}
		pos = pixel + (nModes-1)*d_arrImageInfo[ImageInfoPixelCount];

		if (nModes == 1)
			d_GMMData.rWeight[pos] = 1;
		else
			d_GMMData.rWeight[pos] = d_GMMParams.fAlphaT;

		//renormalize weights
		for (int iLocal = 0; iLocal < nModes-1; iLocal++)
		{
			d_GMMData.rWeight[pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount]] *= m_fOneMinAlpha;
		}

		float4 cGauss;
		cGauss.x = red;
		cGauss.y = green;
		cGauss.z = blue;
		cGauss.w = d_GMMParams.fSigma;
		d_GMMData.ucGaussian[pos] = cGauss;

		//sort
		//find the new place for it
		for (int iLocal = nModes - 1; iLocal>0; iLocal--)
		{
			int posLocal = pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount];
			if (d_GMMParams.fAlphaT < (d_GMMData.rWeight[posLocal - d_arrImageInfo[ImageInfoPixelCount]]))
			{
				break;
			}
			else
			{
				//swap
				float4 tmpuChar;
				float tmpVal;
				SWAP(d_GMMData.ucGaussian[posLocal],
					d_GMMData.ucGaussian[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
					tmpuChar);
				SWAP(d_GMMData.rWeight[posLocal],
					d_GMMData.rWeight[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
					tmpVal);
			}
		}
	}

	//set the number of modes
	*pModesUsed=nModes;

	return bBackground;
}

/*=======================================================================================*/

/*=======================================================================================*/
__device__ int _cudaRemoveShadowGMM(int pixel, 
									float red, float green, float blue, 
									int nModes)
{
	//calculate distances to the modes (+ sort)
	//here we need to go in descending order!!!
	//	long posPixel = pixel * m_nM;
	int pos;
	float tWeight = 0;
	float numerator, denominator;

	// check all the distributions, marked as background:
	for (int iModes=0;iModes<nModes;iModes++)
	{
		pos=pixel+iModes*d_arrImageInfo[ImageInfoPixelCount];
		float4 cGauss = d_GMMData.ucGaussian[pos];
		float weight = d_GMMData.rWeight[pos];
		tWeight += weight;

		numerator = red * cGauss.x + green * cGauss.y + blue * cGauss.z;
		denominator = cGauss.x * cGauss.x + cGauss.y * cGauss.y + cGauss.z * cGauss.z;
		// no division by zero allowed
		if (denominator == 0)
		{
			break;
		}
		float a = numerator / denominator;

		// if tau < a < 1 then also check the color distortion
		if ((a <= 1) && (a >= d_GMMParams.fTau))//m_nBeta=1
		{
			float dR=a * cGauss.x - red;
			float dG=a * cGauss.y - green;
			float dB=a * cGauss.z - blue;

			//square distance -slower and less accurate
			//float maxDistance = cvSqrt(m_fTb*var);
			//if ((fabs(dR) <= maxDistance) && (fabs(dG) <= maxDistance) && (fabs(dB) <= maxDistance))
			//circle
			float dist=(dR*dR+dG*dG+dB*dB);
			if (dist<d_GMMParams.fTb*cGauss.w*a*a)
			{
				return 2;
			}
		}
		if (tWeight > d_GMMParams.fTB)
		{
			break;
		}
	}
	return 0;
}

/*=======================================================================================*/

/*=======================================================================================*/

__device__ void _cudaReplacePixelBackgroundGMM(int pixel, uchar4* pData)
{
	uchar4 tmp;
	float4 cGauss = d_GMMData.ucGaussian[pixel];
	tmp.z = (unsigned char) cGauss.x;
	tmp.y = (unsigned char) cGauss.y;
	tmp.x = (unsigned char) cGauss.z;
	(*pData) = tmp;
}
/*=======================================================================================*/

/*=======================================================================================*/
extern __shared__ int sharedInfo[];

template <int BLOCK_SIZE>
__global__ void cudaUpdateFastBgGMM(unsigned char* data, unsigned char* output)
{
	if(threadIdx.x == 0)
	{
		// the start pixel for current block
		sharedInfo[0] = (blockIdx.x * BLOCK_SIZE)*d_arrImageInfo[ImageInfoPixelsPerThread];
	}
	__syncthreads();

	int iPxStart = sharedInfo[0] + threadIdx.x;
	int iPxEnd = min( d_arrImageInfo[ImageInfoPixelCount], 
		sharedInfo[0] + (BLOCK_SIZE * d_arrImageInfo[ImageInfoPixelsPerThread]));

	uchar4* pGlobalInput = ((uchar4*)data) + iPxStart;
	unsigned char* pGlobalOutput = output + iPxStart;

	int* pUsedModes = d_GMMData.rnUsedModes + iPxStart;
	unsigned char fRed, fGreen, fBlue;
	uchar4 currentInputPx;

	for(int i = iPxStart; i < iPxEnd; i += BLOCK_SIZE)
	{
		// retrieves the color
		currentInputPx = *pGlobalInput;
		fBlue = currentInputPx.x;
		fGreen = currentInputPx.y;
		fRed = currentInputPx.z;
		pGlobalInput += BLOCK_SIZE;

		// update model + background subtract
		int result = _cudaUpdateFastBgGMM(i, fRed, fGreen, fBlue, pUsedModes);
		int nMLocal = *pUsedModes;
		pUsedModes += BLOCK_SIZE;

		if (d_GMMParams.bShadowDetection)
		{
			if (!result)
			{
				result= _cudaRemoveShadowGMM(i, fRed, fGreen, fBlue, nMLocal);
			}
		}

		switch (result)
		{
		case 0:

			//foreground
			(*pGlobalOutput) = 255;
			if (d_GMMParams.bRemoveForeground) 
			{
				_cudaReplacePixelBackgroundGMM(i, pGlobalInput-BLOCK_SIZE);
			}
			break;

		case 1:

			//background
			(*pGlobalOutput) = 0;
			break;

		case 2:

			//shadow
			(*pGlobalOutput) = 128;
			if (d_GMMParams.bRemoveForeground) 
			{
				_cudaReplacePixelBackgroundGMM(i, pGlobalInput-BLOCK_SIZE);
			}

			break;
		}
		pGlobalOutput += BLOCK_SIZE;
	}
}
#endif

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "CvFastBgGMM.h"

#if(CUDAGMM_VERSION == 4)

#define SWAP(a, b, t)	t = (a); a = (b); b = (t)

__constant__ CvFastBgGMMParams d_GMMParams;
__constant__ CvFastBgGMMData d_GMMData;
__constant__ int d_arrImageInfo[ImageInfoCount];

/*====================================================================================*/
// forward declarations
/*====================================================================================*/

int InitCUDA(CvFastBgGMM* pGMM);
__global__ void cudaUpdateFastBgGMM(unsigned char* data, unsigned char* output);

/*====================================================================================*/

/*====================================================================================*/

CvFastBgGMMParams* cvCreateFastBgGMMParams(int width, int height)
{
	CvFastBgGMMParams* pGMMParams = new CvFastBgGMMParams();

	int size = width*height;
	pGMMParams->nWidth = width;
	pGMMParams->nHeight = height;
	pGMMParams->nSize = size;

	pGMMParams->nNBands=3;	//always 3 - not implemented for other values!

	//set parameters
	// K - max number of Gaussians per pixel
	pGMMParams->nM = 4;			
	// Tb - the threshold - n var
	pGMMParams->fTb = 4*4;
	// Tbf - the threshold
	pGMMParams->fTB = 0.9f;//1-cf from the paper 
	// Tgenerate - the threshold
	pGMMParams->fTg = 3.0f*3.0f;//update the mode or generate new
	pGMMParams->fSigma= 11.0f;//sigma for the new mode
	// alpha - the learning factor
	pGMMParams->fAlphaT=0.001f;
	// complexity reduction prior constant
	pGMMParams->fCT=0.05f;

	//shadow
	// Shadow detection
	pGMMParams->bShadowDetection = 0;//turn on
	pGMMParams->fTau = 0.5f;// Tau - shadow threshold

	pGMMParams->bRemoveForeground = 0;
	return pGMMParams;
}
/*====================================================================================*/

/*====================================================================================*/

template <bool toPinned>
void copyImageData(IplImage* h_img, unsigned char* d_pinnedMem, int channels)
{
	if(h_img->widthStep == channels*h_img->width)
	{
		memcpy(
			toPinned ? (d_pinnedMem) : (unsigned char*)(h_img->imageData), 
			toPinned ? (unsigned char*)(h_img->imageData) : (d_pinnedMem),
			h_img->widthStep*h_img->height);
	}
	else
	{
		unsigned char* d_curData = d_pinnedMem;
		if(toPinned)
		{
			for(int i = 0; i < h_img->height; ++i)
			{
				memcpy(
					d_curData,
					&CV_IMAGE_ELEM(h_img, unsigned char, i, 0),
					channels*h_img->width);
				d_curData += (channels*h_img->width);
			}
		}
		else
		{
			for(int i = 0; i < h_img->height; ++i)
			{
				memcpy(
					&CV_IMAGE_ELEM(h_img, unsigned char, i, 0),
					d_curData,
					channels*h_img->width);
				d_curData += (channels*h_img->width);
			}
		}
	}
}

/*====================================================================================*/

/*====================================================================================*/

CvFastBgGMM* cvCreateFastBgGMM(CvFastBgGMMParams* pGMMParams, IplImage* frame0)
{
	CvFastBgGMM* h_pGMMRet = new CvFastBgGMM();

	if(InitCUDA(h_pGMMRet) < 0)
	{
		delete h_pGMMRet;
		return NULL;
	}

	CvFastBgGMMData* h_pGMMData = new CvFastBgGMMData();

	// allocate device global memory
	int iElemCount = pGMMParams->nSize * pGMMParams->nM * sizeof(float);
	int iSizeCount = pGMMParams->nSize * sizeof(int);

	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMData->ucGaussian), 4*iElemCount));
	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMData->rWeight), iElemCount));
	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMData->rnUsedModes), iSizeCount));
	CUDAGMM_SAFE_CALL(hipMemset(h_pGMMData->rnUsedModes, 0, iSizeCount));

	CUDAGMM_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_GMMData), h_pGMMData, sizeof(CvFastBgGMMData), 0, hipMemcpyHostToDevice));

	h_pGMMRet->internal_data = h_pGMMData;

	// we will use 4-channels image as input data!
	h_pGMMRet->inputFrame = cvCreateImage(cvSize(pGMMParams->nWidth, pGMMParams->nHeight), IPL_DEPTH_8U, 4);
	h_pGMMRet->nInputImgSize = 4 * frame0->width *  pGMMParams->nHeight;
	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMRet->d_inputImg), h_pGMMRet->nInputImgSize));
	h_pGMMRet->h_outputImg = cvCreateImage(cvSize(pGMMParams->nWidth, pGMMParams->nHeight), IPL_DEPTH_8U, 1);
	h_pGMMRet->nOutputImgSize = pGMMParams->nWidth *  pGMMParams->nHeight;
	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMRet->d_outputImg), h_pGMMRet->nOutputImgSize));
	
	// d_arrImageInfo constant (device mem.)
	int inpPixelCnt = pGMMParams->nWidth * pGMMParams->nHeight;

	// number of pixels per thread must be 4k, i.e. 4, 8, 12, 16, 20...
	int iPixelsPerThread = min(32, (int)ceil(inpPixelCnt *1.0 / (h_pGMMRet->nBlocksPerGrid * h_pGMMRet->nThreadsPerBlock)));
	iPixelsPerThread = 4*(int)ceil(iPixelsPerThread/4.0f);
	h_pGMMRet->nBlocksPerGrid = (int)ceil(inpPixelCnt*1.0 / ((h_pGMMRet->nThreadsPerBlock) * iPixelsPerThread));

	printf("%d pixels/thread, %d threads/block, %d blocks\r\n", 
		iPixelsPerThread, h_pGMMRet->nThreadsPerBlock, h_pGMMRet->nBlocksPerGrid);

	int arrImgInfo[ImageInfoCount] = {	inpPixelCnt, iPixelsPerThread };
	CUDAGMM_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_arrImageInfo), arrImgInfo, ImageInfoCount*sizeof(int),
		0, hipMemcpyHostToDevice));

	CUDAGMM_SAFE_CALL(hipStreamCreate(&(h_pGMMRet->copyStream)));
	CUDAGMM_SAFE_CALL(hipStreamCreate(&(h_pGMMRet->execStream)));

	// hipHostMallocMapped might fail
	CUDAGMM_SAFE_CALL(hipHostAlloc((void**)&(h_pGMMRet->h_pinnedIn), 
		h_pGMMRet->nInputImgSize, hipHostMallocWriteCombined));
	CUDAGMM_SAFE_CALL(hipHostAlloc((void**)&(h_pGMMRet->h_pinnedOut), 
		h_pGMMRet->nOutputImgSize, hipHostMallocDefault));
	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMRet->d_inputImg2), h_pGMMRet->nInputImgSize));
	CUDAGMM_SAFE_CALL(hipMalloc((void**)&(h_pGMMRet->d_outputImg2), h_pGMMRet->nOutputImgSize));

	// copy the algorithm parameters to Constant memory
	pGMMParams->fPrune = -(pGMMParams->fAlphaT) * (pGMMParams->fCT);
	CUDAGMM_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_GMMParams), pGMMParams, sizeof(CvFastBgGMMParams),
		0, hipMemcpyHostToDevice));

	// setup the initial state for asynchronous execution
	cvCvtColor(frame0, h_pGMMRet->inputFrame, CV_BGR2BGRA);
	copyImageData<true>(h_pGMMRet->inputFrame, h_pGMMRet->h_pinnedIn, 4);
	CUDAGMM_SAFE_CALL(hipMemcpy(h_pGMMRet->d_inputImg2, h_pGMMRet->h_pinnedIn, h_pGMMRet->nInputImgSize, hipMemcpyHostToDevice));
	cudaUpdateFastBgGMM<<< (h_pGMMRet->nBlocksPerGrid), (h_pGMMRet->nThreadsPerBlock), 4, h_pGMMRet->execStream >>>
		( h_pGMMRet->d_inputImg2, h_pGMMRet->d_outputImg2 );

	CUDAGMM_SAFE_CALL(hipMemcpyAsync(h_pGMMRet->d_inputImg, h_pGMMRet->h_pinnedIn, h_pGMMRet->nInputImgSize, 
		hipMemcpyHostToDevice, h_pGMMRet->copyStream));
	CUDAGMM_SAFE_CALL(hipStreamSynchronize(h_pGMMRet->execStream));
	CUDAGMM_SAFE_CALL(hipMemcpy(h_pGMMRet->h_pinnedOut, h_pGMMRet->d_outputImg2, h_pGMMRet->nOutputImgSize, hipMemcpyDeviceToHost));

	CUDAGMM_SAFE_CALL(hipStreamSynchronize(h_pGMMRet->copyStream));
	cudaUpdateFastBgGMM<<< (h_pGMMRet->nBlocksPerGrid), (h_pGMMRet->nThreadsPerBlock), 4, h_pGMMRet->execStream >>>
		( h_pGMMRet->d_inputImg, h_pGMMRet->d_outputImg );

	CUDAGMM_SAFE_CALL(hipMemcpyAsync(h_pGMMRet->d_inputImg2, h_pGMMRet->h_pinnedIn, h_pGMMRet->nInputImgSize, 
		hipMemcpyHostToDevice, h_pGMMRet->copyStream));

	return h_pGMMRet;
}

/*====================================================================================*/

/*====================================================================================*/

void cvReleaseFastBgGMM(CvFastBgGMM** h_ppGMM)
{
	CvFastBgGMM* h_pGMM = *h_ppGMM;

	cvReleaseImage(&(h_pGMM->h_outputImg));
	cvReleaseImage(&(h_pGMM->inputFrame));
	CUDAGMM_SAFE_CALL( hipStreamSynchronize(h_pGMM->copyStream));
	CUDAGMM_SAFE_CALL( hipStreamSynchronize(h_pGMM->execStream));
	CUDAGMM_SAFE_CALL( hipFree(h_pGMM->d_inputImg));
	CUDAGMM_SAFE_CALL( hipFree(h_pGMM->d_outputImg));
	CUDAGMM_SAFE_CALL( hipFree(h_pGMM->d_inputImg2));
	CUDAGMM_SAFE_CALL( hipFree(h_pGMM->d_outputImg2));
	CUDAGMM_SAFE_CALL( hipHostFree(h_pGMM->h_pinnedIn));
	CUDAGMM_SAFE_CALL( hipHostFree(h_pGMM->h_pinnedOut));
	CUDAGMM_SAFE_CALL( hipStreamDestroy(h_pGMM->copyStream));
	CUDAGMM_SAFE_CALL( hipStreamDestroy(h_pGMM->execStream));

	CvFastBgGMMData* h_pGMMData = h_pGMM->internal_data;
	CUDAGMM_SAFE_CALL( hipFree(h_pGMMData->ucGaussian));
	CUDAGMM_SAFE_CALL( hipFree(h_pGMMData->rWeight));
	CUDAGMM_SAFE_CALL( hipFree(h_pGMMData->rnUsedModes));
	
	delete h_pGMM->internal_data;
	delete h_pGMM;
	(*h_ppGMM) = 0;
}

/*====================================================================================*/

/*====================================================================================*/

void cvUpdateFastBgGMM(CvFastBgGMM* pGMM, IplImage* inputImg)
{
	cvCvtColor(inputImg, pGMM->inputFrame, CV_BGR2BGRA);
	CUDAGMM_SAFE_CALL(hipStreamSynchronize(pGMM->copyStream));
	copyImageData<true>(pGMM->inputFrame, pGMM->h_pinnedIn, 4);
	copyImageData<false>(pGMM->h_outputImg, pGMM->h_pinnedOut, 1);
	
	CUDAGMM_SAFE_CALL(hipStreamSynchronize(pGMM->execStream));
	unsigned char* pTmp;
	SWAP(pGMM->d_inputImg, pGMM->d_inputImg2, pTmp);
	SWAP(pGMM->d_outputImg, pGMM->d_outputImg2, pTmp);

	CUDAGMM_SAFE_CALL(hipMemcpyAsync(pGMM->d_inputImg, pGMM->h_pinnedIn, pGMM->nInputImgSize, hipMemcpyHostToDevice, pGMM->copyStream));
	CUDAGMM_SAFE_CALL(hipMemcpyAsync(pGMM->h_pinnedOut, pGMM->d_outputImg, pGMM->nOutputImgSize, hipMemcpyDeviceToHost, pGMM->copyStream));

	cudaUpdateFastBgGMM<<< (pGMM->nBlocksPerGrid), (pGMM->nThreadsPerBlock), 4, pGMM->execStream >>>
		( pGMM->d_inputImg2, pGMM->d_outputImg2 );

#ifdef _DEBUG
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("CUDA error: %d: %s\r\n", error, hipGetErrorString(error));
	}
#endif
}

float cvUpdateFastBgGMMTimer(CvFastBgGMM* pGMM, IplImage* inputImg)
{
	hipEvent_t start, stop;
	float time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );
	
	cvUpdateFastBgGMM(pGMM, inputImg);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	return time;
}

/*============================================================================*/
// CUDA-related functions
/*============================================================================*/

int InitCUDA(CvFastBgGMM* pGMM)
{
#if __DEVICE_EMULATION__

	pGMM->nThreadsPerBlock = pGMM->nBlocksPerGrid = 256;
	return 0;

#else

	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0)
	{
		fprintf(stderr, "There is no device.\n");
		return -1;
	}

	for(i = 0; i < count; i++)
	{
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if(prop.major >= 1)
			{
				pGMM->nThreadsPerBlock = prop.maxThreadsPerBlock / 4;

				// temporarily hard-code a little here...
				pGMM->nBlocksPerGrid = 256;
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return -1;
	}
	hipSetDevice(i);
	return i;

#endif
}

/*=======================================================================================*/

/*=======================================================================================*/

__device__ int _cudaUpdateFastBgGMM(int pixel, 
									float red, float green, float blue,
									int* pModesUsed
									)
{
	//calculate distances to the modes (+ sort???)
	//here we need to go in descending order!!!

	int pos;
	bool bFitsPDF = 0;
	int bBackground = 0;
	float m_fOneMinAlpha = 1 - d_GMMParams.fAlphaT;
	int nModes = (*pModesUsed);
	float weight, totalWeight = 0.0f;
	float dR, dG, dB;
	float dist, k, sigmanew;

	//go through all modes
	for (int iModes = 0; iModes < nModes; iModes++)
	{
		pos = pixel + iModes*d_arrImageInfo[ImageInfoPixelCount];
		weight = d_GMMData.rWeight[pos];

		//fit not found yet
		if (!bFitsPDF)
		{
			//check if it belongs to some of the modes
			//calculate distance
			float4 cGauss = d_GMMData.ucGaussian[pos];

			dR = cGauss.x - red;
			dG = cGauss.y - green;
			dB = cGauss.z - blue;

			//check if it fits the current mode (Factor * sigma)

			//square distance -slower and less accurate
			//float maxDistance = cvSqrt(m_fTg*var);
			//if ((fabs(dR) <= maxDistance) && (fabs(dG) <= maxDistance) && (fabs(dB) <= maxDistance))
			//circle
			dist = dR*dR + dG*dG + dB*dB;

			//background? - m_fTb
			if ((totalWeight < d_GMMParams.fTB) && (dist < d_GMMParams.fTb * cGauss.w))
				bBackground = 1;

			//check fit
			if (dist < d_GMMParams.fTg * cGauss.w)
			{
				//belongs to the mode
				bFitsPDF = 1;

				//update distribution
				k = d_GMMParams.fAlphaT/weight;
				weight = m_fOneMinAlpha * weight + d_GMMParams.fPrune;
				weight += d_GMMParams.fAlphaT;
				cGauss.x -= k*(dR);
				cGauss.y -= k*(dG);
				cGauss.z -= k*(dB);

				//limit update speed for cov matrice
				//not needed
				sigmanew = cGauss.w + k*(dist - cGauss.w);

				//limit the variance
				cGauss.w = sigmanew < 4 ? 4 : 
					sigmanew > 5 * d_GMMParams.fSigma ? 5 * d_GMMParams.fSigma : sigmanew;

				d_GMMData.ucGaussian[pos] = cGauss;

				//sort
				//all other weights are at the same place and 
				//only the matched (iModes) is higher -> just find the new place for it

				for (int iLocal = iModes; iLocal > 0; iLocal--)
				{
					int posLocal = pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount];
					if (weight < (d_GMMData.rWeight[posLocal-d_arrImageInfo[ImageInfoPixelCount]]))
					{
						break;
					}
					else
					{
						//swap
						float tmpVal;
						float4 tmpuChar;
						SWAP(d_GMMData.ucGaussian[posLocal],
							d_GMMData.ucGaussian[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
							tmpuChar);
						SWAP(d_GMMData.rWeight[posLocal],
							d_GMMData.rWeight[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
							tmpVal);
					}
				}

				//belongs to the mode
			}
			else
			{
				weight = m_fOneMinAlpha * weight + d_GMMParams.fPrune;

				//check prune
				if (weight < -(d_GMMParams.fPrune))
				{
					weight = 0.0f;
					nModes--;
					//	bPrune=1;
					//break;//the components are sorted so we can skip the rest
				}
			}
			//check if it fits the current mode (2.5 sigma)
			///////
		}	//fit not found yet
		else
		{
			weight = m_fOneMinAlpha * weight + d_GMMParams.fPrune;

			if (weight < -(d_GMMParams.fPrune))
			{
				weight=0.0;
				nModes--;
				//bPrune=1;
				//break;//the components are sorted so we can skip the rest
			}
		}
		totalWeight += weight;
		d_GMMData.rWeight[pos] = weight;
	}
	//go through all modes
	//////

	//renormalize weights
	for (int iLocal = 0; iLocal < nModes; iLocal++)
	{
		d_GMMData.rWeight[pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount]] /= totalWeight;
	}

	//make new mode if needed and exit
	if (!bFitsPDF)
	{
		if (nModes == d_GMMParams.nM)
		{
			//replace the weakest
		}
		else
		{
			//add a new one
			//totalWeight+=m_fAlphaT;
			//pos++;
			nModes++;
		}
		pos = pixel + (nModes-1)*d_arrImageInfo[ImageInfoPixelCount];

		if (nModes == 1)
			d_GMMData.rWeight[pos] = 1;
		else
			d_GMMData.rWeight[pos] = d_GMMParams.fAlphaT;

		//renormalize weights
		for (int iLocal = 0; iLocal < nModes-1; iLocal++)
		{
			d_GMMData.rWeight[pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount]] *= m_fOneMinAlpha;
		}

		float4 cGauss;
		cGauss.x = red;
		cGauss.y = green;
		cGauss.z = blue;
		cGauss.w = d_GMMParams.fSigma;
		d_GMMData.ucGaussian[pos] = cGauss;

		//sort
		//find the new place for it
		for (int iLocal = nModes - 1; iLocal>0; iLocal--)
		{
			int posLocal = pixel + iLocal*d_arrImageInfo[ImageInfoPixelCount];
			if (d_GMMParams.fAlphaT < (d_GMMData.rWeight[posLocal - d_arrImageInfo[ImageInfoPixelCount]]))
			{
				break;
			}
			else
			{
				//swap
				float4 tmpuChar;
				float tmpVal;
				SWAP(d_GMMData.ucGaussian[posLocal],
					d_GMMData.ucGaussian[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
					tmpuChar);
				SWAP(d_GMMData.rWeight[posLocal],
					d_GMMData.rWeight[posLocal - d_arrImageInfo[ImageInfoPixelCount]],
					tmpVal);
			}
		}
	}

	//set the number of modes
	*pModesUsed=nModes;

	return bBackground;
}

/*=======================================================================================*/

/*=======================================================================================*/
__device__ int _cudaRemoveShadowGMM(int pixel, 
									float red, float green, float blue, 
									int nModes)
{
	//calculate distances to the modes (+ sort???)
	//here we need to go in descending order!!!
	//	long posPixel = pixel * m_nM;
	int pos;
	float tWeight = 0;
	float numerator, denominator;

	// check all the distributions, marked as background:
	for (int iModes=0;iModes<nModes;iModes++)
	{
		pos=pixel+iModes*d_arrImageInfo[ImageInfoPixelCount];
		float4 cGauss = d_GMMData.ucGaussian[pos];
		float weight = d_GMMData.rWeight[pos];
		tWeight += weight;

		numerator = red * cGauss.x + green * cGauss.y + blue * cGauss.z;
		denominator = cGauss.x * cGauss.x + cGauss.y * cGauss.y + cGauss.z * cGauss.z;
		// no division by zero allowed
		if (denominator == 0)
		{
			break;
		}
		float a = numerator / denominator;

		// if tau < a < 1 then also check the color distortion
		if ((a <= 1) && (a >= d_GMMParams.fTau))//m_nBeta=1
		{
			float dR=a * cGauss.x - red;
			float dG=a * cGauss.y - green;
			float dB=a * cGauss.z - blue;

			//square distance -slower and less accurate
			//float maxDistance = cvSqrt(m_fTb*var);
			//if ((fabs(dR) <= maxDistance) && (fabs(dG) <= maxDistance) && (fabs(dB) <= maxDistance))
			//circle
			float dist=(dR*dR+dG*dG+dB*dB);
			if (dist<d_GMMParams.fTb*cGauss.w*a*a)
			{
				return 2;
			}
		}
		if (tWeight > d_GMMParams.fTB)
		{
			break;
		}
	}
	return 0;
}

/*=======================================================================================*/

/*=======================================================================================*/

__device__ void _cudaReplacePixelBackgroundGMM(int pixel, uchar4* pData)
{
	uchar4 tmp;
	float4 cGauss = d_GMMData.ucGaussian[pixel];
	tmp.z = (unsigned char) cGauss.x;
	tmp.y = (unsigned char) cGauss.y;
	tmp.x = (unsigned char) cGauss.z;
	(*pData) = tmp;
}
/*=======================================================================================*/

/*=======================================================================================*/
extern __shared__ int sharedInfo[];

__global__ void cudaUpdateFastBgGMM(unsigned char* data, unsigned char* output)
{
	if(threadIdx.x == 0)
	{
		// the start pixel for current block
		sharedInfo[0] = (blockIdx.x * blockDim.x)*d_arrImageInfo[ImageInfoPixelsPerThread];
	}
	__syncthreads();

	int iPxStart = sharedInfo[0] + threadIdx.x;
	int iPxEnd = min( d_arrImageInfo[ImageInfoPixelCount], 
		sharedInfo[0] + (blockDim.x * d_arrImageInfo[ImageInfoPixelsPerThread]));

	uchar4* pGlobalInput = ((uchar4*)data) + iPxStart;
	unsigned char* pGlobalOutput = output + iPxStart;

	int* pUsedModes = d_GMMData.rnUsedModes + iPxStart;
	uchar fRed, fGreen, fBlue;
	uchar4 currentInputPx;

	for(int i = iPxStart; i < iPxEnd; i += blockDim.x)
	{
		// retrieves the color
		currentInputPx = *pGlobalInput;
		fBlue = currentInputPx.x;
		fGreen = currentInputPx.y;
		fRed = currentInputPx.z;
		pGlobalInput += blockDim.x;

		// update model + background subtract
		int result = _cudaUpdateFastBgGMM(i, fRed, fGreen, fBlue, pUsedModes);
		int nMLocal = *pUsedModes;
		pUsedModes += blockDim.x;

		if (d_GMMParams.bShadowDetection)
		{
			if (!result)
			{
				result= _cudaRemoveShadowGMM(i, fRed, fGreen, fBlue, nMLocal);
			}
		}

		switch (result)
		{
		case 0:

			//foreground
			(*pGlobalOutput) = 255;
			if (d_GMMParams.bRemoveForeground) 
			{
				_cudaReplacePixelBackgroundGMM(i, pGlobalInput-blockDim.x);
			}
			break;

		case 1:

			//background
			(*pGlobalOutput) = 0;
			break;

		case 2:

			//shadow
			(*pGlobalOutput) = 128;
			if (d_GMMParams.bRemoveForeground) 
			{
				_cudaReplacePixelBackgroundGMM(i, pGlobalInput-blockDim.x);
			}

			break;
		}
		pGlobalOutput += blockDim.x;
	}
}

#endif
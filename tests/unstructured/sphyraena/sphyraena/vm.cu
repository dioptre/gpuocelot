#include "hip/hip_runtime.h"
/* vim: set filetype=c : */
/* vim: set ts=6 : */
/* ===========================================================================
 * Copyright 2010 Peter Brownlee Bakkum
 * Permission is provided to use this for research only.
 * ------------------------------------------------------------------------ */



#include <hip/hip_runtime.h>

extern "C" {
#include "sphyraena.h"
}

// these are operator templates. I did this this way so that I could use the same code but
// switch the operator
#define SPHYRAENA_VM_COMPARE(x)										\
	type = max(reg[op->p3].type, reg[op->p1].type);							\
	switch(type) {												\
		case SPHYRAENA_INT :										\
			jump = (castToInt(&reg[op->p3]) x castToInt(&reg[op->p1]));			\
			break;											\
		case SPHYRAENA_FLOAT :										\
			jump = (castToFloat(&reg[op->p3]) x castToFloat(&reg[op->p1]));		\
			break;											\
		case SPHYRAENA_INT64 :										\
			jump = (castToInt64(&reg[op->p3]) x castToInt64(&reg[op->p1]));		\
			break;											\
		case SPHYRAENA_DOUBLE :										\
			jump = (castToDouble(&reg[op->p3]) x castToDouble(&reg[op->p1]));		\
			break;											\
	}														\
	if(jump)													\
		pc = op->p2;											\
	else														\
		pc++;

#define SPHYRAENA_VM_MATH(x)												\
	type = max(reg[op->p2].type, reg[op->p1].type);								\
	reg[op->p3].type = type;											\
	switch(type) {													\
		case SPHYRAENA_INT :											\
			reg[op->p3].mem.i = (castToInt(&reg[op->p2]) x castToInt(&reg[op->p1]));	\
			break;												\
		case SPHYRAENA_FLOAT :											\
			reg[op->p3].mem.f = (castToFloat(&reg[op->p2]) x castToFloat(&reg[op->p1]));	\
			break;												\
		case SPHYRAENA_INT64 :											\
			reg[op->p3].mem.li = (castToInt64(&reg[op->p2]) x castToInt64(&reg[op->p1]));	\
			break;												\
		case SPHYRAENA_DOUBLE :											\
			reg[op->p3].mem.d = (castToDouble(&reg[op->p2]) x castToDouble(&reg[op->p1]));\
			break;												\
	}															\

// these could probably be done in functions but you would have to pass a lot of arguments
// this makes it a bit simpler
#define SPHYRAENA_VM_LOGICAL_LOAD(x)					\
	switch(reg[op->x].type) {						\
		case SPHYRAENA_INT :						\
			val_##x = reg[op->x].mem.i;				\
			break;							\
		case SPHYRAENA_FLOAT :						\
			val_##x = (reg[op->x].mem.f == 0) ? 0 : 1;	\
			break;							\
		case SPHYRAENA_INT64 :						\
			val_##x = (reg[op->x].mem.li == 0) ? 0 : 1;	\
			break;							\
		case SPHYRAENA_DOUBLE :						\
			val_##x = (reg[op->x].mem.d == 0) ? 0 : 1;	\
			break;							\
	}

#define SPHYRAENA_VM_LOGICAL(x)			\
	SPHYRAENA_VM_LOGICAL_LOAD(p1)			\
	SPHYRAENA_VM_LOGICAL_LOAD(p2)			\
	reg[op->p3].type = SPHYRAENA_INT;		\
	reg[op->p3].mem.i = (val_p1 x val_p2);


__device__ void copyDataToReg(sphyraena_mem *reg, char *data, char* d, int column, int row);
__device__ int castToInt(sphyraena_mem *m);
__device__ float castToFloat(sphyraena_mem *m);
__device__ i64 castToInt64(sphyraena_mem *m);
__device__ double castToDouble(sphyraena_mem *m);

// the sql statement
__constant__ sphyraena_stmt cstmt;
// the data block meta information
__constant__ sphyraena_data_gpu cdata;

__shared__ int result_start;
__shared__ int block_start;
__shared__ int reductionblock[SPHYRAENA_THREADSPERBLOCK];
//__shared__ sphyraena_mem registerspace[SPHYRAENA_THREADSPERBLOCK * SPHYRAENA_REGISTERS];
//__device__ sphyraena_mem gregisterspace[SPHYRAENA_THREADSPERBLOCK * SPHYRAENA_REGISTERS];
//__shared__ sphyraena_mem reg_shared[SPHYRAENA_GLOBAL_REGISTERS];

__device__ sphyraena_mem reg_global[SPHYRAENA_GLOBAL_REGISTERS];
__device__ unsigned long long int count_global[SPHYRAENA_GLOBAL_REGISTERS];

__device__ unsigned int block_order;

texture<float, 1, hipReadModeElementType> texRef;

__global__ void VmKernel(char *data, sphyraena_results *results, int start_row, int num_blocks, int rows)
{
	// used for instances where a single thread handles multiple rows, for large data sets
	int curr_row = 0;
	block_start = 0;

	// sets the global registers used for aggregates
	if(threadIdx.x < SPHYRAENA_GLOBAL_REGISTERS) {
		//reg_shared[threadIdx.x].mem.li = 0;
		if(blockIdx.x == 0) {
			reg_global[threadIdx.x].mem.li = 0;
			reg_global[threadIdx.x].type = SPHYRAENA_NULL;
		}
	}

new_row:

	int row = (blockIdx.x + num_blocks * curr_row) * blockDim.x + threadIdx.x + start_row;

	sphyraena_mem reg[SPHYRAENA_REGISTERS];
	//sphyraena_mem *reg = (sphyraena_mem*) &registerspace[threadIdx.x * SPHYRAENA_REGISTERS];
	//sphyraena_mem *reg = (sphyraena_mem*) &gregisterspace[threadIdx.x * SPHYRAENA_REGISTERS];

	// program counter for opcode program
	int pc = 0;
	// stride of results block
	int stride = 0;
	// hardcoded registers for logical operations
	int val_p1, val_p2;
	// pointer to current operation
	sphyraena_op *op;

#ifdef COLUMNROW
	char *d = data;
#else
	char *d = data + row * cdata.stride;
#endif

	// pointer to results block
	char *r = (char*) &results->r;
	// used to store the current variable type
	int type;
	int jump = 0;
	int aggfinal_num = 0;
	int i, j;

	if(row >= cdata.rows)
		return;

// there are two modes of operation: divergent and synchronized
// the divergent block allows threads to diverge based on opcodes
// divergent is normally used, then we jump to synchronized on OP_Next
divergent:
	while(pc < SPHYRAENA_MAX_OPS) {
		op = &cstmt.op[pc];

		// this is a massive switch statement of opcodes. It is in alphabetical order other than
		// OP_Column, which is called frequently enough to make see a performance distance in moving
		// it to the front. Depending on the opcodes and size of the opcode programs the total switching
		// overhead for a query could be as high as 30%. On a CPU you could avoid this overhead with a 
		// jump table accessed with the opcode, but this is not supported on current (Tesla C1060)
		// hardware, so we are stuck with the overhead, but this is a major optimization for future
		// hardware.
		// The opcodes are documented on the SQLite website
		switch(op->opcode)
		{
			case OP_Column :
				copyDataToReg(&reg[op->p3], data, d, op->p2, row);
				//fprintf(stderr, "Column reg[%i] = %i  %i\n", op->p3, ((int*)(d + data->offsets[op->p2]))[0]);
				pc++;
				break;

			case OP_Add :
				SPHYRAENA_VM_MATH(+)
				pc++;
				break;

			case OP_AddImm :
				reg[op->p1].mem.i = castToInt(&reg[op->p1]) + op->p2;
				pc++;
				break;

			case OP_AggStep :
				switch(op->p4.i) {
					case SPHYRAENA_AGG_COUNT :
						reg[op->p3].mem.i = 1;
						break;

					case SPHYRAENA_AGG_SUM :
					case SPHYRAENA_AGG_MIN :
					case SPHYRAENA_AGG_MAX :
					case SPHYRAENA_AGG_AVG :
						reg[op->p3].mem.li = reg[op->p2].mem.li;
						reg[op->p3].type = reg[op->p2].type;
						break;
				}
				pc++;
				break;

			case OP_And :
				SPHYRAENA_VM_LOGICAL(&&)
				pc++;
				break;

			case OP_BitAnd :
				reg[op->p3].mem.li = reg[op->p1].mem.li & reg[op->p2].mem.li;
				pc++;
				break;

			case OP_BitNot :
				reg[op->p2].mem.li = ~reg[op->p1].mem.li;
				pc++;
				break;

			case OP_BitOr :
				reg[op->p3].mem.li = reg[op->p1].mem.li | reg[op->p2].mem.li;
				pc++;
				break;

			case OP_Copy :
			case OP_SCopy :
				reg[op->p2].type = reg[op->p1].type;
				reg[op->p2].mem.li = reg[op->p1].mem.li;
				pc++;
				break;

			case OP_Divide :
				SPHYRAENA_VM_MATH(/)
				pc++;
				break;

			case OP_Eq :
				SPHYRAENA_VM_COMPARE(==)
				break;

			case OP_Ge :
				SPHYRAENA_VM_COMPARE(>=)
				break;

			case OP_Gt :
				SPHYRAENA_VM_COMPARE(>)
				break;

			case OP_Halt :
				goto finish;

			case OP_If :
				SPHYRAENA_VM_LOGICAL_LOAD(p1)
				if(val_p1)
					pc = op->p2;
				else
					pc++;
				break;

			case OP_IfNeg :
				if(reg[op->p1].mem.i < 0)
					pc = op->p2;
				else
					pc++;
				break;

			case OP_IfNot :
				SPHYRAENA_VM_LOGICAL_LOAD(p1)
				if(!val_p1)
					pc = op->p2;
				else
					pc++;
				break;

			case OP_IfPos :
				if(reg[op->p1].mem.i < 0)
					pc = op->p2;
				else
					pc++;
				break;
			
			case OP_IfZero :
				if(reg[op->p1].mem.i == 0)
					pc = op->p2;
				else
					pc++;
				break;

			case OP_Int64 :
				reg[op->p2].type = SPHYRAENA_INT64;
				reg[op->p2].mem.li = op->p4.li;
				pc++;
				break;

			case OP_Integer :
				reg[op->p2].type = SPHYRAENA_INT;
				reg[op->p2].mem.i = op->p1;
				pc++;
				break;

			case OP_Le :
				SPHYRAENA_VM_COMPARE(<=)
				break;

			case OP_Lt :
				SPHYRAENA_VM_COMPARE(<)
				break;

			case OP_Multiply :
				SPHYRAENA_VM_MATH(*)
				pc++;
				break;

			case OP_Ne :
				SPHYRAENA_VM_COMPARE(!=)
				break;

			case OP_Next :
				goto coordinated;

			case OP_Not :
				SPHYRAENA_VM_LOGICAL_LOAD(p1)
				reg[op->p2].type = SPHYRAENA_INT;
				reg[op->p2].mem.i = !val_p1;
				pc++;
				break;

			case OP_Null :
				reg[op->p2].mem.li = 0;
				reg[op->p2].type = SPHYRAENA_NULL;
				pc++;
				break;

			case OP_Or :
				SPHYRAENA_VM_LOGICAL(||)
				pc++;
				break;

			case OP_Real :
				reg[op->p2].type = SPHYRAENA_DOUBLE;
				reg[op->p2].mem.d = op->p4.d;
				pc++;
				break;

			case OP_Remainder :
				break;

			case OP_ResultRow :
				//fprintf(stderr, "resultrow    %i   %i\n", reg[7].mem.i, reg[8].mem.i);
				
				if(block_start != -1)
					j = atomicAdd(&block_start, 1);

				for(i = op->p1; i < op->p1 + op->p2; i++) {
					results->types[i - op->p1] = reg[i].type;
					results->offsets[i - op->p1] = stride;

					switch(reg[i].type) {
						case SPHYRAENA_INT :
							stride += sizeof(int);
							break;
						case SPHYRAENA_FLOAT :
							stride += sizeof(float);
							break;
						case SPHYRAENA_INT64 :
							stride += sizeof(i64);
							break;
						case SPHYRAENA_DOUBLE :
							stride += sizeof(double);
							break;
					}
				}

				// this is a slight abuse of syncthreads. According to the documentation
				// syncthreads should never be called in a conditional not executed uniformly
				// across a threadblock. It turns out that synchthreads still works.
				// I implemented it this way because I had to use atomic functions for the reduction,
				// since even though every thread in the threadblock executes OP_Next, syncthreads
				// does not wait for divergent threads to catch up, so a normal reduction cannot be
				// performed. In the event that the implementation changes, this code can be moved
				// to OP_Next, and a proper coordinated reduction can be performed.
				__syncthreads();

				if(j == 0 && block_start != 0) {
					result_start = atomicAdd(&results->rows, block_start);
					block_start = 0;
				}

				__syncthreads();

				// round stride up to a power of 2
				stride--;
				stride |= stride >> 1;
				stride |= stride >> 2;
				stride |= stride >> 4;
				stride |= stride >> 8;
				stride |= stride >> 16;
				stride++;
				stride *= 4;

				results->stride = stride;
				results->columns = op->p2;

				r += (result_start + j) * stride;

				for(i = op->p1; i < op->p1 + op->p2; i++) {
					switch(reg[i].type) {
						case SPHYRAENA_INT :
							((int*)r)[0] = reg[i].mem.i;
							r += sizeof(int);
							break;
						case SPHYRAENA_FLOAT :
							((float*)r)[0] = reg[i].mem.f;
							r += sizeof(float);
							break;
						case SPHYRAENA_INT64 :
							//((i64*)r)[0] = reg[i].mem.li;
							((i64*)r)[0] = reg[i].mem.li;
							r += sizeof(i64);
							break;
						case SPHYRAENA_DOUBLE :
						((double*)r)[0] = reg[i].mem.d;
							r += sizeof(double);
							break;
					}
				}

				pc++;
				break;

			case OP_Rowid :
				// TODO change to actual pkey
				copyDataToReg(&reg[op->p2], data, d, 0, row);
				pc++;
				break;

			case OP_ShiftLeft :
				reg[op->p3].type = reg[op->p2].type;
				reg[op->p3].mem.li = reg[op->p2].mem.li << op->p1;
				pc++;
				break;

			case OP_ShiftRight :
				reg[op->p3].type = reg[op->p2].type;
				reg[op->p3].mem.li = reg[op->p2].mem.li >> op->p1;
				pc++;
				break;

			case OP_Subtract :
				SPHYRAENA_VM_MATH(-);
				pc++;
				break;
			
			default :
				pc++;
				break;
		}


	}

// this is the coordinated opcode block, for when it is essential that every thread acts in
// concert, as in the global aggregate reductions, note that several other opcodes are implemented
// in this block. These are used for post-aggregate operations, such as AVG(col1) + AVG(col2).
// Since all operation is done within a single kernel launch it is necessary to use atomics for
// the aggregate reductions, and establish threadblock order for ex post operations. Because
// of the threadblock ordering, performing multiple aggregates in the same query probably wont work,
// since there is no way to synchronize threadblocks. This should be avoided. All these are reasons
// that a multiple kernel launch model would be better, but that would erase SQLite registers,
// so a good deal more work is needed to accomplish that.
coordinated:

	while(pc < SPHYRAENA_MAX_OPS) {
		op = &cstmt.op[pc];

		switch(op->opcode)
		{
			case OP_AggFinal :
				aggfinal_num++;

				switch(op->p4.i) {
					case SPHYRAENA_AGG_COUNT :
						// find the the next lowest power of 2 from the 
						// thread block size, including the current size
						i = SPHYRAENA_THREADSPERBLOCK;
						SPHYRAENA_ROUNDTOPWR2(i)
						j = i;
						i = i >> 1;
						i = 128;

						reductionblock[threadIdx.x] = reg[op->p1].mem.i;

						for( ; i > 0; i >>= 1, j >>= 1) {
							if(i >= 32)
								__syncthreads();
							if(threadIdx.x >= i && threadIdx.x < j)
								reductionblock[i - (threadIdx.x - i) - 1] += reductionblock[threadIdx.x];
						}

						//int tid = threadIdx.x;

						/*if(tid >= i)
							reductionblock[i - (tid - i) - 1] += reductionblock[tid];
						__syncthreads();

						if(i >= 512) {
							if(tid < 256)
								reductionblock[tid] += reductionblock[tid + 256];
							__syncthreads();
						}
						if(i >= 256) {
							if(tid < 128)
								reductionblock[tid] += reductionblock[tid + 128];
							__syncthreads();
						}
						if(i >= 128) {
							if(tid < 64)
								reductionblock[tid] += reductionblock[tid + 64];
							__syncthreads();
						}

						if(tid < 32) {
							if(i >= 64)
								reductionblock[tid] += reductionblock[tid + 32];
							if(i >= 32)
								reductionblock[tid] += reductionblock[tid + 16];
							if(i >= 16)
								reductionblock[tid] += reductionblock[tid +  8];
							if(i >=  8)
								reductionblock[tid] += reductionblock[tid +  4];
							if(i >=  4)
								reductionblock[tid] += reductionblock[tid +  2];
							if(i >=  2)
								reductionblock[tid] += reductionblock[tid +  1];
						}*/

						if(threadIdx.x == 0) {
							unsigned long long int ulli = atomicAdd(&reg_global[op->p1].mem.ulli, reductionblock[0]);
							reg[op->p1].mem.li = ulli + reductionblock[0];
							reg[op->p1].type = SPHYRAENA_INT;
						}
						break;

					case SPHYRAENA_AGG_SUM :
                                    i = SPHYRAENA_THREADSPERBLOCK;
                                    SPHYRAENA_ROUNDTOPWR2(i)
                                    j = i;
                                    i = i >> 1;

                                    reductionblock[threadIdx.x] = reg[op->p1].mem.i;

                                    for( ; i > 0; i >>= 1, j >>= 1) {
							if(i >= 32)
                                          	__syncthreads();
                                          if(threadIdx.x >= i && threadIdx.x < j)
                                                reductionblock[i - (threadIdx.x - i) - 1] += reductionblock[threadIdx.x];
                                    }

						if(threadIdx.x == 0) {
							unsigned long long int ulli = atomicAdd(&reg_global[op->p1].mem.ulli, reductionblock[0]);
							reg[op->p1].mem.li = ulli + reductionblock[0];
							reg[op->p1].type = SPHYRAENA_INT;
                                    }
						break;

					case SPHYRAENA_AGG_MIN :
                                    i = SPHYRAENA_THREADSPERBLOCK;
                                    SPHYRAENA_ROUNDTOPWR2(i)
                                    j = i;
                                    i = i >> 1;

                                    reductionblock[threadIdx.x] = reg[op->p1].mem.i;

                                    for( ; i > 0; i >>= 1, j >>= 1) {
							if(i >= 32)
                                          	__syncthreads();
                                          if(threadIdx.x >= i && threadIdx.x < j)
                                                SPHYRAENA_MIN(reductionblock[i - (threadIdx.x - i) - 1], reductionblock[threadIdx.x])
                                    }

                                    if(threadIdx.x == 0) {
                                          i = atomicMin(&reg_global[op->p1].mem.i, reductionblock[0]);
                                          SPHYRAENA_MIN(i, reductionblock[0]);
                                          reg[op->p1].mem.li = i;
                                          reg[op->p1].type = SPHYRAENA_INT;
                                    }
                                    break;


                              case SPHYRAENA_AGG_MAX :
                                    i = SPHYRAENA_THREADSPERBLOCK;
                                    SPHYRAENA_ROUNDTOPWR2(i)
                                    j = i;
                                    i = i >> 1;

                                    reductionblock[threadIdx.x] = reg[op->p1].mem.i;

                                    for( ; i > 0; i >>= 1, j >>= 1) {
							if(i >= 32)
                                          	__syncthreads();
                                          if(threadIdx.x >= i && threadIdx.x < j)
                                                SPHYRAENA_MAX(reductionblock[i - (threadIdx.x - i) - 1], reductionblock[threadIdx.x])
                                    }

                                    if(threadIdx.x == 0) {
                                          i = atomicMax(&reg_global[op->p1].mem.i, reductionblock[0]);
                                          SPHYRAENA_MAX(i, reductionblock[0]);
                                          reg[op->p1].mem.li = i;
                                          reg[op->p1].type = SPHYRAENA_INT;
                                    }
                                    break;

					case SPHYRAENA_AGG_AVG :
                                    i = SPHYRAENA_THREADSPERBLOCK;
                                    SPHYRAENA_ROUNDTOPWR2(i)
                                    j = i;
                                    i = i >> 1;

                                    reductionblock[threadIdx.x] = (reg[op->p1].type != SPHYRAENA_NULL) ? 1 : 0;

                                    for( ; i > 0; i >>= 1, j >>= 1) {
							if(i >= 32)
                                          	__syncthreads();
                                          if(threadIdx.x >= i && threadIdx.x < j)
								reductionblock[i - (threadIdx.x - i) - 1] += reductionblock[threadIdx.x];
                                    }

						int sum;

                                    if(threadIdx.x == 0) {
                                          sum = atomicAdd(&reg_global[0].mem.i, reductionblock[0]);
							sum += reductionblock[0];
                                    }


                                    i = SPHYRAENA_THREADSPERBLOCK;
                                    SPHYRAENA_ROUNDTOPWR2(i)
                                    j = i;
                                    i = i >> 1;

						// int overflow?
                                    reductionblock[threadIdx.x] = reg[op->p1].mem.i;

                                    for( ; i > 0; i >>= 1, j >>= 1) {
							if(i >= 32)
                                          	__syncthreads();
                                          if(threadIdx.x >= i && threadIdx.x < j)
								reductionblock[i - (threadIdx.x - i) - 1] += reductionblock[threadIdx.x];
                                    }

						double avg;

                                    if(threadIdx.x == 0) {
                                          avg = (double)atomicAdd(&reg_global[op->p1].mem.ulli, reductionblock[0]);
							avg += reductionblock[0];
							avg /= sum;
							reg[op->p1].mem.f = (float)avg;
							reg[op->p1].type = SPHYRAENA_FLOAT;
                                    }
                                    break;
				}

				pc++;
				break;
					
			case OP_Column :
				goto divergent;

			case OP_Copy :
			case OP_SCopy :
				reg[op->p2].type = reg[op->p1].type;
				reg[op->p2].mem.li = reg[op->p1].mem.li;
				pc++;
				break;

			case OP_Halt :
				goto finish;

			case OP_ResultRow :
				if(threadIdx.x == 0)
					//i = atomicInc(&block_order, gridDim.x);
					i = atomicAdd(&block_order, 1);
				else
					i = 0;

				if(i == gridDim.x - 1) {
					block_order = 0;
				
					for(i = op->p1; i < op->p1 + op->p2; i++) {
						results->types[i - op->p1] = reg[i].type;
						results->offsets[i - op->p1] = stride;

						switch(reg[i].type) {
							case SPHYRAENA_INT :
								stride += sizeof(int);
								break;
							case SPHYRAENA_FLOAT :
								stride += sizeof(float);
								break;
							case SPHYRAENA_INT64 :
								stride += sizeof(i64);
								break;
							case SPHYRAENA_DOUBLE :
								stride += sizeof(double);
								break;
						}
					}

					stride--;
					stride |= stride >> 1;
					stride |= stride >> 2;
					stride |= stride >> 4;
					stride |= stride >> 8;
					stride |= stride >> 16;
					stride++;

					results->stride = stride;
					results->columns = op->p2;
					results->rows = 1;

					for(i = op->p1; i < op->p1 + op->p2; i++) {
						switch(reg[i].type) {
							case SPHYRAENA_INT :
								((int*)r)[0] = reg[i].mem.i;
								r += sizeof(int);
								break;
							case SPHYRAENA_FLOAT :
								((float*)r)[0] = reg[i].mem.f;
								r += sizeof(float);
								break;
							case SPHYRAENA_INT64 :
								((i64*)r)[0] = reg[i].mem.li;
								r += sizeof(i64);
								break;
							case SPHYRAENA_DOUBLE :
							((double*)r)[0] = reg[i].mem.d;
								r += sizeof(double);
								break;
						}
					}

				}
				pc++;
				break;

			default :
				pc++;
				break;
		}
	}

finish:

	curr_row++;
	if(curr_row < rows)
		goto new_row;

}

__device__ void copyDataToReg(sphyraena_mem *reg, char *data, char* d, int column, int row)
{
	reg->type = cdata.types[column];

#ifdef COLUMNROW
	/*if(reg->type <= SPHYRAENA_FLOAT) {
		reg->mem.f = tex1Dfetch(texRef, (cdata.offsets[column] + row * sizeof(int)) / sizeof(int));
	}
	else {
		reg->mem.segment.hi = tex1Dfetch(texRef,
			(cdata.offsets[column] + row * sizeof(i64)) / sizeof(int));
		reg->mem.segment.lo = tex1Dfetch(texRef,
			(cdata.offsets[column] + row * sizeof(i64) + sizeof(int)) / sizeof(int));
	}*/

	char* p = d + cdata.offsets[column] + row * sizeof(int);

	switch(cdata.types[column]) {
		case SPHYRAENA_INT :
			reg->mem.i = ((int*)p)[0];
			break;
		case SPHYRAENA_FLOAT :
			reg->mem.f = ((float*)p)[0];
			break;
		case SPHYRAENA_INT64 :
			reg->mem.li = ((i64*)p)[0];
			break;
		case SPHYRAENA_DOUBLE :
			reg->mem.d = ((double*)p)[0];
			break;
	}
#else
	/*reg->mem.f = tex1Dfetch(texRef, (row * cdata.stride + cdata.offsets[column]) / sizeof(int));

	if(reg->type > SPHYRAENA_FLOAT)
		reg->mem.segment.lo = tex1Dfetch(texRef,
			(row * cdata.stride + cdata.offsets[column]) / sizeof(int) + sizeof(int));*/


	//printf("%i\n", reg->mem.i);
	char* p = d + cdata.offsets[column];

	switch(cdata.types[column]) {
		case SPHYRAENA_INT :
			reg->mem.i = ((int*)p)[0];
			break;
		case SPHYRAENA_FLOAT :
			reg->mem.f = ((float*)p)[0];
			break;
		case SPHYRAENA_INT64 :
			reg->mem.li = ((i64*)p)[0];
			//reg->mem.li = 0;
			//reg->mem.segment.lo = ((int*)p)[0];
			//reg->mem.segment.hi = ((int*)p)[1];
			break;
		case SPHYRAENA_DOUBLE :
			reg->mem.d = ((double*)p)[0];
			break;
	}
#endif

}

__device__ int castToInt(sphyraena_mem *m)
{
	switch(m->type) {
		case SPHYRAENA_INT :
			return m->mem.i;
		case SPHYRAENA_FLOAT :
			return (int)m->mem.f;
		case SPHYRAENA_INT64 :
			return (int)m->mem.li;
		case SPHYRAENA_DOUBLE :
			return (int)m->mem.d;
	}
	return 0;
}

__device__ float castToFloat(sphyraena_mem *m)
{
	switch(m->type) {
		case SPHYRAENA_FLOAT :
			return m->mem.f;
		case SPHYRAENA_INT :
			return (float)m->mem.i;
		case SPHYRAENA_INT64 :
			return (float)m->mem.li;
		case SPHYRAENA_DOUBLE :
			return (float)m->mem.d;
	}
	return 0;
}

__device__ i64 castToInt64(sphyraena_mem *m)
{
	switch(m->type) {
		case SPHYRAENA_INT64 :
			return m->mem.li;
		case SPHYRAENA_INT :
			return (i64)m->mem.i;
		case SPHYRAENA_FLOAT :
			return (i64)m->mem.f;
		case SPHYRAENA_DOUBLE :
			return (i64)m->mem.d;
	}
	return 0;
}

__device__ double castToDouble(sphyraena_mem *m)
{
	switch(m->type) {
		case SPHYRAENA_DOUBLE :
			return m->mem.d;
		case SPHYRAENA_INT :
			return (double)m->mem.i;
		case SPHYRAENA_FLOAT :
			return (double)m->mem.f;
		case SPHYRAENA_INT64 :
			return (double)m->mem.li;
	}
	return 0;
}


// Performs the kernel call
// You'll see that I have commented out some texture code, I tested this but
// found negligible performance improvement, probably because the global
// memory access is a very small part of program execution since the SQLite registers
// are stored in local (global) memory. If that wasn't the case, texture memory
// may make a difference.
extern "C"
int sphyraena_vm(sphyraena *s)
{
	hipError_t r;
	hipMemcpyToSymbol((char*)&cstmt, s->stmt_cpu,
		sizeof(sphyraena_stmt), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol((char*)&cdata, s->data_cpu,
		sizeof(sphyraena_data_gpu), 0, hipMemcpyHostToDevice);
	hipMemset(s->results_gpu, 0, sizeof(int));
	int zero = 0;
	hipMemcpyToSymbol((char*)&block_order, &zero, sizeof(int),
		0, hipMemcpyHostToDevice);

	/*const struct textureReference *texRefPtr;
	r = hipGetTextureReference(&texRefPtr, HIP_SYMBOL("texRef"));

	if(r != hipSuccess) {
		fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(r));
		return SPHYRAENA_ERR_TEXTURE;
	}

	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc<float>();

	r = hipBindTexture(0, texRefPtr, (char*)s->data_gpu,
		&channelDesc, s->data_size);

	//texRef.filterMode = hipFilterModeLinear;

	if(r != hipSuccess) {
		fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(r));
		return SPHYRAENA_ERR_TEXTURE;
	}*/

	int blocks = (s->data_cpu->rows + s->threads_per_block - 1) / s->threads_per_block;
	int thread_rows = 1;

	if(blocks >= 65536) {
		thread_rows = (int) ceilf((float)blocks / (float)65536);
		blocks = (int) ceilf((float)blocks / (float)thread_rows);
	}

	VmKernel<<<blocks, s->threads_per_block>>>((char*)s->data_gpu, s->results_gpu, 0, blocks, thread_rows);

	if((r = hipGetLastError()) != hipSuccess) {
		fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(r));
		return SPHYRAENA_ERR_KERNEL;
	}

	r = hipDeviceSynchronize();

	if(r != hipSuccess) {
		fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(r));
		return SPHYRAENA_ERR_KERNEL;
	}

	return SPHYRAENA_SUCCESS;
}


// performs the kernel call with streaming blocks, using s->stream_width to determine
// the number of streaming blocks
extern "C"
int sphyraena_vm_streaming(sphyraena *s)
{
	if(s->data_cpu->rows < 1000)
		return sphyraena_vm(s);

	hipError_t r;
	hipMemcpyToSymbol((char*)&cstmt, s->stmt_cpu,
		sizeof(sphyraena_stmt), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol((char*)&cdata, s->data_cpu,
		sizeof(sphyraena_data_gpu), 0, hipMemcpyHostToDevice);
	hipMemset(s->results_gpu, 0, sizeof(int));
	int zero = 0;
	hipMemcpyToSymbol((char*)&block_order, &zero, sizeof(int),
		0, hipMemcpyHostToDevice);

	int rows_per_stream = (s->data_cpu->rows + s->stream_width - 1) / s->stream_width;

	int blocks = (rows_per_stream + s->threads_per_block - 1) / s->threads_per_block;
	int block_size = s->data_cpu->stride * rows_per_stream;

	int thread_rows = 0;

	if(blocks >= 65536) {
		thread_rows = (int) ceilf((float)blocks / (float)65536);
		blocks = (int) ceilf((float)blocks / (float)thread_rows);
	}

	//printf("reg size   %i        block_size  %i\n", s->data_cpu->rows * s->data_cpu->stride, block_size);

	//printf("rps %i     blocks %i   block_size %i\n", rows_per_stream, blocks, block_size);

	hipStream_t stream[s->stream_width];
	for(int i = 0; i < s->stream_width; i++)
		hipStreamCreate(&stream[i]);

	for(int i = 0; i < s->stream_width; i++)
		hipMemcpyAsync(s->data_gpu + block_size * i, s->data_cpu->d + block_size * i,
			block_size, hipMemcpyHostToDevice, stream[i]);

	for(int i = 0; i < s->stream_width; i++) {
		VmKernel<<<blocks, s->threads_per_block, 0, stream[i]>>>
			((char*)s->data_gpu, s->results_gpu, i * blocks * s->threads_per_block, blocks, thread_rows);
	}

	for(int i = 0; i < s->stream_width; i++)
		hipStreamDestroy(stream[i]);

	if((r = hipGetLastError()) != hipSuccess) {
		fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(r));
		return SPHYRAENA_ERR_KERNEL;
	}

	r = hipDeviceSynchronize();

	if(r != hipSuccess) {
		fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(r));
		return SPHYRAENA_ERR_KERNEL;
	}

	return SPHYRAENA_SUCCESS;
}



#include "hip/hip_runtime.h"

/* vim: set filetype=c : */

#include <stdio.h>

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=0;i<num_iterations;i++)
	    g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

int correct_data(int *a, const int n, const int c)
{
    for(int i = 0; i < n; i++)
        if(a[i] != c)
		{
			printf("%d: %d %d\n", i, a[i], c);
            return 0;
		}

    return 1;
}

int main(int argc, char *argv[])
{
    int cuda_device = 0;
    int nstreams = 4;               // number of streams for CUDA calls
    int nreps = 10;                 // number of times each experiment is repeated
    int n = 16 * 1024 * 1024;       // number of ints in the data set
    int nbytes = n * sizeof(int);   // number of data bytes
    dim3 threads, blocks;           // kernel launch configuration
    float elapsed_time, time_memcpy, time_kernel;   // timing variables

	int niterations;	// number of iterations for the loop inside the kernel

    if( argc > 1 )
        cuda_device = atoi( argv[1] );

#ifdef __DEVICE_EMULATION__
    n = 4096;   // reduced workload for emulation (n should be divisible by 512*nstreams)
#endif

    // check the compute capability of the device
    int num_devices=0;
    hipGetDeviceCount(&num_devices);
    if(0==num_devices)
    {
        printf("your system does not have a CUDA capable device\n");
        return 1;
	}
	
    // check if the command-line chosen device ID is within range, exit if not
    if( cuda_device >= num_devices )
	{
        printf("choose device ID between 0 and %d\n", num_devices-1);
        return 1;
	}

    hipSetDevice( cuda_device );

    hipDeviceProp_t device_properties;
	
    hipGetDeviceProperties(&device_properties, cuda_device);
    if( (1 == device_properties.major) && (device_properties.minor < 1))
        printf("%s does not have compute capability 1.1 or later\n\n", device_properties.name);
    if(device_properties.minor > 1)
        niterations = 5;
    else
        niterations = 1;    // reduced workload for compute capability 1.0 and 1.1

	printf("running on: %s\n\n", device_properties.name );

    // allocate host memory
    int c = 5;                      // value to which the array will be initialized
    int *a = 0;                     // pointer to the array data in host memory
    // allocate host memory (pinned is required for achieve asynchronicity)
    hipHostMalloc((void**)&a, nbytes); 

    // allocate device memory
    int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
    hipMalloc((void**)&d_a, nbytes);
    hipMalloc((void**)&d_c, sizeof(int));
    hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice);

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
        hipStreamCreate(&(streams[i]));

    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    // time memcopy from device
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, streams[0]);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    hipEventElapsedTime(&time_memcpy, start_event, stop_event);
    printf("memcopy:\t%.2f\n", time_memcpy);
    
    // time kernel
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&time_kernel, start_event, stop_event);
    printf("kernel:\t\t%.2f\n", time_kernel);

    //////////////////////////////////////////////////////////////////////
    // time non-streamed execution for reference
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        init_array<<<blocks, threads>>>(d_a, d_c, niterations);
        hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("non-streamed:\t%.2f (%.2f expected)\n", elapsed_time / nreps, time_kernel + time_memcpy);

    //////////////////////////////////////////////////////////////////////
    // time execution with nstreams streams
    threads=dim3(512,1);
    blocks=dim3(n/(nstreams*threads.x),1);
    memset(a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
    hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        // asynchronously launch nstreams kernels, each operating on its own portion of data
        for(int i = 0; i < nstreams; i++)
            init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);

        // asynchronoously launch nstreams memcopies.  Note that memcopy in stream x will only
        //   commence executing when all previous CUDA calls in stream x have completed
        for(int i = 0; i < nstreams; i++)
            hipMemcpyAsync(a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("%d streams:\t%.2f (%.2f expected with compute capability 1.1 or later)\n", nstreams, elapsed_time / nreps, time_kernel + time_memcpy / nstreams);

    // check whether the output is correct
    printf("-------------------------------\n");
    if(correct_data(a, n, c*nreps*niterations))	// each element should have been incremented by c a total of npres*niterations times
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // release resources
    for(int i = 0; i < nstreams; i++)
        hipStreamDestroy(streams[i]);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipHostFree(a);
    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();

    return 0;
}


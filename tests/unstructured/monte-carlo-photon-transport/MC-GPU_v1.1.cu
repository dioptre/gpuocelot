#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////////////////////
//
//               ****************************
//               *** MC-GPU , version 1.1 ***
//               ****************************
//
/**
 *      \mainpage  MC-GPU v1.1
 *
 *
 * \b MC-GPU is an x ray transport simulation code that can generate radiographic
 * projection images and computed tomography (CT) scans of voxelized objects,
 * including realistic human anatomy phantoms.
 * 
 * The code implements a massively multi-threaded Monte Carlo simulation algorithm
 * for the transport of x rays in a voxelized geometry. The program has been
 * developed using the \b CUDA programming model and the simulation can be executed in
 * parallel in a state-of-the-art GPU from \b NVIDIA, giving an speed up of the order
 * of 15-25 times, compared to a CPU execution. The x ray interaction models and
 * cross sections have been adapted from \b PENELOPE \b 2006.
 * Currently, the code does not transport secondary electrons and the electrons
 * that would be created in photoelectric and Compton events are assumed to be
 * locally absorbed (dose is not reported).
 * 
 * The MC-GPU code has been described in different scientific publications. A brief
 * description of the code features is given below. This description has been taken
 * from the main paper that can be cited to refer to this code:
 *
 * \code
 *  Andreu Badal and Aldo Badano, "Accelerating Monte Carlo simulations of photon transport in a voxelized
 *  geometry using a massively parallel Graphics Processing Unit", Medical Physics 36, pp. 4878–4880 (2009)
 * \endcode
 *
 * This code is still in development, please report to the authors any issue/bug
 * that you may encounter. Feel free to suggest improvements to the code too. 
 *
 *
 *    \section  sec_disc  DISCLAIMER
 *
 *  This software and documentation (the "Software") were developed at the Food and
 *  Drug Administration (\b FDA) by employees of the Federal Government in the course
 *  of their official duties. Pursuant to Title 17, Section 105 of the United States
 *  Code, this work is not subject to copyright protection and is in the public
 *  domain. Permission is hereby granted, free of charge, to any person obtaining a
 *  copy of the Software, to deal in the Software without restriction, including
 *  without limitation the rights to use, copy, modify, merge, publish, distribute,
 *  sublicense, or sell copies of the Software or derivatives, and to permit persons
 *  to whom the Software is furnished to do so. FDA assumes no responsibility
 *  whatsoever for use by other parties of the Software, its source code,
 *  documentation or compiled executables, and makes no guarantees, expressed or
 *  implied, about its quality, reliability, or any other characteristic. Further,
 *  use of this code in no way implies endorsement by the FDA or confers any
 *  advantage in regulatory decisions.  Although this software can be redistributed
 *  and/or modified freely, we ask that any derivative works bear some notice that
 *  they are derived from it, and any modified versions bear some notice that they
 *  have been modified.
 * 
 *     \section sec_Intro Code features
 *  
 * MC-GPU does not currently simulate the transport of electrons.
 * The interactions between the photons and the material objects are simulated
 * using the well-known interaction sampling models from the PENELOPE 2006
 * subroutine package.
 * 
 * In order to speed up the ray-tracing of the code and minimize the access to the
 * slow GPU main memory, the photon trajectories across the voxels are computed
 * using the Woodcock tracking algorithm.
 * With this technique the photons perceive the geometry as a uniform medium
 * composed of the material of the most attenuating voxel.
 * In this way, the voxel boundaries do not have to be explicitly calculated and
 * multiple voxels can be crossed in a single step.
 * To keep the simulation unbiased, some of the interactions are considered
 * "virtual" (i.e., do not change the photon energy or direction of movement),
 * depending on the actual energy and the material at the interaction site.
 * In a typical simulation, several thousand threads are launched simultaneously in
 * the GPU, each one of them simulating a batch of 10000, or more, photon tracks.
 * 
 * The random number generator used in PENELOPE, ranecu, is also used in the GPU
 * program.
 * To ensure that the simulated tracks are not correlated, each thread initializes
 * the generator to a unique position in the random sequence, far enough from the
 * other threads, using the algorithm implemented in the seedsMLCG code.
 * 
 * The new code is currently used in the study of scatter in x-ray imaging and
 * includes a tally to generate radiographic images.
 * The image is formed by counting the energy that enters a user-defined 2D grid of
 * pixels, which is a simple approximation to a noise-free flat-panel detector with
 * 100% detection efficiency; the pixel values have units of eV/cm^2.
 * Four different images are reported at the end of the simulation, corresponding
 * to the signal produced by non-scattered, single Compton, single Rayleigh, and
 * multi-scattered photons.
 * The radiation source is implemented as a point source emitting monoenergetic
 * photons within a fan beam, producing a rectangular field on the detector
 * equivalent to a collimated cone beam.
 * 
 * 
 *    \section sec_CPU Code compilation and execution
 *
 *   MC-GPU has been tested only in the Linux operating system.
 *   A Makefile script is provided to compile the MC-GPU code in Linux.
 *   The CUDA libraries and the GNU GCC compiler must be previously installed.
 *   The Makefile may have to be edited to modify the library path.
 *
 *   A README text file is provided with the MC-GPU source code. Read this file for
 *   more information on the code usage. An example simulation input file is also
 *   provided.
 *
 *   MC-GPU uses CUDA to access the GPU but all the actual computations are coded
 *   in standard C code.
 *   All the CUDA specific commands are enclosed within preprocessor if statements.
 *   Defining the pre-processor variable "USING_CUDA" (i.e., compiling with
 *   "-DUSING_CUDA") the particle transport is executed in parallel in an
 *   NVIDIA GPU using CUDA. Otherwise, the code is sequentially executed in
 *   the CPU.
 * 
 * 
 *    \section sec_CT Parallel simulation of CT scans:
 * 
 *   From version 1.1, MC-GPU allows the simulation of a CT scan. The CT is
 *   simulated generating multiple projection images around the static voxelized
 *   geometry. To speed up the CT simulation, the MPI library is used to address
 *   multiple GPUs and obtain multiple projections in parallel. In order to
 *   activate the MPI code, the pre-processor variable "USING_MPI" has to be
 *   defined (ie, compiling with "-DUSING_MPI").
 *   To use the code in parallel in N GPUs (in a single computer), the user
 *   has to run the program with N MPI threads in the CPU (eg,
 *   "mpirun -np 4 ./MC-GPU.x MC-GPU.in"). Each thread will get a unique id in
 *   the CPU (myID=0->N) and will address a unique GPU. The CT simulation will
 *   then be split so that the threads simulate consecutive projections
 *   independently, avoiding any intercommunication between threads.
 * 
 * 
 * 
 * 
 * 
 *                      @file    MC-GPU_v1.1.cu
 *                      @author  Andreu Badal (Andreu.Badal-Soler@fda.hhs.gov)
 *                      @date    2010/06/25
 *                        -- First version:  2009/03/17
 */ 
////////////////////////////////////////////////////////////////////////////////////////







// *** Include header file with the structures and functions declarations
#include <MC-GPU_v1.1.h>

// *** Include the computing kernel:
#include <MC-GPU_kernel_v1.1.cu>


////////////////////////////////////////////////////////////////////////////////
//!  Main program to transport x rays in a 3D voxel geometry using the GPU.
//!  This function reads the description of the simulation from an external file
//!  given in the command line. This input file defines the number of particles to
//!  simulate, the characteristics of the x-ray source and the detector, the number
//!  and spacing of the projections (if simulating a CT), the location of the
//!  material files containing the interaction mean free paths, and the location
//!  of the voxelized geometry file.
//!
//!                            @author  Andreu Badal
//!                            @date    2010/03/19
//!
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

#ifdef USING_MPI
// -- Using MPI to address multiple GPUs in the same workstation (use only to simulate a CT scan).
  int myID = -88, numprocs = -99;
  MPI_Init(&argc, &argv);                   // Init MPI and get the current thread ID 
  MPI_Comm_rank(MPI_COMM_WORLD, &myID);
  MPI_Comm_size(MPI_COMM_WORLD, &numprocs);
  printf("            *** MPI run:  myId=%d , numprocs=%d\n",myID,numprocs);
  fflush(stdout);   // Clear the screen output buffer
  MPI_Barrier(MPI_COMM_WORLD);   // Synchronize MPI threads  
#else
  int myID = 0, numprocs = 1;   // Only one CPU thread used when MPI is not activated (multiple projections will be simulated sequentially).
#endif
  

  // *** Declare the arrays and structures that will contain the simulation data:

  struct voxel_struct voxel_data;          // Define the geometric constants of the voxel file
  struct source_struct source_data;        // Define the particles source
  struct detector_struct detector_data;    // Define an x ray detector
  struct linear_interp mfp_table_data;     // Constant data for the linear interpolation

  struct compton_struct  compton_table;    // Structure containing Compton sampling data (to be copied to CONSTANT memory)
  struct rayleigh_struct rayleigh_table;   // Structure containing Rayleigh sampling data (to be copied to CONSTANT memory)
  
  float2 *voxel_mat_dens = NULL;           // Poiter where voxels array will be allocated
  unsigned int voxel_mat_dens_bytes = 0;   // Size (in bytes) of the voxels array (using unsigned int to allocate up to 4.2GBytes)
  float density_max[MAX_MATERIALS];
  float density_nominal[MAX_MATERIALS];
  unsigned long long int *image = NULL;    // Poiter where image array will be allocated
  int image_bytes = -1;                    // Size of the image array
  int mfp_table_bytes = -1, mfp_Woodcock_table_bytes = -1;   // Size of the table arrays
  float2 *mfp_Woodcock_table = NULL;                // Linear interpolation data for the Woodcock mean free path [cm]
  float3 *mfp_table_a = NULL, *mfp_table_b = NULL;  // Linear interpolation data for 3 different interactions:
                                              //  (1) inverse total mean free path (divided by density, cm^2/g)
                                              //  (2) inverse Compton mean free path (divided by density, cm^2/g)
                                              //  (3) inverse Rayleigh mean free path (divided by density, cm^2/g)
  
#ifdef USING_CUDA
  // - MASTER_THREAD == "if(0==myID)": macro to print the messages just once when using MPI threads; it has no effect if MPI is not used.
  MASTER_THREAD printf  ("\n             *** CUDA SIMULATION IN THE GPU ***\n");
#else
  MASTER_THREAD printf  ("\n             *** SIMULATION IN THE CPU ***\n");
#endif

  time_t current_time = time(NULL);     // Get current time (in seconds)
  MASTER_THREAD printf("\n****** Code execution started on: %s\n", ctime(&current_time));

  MASTER_THREAD printf("    -- INITIALIZATION phase:\n");

  // -- Start time counter:
  clock_t clock_start, clock_end;      // (requires standard header <time.h>)
  clock_start = clock();               // Get current clock counter


  unsigned long long int total_histories;
  int histories_per_thread, seed_input, num_threads_per_block, gpu_id, num_projections;
  double D_angle, angularROI_0, angularROI_1, initial_angle;
  char file_name_voxels[250], file_name_materials[MAX_MATERIALS][250], file_name_output[250];  

  // *** Read the input file given in the command line and return the significant data:
  read_input(argc, argv, myID, &total_histories, &seed_input, &gpu_id, &num_threads_per_block, &histories_per_thread, &detector_data, &image, &image_bytes, &source_data, file_name_voxels, file_name_materials, file_name_output, &num_projections, &D_angle, &angularROI_0, &angularROI_1, &initial_angle);

        float3 detector_center;
        detector_center.x = source_data.position[0].x + source_data.direction[0].x * detector_data.sdd;   // Set the center of the detector straight ahead of the focal spot.
        detector_center.y = source_data.position[0].y + source_data.direction[0].y * detector_data.sdd;
        detector_center.z = source_data.position[0].z + source_data.direction[0].z * detector_data.sdd;

  MASTER_THREAD
  {
        printf("                       total_histories = %lld\n", total_histories);
        printf("                           random seed = %d\n", seed_input);
        printf("                         source energy = %.3f keV\n", (source_data.energy/1.0e3f));
        
//         printf("            azimuthal, polar apertures = %.6f , %.6f degrees\n", 2.0*atan(source_data.tan_phi_semiaperture)*RAD2DEG, 2.0*atan(source_data.tan_theta_semiaperture)*RAD2DEG);
        printf("      azimuthal (phi), polar apertures = %.6f , %.6f degrees\n", source_data.D_phi*RAD2DEG, 2.0*(90.0 - acos(source_data.cos_theta_low)*RAD2DEG) );

         
        printf("                   focal spot position = (%f, %f, %f)\n", source_data.position[0].x, source_data.position[0].y, source_data.position[0].z);
        printf("                      source direction = (%f, %f, %f)\n", source_data.direction[0].x, source_data.direction[0].y, source_data.direction[0].z);
        printf("                  initial angle from X = %lf\n", initial_angle*RAD2DEG);
        printf("                       detector center = (%f, %f, %f)\n", detector_center.x, detector_center.y, detector_center.z);
        printf("           detector low corner (at +Y) = (%f, %f, %f)\n", detector_data.corner_min_rotated_to_Y[0].x, detector_data.corner_min_rotated_to_Y[0].y, detector_data.corner_min_rotated_to_Y[0].z);
        printf("              source-detector distance = %f cm\n", detector_data.sdd);
        printf("                number of pixels image = %dx%d\n", detector_data.num_pixels.x, detector_data.num_pixels.y);
        printf("                            pixel size = %.3fx%.3f cm\n", 1.0f/detector_data.inv_pixel_size_X, 1.0f/detector_data.inv_pixel_size_Z);
        printf("                 number of projections = %d\n", num_projections);
        if (num_projections!=1)
        {
          printf("             angle between projections = %lf\n", D_angle*RAD2DEG);
          printf("            angular region of interest = [%lf,%lf] degrees\n", angularROI_0*RAD2DEG, angularROI_1*RAD2DEG);
        }
        printf("                      Input voxel file = %s\n", file_name_voxels);
        printf("                     Output image file = %s\n", file_name_output);
        fflush(stdout);
}


  // *** Set the detectors and sources for the CT trajectory (if needed, ie, for more than one projection):
  if (num_projections != 1)
    set_CT_trajectory(myID, num_projections, D_angle, angularROI_0, angularROI_1, &source_data, &detector_data);

  fflush(stdout);
        

  // *** Read the voxel data and allocate the density map matrix. Return the maximum density:
  load_voxels(myID, file_name_voxels, density_max, &voxel_data, &voxel_mat_dens, &voxel_mat_dens_bytes);  
  MASTER_THREAD printf("       Total CPU memory allocated for voxels vector and data structures = %f Mbytes\n", (voxel_mat_dens_bytes+image_bytes+sizeof(struct voxel_struct)+sizeof(struct source_struct)+sizeof(struct detector_struct)+sizeof(struct linear_interp)+2*mfp_table_bytes+sizeof(struct rayleigh_struct)+sizeof(struct compton_struct))/(1024.f*1024.f));
  MASTER_THREAD fflush(stdout);

  // *** Read the material mean free paths and set the interaction table in a "linear_interp" structure:
  load_material(myID, file_name_materials, density_max, density_nominal, &mfp_table_data, &mfp_Woodcock_table, &mfp_Woodcock_table_bytes, &mfp_table_a, &mfp_table_b, &mfp_table_bytes, &rayleigh_table, &compton_table);

  // -- Check that the input material tables and the x-ray source are consistent:
  if ( (source_data.energy < mfp_table_data.e0) || (source_data.energy > (mfp_table_data.e0 + (mfp_table_data.num_values-1)/mfp_table_data.ide)) )
  {
    MASTER_THREAD printf("\n !!ERROR!! The input x-ray source energy (%f eV) is outside the tabulated energy interval (from %f to %f eV)!!\n\n", source_data.energy, mfp_table_data.e0, (mfp_table_data.e0+(mfp_table_data.num_values-1)/mfp_table_data.ide));
    exit(-1);
  }


#ifdef USING_MPI
  if (myID != gpu_id)
    gpu_id = myID;     // Use the GPU with the same number as the CPU thread number (from '0' to 'numprocs-1')
  else                 // but skip the GPU number given in the input file (useful if GPU 0 is attached to a display).
    gpu_id = numprocs;
#endif


#ifdef USING_CUDA
  // -- Declare the pointers to the device global memory, when using the GPU:
  float2 *voxel_mat_dens_device     = NULL,
         *mfp_Woodcock_table_device = NULL;
  float3 *mfp_table_a_device        = NULL,
         *mfp_table_b_device        = NULL;
  unsigned long long int *image_device          = NULL;
  struct rayleigh_struct *rayleigh_table_device = NULL;
  struct compton_struct  *compton_table_device  = NULL;

  // -- Sets the CUDA enabled GPU that will be used in the simulation, and allocate and copies the simulation data in the GPU global and constant memories.
  init_CUDA_device(&gpu_id, myID, &voxel_data, &source_data, &detector_data, &mfp_table_data,  /*Variables GPU constant memory*/
        voxel_mat_dens, &voxel_mat_dens_device, voxel_mat_dens_bytes,                          /*Variables GPU global memory*/
        image, &image_device, image_bytes,
        mfp_Woodcock_table, &mfp_Woodcock_table_device, mfp_Woodcock_table_bytes,
        mfp_table_a, mfp_table_b, &mfp_table_a_device, &mfp_table_b_device, mfp_table_bytes,
        &rayleigh_table, &rayleigh_table_device,
        &compton_table, &compton_table_device);
#endif

  clock_end = clock();
  double time_elapsed_1 = ((double)(clock_end-clock_start))/CLOCKS_PER_SEC;
  double time_elapsed_2 = 0.0, time_total = 0.0;
  MASTER_THREAD printf("\n    -- INITIALIZATION finished: elapsed time = %.3f s\n", time_elapsed_1);

#ifdef USING_MPI
  fflush(stdout);
  MPI_Barrier(MPI_COMM_WORLD);   // Synchronize MPI threads before starting the MC phase.
#endif

  MASTER_THREAD printf("\n\n    -- MONTE CARLO LOOP phase:\n\n");
  MASTER_THREAD fflush(stdout);   // Clear the screen output buffer

  // -- Scale the input number of particles taking into account that "histories_per_thread" tracks will be
  //    simulated for each call to "track_particles", ie, each GPU thread.
  //    The total number of particles simulated will be increased to the nearest multiple histories_per_thread.
  if(((unsigned long long int)(total_histories/histories_per_thread))>=2147483647)
  {
    MASTER_THREAD printf("\n\n   !!ERROR!! \"total_histories/histories_per_thread = %lld > 2147483647\": this value can not be represented with a 4 byte integer.\n     Please, increase histories_per_thread until the fraction is acceptable. Sorry about that.\n\n", ((unsigned long long int)(total_histories/histories_per_thread)));
    exit(-1);
  }
  int total_history_batch = (int)(total_histories/histories_per_thread);
  if (0!=(total_histories%histories_per_thread))
  {
    total_history_batch++;        // Input value is not multiple of HISTORIES_PER_THREAD: make an extra iteration
    total_histories = (unsigned long long int)total_history_batch*histories_per_thread;   // Total histories will be higher than input value
  }


  // *** CT simulation: simulate multiple projections if requested:

  double current_angle;
  int num_p;  // == current projection number
  for (num_p=0; num_p<num_projections; num_p++)
  {
    #ifdef USING_MPI
      // -- Distribute the projections to the multiple threads evenly: each thread simulates an angle and skip the following 'numprocs' angles
      if( (num_p % numprocs) != myID )
      {
        // printf(" ... Thread %d skipped projection=%d, modulo=%d\n", myID, num_p, (num_p % numprocs));
        continue;
      }
    #endif
    
    // -- Check if this projection is inside the angular region of interest
    current_angle = initial_angle + num_p * D_angle;    
    if (current_angle<0.0)
      current_angle += 2.0*PI;   // Make sure the angle is not negative.
    else if (current_angle>=(2.0*PI-0.0001))
      current_angle -= 2.0*PI;   // Make sure the angle is not above or equal to 360 degrees.

    if ((current_angle < angularROI_0) || (current_angle > angularROI_1))
    {
      printf("\n         << Skipping projection #%d >> Angle: %f degrees --> outside the angular region of interest.\n", num_p, current_angle*RAD2DEG);
      continue;   // Cycle loop: do not simulate this projection!
    }
      
    if (num_projections!=1)
      printf("\n   << Simulating Projection %d of %d >> Angle: %lf degrees.\n\n", num_p, num_projections-1, current_angle*RAD2DEG);
      
    #ifdef USING_CUDA
      // --Executing the kernel in the GPU:
      int total_history_batch_blocks = (int)(total_history_batch/num_threads_per_block);
      if (0!=(total_history_batch%num_threads_per_block))
      {
        total_history_batch_blocks++;        // Value is not multiple of num_threads: execute an extra block of threads
        total_histories = (unsigned long long int)total_history_batch_blocks*num_threads_per_block*histories_per_thread;   // Total histories will be higher than input value
      }
      // -- Setup the execution parameters, taking into account the current CUDA specifications:
      //    Warp size: 32, Max number threads per block: 512, Max sizes each dimension of block: 512x512x64, Max sizes each dimension of grid: 65535x65535x1
      if (total_history_batch_blocks>65535)
      {
        MASTER_THREAD printf("\n\n   !!ERROR!! \"total_histories/histories_per_thread/num_threads_per_block ~ %d > 65535\": the GPU can not simulate so many blocks (with a 1D grid).\n             Please, increase histories_per_thread or num_threads_per_block until the fraction is acceptable. Sorry about that.\n\n", total_history_batch_blocks);
        exit(-1);
      }
      dim3 blocks(total_history_batch_blocks, 1);
      dim3 threads(num_threads_per_block, 1);
      printf("       Executing %d blocks of %d threads, with %d histories in each thread: %lld histories in total.\n", total_history_batch_blocks, num_threads_per_block, histories_per_thread, total_histories);
      fflush(stdout); 

      // -- Execute the kernel
      unsigned int timer = 0;
      cutilCheckError(cutCreateTimer(&timer));
      cutilCheckError(cutStartTimer(timer));

      clock_start = clock();
      track_particles<<<blocks,threads>>>(histories_per_thread, num_p, seed_input, image_device, voxel_mat_dens_device, mfp_Woodcock_table_device, mfp_table_a_device, mfp_table_b_device, rayleigh_table_device, compton_table_device);
      hipDeviceSynchronize();    // Force the runtime to wait until all device tasks have completed

      // -- Check if kernel execution generated any error:
      cutilCheckMsg(" !!Kernel execution failed while simulating particle tracks!! ");

      cutilCheckError( cutStopTimer( timer));
      printf("       ==> CUDA: Kernel execution time in the device: %.3f s \n", 0.001f*cutGetTimerValue( timer));
      cutilCheckError( cutDeleteTimer( timer));
      cutilCheckError( cutCreateTimer( &timer));
      cutilCheckError( cutStartTimer( timer));
      cutilSafeCall( hipMemcpy( image, image_device, image_bytes, hipMemcpyDeviceToHost) );  // Copy final results to host
      cutilCheckError( cutStopTimer( timer));
      printf("                 Time copying results from device to host: %.3f s\n", 0.001f*cutGetTimerValue( timer));
      cutilCheckError( cutDeleteTimer( timer));

    #else
      // --Executing the kernel in the CPU:
      printf("       Executing %d history batches, with %d histories in each batch: %lld histories in total.\n", total_history_batch, histories_per_thread, total_histories);

      // -- Copy local structures to global struct variables accessible from "track_particles" (__constant__ variables in the GPU):
      source_data_CONST = source_data;
      detector_data_CONST = detector_data;
      voxel_data_CONST = voxel_data;
      mfp_table_data_CONST = mfp_table_data;

      clock_start = clock();
      int n;
      for(n=0; n<total_history_batch; n++)
      {
        // -- Simulate a particle track initializing the PRNG with the particle number 'n':
        track_particles(n, histories_per_thread, num_p, seed_input, image, voxel_mat_dens, mfp_Woodcock_table, mfp_table_a, mfp_table_b, &rayleigh_table, &compton_table);
      }
    #endif

    // Get final time and calculate loop execution time:
    clock_end = clock();
    time_elapsed_2 = ((double)(clock_end-clock_start))/CLOCKS_PER_SEC;
    time_total += time_elapsed_2;   // Count total time (in seconds).
        //  printf("\n    -- MONTE CARLO LOOP finished: time tallied in MAIN program: %.3f s\n\n", time_elapsed_2);

    // *** Report the final results:
    char file_name_output_num_p[253];
    if (1==num_projections)
      strcpy(file_name_output_num_p, file_name_output);   // Use the input name for single projection
    else
      sprintf(file_name_output_num_p, "%s_%03d", file_name_output, num_p);   // Create the output file name with the input name + projection number (3 digits, padding with 0)

    report_host(file_name_output_num_p, &detector_data, &source_data, image, time_elapsed_2, total_histories, num_p, num_projections, D_angle, initial_angle, myID, numprocs);

    // *** Clear the image after reporting, unless this is the last projection to simulate:
    if (num_p<(num_projections-1))
    {
      int pixels_per_image = detector_data.num_pixels.x * detector_data.num_pixels.y;
      #ifdef USING_CUDA
        printf("       ==> CUDA: Launching kernel to reset the device image to 0: number of blocks = %d, threads per block = 128\n", (int)ceil(pixels_per_image/128.0f) );
        init_image_array_GPU<<<(int)(ceil(pixels_per_image/128.0f)),128>>>(image_device, pixels_per_image);
        hipDeviceSynchronize();
        cutilCheckMsg(" !!Kernel execution failed initializing the image array!! ");  // Check if kernel execution generated any error:
      #else
        int j;
        for (j=0; j<pixels_per_image; j++)   // INIT IMAGE ARRAY IN THE CPU
        {
          image[j                   ] = (unsigned long long int)(0);
          image[j+  pixels_per_image] = (unsigned long long int)(0);
          image[j+2*pixels_per_image] = (unsigned long long int)(0);
          image[j+3*pixels_per_image] = (unsigned long long int)(0);
        }
      #endif
    }
  }  // [Loop end: iterate for next CT projection]

  MASTER_THREAD printf("\n    -- MONTE CARLO LOOP finished: total time tallied in MAIN program: %.3f s\n", time_total);


  // *** Clean up RAM memory:
  free(voxel_mat_dens);
  free(image);
  free(mfp_Woodcock_table);
  free(mfp_table_a);
  free(mfp_table_b);
  
#ifdef USING_CUDA
  unsigned int timer = 0;
  cutilCheckError( cutCreateTimer( &timer));
  cutilCheckError( cutStartTimer( timer));
  
  // -- Clean up GPU device memory:
  cutilSafeCall(hipFree(voxel_mat_dens_device));
  cutilSafeCall(hipFree(image_device));
  cutilSafeCall(hipFree(mfp_Woodcock_table_device));
  cutilSafeCall(hipFree(mfp_table_a_device));
  cutilSafeCall(hipFree(mfp_table_b_device));

  hipDeviceReset();

  cutilCheckError( cutStopTimer( timer));
  MASTER_THREAD printf("       ==> CUDA: Time freeing the device memory and ending the GPU threads: %.6f s\n", 0.001f*cutGetTimerValue( timer));
  cutilCheckError( cutDeleteTimer( timer));
#endif

#ifdef USING_MPI
  current_time=time(NULL);     // Get current time (in seconds)
  printf("       MPI node %d done! Time: %s\n", myID, ctime(&current_time));
  fflush(stdout);   // Clear the screen output buffer
  MPI_Barrier(MPI_COMM_WORLD);   // Synchronize MPI threads
  MPI_Finalize();   // Finalize MPI library: no more MPI calls allowed below.
#endif

  current_time=time(NULL);     // Get current time (in seconds)
  MASTER_THREAD printf("\n****** Code execution finished on: %s\n", ctime(&current_time));
  
  return 0;
}



////////////////////////////////////////////////////////////////////////////////
//! Read the input file given in the command line and return the significant data.
//! Example input file:
//!
//!    1000000          [Total number of histories to simulate]
//!    geometry.vox     [Voxelized geometry file name]
//!    material.mat     [Material data file name]
//!
//!       @param[in] argc   Command line parameters
//!       @param[in] argv   Command line parameters: name opf input file
//!       @param[out] total_histories  Total number of particles to simulate
//!       @param[out] seed_input   Input random number generator seed
//!       @param[out] num_threads_per_block   Number of CUDA threads for each GPU block
//!       @param[out] detector_data
//!       @param[out] image
//!       @param[out] source_data
//!       @param[out] file_name_voxels
//!       @param[out] file_name_materials
//!       @param[out] file_name_output
////////////////////////////////////////////////////////////////////////////////
void read_input(int argc, char** argv, int myID, unsigned long long int* total_histories, int* seed_input, int* gpu_id, int* num_threads_per_block, int* histories_per_thread, struct detector_struct* detector_data, unsigned long long int** image_ptr, int* image_bytes, struct source_struct* source_data, char* file_name_voxels, char file_name_materials[MAX_MATERIALS][250] , char* file_name_output, int* num_projections, double* D_angle, double* angularROI_0, double* angularROI_1, double* initial_angle)
{
  FILE* file_ptr = NULL;
  char new_line[250];
  char *new_line_ptr = NULL;
  double dummy_double;

  // -- Read the input file name from command line, if given (otherwise keep default value):
  if (2==argc)
  {
    file_ptr = fopen(argv[1], "r");
    if (NULL==file_ptr)
    {
      printf("\n\n   !!read_input ERROR!! Input file not found or not readable. Input file name: \'%s\'\n\n", argv[1]);
      exit(-1);
    }
  }
  else if (argc>2)
  {
    printf("\n\n   !!read_input ERROR!! Too many input parameter (argc=%d)!! Provide only the input file name.\n\n", argc);
    exit(-1);
  }
  else
  {
    printf("\n\n   !!read_input ERROR!! Input file name not given as an execution parameter!! Try again...\n\n");
    exit(-1);
  }

  MASTER_THREAD printf("\n    -- Reading the input file \'%s\':\n", argv[1]);

  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);    // Read full line (max. 250 characters).
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!read_input ERROR!! Input file is not readable or does not contain the string \'SECTION SIMULATION\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"SECTION SIMULATION")==NULL);   // Skip comments and empty lines until the section begins
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%lf", &dummy_double);
    *total_histories = (unsigned long long int) (dummy_double+0.0001);  // Maximum unsigned long long value: 18446744073709551615
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%d", seed_input);   // Set the RANECU PRNG seed (the same seed will be used to init the 2 MLCGs in RANECU)
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%d", gpu_id);       // GPU NUMBER WHERE SIMULATION WILL RUN
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%d", num_threads_per_block);  // GPU THREADS PER CUDA BLOCK
  if ((*num_threads_per_block%32)!=0)
  {
    printf("\n\n   !!read_input ERROR!! The input number of GPU threads per CUDA block must be a multiple of 32 (warp size). Input value: %d !!\n\n", *num_threads_per_block);
    exit(-2);
  }
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%d", histories_per_thread);   // HISTORIES PER GPU THREAD

  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!read_input ERROR!! Input file is not readable or does not contain the string \'SECTION SOURCE\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"SECTION SOURCE")==NULL);   // Skip comments and empty lines until the section begins
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%f", &(source_data->energy));  // X-RAY ENERGY [eV]
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%f %f %f", &source_data->position[0].x, &source_data->position[0].y, &source_data->position[0].z);   // SOURCE POSITION: X Y Z [cm]
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%f %f %f", &source_data->direction[0].x, &source_data->direction[0].y, &source_data->direction[0].z);   // SOURCE DIRECTION COSINES: U V W
    // -- Normalize the input beam direction to 1:
    dummy_double = 1.0/sqrt((double)(source_data->direction[0].x*source_data->direction[0].x + source_data->direction[0].y*source_data->direction[0].y + source_data->direction[0].z*source_data->direction[0].z));
    source_data->direction[0].x = (float)(((double)source_data->direction[0].x)*dummy_double);
    source_data->direction[0].y = (float)(((double)source_data->direction[0].y)*dummy_double);
    source_data->direction[0].z = (float)(((double)source_data->direction[0].z)*dummy_double);
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
  

/* !!DeBuG!! OLD WRONG FAN BEAM:  
    sscanf(new_line, "%f %f", &source_data->tan_phi_semiaperture, &source_data->tan_theta_semiaperture);
    if ( (source_data->tan_phi_semiaperture>-1.0e-6) && (source_data->tan_theta_semiaperture>-1.0e-6) )   // If we enter a negative angle, the fan beam will cover exactly the detector surface (see below).
    {
      source_data->tan_phi_semiaperture   = tan(0.5 * source_data->tan_phi_semiaperture*DEG2RAD);    // Divide by 2 the input apertures to get semiapertures.
      source_data->tan_theta_semiaperture = tan(0.5 * source_data->tan_theta_semiaperture*DEG2RAD);  // The tan of the semiaperture is the size of the field at distance 1, where we will sample uniform points.
    }
*/


  // Read input fan beam polar (theta) and azimuthal (phi) aperture angles (deg):
  double phi_aperture, theta_aperture;
  sscanf(new_line, "%lf %lf", &phi_aperture, &theta_aperture);

  if (theta_aperture > 180.0)
  {
    printf("\n\n   !!read_input ERROR!! Input polar aperture must be in [0,180] deg.!\n");
    printf("                       theta_aperture = %lf, phi_aperture = %lf\n", theta_aperture, phi_aperture);
    exit(-2);
  }
  if (phi_aperture > 360.0)
  {
    printf("\n\n   !!read_input ERROR!! Input azimuthal aperture must be in [0,360] deg.!\n");
    printf("                       theta_aperture = %lf, phi_aperture = %lf\n", theta_aperture, phi_aperture);
    exit(-2);
  }
  // Entering a negative theta_aperture or phi_aperture, the emitted fan beam will cover exactly the detector: see below
  
  // *** RECTANGULAR BEAM INITIALIZATION: aperture initially centered at (0,1,0), ie, THETA_0=90, PHI_0=90
  //     Using the algorithm used in PENMAIN.f, from penelope 2008 (by F. Salvat).
  source_data->cos_theta_low = (float)( cos((90.0 - 0.5*theta_aperture)*DEG2RAD) );
  source_data->D_cos_theta   = (float)( -2.0*source_data->cos_theta_low );      // Theta aperture is symetric above and below 90 deg
  source_data->phi_low       = (float)( (90.0 - 0.5*phi_aperture)*DEG2RAD );
  source_data->D_phi         = (float)( phi_aperture*DEG2RAD );

  source_data->max_height_at_y1cm = (float) ( tan(0.5*theta_aperture*DEG2RAD) );
          
  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!read_input ERROR!! Input file is not readable or does not contain the string \'SECTION DETECTOR\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"SECTION DETECTOR")==NULL);   // Skip comments and empty lines until the section begins
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    trim_name(new_line, file_name_output);   // OUTPUT IMAGE FILE NAME (no spaces)
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%d %d", &detector_data->num_pixels.x, &detector_data->num_pixels.y);   // NUMBER OF PIXELS IN THE IMAGE: Nx Nz
    detector_data->total_num_pixels = detector_data->num_pixels.x * detector_data->num_pixels.y;
  
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
  sscanf(new_line, "%f %f", &detector_data->width_X, &detector_data->height_Z);   // IMAGE SIZE (width, height): Dx Dz [cm]
    detector_data->inv_pixel_size_X = detector_data->num_pixels.x / detector_data->width_X;
    detector_data->inv_pixel_size_Z = detector_data->num_pixels.y / detector_data->height_Z;

  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    sscanf(new_line, "%f", &detector_data->sdd);            // SOURCE-TO-DETECTOR DISTANCE [cm] (detector set in front of the source, normal to the input direction)

    float3 detector_center;   // Center of the detector straight ahead of the focal spot.
    detector_center.x = source_data->position[0].x + source_data->direction[0].x * detector_data->sdd;
    detector_center.y = source_data->position[0].y + source_data->direction[0].y * detector_data->sdd;
    detector_center.z = source_data->position[0].z + source_data->direction[0].z * detector_data->sdd;
        
    if ((detector_data->sdd)<1.0e-6)
    {
      printf("\n\n   !!read_input ERROR!! The source-to-detector distance must be positive. Input: ssd=%f!!\n\n", detector_data->sdd);
      exit(-2);
    }

/* !!DeBuG!! OLD WRONG FAN BEAM:  
    if ( (source_data->tan_phi_semiaperture < -1.0e-5) || (source_data->tan_theta_semiaperture < -1.0e-5) )   // If we enter a negative angle, the fan beam will cover exactly the detector surface.
    {
      source_data->tan_phi_semiaperture   = 0.5 * detector_data->width_X  / (detector_data->sdd);  // Set the aperture to cover the whole detector exactly (detector is at distance ssd, move to 1cm).
      source_data->tan_theta_semiaperture = 0.5 * detector_data->height_Z / (detector_data->sdd);
    }
*/
  if ( (theta_aperture < -1.0e-7) || (phi_aperture < -1.0e-7) )   // If we enter a negative angle, the fan beam will cover exactly the detector surface.
  {
    theta_aperture= 2.0 * atan(0.5*detector_data->height_Z/(detector_data->sdd)) * RAD2DEG;   // Optimum angles
    phi_aperture  = 2.0 * atan(0.5*detector_data->width_X/(detector_data->sdd)) * RAD2DEG;

    source_data->cos_theta_low = (float)( cos((90.0 - 0.5*theta_aperture)*DEG2RAD) );
    source_data->D_cos_theta   = (float)( -2.0*source_data->cos_theta_low );      // Theta aperture is symetric above and below 90 deg
    source_data->phi_low       = (float)( (90.0 - 0.5*phi_aperture)*DEG2RAD );
    source_data->D_phi         = (float)( phi_aperture*DEG2RAD );
    source_data->max_height_at_y1cm = (float) ( tan(0.5*theta_aperture*DEG2RAD) ); 
  }

  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!read_input ERROR!! Input file is not readable or does not contain the string \'SECTION CT SCAN\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"SECTION CT")==NULL);  // Skip comments and empty lines until the section begins
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
  sscanf(new_line, "%d", num_projections);     // NUMBER OF PROJECTIONS (beam must be perpendicular to Z axis)
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
  sscanf(new_line, "%lf", D_angle);   // ANGLE BETWEEN PROJECTIONS [degrees] (360/num_projections for full CT)
  *D_angle = (*D_angle)*DEG2RAD;      // store the angle in radians

  // Calculate initial source angle:
  *initial_angle = acos((double)(source_data->direction[0].x));
  if (source_data->direction[0].y<0)
    *initial_angle = -(*initial_angle);     // Correct for the fact that positive and negative angles have the same ACOS
  if (*initial_angle<0.0)
    *initial_angle = (*initial_angle) + 2.0*PI;   // Make sure the angle is not negative, between [0,360) degrees.
  *initial_angle = (*initial_angle) - PI;   // Correct the fact that the source is opposite to the detector (180 degrees difference).
  if (*initial_angle<0.0)
    *initial_angle = (*initial_angle) + 2.0*PI;   // Make sure the initial angle is not negative, between [0,360) degrees..

  
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
  sscanf(new_line, "%lf %lf", angularROI_0, angularROI_1);   // ANGLES OF INTEREST (projections outside this interval will be skipped)
  if (*angularROI_0<-0.001 || *angularROI_1>360.001)
  {
    printf("\n\n   !!read_input ERROR!! The angles in the angular region of interest must be in the interval [0,360]. Input: %f, %f.\n\n", *angularROI_0, *angularROI_1);   // The reconstructed planes are always parallel to the XY plane.\n");
    exit(-2);
  }
  *angularROI_0 = (*angularROI_0 - 0.0001)*DEG2RAD;   // Store the angles of interest in radians, increasing a little the interval to avoid precission problems
  *angularROI_1 = (*angularROI_1 + 0.0001)*DEG2RAD;

  
  if (0 == (*num_projections))
    *num_projections = 1;      // Zero projections has the same effect as 1 projection (ie, no CT scan rotation). Negative values are allowed and the source rotates in opposite rotation.
  if ( (fabs(*num_projections) > 1) && (fabs(source_data->direction[0].z)>0.00001f) )
  {
    printf("\n\n   !!read_input ERROR!! Sorry, but currently we can only simulate CT scans when the source direction is perpendicular to the Z axis (ie, w=0).\n\n");   // The reconstructed planes are always parallel to the XY plane.\n");
    exit(-2);
  }
  

  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!read_input ERROR!! Input file is not readable or does not contain the string \'SECTION VOXELIZED GEOMETRY FILE\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"SECTION VOXEL")==NULL);   // Skip comments and empty lines until the section begins
  new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
  trim_name(new_line, file_name_voxels);   // VOXEL GEOMETRY FILE (penEasy 2008 format)

  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!read_input ERROR!! Input file is not readable or does not contain the string \'SECTION MATERIAL FILE LIST\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"SECTION MATERIAL")==NULL);   // Skip comments and empty lines until the section begins

  int i;
  for (i=0; i<MAX_MATERIALS; i++)
  {
    new_line_ptr = fgets_trimmed(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
      file_name_materials[i][0]='\n';   // The input file is allowed to finish without defining all the materials
    else
      trim_name(new_line, file_name_materials[i]);
  }
  // [Finish reading input file]

  /////////////////////////////////////////////////////////////////////////////

  // *** Set the rotation that will bring particles from the detector plane to +Y=(0,+1,0) through a rotation around X and around Z (counter-clock):
  double rotX, rotZ, cos_rX, cos_rZ, sin_rX, sin_rZ;
      // rotX = 1.5*PI - acos(source_data->direction.z);  // Rotate to +Y = (0,+1,0) --> rotX_0 = 3/2*PI == -PI/2
    rotX = acos(source_data->direction[0].z) - 0.5*PI;  // Rotate to +Y = (0,+1,0) --> rotX_0 =  -PI/2
      // rotX = 0.5*PI - acos(source_data->direction.z);  // Rotate to +Y = (0,+1,0) --> rotX_0 =  PI/2
  if ( (source_data->direction[0].x*source_data->direction[0].x + source_data->direction[0].y*source_data->direction[0].y) > 1.0e-8 )   // == u^2+v^2 > 0
  {
      // rotZ = 0.5*PI - acos(source_data->direction.x/sqrt(source_data->direction.x*source_data->direction.x + source_data->direction.y*source_data->direction.y));
    if (source_data->direction[0].y >= 0.0f)
      rotZ = 0.5*PI - acos(source_data->direction[0].x/sqrt(source_data->direction[0].x*source_data->direction[0].x + source_data->direction[0].y*source_data->direction[0].y));
    else
      rotZ = 0.5*PI - (-acos(source_data->direction[0].x/sqrt(source_data->direction[0].x*source_data->direction[0].x + source_data->direction[0].y*source_data->direction[0].y)));
  }
  else
    rotZ = 0.0;   // Vector pointing to +Z, do not rotate around Z then.
 
  // -- Set the rotation matrix RzRx (called inverse because moves from the correct position to the reference at +Y):
  cos_rX = cos(rotX);
  cos_rZ = cos(rotZ);
  sin_rX = sin(rotX);
  sin_rZ = sin(rotZ);

  // Rotation matrix RxRz:
  detector_data->rot_inv[0][0] =  cos_rZ;
  detector_data->rot_inv[0][1] = -sin_rZ;
  detector_data->rot_inv[0][2] =  0.0f;
  detector_data->rot_inv[0][3] =  cos_rX*sin_rZ;
  detector_data->rot_inv[0][4] =  cos_rX*cos_rZ;
  detector_data->rot_inv[0][5] = -sin_rX;
  detector_data->rot_inv[0][6] =  sin_rX*sin_rZ;
  detector_data->rot_inv[0][7] =  sin_rX*cos_rZ;
  detector_data->rot_inv[0][8] =  cos_rX;



  if ((source_data->direction[0].y > 0.99995f) && (*num_projections==1))
  {
    // Simulating a single projection and initial beam pointing to +Y: no rotation needed!!
    detector_data->rotation_flag = 0;
    detector_data->corner_min_rotated_to_Y[0].x = detector_center.x;
    detector_data->corner_min_rotated_to_Y[0].y = detector_center.y;
    detector_data->corner_min_rotated_to_Y[0].z = detector_center.z;

    MASTER_THREAD printf("       Source pointing to (0,1,0): maximizing code efficiency -> detector not rotated, initial location in voxels found faster.\n");  // -> the simulation will be faster than for other angles.");

  }
  else
  { // Rotation needed to set the detector perpendicular to +Y:
    detector_data->rotation_flag = 1;
    // -- Rotate the detector center to +Y:
    detector_data->corner_min_rotated_to_Y[0].x = detector_center.x*detector_data->rot_inv[0][0] + detector_center.y*detector_data->rot_inv[0][1] + detector_center.z*detector_data->rot_inv[0][2];
    detector_data->corner_min_rotated_to_Y[0].y = detector_center.x*detector_data->rot_inv[0][3] + detector_center.y*detector_data->rot_inv[0][4] + detector_center.z*detector_data->rot_inv[0][5];
    detector_data->corner_min_rotated_to_Y[0].z = detector_center.x*detector_data->rot_inv[0][6] + detector_center.y*detector_data->rot_inv[0][7] + detector_center.z*detector_data->rot_inv[0][8];

    MASTER_THREAD printf("       Rotations from the input direction to +Y [deg]: rotZ = %f , rotX = %f\n", rotZ*RAD2DEG, rotX*RAD2DEG);
    
  }
  // -- Set the lower corner (minimum) coordinates at the normalized orientation: +Y. The detector has thickness 0.
  detector_data->corner_min_rotated_to_Y[0].x = detector_data->corner_min_rotated_to_Y[0].x - 0.5*detector_data->width_X;
  detector_data->corner_min_rotated_to_Y[0].y = detector_data->corner_min_rotated_to_Y[0].y;
  detector_data->corner_min_rotated_to_Y[0].z = detector_data->corner_min_rotated_to_Y[0].z - 0.5*detector_data->height_Z;

  /////////////////////////////////////////////////////////////////////////////

  // *** Init the fan beam source model:

  if (1 == detector_data->rotation_flag)
  {
    // Initial beam NOT pointing to +Y: rotation is needed to move the sampled vector from (0,1,0) to the given direction!!
    rotX = 0.5*PI - acos(source_data->direction[0].z);         // ! Rotation about X: acos(wsrc)==theta, theta=90 for alpha=0, ie, +Y.
    rotZ = atan2(source_data->direction[0].y, source_data->direction[0].x) - 0.5*PI;   // ! Rotation about Z:  initial phi = 90 (+Y).  [ATAN2(v,u) = TAN(v/u), with the angle in the correct quadrant.
    cos_rX = cos(rotX);
    cos_rZ = cos(rotZ);
    sin_rX = sin(rotX);
    sin_rZ = sin(rotZ);
    // --Rotation around X (alpha) and then around Z (phi): Rz*Rx (oposite of detector rotation)
    source_data->rot_fan[0][0] =  cos_rZ;
    source_data->rot_fan[0][1] = -cos_rX*sin_rZ;
    source_data->rot_fan[0][2] =  sin_rX*sin_rZ;
    source_data->rot_fan[0][3] =  sin_rZ;
    source_data->rot_fan[0][4] =  cos_rX*cos_rZ;
    source_data->rot_fan[0][5] = -sin_rX*cos_rZ;
    source_data->rot_fan[0][6] =  0.0f;
    source_data->rot_fan[0][7] =  sin_rX;
    source_data->rot_fan[0][8] =  cos_rX;
    
    MASTER_THREAD printf("       Rotations from +Y to the input direction for the fan beam source model [deg]: rotZ = %f , rotX = %f\n", rotZ*RAD2DEG, rotX*RAD2DEG);
  }      

  /////////////////////////////////////////////////////////////////////////////

  // *** Allocate array for the 4 detected images (non-scattered, Compton, Rayleigh, multiple-scatter):
  *image_bytes = 4 * sizeof(unsigned long long int)*(detector_data->num_pixels.x)*(detector_data->num_pixels.y);
  (*image_ptr) = (unsigned long long int*) malloc(*image_bytes);
  if (*image_ptr==NULL)
  {
    printf("\n\n   !!malloc ERROR!! Not enough memory to allocate %d pixels for the 4 scatter images (%f Mbytes)!!\n\n", ((detector_data->num_pixels.x)*(detector_data->num_pixels.y)), (*image_bytes)/(1024.f*1024.f));
    exit(-2);
  }
  else
  {
    MASTER_THREAD printf("       Array for 4 scatter images correctly allocated (%f Mbytes)\n", (*image_bytes)/(1024.f*1024.f));
  }

  // *** Initialize the images to 0 in the CPU.
  register int j, pixels_per_image = (detector_data->num_pixels.x * detector_data->num_pixels.y);
  for (j=0; j<pixels_per_image; j++)
  {
    (*image_ptr)[j                   ] = (unsigned long long int)(0);   // Initialize non-scattered image
    (*image_ptr)[j+  pixels_per_image] = (unsigned long long int)(0);   // Initialize Compton image
    (*image_ptr)[j+2*pixels_per_image] = (unsigned long long int)(0);   // Initialize Rayleigh image
    (*image_ptr)[j+3*pixels_per_image] = (unsigned long long int)(0);   // Initialize multiple scattering image
  }
}



////////////////////////////////////////////////////////////////////////////////
//! Extract a file name from an input text line, trimming the initial blanks,
//! trailing comment (#) and stopping at the first blank (the file name should
//! not contain blanks).
//!
//!       @param[in] input_line   Input sentence with blanks and a trailing comment
//!       @param[out] file_name   Trimmed file name
////////////////////////////////////////////////////////////////////////////////
void trim_name(char* input_line, char* file_name)
{
  int a=0, b=0;
  
  // Discard initial blanks:
  while(' '==input_line[a])
  {
    a++;
  }

  // Read file name until a blank or a comment symbol (#) is found:
  while ((' '!=input_line[a])&&('#'!=input_line[a]))
  {
    file_name[b] = input_line[a];
    b++;
    a++;
  }
  
  file_name[b] = '\0';    // Terminate output string
}

////////////////////////////////////////////////////////////////////////////////
//! Read a line of text and trim initial blancks and trailing comments (#).
//!
//!       @param[in] num   Characters to read
//!       @param[in] file_ptr   Pointer to the input file stream
//!       @param[out] trimmed_line   Trimmed line from input file, skipping empty lines and comments
////////////////////////////////////////////////////////////////////////////////
char* fgets_trimmed(char* trimmed_line, int num, FILE* file_ptr)
{
  char  new_line[250];
  char *new_line_ptr = NULL;
  int a=0, b=0;
  trimmed_line[0] = '\0';   //  Init with a mark that means no file input
  
  do
  {
    a=0; b=0;
    new_line_ptr = fgets(new_line, num, file_ptr);   // Read new line
    if (new_line_ptr != NULL)
    {
      // Discard initial blanks:
      while(' '==new_line[a])
      {
        a++;
      }
      // Read file until a comment symbol (#) or end-of-line are found:
      while (('\n'!=new_line[a])&&('#'!=new_line[a]))
      {
        trimmed_line[b] = new_line[a];
        b++;
        a++;
      }
    }
  } while(new_line_ptr!=NULL &&  '\0'==trimmed_line[0]);   // Keep reading lines until end-of-file or a line that is not empty or only comment is found
  
  trimmed_line[b] = '\0';    // Terminate output string
  return new_line_ptr;
}



////////////////////////////////////////////////////////////////////////////////
//! Read the voxel data and allocate the material and density matrix.
//! Also find and report the maximum density defined in the geometry.
//!
// -- Sample voxel geometry file:
//
//   #  (comment lines...)
//   #
//   #   Voxel order: X runs first, then Y, then Z.
//   #
//   [SECTION VOXELS HEADER v.2008-04-13]
//   411  190  113      No. OF VOXELS IN X,Y,Z
//   5.000e-02  5.000e-02  5.000e-02    VOXEL SIZE (cm) ALONG X,Y,Z
//   1                  COLUMN NUMBER WHERE MATERIAL ID IS LOCATED
//   2                  COLUMN NUMBER WHERE THE MASS DENSITY IS LOCATED
//   1                  BLANK LINES AT END OF X,Y-CYCLES (1=YES,0=NO)
//   [END OF VXH SECTION]
//   1 0.00120479
//   1 0.00120479
//   ...
//
//!       @param[in] file_name_voxels  Name of the voxelized geometry file.
//!       @param[out] density_max  Array with the maximum density for each material in the voxels.
//!       @param[out] voxel_data   Pointer to a structure containing the voxel number and size.
//!       @param[out] voxel_mat_dens_ptr   Pointer to the vector with the voxel materials and densities.
////////////////////////////////////////////////////////////////////////////////
void load_voxels(int myID, char* file_name_voxels, float* density_max, struct voxel_struct* voxel_data, float2** voxel_mat_dens_ptr, unsigned int* voxel_mat_dens_bytes)
{
  char new_line[250];
  char *new_line_ptr = NULL;
  
  FILE* file_ptr = fopen(file_name_voxels, "r");
  if (file_ptr==NULL)
  {
    printf("\n\n   !!fopen ERROR!! File %s does not exist!!\n", file_name_voxels);
    exit(-2);
  }
  MASTER_THREAD printf("\n    -- Reading voxel file \'%s\':\n",file_name_voxels);
  MASTER_THREAD fflush(stdout);
  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!Reading ERROR!! File is not readable or does not contain the string \'[SECTION VOXELS HEADER\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"[SECTION VOXELS")==NULL);   // Skip comments and empty lines until the header begins

  new_line_ptr = fgets(new_line, 250, file_ptr);   // Read full line (max. 250 characters).
  sscanf(new_line, "%d %d %d",&voxel_data->num_voxels.x, &voxel_data->num_voxels.y, &voxel_data->num_voxels.z);
  new_line_ptr = fgets(new_line, 250, file_ptr);
  sscanf(new_line, "%f %f %f", &voxel_data->inv_voxel_size.x, &voxel_data->inv_voxel_size.y, &voxel_data->inv_voxel_size.z);
  do
  {
    new_line_ptr = fgets(new_line, 250, file_ptr);
    if (new_line_ptr==NULL)
    {
      printf("\n\n   !!Reading ERROR!! File is not readable or does not contain the string \'[END OF VXH SECTION]\'!!\n");
      exit(-2);
    }
  }
  while(strstr(new_line,"[END OF VXH SECTION")==NULL);   // Skip rest of the header

  // -- Store the size of the voxel bounding box (used in the source function):
  voxel_data->size_bbox.x = voxel_data->num_voxels.x * voxel_data->inv_voxel_size.x;
  voxel_data->size_bbox.y = voxel_data->num_voxels.y * voxel_data->inv_voxel_size.y;
  voxel_data->size_bbox.z = voxel_data->num_voxels.z * voxel_data->inv_voxel_size.z;

  
  MASTER_THREAD printf("       Number of voxels in the input geometry file: %d x %d x %d =  %d\n", voxel_data->num_voxels.x, voxel_data->num_voxels.y, voxel_data->num_voxels.z, (voxel_data->num_voxels.x*voxel_data->num_voxels.y*voxel_data->num_voxels.z));
  MASTER_THREAD printf("       Size of the input voxels: %f x %f x %f cm\n", voxel_data->inv_voxel_size.x, voxel_data->inv_voxel_size.y, voxel_data->inv_voxel_size.z);
  MASTER_THREAD printf("       Voxel bounding box size: %f x %f x %f cm\n", voxel_data->size_bbox.x, voxel_data->size_bbox.y,  voxel_data->size_bbox.z);
  // printf("       The geometry must be given in two columns, with the voxel density in the second column.\n");
  // printf("       The  X,Y-cycles may, or may not, be separated by blank lines.\n");

  
  // -- Store the inverse of the pixel sides (in cm) to speed up the particle location in voxels.
  voxel_data->inv_voxel_size.x = 1.0f/(voxel_data->inv_voxel_size.x);
  voxel_data->inv_voxel_size.y = 1.0f/(voxel_data->inv_voxel_size.y);
  voxel_data->inv_voxel_size.z = 1.0f/(voxel_data->inv_voxel_size.z);

  // -- Allocate the voxel matrix and store array size:
  *voxel_mat_dens_bytes = sizeof(float2)*(voxel_data->num_voxels.x)*(voxel_data->num_voxels.y)*(voxel_data->num_voxels.z);
  *voxel_mat_dens_ptr    = (float2*) malloc(*voxel_mat_dens_bytes);
  if (*voxel_mat_dens_ptr==NULL)
  {
    printf("\n\n   !!malloc ERROR!! Not enough memory to allocate %d voxels (%f Mbytes)!!\n\n", (voxel_data->num_voxels.x*voxel_data->num_voxels.y*voxel_data->num_voxels.z), (*voxel_mat_dens_bytes)/(1024.f*1024.f));
    exit(-2);
  }
  MASTER_THREAD printf("\n    -- Initializing the voxel material and density vector (%f Mbytes)\n", (*voxel_mat_dens_bytes)/(1024.f*1024.f));

  // -- Read the voxel densities:
  MASTER_THREAD printf("       Reading the voxel densities... ");
      
  int i, j, k, read_lines=0, dummy_material, read_items;
  float dummy_density;
  float2 *voxels_ptr = *voxel_mat_dens_ptr;

  for (k=0; k<MAX_MATERIALS; k++)
    density_max[k] = -999.0f;   // Init array with an impossible low density value
  

  for(k=0; k<(voxel_data->num_voxels.z); k++)
  {
    for(j=0; j<(voxel_data->num_voxels.y); j++)
    {
      for(i=0; i<(voxel_data->num_voxels.x); i++)
      {
        read_items = fscanf(file_ptr,"%d %f", &dummy_material, &dummy_density);    // Read the next 2 numbers
        if (read_items!=2)
          printf("\n   !!WARNING!! Expecting to read 2 items (material and density). read_items=%d, read_lines=%d \n", read_items, read_lines);
          
        new_line_ptr = fgets(new_line, 250, file_ptr);                             // Continue reading until end-of-line

        if (dummy_material>MAX_MATERIALS)
        {
          printf("\n\n   !!ERROR!! Voxel material number too high: #mat=%d, MAX_MATERIALS=%d\n\n", dummy_material, MAX_MATERIALS);
          exit(-2);
        }
        
        if (dummy_density > density_max[dummy_material-1])
          density_max[dummy_material-1] = dummy_density;  // Store maximum density for each material

        (*voxels_ptr).x = (float)(dummy_material)+0.0001f;  // Assign material value as float (the integer value will be recovered by truncation)
        (*voxels_ptr).y = dummy_density;      // Assign density value
        voxels_ptr++;                         // Move to next voxel

        read_lines++;
      }
    }
  }
  MASTER_THREAD printf("Total number of voxels read: %d\n",read_lines);
  fclose(file_ptr);     // Close input file
}


////////////////////////////////////////////////////////////////////////////////
//! Read the material input files and set the mean free paths and the "linear_interp" structures.
//! Find the material nominal density. Set the Woodcock trick data.
//
// -- Sample material data file (data obtained from the PENELOPE 2006 database and models):
//
//    [MATERIAL NAME]
//     Water
//    [NOMINAL DENSITY (g/cm^3)]
//     1.000
//    [NUMBER OF DATA VALUES]
//     4096
//    [MEAN FREE PATHS :: Energy (eV) || Rayleigh | Compton | Photoelectric | Pair-production | TOTAL (cm)]
//     1.00000E+03  7.27451E-01  9.43363E+01  2.45451E-04  1.00000E+35  2.45367E-04
//     5.00000E+03  1.80004E+00  8.35996E+00  2.38881E-02  1.00000E+35  2.35089E-02
//     1.00000E+04  4.34941E+00  6.26746E+00  2.02568E-01  1.00000E+35  1.87755E-01
//     ...
//     #[RAYLEIGH INTERACTIONS (RITA sampling  of atomic form factor from EPDL database)]
//     ...
//     #[COMPTON INTERACTIONS (relativistic impulse model with approximated one-electron analytical profiles)]
//     ...
//
//!       @param[in] file_name_materials    Array with the names of the material files.
//!       @param[in] density_max   maximum density in the geometry (needed to set Woodcock trick)
//!       @param[out] density_nominal   Array with the nominal density of the materials read
//!       @param[out] mfp_table_data   Constant values for the linear interpolation
//!       @param[out] mfp_table_a_ptr   First element for the linear interpolation.
//!       @param[out] mfp_table_b_ptr   Second element for the linear interpolation.
////////////////////////////////////////////////////////////////////////////////
void load_material(int myID, char file_name_materials[MAX_MATERIALS][250], float* density_max, float* density_nominal, struct linear_interp* mfp_table_data, float2** mfp_Woodcock_table_ptr, int* mfp_Woodcock_table_bytes, float3** mfp_table_a_ptr, float3** mfp_table_b_ptr, int* mfp_table_bytes, struct rayleigh_struct *rayleigh_table_ptr, struct compton_struct *compton_table_ptr)
{
  char new_line[250];
  char *new_line_ptr = NULL;
  int mat, i, bin, input_num_values = 0, input_rayleigh_values = 0, input_num_shells = 0;
  double delta_e=-99999.0;

  // -- Init the number of shells to 0 for all materials
  for (mat=0; mat<MAX_MATERIALS; mat++)
    compton_table_ptr->noscco[mat] = 0;
    
          
  // --Read the material data files:
  MASTER_THREAD printf("\n    -- Reading the material data files (MAX_MATERIALS=%d):\n", MAX_MATERIALS);
  for (mat=0; mat<MAX_MATERIALS; mat++)
  {
    if ((file_name_materials[mat][0]=='\0') || (file_name_materials[mat][0]=='\n') || (density_max[mat]<0))  //  Empty file name, or material not found in the voxels
       continue;   // Re-start loop for next material

    MASTER_THREAD printf("         Mat %d: File \'%s\'\n", mat, file_name_materials[mat]);
//     printf("    -- Reading material file #%d: \'%s\'\n", mat, file_name_materials[mat]);

    FILE* file_ptr = fopen(file_name_materials[mat], "r");
    if (file_ptr==NULL)
    {
      printf("\n\n   !!fopen ERROR!! File %d \'%s\' does not exist!!\n", mat, file_name_materials[mat]);
      exit(-2);
    }
    do
    {
      new_line_ptr = fgets(new_line, 250, file_ptr);   // Read full line (max. 250 characters).
      if (new_line_ptr==NULL)
      {
        printf("\n\n   !!Reading ERROR!! File is not readable or does not contain the string \'[NOMINAL DENSITY\'!!\n");
        exit(-2);
      }
    }
    while(strstr(new_line,"[NOMINAL DENSITY")==NULL);   // Skip rest of the header

    // Read the material nominal density:
    new_line_ptr = fgets(new_line, 250, file_ptr);
    sscanf(new_line, "# %f", &density_nominal[mat]);
    MASTER_THREAD printf("                Nominal density = %f g/cm^3; Max mat density in voxels = %f\n", density_nominal[mat], density_max[mat]);

    // --For the first material, set the number of energy values and allocate table arrays:
    new_line_ptr = fgets(new_line, 250, file_ptr);
    new_line_ptr = fgets(new_line, 250, file_ptr);
    sscanf(new_line, "# %d", &input_num_values);
    if (0==mat)
    {
      mfp_table_data->num_values = input_num_values;
      MASTER_THREAD printf("                Number of energy values in the mean free path database: %d.\n", input_num_values);

      // Allocate memory for the linear interpolation arrays:
      *mfp_Woodcock_table_bytes = sizeof(float2)*input_num_values;
      *mfp_Woodcock_table_ptr   = (float2*) malloc(*mfp_Woodcock_table_bytes);  // Allocate space for the 2 parameter table
      *mfp_table_bytes = sizeof(float3)*input_num_values*MAX_MATERIALS;
      *mfp_table_a_ptr = (float3*) malloc(*mfp_table_bytes);  // Allocate space for the 4 MFP tables
      *mfp_table_b_ptr = (float3*) malloc(*mfp_table_bytes);
      *mfp_table_bytes = sizeof(float3)*input_num_values*MAX_MATERIALS;

      if (input_num_values>MAX_ENERGYBINS)
      {
        printf("\n\n   !!load_material ERROR!! Too many energy bins (Input bins=%d): increase parameter MAX_ENERGYBINS=%d!!\n\n", input_num_values, MAX_ENERGYBINS);
        exit(-2);
      }
      
      if ((NULL==*mfp_Woodcock_table_ptr)||(NULL==*mfp_table_a_ptr)||(NULL==*mfp_table_b_ptr))
      {
        printf("\n\n   !!malloc ERROR!! Not enough memory to allocate the linear interpolation data: %d bytes!!\n\n", (*mfp_Woodcock_table_bytes+2*(*mfp_table_bytes)));
        exit(-2);
      }
      else
      {
        MASTER_THREAD printf("                Linear interpolation data correctly allocated (%f Mbytes)\n", (*mfp_Woodcock_table_bytes+2*(*mfp_table_bytes))/(1024.f*1024.f));
      }
      for (i=0; i<input_num_values; i++)
      {
        (*mfp_Woodcock_table_ptr)[i].x = 99999999.99f;    // Init this array with a huge MFP, the minimum values are calculated below
      }
    }
    else   // Materials after first
    {
      if (input_num_values != mfp_table_data->num_values)
      {
        printf("\n\n   !!load_material ERROR!! Incorrect number of energy values given in material \'%s\': input=%d, expected=%d\n",file_name_materials[mat], input_num_values, mfp_table_data->num_values);
        exit(-2);
      }
    }

    // -- Read the mean free paths (and Rayleigh cumulative prob):
    new_line_ptr = fgets(new_line, 250, file_ptr);
    new_line_ptr = fgets(new_line, 250, file_ptr);
    double d_energy, d_rayleigh, d_compton, d_photelectric, d_total_mfp, d_pmax, e_last=-1.0;
    
    for (i=0; i<input_num_values; i++)
    {

      new_line_ptr = fgets(new_line, 250, file_ptr);
      sscanf(new_line,"  %le  %le  %le  %le  %le  %le", &d_energy, &d_rayleigh, &d_compton, &d_photelectric, &d_total_mfp, &d_pmax);

      // Find and store the minimum total MFP at the current energy, for every material's maximum density:
      float temp_mfp = d_total_mfp*(density_nominal[mat])/(density_max[mat]);
      if (temp_mfp < (*mfp_Woodcock_table_ptr)[i].x)
        (*mfp_Woodcock_table_ptr)[i].x = temp_mfp;       // Store minimum total mfp [cm]

      // Store the inverse MFP data points with [num_values rows]*[MAX_MATERIALS columns]
      (*mfp_table_a_ptr)[i*(MAX_MATERIALS)+mat].x = 1.0/(d_total_mfp*density_nominal[mat]);   // inverse TOTAL mfp * nominal density
      (*mfp_table_a_ptr)[i*(MAX_MATERIALS)+mat].y = 1.0/(d_compton  *density_nominal[mat]);   // inverse Compton mfp * nominal density
      (*mfp_table_a_ptr)[i*(MAX_MATERIALS)+mat].z = 1.0/(d_rayleigh *density_nominal[mat]);   // inverse Rayleigh mfp * nominal density

      rayleigh_table_ptr->pmax[i*(MAX_MATERIALS)+mat] = d_pmax;    // Store the maximum cumulative probability of atomic form factor F^2 for

      if (0==i && 0==mat)
      {
        mfp_table_data->e0  = d_energy;   // Store the first energy of the first material
      }

      if (0==i)
      {
        if (fabs(d_energy-mfp_table_data->e0)>1.0e-9)
        {
          printf("\n\n   !!load_material ERROR!! Incorrect first energy value given in material \'%s\': input=%f, expected=%f\n", file_name_materials[mat], d_energy, mfp_table_data->e0);
          exit(-2);
        }
      }
      else if (1==i)
      {
        delta_e = d_energy-e_last;
      }
      else if (i>1)
      {
        if (((fabs((d_energy-e_last)-delta_e))/delta_e)>0.001)  // Tolerate up to a 0.1% relative variation in the delta e (for each bin) to account for possible precission errors reading the energy values
        {
          printf("  !!ERROR reading material data!! The energy step between mean free path values is not constant!!\n      (maybe not enough decimals given for the energy values)\n      #value = %d, First delta: %f , New delta: %f, Energy: %f ; Rel.Dif=%f\n", i, delta_e, (d_energy-e_last), d_energy,((fabs((d_energy-e_last)-delta_e))/delta_e));
          exit(-2);
        }
      }
      e_last = d_energy;
    }
    
    if (0==mat) MASTER_THREAD printf("                Minimum energy = %f, Maximum = %f; Delta E (1st bin) = %f\n", (mfp_table_data->e0), e_last, delta_e);

    // -- Store the inverse of delta energy:
    mfp_table_data->ide = 1.0f/delta_e;

    // -- Store MFP data slope 'b' (.y for Woodcock):
    for (i=0; i<(input_num_values-1); i++)
    {
      bin = i*MAX_MATERIALS+mat;                   // Set current bin, skipping MAX_MATERIALS columns
      (*mfp_table_b_ptr)[bin].x = ((*mfp_table_a_ptr)[bin+MAX_MATERIALS].x - (*mfp_table_a_ptr)[bin].x) / delta_e;
      (*mfp_table_b_ptr)[bin].y = ((*mfp_table_a_ptr)[bin+MAX_MATERIALS].y - (*mfp_table_a_ptr)[bin].y) / delta_e;
      (*mfp_table_b_ptr)[bin].z = ((*mfp_table_a_ptr)[bin+MAX_MATERIALS].z - (*mfp_table_a_ptr)[bin].z) / delta_e;
    }
    // After maximum energy (last bin), assume constant slope:
    (*mfp_table_b_ptr)[(input_num_values-1)*MAX_MATERIALS+mat] = (*mfp_table_b_ptr)[(input_num_values-2)*MAX_MATERIALS+mat];

    // -- Rescale the 'a' parameter (.x for Woodcock) as if the bin started at energy = 0: we will not have to rescale to the bin minimum energy every time
    for (i=0; i<input_num_values; i++)
    {
      d_energy = mfp_table_data->e0 + i*delta_e;   // Set current bin lowest energy value
      bin = i*MAX_MATERIALS+mat;                   // Set current bin, skipping MAX_MATERIALS columns
      (*mfp_table_a_ptr)[bin].x = (*mfp_table_a_ptr)[bin].x - d_energy*(*mfp_table_b_ptr)[bin].x;
      (*mfp_table_a_ptr)[bin].y = (*mfp_table_a_ptr)[bin].y - d_energy*(*mfp_table_b_ptr)[bin].y;
      (*mfp_table_a_ptr)[bin].z = (*mfp_table_a_ptr)[bin].z - d_energy*(*mfp_table_b_ptr)[bin].z;
    }

    // -- Reading data for RAYLEIGH INTERACTIONS (RITA sampling  of atomic form factor from EPDL database):
    do
    {
      new_line_ptr = fgets(new_line, 250, file_ptr);
      if (feof(file_ptr)!=0)
      {
        printf("\n\n   !!End-of-file ERROR!! Rayleigh data not found: \"#[DATA VALUES...\" in file \'%s\'. Last line read: %s\n\n", file_name_materials[mat], new_line);
        exit(-2);
      }
    }
    while(strstr(new_line,"[DATA VALUES")==NULL);   // Skip all lines until this text is found
      
    new_line_ptr = fgets(new_line, 250, file_ptr);     // Read the number of data points in Rayleigh
    sscanf(new_line, "# %d", &input_rayleigh_values);
        
    if (input_rayleigh_values != NP_RAYLEIGH)
    {
      printf("\n\n   !!ERROR!! The number of values for Rayleigh sampling is different than the allocated space: input=%d, NP_RAYLEIGH=%d. File=\'%s\'\n", input_rayleigh_values, NP_RAYLEIGH, file_name_materials[mat]);
      exit(-2);
    }
    new_line_ptr = fgets(new_line, 250, file_ptr);    // Comment line:  #[SAMPLING DATA FROM COMMON/CGRA/: X, P, A, B, ITL, ITU]
    for (i=0; i<input_rayleigh_values; i++)
    {
      int itlco_tmp, ituco_tmp;
      bin = NP_RAYLEIGH*mat + i;

      new_line_ptr = fgets(new_line, 250, file_ptr);
      sscanf(new_line,"  %e  %e  %e  %e  %d  %d", &(rayleigh_table_ptr->xco[bin]), &(rayleigh_table_ptr->pco[bin]),
                                                  &(rayleigh_table_ptr->aco[bin]), &(rayleigh_table_ptr->bco[bin]),
                                                  &itlco_tmp, &ituco_tmp);

      rayleigh_table_ptr->itlco[bin] = (unsigned char) itlco_tmp;
      rayleigh_table_ptr->ituco[bin] = (unsigned char) ituco_tmp;
                                                  
    }
    //  printf("    -- Rayleigh sampling data read. Input values = %d\n",input_rayleigh_values);

    // -- Reading COMPTON INTERACTIONS data (relativistic impulse model with approximated one-electron analytical profiles):
    do
    {
      new_line_ptr = fgets(new_line, 250, file_ptr);
      if (feof(file_ptr)!=0)
      {
        printf("\n\n   !!End-of-file ERROR!! Compton data not found: \"[NUMBER OF SHELLS]\" in file \'%s\'. Last line read: %s\n\n", file_name_materials[mat], new_line);
        exit(-2);
      }
    }
    while(strstr(new_line,"[NUMBER OF SHELLS")==NULL);   // Skip all lines until this text is found
    new_line_ptr = fgets(new_line, 250, file_ptr);
    sscanf(new_line, "# %d", &input_num_shells);      // Read the NUMBER OF SHELLS
    if (input_num_shells>MAX_SHELLS)
    {
      printf("\n\n   !!ERROR!! Too many shells for Compton interactions in file \'%s\': input=%d, MAX_SHELLS=%d\n", file_name_materials[mat], input_num_shells, MAX_SHELLS);
      exit(-2);
    }
    compton_table_ptr->noscco[mat] = input_num_shells;   // Store number of shells for this material in structure
    new_line_ptr = fgets(new_line, 250, file_ptr);    // Comment line:  #[SHELL INFORMATION FROM COMMON/CGCO/: FCO, UICO, FJ0, KZCO, KSCO]
    int kzco_dummy, ksco_dummy;
    for (i=0; i<input_num_shells; i++)
    {

      bin = mat + i*MAX_MATERIALS;

      new_line_ptr = fgets(new_line, 250, file_ptr);
      sscanf(new_line," %e  %e  %e  %d  %d", &(compton_table_ptr->fco[bin]), &(compton_table_ptr->uico[bin]),
                                              &(compton_table_ptr->fj0[bin]), &kzco_dummy, &ksco_dummy);
    }
  
    fclose(file_ptr);    // Material data read. Close the current material input file
    
  }  // ["for" loop: continue with next material]


  // -- Store Woodcock MFP slope in component '.y':
  for (i=0; i<(mfp_table_data->num_values-1); i++)
    (*mfp_Woodcock_table_ptr)[i].y = ((*mfp_Woodcock_table_ptr)[i+1].x - (*mfp_Woodcock_table_ptr)[i].x)/delta_e;

  // -- Rescale the first parameter in component .x for Woodcock
  for (i=0; i<mfp_table_data->num_values; i++)
  {
    (*mfp_Woodcock_table_ptr)[i].x = (*mfp_Woodcock_table_ptr)[i].x - (mfp_table_data->e0 + i*delta_e)*(*mfp_Woodcock_table_ptr)[i].y;
  }
  
}
////////////////////////////////////////////////////////////////////////////////



#ifdef USING_CUDA
////////////////////////////////////////////////////////////////////////////////
//!  Sets the CUDA enabled GPU that will be used in the simulation.
//!  Allocates and copies the simulation data in the GPU global and constant memories.
//!
////////////////////////////////////////////////////////////////////////////////
void init_CUDA_device( int* gpu_id, int myID,
      /*Variables to GPU constant memory:*/ struct voxel_struct* voxel_data, struct source_struct* source_data, struct detector_struct* detector_data, struct linear_interp* mfp_table_data,
      /*Variables to GPU global memory:*/ float2* voxel_mat_dens, float2** voxel_mat_dens_device, unsigned int voxel_mat_dens_bytes,
        unsigned long long int* image, unsigned long long int** image_device, int image_bytes,
        float2* mfp_Woodcock_table, float2** mfp_Woodcock_table_device, int mfp_Woodcock_table_bytes,
        float3* mfp_table_a, float3* mfp_table_b, float3** mfp_table_a_device, float3** mfp_table_b_device, int mfp_table_bytes,
        struct rayleigh_struct* rayleigh_table, struct rayleigh_struct** rayleigh_table_device,
        struct compton_struct* compton_table, struct compton_struct** compton_table_device )
{
  int deviceCount, coresPerSM;
  cutilSafeCall(hipGetDeviceCount(&deviceCount));
  if (0==deviceCount)
  {
    printf("\n  !!ERROR!! No CUDA enabled GPU detected!!\n\n");
    exit(-1);
  }
  if ( ((*gpu_id)>(deviceCount-1)) || ((*gpu_id)<0) )
  {
    printf("\n  !!WARNING!! The input GPU number is not valid: input_GPU=%d, maximum GPU number=%d.\n", (*gpu_id), deviceCount-1);
    // printf("              The device with the maximum Gflop count will be automatically selected.\n");
    // (*gpu_id) = cutGetMaxGflopsDeviceId();
    exit(-3);
  }

  cutilSafeCall(hipSetDevice(*gpu_id));   // Set the GPU device.

  hipDeviceProp_t deviceProp;
  cutilSafeCall(hipGetDeviceProperties(&deviceProp, *gpu_id));
  if (deviceProp.major>99 || deviceProp.minor>99)
  {
    printf("\n  !!ERROR!! The selected GPU device does not support CUDA!! GPU_id=%d, deviceCount=%d, compute capability=%d.%d\n\n", (*gpu_id), deviceCount, deviceProp.major,deviceProp.minor);
    exit(-1);
  }

  if (deviceProp.major>1)
  {
    coresPerSM = 32;   // Set number of cores per microprocessor for Fermi GPUs (compute capability 2.x)
    
#ifdef LARGE_CACHE  
    // -- Compute capability > 1: set a large L1 cache for the global memory, reducing the size of the shared memory:
    //       hipFuncCachePreferShared: shared memory is 48 KB
    //       hipFuncCachePreferL1: shared memory is 16 KB
    //       hipFuncCachePreferNone: no preference
    printf("\n       ==> CUDA: LARGE_CACHE defined --> setting a large global memory cache (L1) and a small shared memory (hipFuncCachePreferL1).\n");
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(track_particles), hipFuncCachePreferL1);            // !!DeBuG!! Set a large cache instead of a large shared memory.
// #else
// -- Using default:
// printf("\n       ==> CUDA: LARGE_CACHE not defined --> setting a large shared memory and a small global memory cache (hipFuncCachePreferShared).\n");
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(track_particles), hipFuncCachePreferShared);            //!!DeBuG!! Setting size of shared memory/global cache
#endif

  }
  else
    coresPerSM = 8;   // Set number of cores per microprocessor before Fermi (compute capability 1.x)

  // -- Reading the device properties as shown in NVIDIA's SDK sample code "deviceQuery"
  printf("\n       ==> CUDA: %d CUDA enabled GPU detected! Using device #%d: \"%s\"\n", deviceCount, (*gpu_id), deviceProp.name);    
  printf("                 Compute capability: %d.%d, Number multiprocessors: %d, Number cores: %d\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount, coresPerSM*deviceProp.multiProcessorCount);
  printf("                 Clock rate: %.2f GHz, Global memory: %.3f Mbyte, Constant memory: %.2f kbyte\n", deviceProp.clockRate*1.0e-6f, deviceProp.totalGlobalMem/(1024.f*1024.f), deviceProp.totalConstMem/1024.f);
  printf("                 Shared memory per block: %.2f kbyte, Registers per block: %.2f kbyte\n", deviceProp.sharedMemPerBlock/1024.f, deviceProp.regsPerBlock/1024.f);
  int driverVersion = 0, runtimeVersion = 0;  
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("                 CUDA Driver Version: %d.%d, Runtime Version: %d.%d\n", driverVersion/1000, driverVersion%100, runtimeVersion/1000, runtimeVersion%100);

  if (0!=deviceProp.kernelExecTimeoutEnabled)
  {
    printf("\n   !!ERROR!! The selected GPU is connected to an X-Win server and the kernel run time is limited to 5 sec. Aborting execution!!");
    exit(-1);
  }    

  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));

  // -- Allocate the constant variables in the device:
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("voxel_data_CONST"),     voxel_data,     sizeof(struct voxel_struct)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("source_data_CONST"),    source_data,    sizeof(struct source_struct)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("detector_data_CONST"),  detector_data,  sizeof(struct detector_struct)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("mfp_table_data_CONST"), mfp_table_data, sizeof(struct linear_interp)));

  double total_mem = sizeof(struct voxel_struct)+sizeof(struct source_struct)+sizeof(struct detector_struct)+sizeof(struct linear_interp);
  MASTER_THREAD printf("       ==> CUDA: Structures successfully copied to the device. CONSTANT memory used: %lf kbytes (%.1lf%%)\n", total_mem/1024.0, 100.0*total_mem/deviceProp.totalConstMem);

  // -- Allocate the device memory:
  cutilSafeCall(hipMalloc((void**) voxel_mat_dens_device, voxel_mat_dens_bytes));
  cutilSafeCall(hipMalloc((void**) image_device, image_bytes));
  cutilSafeCall(hipMalloc((void**) mfp_Woodcock_table_device, mfp_Woodcock_table_bytes));
  cutilSafeCall(hipMalloc((void**) mfp_table_a_device, mfp_table_bytes));
  cutilSafeCall(hipMalloc((void**) mfp_table_b_device, mfp_table_bytes));
  cutilSafeCall(hipMalloc((void**) rayleigh_table_device, sizeof(struct rayleigh_struct)));
  cutilSafeCall(hipMalloc((void**) compton_table_device,  sizeof(struct compton_struct)));

  total_mem = voxel_mat_dens_bytes+image_bytes+mfp_Woodcock_table_bytes+2*mfp_table_bytes+sizeof(struct compton_struct)+sizeof(struct rayleigh_struct);
  if (*voxel_mat_dens_device==NULL || *image_device==NULL || *mfp_Woodcock_table_device==NULL ||
      *mfp_table_a_device==NULL || *mfp_table_a_device==NULL || *rayleigh_table_device==NULL || *compton_table_device==NULL)
  {
    printf("\n hipMalloc ERROR!! Device global memory not correctly allocated!! (%lf Mbytes)\n", total_mem/(1024.0*1024.0));
    exit(-1);
  }
  else
  {
    MASTER_THREAD printf("       ==> CUDA: Device global memory correctly allocated. GLOBAL memory used: %lf Mbytes (%.1lf%%)\n", total_mem/(1024.0*1024.0), 100.0*total_mem/deviceProp.totalGlobalMem);
  }

  // --Copy the host memory to the device:
  cutilSafeCall(hipMemcpy(*voxel_mat_dens_device, voxel_mat_dens, voxel_mat_dens_bytes, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(*mfp_Woodcock_table_device, mfp_Woodcock_table, mfp_Woodcock_table_bytes, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(*mfp_table_a_device, mfp_table_a, mfp_table_bytes, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(*mfp_table_b_device, mfp_table_b, mfp_table_bytes, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(*rayleigh_table_device, rayleigh_table, sizeof(struct rayleigh_struct), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(*compton_table_device, compton_table, sizeof(struct compton_struct), hipMemcpyHostToDevice));


  //   --Init the image array to 0 using a GPU kernel instead of hipMemcpy:
  //     Simple version: cutilSafeCall( hipMemcpy( image_device, image, image_bytes, hipMemcpyHostToDevice) );

  int pixels_per_image = detector_data->num_pixels.x * detector_data->num_pixels.y;
  MASTER_THREAD printf("       ==> CUDA: Launching kernel to initialize device image to 0: number of blocks = %d, threads per block = 128\n", (int)ceil(pixels_per_image/128.0f) );

  init_image_array_GPU<<<(int)(ceil(pixels_per_image/128.0f)),128>>>(*image_device, pixels_per_image);
    hipDeviceSynchronize();      // Force the runtime to wait until all device tasks have completed
    cutilCheckMsg(" !!Kernel execution failed initializing the image array!! ");  // Check if kernel execution generated any error:

  cutilCheckError(cutStopTimer(timer));
  MASTER_THREAD printf("                 Time spent allocating and copying memory to the device: %.6f s\n", 0.001f*cutGetTimerValue( timer));
  cutilCheckError(cutDeleteTimer(timer));

}
#endif
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
//! Report the final results, from the host CPU.
//!
//!       @param[in] file_name_output   File where tallied image is reported
//!       @param[in] detector_data   Detector description read from the input file (pointer to detector_struct)
//!       @param[in] image  Tallied image (in meV per pixel)
//!       @param[in] time_elapsed   Time elapsed during the main loop execution (in seconds)
//!       @param[in] total_histories   Total number of x-rays simulated
////////////////////////////////////////////////////////////////////////////////
int report_host(char* file_name_output, struct detector_struct* detector_data, struct source_struct* source_data, unsigned long long int* image, double time_elapsed, unsigned long long int total_histories, int current_projection, int num_projections, double D_angle, double initial_angle, int myID, int numprocs)
{
    // printf("\n           ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");
    // printf("           ~~ Simulation performance quick report ~~\n");
    // printf("           ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n");


  //  -Find current angle
  double current_angle = initial_angle+current_projection*D_angle;
    if (current_angle<0.0)
    current_angle += 2.0*PI;   // Make sure the angle is not negative.
  else if (current_angle>=2.0*PI)
    current_angle -= 2.0*PI;   // Make sure the angle is not above 360 degrees.

  // -- Report data:
  printf("\n          *** SIMULATION PERFORMANCE REPORT ***\n");
  if(num_projections!=1)   // Output the projection angle when simulating a CT:
    printf("              Projection %d of %d. Angle from X axis: %lf\n", current_projection, num_projections-1, current_angle*RAD2DEG);
#ifdef USING_MPI
  printf("              Computed by the MPI thread %d of %d\n", myID+1, numprocs);
#endif
  printf("              Simulated x rays:    %lld\n", total_histories);
  printf("              Simulation time [s]: %.2f\n", time_elapsed);
  if (time_elapsed>0.000001)
    printf("              Speed [x-rays/s]:    %.2f\n", ((double)total_histories)/time_elapsed);

  FILE* file_ptr = fopen(file_name_output, "w");
  
  if (file_ptr==NULL)
  {
    printf("\n\n   !!fopen ERROR!! File %s can not be opened!!\n", file_name_output);
    exit(-3);
  }

  fprintf(file_ptr, "# \n");
#ifdef USING_CUDA
  fprintf(file_ptr, "#  *** SIMULATION IN THE GPU USING CUDA ***\n");
#else
  fprintf(file_ptr, "#  *** SIMULATION IN THE CPU ***\n");
#endif
  fprintf(file_ptr, "#\n");
  fprintf(file_ptr, "#  Image created counting the energy arriving at each pixel.\n");
  fprintf(file_ptr, "#  Pixel value units: eV/cm^2 per history\n");
  if(num_projections!=1)   // Output the projection angle when simulating a CT:
    fprintf(file_ptr, "#  CT projection %d of %d. Angle from X axis: %lf\n", current_projection, num_projections-1, current_angle*RAD2DEG);
#ifdef USING_MPI
  fprintf(file_ptr, "#  Computed by the MPI thread %d of %d\n", myID+1, numprocs);
#endif    
  fprintf(file_ptr, "#  Pixel size:  %lf x %lf = %lf cm^2\n", 1.0/(double)(detector_data->inv_pixel_size_X), 1.0/(double)(detector_data->inv_pixel_size_Z), 1.0/(double)(detector_data->inv_pixel_size_X*detector_data->inv_pixel_size_Z));
  
  fprintf(file_ptr, "#  Number of pixels in X and Z:  %d  %d\n", detector_data->num_pixels.x, detector_data->num_pixels.y);
  fprintf(file_ptr, "#  (X rows given first, a blank line separates the different Z values)\n");
  fprintf(file_ptr, "# \n");
  fprintf(file_ptr, "#  [NON-SCATTERED] [COMPTON] [RAYLEIGH] [MULTIPLE-SCATTING]\n");
  fprintf(file_ptr, "# ==========================================================\n");

  const double SCALE = 1.0/SCALE_eV;    // conversion to eV using the inverse of the constant used in the "tally_image" kernel function (defined in the header file)
  const double NORM = SCALE * detector_data->inv_pixel_size_X * detector_data->inv_pixel_size_Z / ((double)total_histories);  // ==> [eV/cm^2 per history]
  double energy_noScatter, energy_compton, energy_rayleigh, energy_multiscatter;
  double energy_integral = 0.0;   // Integrate (add) the energy in the image pixels [meV]

  int pixels_per_image = (detector_data->num_pixels.x*detector_data->num_pixels.y), pixel=0;
  int i, j;
  for(j=0; j<detector_data->num_pixels.y; j++)
  {
    for(i=0; i<detector_data->num_pixels.x; i++)
    {
      energy_noScatter    = (double)(image[pixel]);
      energy_compton      = (double)(image[pixel +   pixels_per_image]);
      energy_rayleigh     = (double)(image[pixel + 2*pixels_per_image]);
      energy_multiscatter = (double)(image[pixel + 3*pixels_per_image]);

      // -- Write the results in an external file; the image corresponding to all particles not written: it has to be infered adding all images
      fprintf(file_ptr, "%.8lf %.8lf %.8lf %.8lf\n", NORM*energy_noScatter, NORM*energy_compton, NORM*energy_rayleigh, NORM*energy_multiscatter);

      energy_integral += energy_noScatter + energy_compton + energy_rayleigh + energy_multiscatter;
      pixel++;
    }
    fprintf(file_ptr, "\n");     // Separate rows with an empty line for visualization with gnuplot.
  }
  fprintf(file_ptr, "#   *** Simulation REPORT: ***\n");
  fprintf(file_ptr, "#       Fraction energy detected: %.3lf%%\n", 100.0*SCALE*(energy_integral/(double)(total_histories))/(double)(source_data->energy));
  fprintf(file_ptr, "#       Simulated x rays:    %lld\n", total_histories);
  fprintf(file_ptr, "#       Simulation time [s]: %.2f\n", time_elapsed);
  if (time_elapsed>0.000001)
    fprintf(file_ptr, "#       Speed [x-rays/sec]:  %.2f\n\n", ((double)total_histories)/time_elapsed);

  fclose(file_ptr);  // Close output file and flush stream

  printf("              Fraction of initial energy arriving at the detector (for a monoenergetic beam):  %.3lf%%\n\n", 100.0*SCALE*(energy_integral/(double)(total_histories))/(double)(source_data->energy));  // !!DeBuG!! Caution: Only valid for a monoenergetic beam!
  fflush(stdout);
  
  return 0;   // Report could return not 0 to continue the simulation...  !!DeBuG!!
}
///////////////////////////////////////////////////////////////////////////////



///////////////////////////////////////////////////////////////////////////////
//!  Sets the CT trajectory: store in memory the source and detector rotations
//!  that are needed to calculate the multiple projections.
//!  The first projection (0) was previously initialized in function "read_input".
//!  
//!
//!  ASSUMPTIONS: the CT scan plane must be perpendicular to the Z axis, ie,
//!               the initial direction of the particles must have w=0!
//!
///////////////////////////////////////////////////////////////////////////////
void set_CT_trajectory(int myID, int num_projections, double D_angle, double angularROI_0, double angularROI_1, struct source_struct* source_data, struct detector_struct* detector_data)
{
  MASTER_THREAD printf("\n    -- Setting the sources and detectors for the %d CT projections:\n", num_projections);
  double cos_rX, cos_rZ, sin_rX, sin_rZ, current_angle;

  // --Set center of rotation exactly half way between source and detector
  float3 center_rotation, detector_center;
  center_rotation.x =  source_data->position[0].x + source_data->direction[0].x * (0.5*detector_data->sdd);
  center_rotation.y =  source_data->position[0].y + source_data->direction[0].y * (0.5*detector_data->sdd);
  center_rotation.z =  source_data->position[0].z;    //  + source_data->direction.z * (0.5*detector_data->sdd);   // !!DeBuG!! w=0 all the time!!

  // --Angular span between projections:

  //  -Set initial angle for the source (180 degress less than the detector pointed by the direction vector; the zero angle is the X axis, increasing to +Y axis).
  current_angle = acos((double)source_data->direction[0].x);
  if (source_data->direction[0].y<0)
    current_angle = -current_angle;     // Correct for the fact that positive and negative angles have the same ACOS
  if (current_angle<0.0)
    current_angle += 2.0*PI;   // Make sure the angle is not negative, between [0,360) degrees.
  current_angle = current_angle - PI;   // Correct the fact that the source is opposite to the detector (180 degrees difference).
  if (current_angle<0.0)
    current_angle += 2.0*PI;   // Make sure the angle is not negative, between [0,360) degrees..

  MASTER_THREAD printf("         << Projection #0 >> initial_angle=%f , D_angle=%f\n", current_angle*RAD2DEG, D_angle*RAD2DEG); 
  MASTER_THREAD printf("                             Source direction=(%f,%f,%f), position=(%f,%f,%f)\n", source_data->direction[0].x,source_data->direction[0].y,source_data->direction[0].z, source_data->position[0].x,source_data->position[0].y,source_data->position[0].z);  // !!DeBuG!! Verbose

  int i;
  for (i=1; i<num_projections; i++)   // The first projection (i=0) was initialized in function "read_input".
  {
    // --Set the new source location and direction, for the current CT projection:
    current_angle += D_angle;
    if (current_angle>=(2.0*PI-0.0001))
      current_angle -= 2.0*PI;   // Make sure the angle is not above or equal to 360 degrees.

      // -- Check if this projection is inside the angular region of interest
      //   if ((current_angle < angularROI_0) || (current_angle > angularROI_1))
      //   {
      //     printf("            Skipping projection #%d: angle %f outside the angular region of interest.\n", i, current_angle*RAD2DEG);  // !!DeBuG!! Verbose
      //     continue;
      //   }

    source_data->position[i].x = center_rotation.x + (0.5*detector_data->sdd)*cos(current_angle);
    source_data->position[i].y = center_rotation.y + (0.5*detector_data->sdd)*sin(current_angle);
    source_data->position[i].z = source_data->position[0].z;   //  !!DeBuG!! The Z position must be constant (w=0)!!   !!DeBuG!!

    source_data->direction[i].x = center_rotation.x - source_data->position[i].x;
    source_data->direction[i].y = center_rotation.y - source_data->position[i].y;
    source_data->direction[i].z = 0.0f;    //  center_rotation.z - source_data->position.z;   !!DeBuG!! w=0 all the time!!   !!DeBuG!!

    double norm = 1.0/sqrt((double)source_data->direction[i].x*(double)source_data->direction[i].x + (double)source_data->direction[i].y*(double)source_data->direction[i].y /* + source_data->direction[i].z*source_data->direction[i].z*/);
    source_data->direction[i].x = (float)(((double)source_data->direction[i].x)*norm);
    source_data->direction[i].y = (float)(((double)source_data->direction[i].y)*norm);
      // source_data->direction[i].z = (float)(((double)source_data->direction[i].z)*norm);

    // --Set the new detector in front of the new source:
    detector_center.x = source_data->position[i].x + source_data->direction[i].x * detector_data->sdd;   // Set the center of the detector straight ahead of the focal spot.
    detector_center.y = source_data->position[i].y + source_data->direction[i].y * detector_data->sdd;
    detector_center.z = source_data->position[i].z;    //  + source_data->direction[i].z * detector_data->sdd;   !!DeBuG!! w=0 all the time!!

    double rotX, rotZ;

      //  detector_data->rotation_flag = 1;   // !!DeBuG!! Already set in read_input!

    // -- Rotate the detector center to +Y:
    //    Set the rotation that will bring particles from the detector plane to +Y=(0,+1,0) through a rotation around X and around Z (counter-clock):
    rotX = 0.0;   // !!DeBuG!! w=0 all the time!!  CORRECT CALCULATION:  acos(source_data->direction.z) - 0.5*PI;  // Rotate to +Y = (0,+1,0) --> rotX_0 =  -PI/2

    if ( (source_data->direction[i].x*source_data->direction[i].x + source_data->direction[i].y*source_data->direction[i].y) > 1.0e-8 )   // == u^2+v^2 > 0
      if (source_data->direction[i].y >= 0.0f)
        rotZ = 0.5*PI - acos(source_data->direction[i].x/sqrt(source_data->direction[i].x*source_data->direction[i].x + source_data->direction[i].y*source_data->direction[i].y));
      else
        rotZ = 0.5*PI - (-acos(source_data->direction[i].x/sqrt(source_data->direction[i].x*source_data->direction[i].x + source_data->direction[i].y*source_data->direction[i].y)));
    else
      rotZ = 0.0;   // Vector pointing to +Z, do not rotate around Z then.

    MASTER_THREAD printf("         << Projection #%d >> current_angle=%f, rotation around Z = %f\n", i, current_angle*RAD2DEG, rotZ*RAD2DEG);  // !!DeBuG!! Verbose
    MASTER_THREAD printf("                             Source direction=(%f,%f,%f), position=(%f,%f,%f)\n", source_data->direction[i].x,source_data->direction[i].y,source_data->direction[i].z, source_data->position[i].x,source_data->position[i].y,source_data->position[i].z);  // !!DeBuG!! Verbose

    cos_rX = cos(rotX);
    cos_rZ = cos(rotZ);
    sin_rX = sin(rotX);
    sin_rZ = sin(rotZ);
    detector_data->rot_inv[i][0] =  cos_rZ;    // Rotation matrix RxRz:
    detector_data->rot_inv[i][1] = -sin_rZ;
    detector_data->rot_inv[i][2] =  0.0f;
    detector_data->rot_inv[i][3] =  cos_rX*sin_rZ;
    detector_data->rot_inv[i][4] =  cos_rX*cos_rZ;
    detector_data->rot_inv[i][5] = -sin_rX;
    detector_data->rot_inv[i][6] =  sin_rX*sin_rZ;
    detector_data->rot_inv[i][7] =  sin_rX*cos_rZ;
    detector_data->rot_inv[i][8] =  cos_rX;


    detector_data->corner_min_rotated_to_Y[i].x = detector_center.x*detector_data->rot_inv[i][0] + detector_center.y*detector_data->rot_inv[i][1] + detector_center.z*detector_data->rot_inv[i][2];
    detector_data->corner_min_rotated_to_Y[i].y = detector_center.x*detector_data->rot_inv[i][3] + detector_center.y*detector_data->rot_inv[i][4] + detector_center.z*detector_data->rot_inv[i][5];
    detector_data->corner_min_rotated_to_Y[i].z = detector_center.x*detector_data->rot_inv[i][6] + detector_center.y*detector_data->rot_inv[i][7] + detector_center.z*detector_data->rot_inv[i][8];

    // -- Set the lower corner (minimum) coordinates at the normalized orientation: +Y. The detector has thickness 0.
    detector_data->corner_min_rotated_to_Y[i].x = detector_data->corner_min_rotated_to_Y[i].x - 0.5*detector_data->width_X;
    detector_data->corner_min_rotated_to_Y[i].y = detector_data->corner_min_rotated_to_Y[i].y;
    detector_data->corner_min_rotated_to_Y[i].z = detector_data->corner_min_rotated_to_Y[i].z - 0.5*detector_data->height_Z;

    // *** Init the fan beam source model:

      rotZ = -rotZ;   // The source rotation is the inverse of the detector.
      cos_rX = cos(rotX);
      cos_rZ = cos(rotZ);
      sin_rX = sin(rotX);
      sin_rZ = sin(rotZ);
      // --Rotation around X (alpha) and then around Z (phi): Rz*Rx (oposite of detector rotation)
      source_data->rot_fan[i][0] =  cos_rZ;
      source_data->rot_fan[i][1] = -cos_rX*sin_rZ;
      source_data->rot_fan[i][2] =  sin_rX*sin_rZ;
      source_data->rot_fan[i][3] =  sin_rZ;
      source_data->rot_fan[i][4] =  cos_rX*cos_rZ;
      source_data->rot_fan[i][5] = -sin_rX*cos_rZ;
      source_data->rot_fan[i][6] =  0.0f;
      source_data->rot_fan[i][7] =  sin_rX;
      source_data->rot_fan[i][8] =  cos_rX;

        // printf("\n    -- Source location and direction for the following CT projection:\n");   // !!DeBuG!! Verbose
        // printf("                 angle between projections = %lf degrees\n", D_angle*RAD2DEG);
        // printf("                             current angle = %lf degrees\n", current_angle*RAD2DEG);
        // printf("                   new focal spot position = (%f, %f, %f)\n", source_data->position[i].x, source_data->position[i].y, source_data->position[i].z);
        // printf("                      new source direction = (%f, %f, %f)\n", source_data->direction[i].x, source_data->direction[i].y, source_data->direction[i].z);
        // printf("                       new detector center = (%f, %f, %f)\n", detector_center.x, detector_center.y, detector_center.z);
        // printf("           new detector low corner (at +Y) = (%f, %f, %f)\n", detector_data->corner_min_rotated_to_Y[i].x, detector_data->corner_min_rotated_to_Y[i].y, detector_data->corner_min_rotated_to_Y[i].z);
        // printf("                        center of rotation = (%f, %f, %f)\n", center_rotation.x, center_rotation.y, center_rotation.z);
        // printf("         detector width (X) and height (Z) = %f , %f cm\n", detector_data->width_X, detector_data->height_Z);
        // printf("            rotations to +Y around Z and X = %f , %f degrees\n", rotZ*RAD2DEG, rotX*RAD2DEG);
  }
}
///////////////////////////////////////////////////////////////////////////////


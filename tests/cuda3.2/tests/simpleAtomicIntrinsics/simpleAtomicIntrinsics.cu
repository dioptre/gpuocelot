/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A simple program demonstrating trivial use of global memory atomic 
   device functions (atomic*() functions).
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "cutil_inline.h"

// includes, kernels
#include "simpleAtomicIntrinsics_kernel.cu"

static char *sSDKsample = "[simpleAtomicIntrinsics]\0";

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
int computeGold( int* gpuData, const int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char **argv)
{
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    int dev;

	printf("%s\n", sSDKsample);

    // This will pick the best possible CUDA capable device
    dev = cutilChooseCudaDevice(argc, argv);

    cutilSafeCall( hipGetDeviceProperties(&deviceProp, dev) );

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n", 
		deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if(version < 0x11) 
    {
        printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKsample);
        printf("PASSED");
        hipDeviceReset();
        cutilExit(argc, argv);
    }

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    unsigned int numThreads = 256;
    unsigned int numBlocks = 64;
    unsigned int numData = 11;
    unsigned int memSize = sizeof(int) * numData;

    //allocate mem for the result on host side
    int *h_odata = (int *)malloc(memSize);

    //initalize the memory
    for(unsigned int i = 0; i < numData; i++)
        h_odata[i] = 0;

    //To make the AND and XOR tests generate something other than 0...
    h_odata[8] = h_odata[10] = 0xff; 

    // allocate device memory for result
    int *d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, memSize));
    // copy host memory to device to initialize to zers
    cutilSafeCall( hipMemcpy( d_odata, h_odata, memSize, hipMemcpyHostToDevice) );


    // execute the kernel
    testKernel<<<numBlocks, numThreads>>>(d_odata);
    cutilCheckMsg("Kernel execution failed");
    //Copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_odata, memSize, hipMemcpyDeviceToHost) );

    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));

	printf("\n%s - Test Summary\n", sSDKsample);

    //compute reference solution
    if(computeGold(h_odata, numThreads * numBlocks))
        printf("PASSED\n");
    else
        printf("FAILED\n");

    // cleanup memory
    free(h_odata);
    cutilSafeCall(hipFree(d_odata));

    hipDeviceReset();
}

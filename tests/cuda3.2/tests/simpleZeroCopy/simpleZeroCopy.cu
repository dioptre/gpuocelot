#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>

/* Add two vectors on the GPU */

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = a[idx] + b[idx];
}

int main(int argc, char **argv)
{
  int n, nelem, idev, deviceCount;
  char *device = NULL;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;                      // Pinned memory allocated on the CPU
  float *d_a, *d_b, *d_c;                // Device pointers for mapped memory  
  float errorNorm, refNorm, ref, diff;
  hipDeviceProp_t deviceProp;

  if(cutCheckCmdLineFlag(argc, (const char **)argv, "help"))
  {
    printf("Usage:  simpleZeroCopy [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    cutilExit(argc, argv);    
  }

  /* Get the device selected by the user or default to 0, and then set it. */

  if(cutGetCmdLineArgumentstr(argc, (const char**)argv, "device", &device))
  {
    hipGetDeviceCount(&deviceCount);
    idev = atoi(device);
    if(idev >= deviceCount || idev < 0)
    {
      fprintf(stderr, "Invalid device number %d, using default device 0.\n",
              idev);
      idev = 0;
    }
  }
  else
  {
    idev = 0;
  }

  cutilSafeCall(hipSetDevice(idev));

  /* Verify the selected device supports mapped memory and set the device
     flags for mapping host memory. */

  cutilSafeCall(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
  if(!deviceProp.canMapHostMemory)
  {
    fprintf(stderr, "Device %d cannot map host memory!\n", idev);
    printf("PASSED");
    cutilExit(argc, argv);
  }
  cutilSafeCall(hipSetDeviceFlags(hipDeviceMapHost));
#else
  fprintf(stderr, "This CUDART version does not support <hipDeviceProp_t.canMapHostMemory> field\n");
  printf("PASSED");
  cutilExit(argc, argv);
#endif

  /* Allocate mapped CPU memory. */

  nelem = 1048576;
  bytes = nelem*sizeof(float);
#if CUDART_VERSION >= 2020
  flags = hipHostMallocMapped;
  cutilSafeCall(hipHostAlloc((void **)&a, bytes, flags));
  cutilSafeCall(hipHostAlloc((void **)&b, bytes, flags));
  cutilSafeCall(hipHostAlloc((void **)&c, bytes, flags));
#endif
  /* Initialize the vectors. */

  for(n = 0; n < nelem; n++)
  {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  /* Get the device pointers for the pinned CPU memory mapped into the GPU
     memory space. */

#if CUDART_VERSION >= 2020
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

  /* Call the GPU kernel using the device pointers for the mapped memory. */ 

  printf("Adding vectors using mapped CPU memory...\n");
  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem/(float)block.x));
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);  
  cutilSafeCall(hipDeviceSynchronize());
  cutilCheckMsg("vectorAddGPU() execution failed");

  /* Compare the results */

  printf("Checking the results...\n");
  errorNorm = 0.f;
  refNorm = 0.f;
  for(n = 0; n < nelem; n++)
  {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff*diff;
    refNorm += ref*ref;
  }
  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);

  /* Memory clean up */

  printf("Releasing CPU memory...\n");
#if CUDART_VERSION >= 2020
  cutilSafeCall(hipHostFree(a));
  cutilSafeCall(hipHostFree(b));
  cutilSafeCall(hipHostFree(c));
#endif

  hipDeviceReset();

  printf("%s\n", (errorNorm/refNorm < 1.e-6f) ? "PASSED" : "FAILED");

  cutilExit(argc, argv);
}

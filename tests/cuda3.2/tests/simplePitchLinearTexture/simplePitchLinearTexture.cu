#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 
/* pitchLinearTexture 
 *
 * This example demostrates how to use textures bound to pitch linear memory.  
 * It performs a shift of matrix elements using wrap addressing mode (aka
 * periodic boundary conditions) on two arrays, a pitch linear and a CUDA array,
 * in order to highlight the differences in using each. 
 *
 * Textures binding to pitch linear memory is a new feature in CUDA 2.2,
 * and allows use of texture features such as wrap addressing mode and 
 * filtering which are not possible with textures bound to regular linear memory 
 */

#include <stdio.h>
#include <cutil_inline.h>


#define NUM_REPS 100  // number of repetitions performed  
#define TILE_DIM 16   // tile/block size

// Texture references

texture<float, 2, hipReadModeElementType> texRefPL;
texture<float, 2, hipReadModeElementType> texRefArray;

// -------
// kernels
// -------
//
// NB: (1) The second argument "pitch" is in elements, not bytes
//     (2) normalized coordinates are used (required for wrap address mode)

__global__ void shiftPitchLinear(float* odata, int pitch, int width, int height, 
				 int shiftX, int shiftY)
{
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;
  
  odata[yid*pitch+xid] = tex2D(texRefPL, 
			       (xid + shiftX)/(float)width, 
			       (yid + shiftY)/(float)height);
}

__global__ void shiftArray(float* odata, int pitch, int width, int height,
			   int shiftX, int shiftY)
{
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;
  
  odata[yid*pitch+xid] = tex2D(texRefArray, 
			       (xid + shiftX)/(float)width, 
			       (yid + shiftY)/(float)height);
}

// ----
// main
// ----

int
main( int argc, char** argv) 
{
  // set array size

 #ifdef __DEVICE_EMULATION__
  const int nx = 64;  
  const int ny = 64;
#else
  const int nx = 2048;  
  const int ny = 2048;
#endif
  // shifts applied to x and y data

  const int x_shift = 5;
  const int y_shift = 7;

  if ((nx%TILE_DIM != 0)  || (ny%TILE_DIM != 0)) {
    printf("nx and ny must be multiples of TILE_DIM\nPASSED\n");
    cutilExit(argc, argv);
  }

  // execution configuration parameters

  dim3 grid(nx/TILE_DIM, ny/TILE_DIM), threads(TILE_DIM, TILE_DIM);

  // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
    cutilDeviceInit(argc, argv);
  else
    hipSetDevice( cutGetMaxGflopsDeviceId() );

  // CUDA events for timing

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ----------------------------------
  // Host allocation and initialization
  // ----------------------------------

  float *h_idata = (float*) malloc(sizeof(float)*nx*ny);
  float *h_odata = (float*) malloc(sizeof(float)*nx*ny);
  float *gold = (float*) malloc(sizeof(float)*nx*ny);

  for(int i = 0; i < nx*ny; ++i) h_idata[i] = (float) i;  

  // ------------------------
  // Device memory allocation
  // ------------------------

  // Pitch linear input data

  float *d_idataPL;
  size_t d_pitchBytes;

  cutilSafeCall(hipMallocPitch((void**) &d_idataPL, &d_pitchBytes, nx*sizeof(float), ny));

  // Array input data

  hipArray *d_idataArray;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  cutilSafeCall(hipMallocArray(&d_idataArray, &channelDesc, nx, ny));

  // Pitch linear output data

  float *d_odata;
  cutilSafeCall(hipMallocPitch( (void**) &d_odata, &d_pitchBytes, nx*sizeof(float), ny));

  // ------------------------
  // copy host data to device
  // ------------------------

  // Pitch linear

  size_t h_pitchBytes = nx*sizeof(float);

  cutilSafeCall(hipMemcpy2D(d_idataPL, d_pitchBytes, h_idata, h_pitchBytes, 
			     nx*sizeof(float), ny, hipMemcpyHostToDevice));

  // Array

  cutilSafeCall(hipMemcpyToArray(d_idataArray, 0, 0, h_idata, 
				  nx*ny*sizeof(float), hipMemcpyHostToDevice));

  // ----------------------
  // Bind texture to memory
  // ----------------------

  // Pitch linear

  texRefPL.normalized = 1;
  texRefPL.filterMode = hipFilterModePoint;
  texRefPL.addressMode[0] = hipAddressModeWrap;
  texRefPL.addressMode[1] = hipAddressModeWrap;
  
  cutilSafeCall(hipBindTexture2D(0, &texRefPL, d_idataPL, &channelDesc, nx, ny, d_pitchBytes));
  
  // Array

  texRefArray.normalized = 1;
  texRefArray.filterMode = hipFilterModePoint;
  texRefArray.addressMode[0] = hipAddressModeWrap;
  texRefArray.addressMode[1] = hipAddressModeWrap;

  cutilSafeCall(hipBindTextureToArray(texRefArray, d_idataArray, channelDesc));

  // ---------------------
  // reference calculation
  // ---------------------

  for (int j = 0; j < ny; j++) {
    int jshift = (j+y_shift)%ny;
    for (int i = 0; i < nx; i++) {
      int ishift = (i+x_shift)%nx;
      gold[j*nx + i] = h_idata[jshift*nx + ishift];
    }
  }

  // ----------------
  // shiftPitchLinear
  // ----------------

  cutilSafeCall(hipMemset2D(d_odata, d_pitchBytes, 0, nx*sizeof(float), ny));
  cutilSafeCall(hipEventRecord(start, 0));
  for (int i=0; i < NUM_REPS; i++) {
    shiftPitchLinear<<<grid, threads>>>(d_odata, d_pitchBytes/sizeof(float), nx, ny, 
					x_shift, y_shift);
  }
  cutilSafeCall(hipEventRecord(stop, 0));
  cutilSafeCall(hipEventSynchronize(stop));
  float timePL;
  cutilSafeCall(hipEventElapsedTime(&timePL, start, stop));    

  // check results

  cutilSafeCall(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes, 
			     nx*sizeof(float), ny, hipMemcpyDeviceToHost));

  CUTBoolean res = cutComparef(gold, h_odata, nx*ny);

  CUTBoolean success = CUTTrue;;
  if (res == CUTFalse) {
    printf("*** shiftPitchLinear failed ***\n");
    success = CUTFalse;
  }
    
  // ----------
  // shiftArray
  // ----------

  cutilSafeCall(hipMemset2D(d_odata, d_pitchBytes, 0, nx*sizeof(float), ny));
  cutilSafeCall(hipEventRecord(start, 0));
  for (int i=0; i < NUM_REPS; i++) {
    shiftArray<<<grid, threads>>>(d_odata, d_pitchBytes/sizeof(float), nx, ny, x_shift, y_shift);
  }
  cutilSafeCall(hipEventRecord(stop, 0));
  cutilSafeCall(hipEventSynchronize(stop));
  float timeArray;
  cutilSafeCall(hipEventElapsedTime(&timeArray, start, stop));    

  // check results

  cutilSafeCall(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes, 
		     nx*sizeof(float), ny, hipMemcpyDeviceToHost));
  res = cutComparef(gold, h_odata, nx*ny);

  if (res == CUTFalse) {
    printf("*** shiftArray failed ***\n");
    success = CUTFalse;
  }
    
  float bandwidthPL = 2.f*1000*nx*ny*sizeof(float)/(1.e+9)/(timePL/NUM_REPS);
  float bandwidthArray = 2.f*1000*nx*ny*sizeof(float)/(1.e+9)/(timeArray/NUM_REPS);
  printf("\nBandwidth (GB/s) for pitch linear: %.2e; for array: %.2e\n", 
	 bandwidthPL, bandwidthArray);

  float fetchRatePL = nx*ny/1.e+6/(timePL/(1000.0f*NUM_REPS));
  float fetchRateArray = nx*ny/1.e+6/(timeArray/(1000.0f*NUM_REPS));
  printf("\nTexture fetch rate (Mpix/s) for pitch linear: %.2e; for array: %.2e\n\n", 
	 fetchRatePL, fetchRateArray);


  printf("%s\n", (success == CUTTrue) ? "PASSED" : "FAILED");

  // cleanup

  free(h_idata);
  free(h_odata);
  free(gold);

  cutilSafeCall(hipUnbindTexture(texRefPL));
  cutilSafeCall(hipUnbindTexture(texRefArray));
  cutilSafeCall(hipFree(d_idataPL));
  cutilSafeCall(hipFreeArray(d_idataArray));
  cutilSafeCall(hipFree(d_odata));

  cutilSafeCall(hipEventDestroy(start));
  cutilSafeCall(hipEventDestroy(stop));
  
  hipDeviceReset();
  cutilExit(argc, argv);
}

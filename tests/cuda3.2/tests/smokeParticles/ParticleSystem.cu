#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 /*
	This file contains simple wrapper functions that call the CUDA kernels
 */

#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "particles_kernel.cu"
#include "ParticleSystem.cuh"

extern "C"
{

hipArray *noiseArray;

void initCuda(bool bUseGL)
{
	if (bUseGL) {
	    cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
	} else {
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	}
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

inline float frand()
{
    return rand() / (float) RAND_MAX;
}

// create 3D texture containing random values
void createNoiseTexture(int w, int h, int d)
{
    hipExtent size = make_hipExtent(w, h, d);
    uint elements = (uint) size.width*size.height*size.depth;

    float *volumeData = (float *)malloc(elements*4*sizeof(float));
    float *ptr = volumeData;
    for(uint i=0; i<elements; i++) {
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
    }


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    cutilSafeCall( hipMalloc3DArray(&noiseArray, &channelDesc, size) );

    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr   = make_hipPitchedPtr((void*)volumeData, size.width*sizeof(float4), size.width, size.height);
    copyParams.dstArray = noiseArray;
    copyParams.extent   = size;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );

    free(volumeData);

    // set texture parameters
    noiseTex.normalized = true;                      // access with normalized texture coordinates
    noiseTex.filterMode = hipFilterModeLinear;      // linear interpolation
    noiseTex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    noiseTex.addressMode[1] = hipAddressModeWrap;
    noiseTex.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(noiseTex, noiseArray, channelDesc));
}

void 
integrateSystem(float4 *oldPos, float4 *newPos,
				float4 *oldVel, float4 *newVel,
                float deltaTime,
                int numParticles)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    integrateD<<< numBlocks, numThreads >>>(newPos, newVel,
                                            oldPos, oldVel,
                                            deltaTime,
											numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");
}


void 
calcDepth(float4*  pos, 
		  float*   keys,		// output
          uint*    indices,		// output 
          float3   sortVector,
          int      numParticles)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcDepthD<<< numBlocks, numThreads >>>(pos, keys, indices, sortVector, numParticles);
    cutilCheckMsg("calcDepthD execution failed");
}

}   // extern "C"

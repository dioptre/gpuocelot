#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* 
 * CUDA Device code for particle simulation.
 */

#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"

texture<float4, 3, hipReadModeElementType> noiseTex;

// simulation parameters
__constant__ SimParams params;

// look up in 3D noise texture
__device__
float3 noise3D(float3 p)
{
    float4 n = tex3D(noiseTex, p.x, p.y, p.z);
    return make_float3(n.x, n.y, n.z);
}

__device__
float3 fractalSum3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += noise3D(p*freq)*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

__device__
float3 turbulence3D(float3 p, int octaves, float lacunarity, float gain)
{
	float freq = 1.0f, amp = 0.5f;
	float3 sum = make_float3(0.0f);	
	for(int i=0; i<octaves; i++) {
		sum += fabs(noise3D(p*freq))*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

// integrate particle attributes
__global__ void
integrateD(float4* newPos, float4* newVel, 
           float4* oldPos, float4* oldVel, 
           float deltaTime,
		   int numParticles)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;

	volatile float4 posData = oldPos[index];	// ensure coalesced reads
    volatile float4 velData = oldVel[index];

    float3 pos = make_float3(posData.x, posData.y, posData.z);
    float3 vel = make_float3(velData.x, velData.y, velData.z);
    
    // update particle age
	float age = posData.w;	
	float lifetime = velData.w;
	if (age < lifetime) {
		age += deltaTime;
	} else {
	    age = lifetime;
    }

    // apply accelerations
    vel += params.gravity * deltaTime;

    // apply procedural noise
    float3 noise = noise3D(pos*params.noiseFreq + params.time*params.noiseSpeed);
    vel += noise * params.noiseAmp;

    // new position = old position + velocity * deltaTime
    pos += vel * deltaTime;

    vel *= params.globalDamping;

    // store new position and velocity
    newPos[index] = make_float4(pos, age);
    newVel[index] = make_float4(vel, velData.w);
}

// calculate sort depth for each particle
__global__ void calcDepthD(float4* pos, float* keys, uint *indices, float3 vector, int numParticles)
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;

	volatile float4 p = pos[index];
	float key = -dot(make_float3(p.x, p.y, p.z), vector);        // project onto sort vector
	
	keys[index] = key;
	indices[index] = index;
}

#endif

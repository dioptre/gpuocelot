
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 
// Work in progress:

// Odd/Even sort.
__device__ void sortColors3(float * v, float3 * colors, int * xrefs)
{
	int tid = threadIdx.x;

	xrefs[tid] = tid;

	float e;
	const bool active = (tid != 0) && (tid != 15);
	
	for (int i = 0; i < 8; i++)
	{
		if (tid & 1) e = min(v[tid], v[tid+1]);
		else e = max(v[tid-1], v[tid]);
		v[tid] = e;

		if (active)
		{
			if (tid & 1) e = max(v[tid-1], v[tid]);
			else e = min(v[tid], v[tid+1]);
			v[tid] = e;
		}
	}
	
	/*for (int i = 0; i < 8; i++)
	{
		int x;
		{
			int odd = tid & 1;
			int x0 = xrefs[tid ^ odd];
			int x1 = xrefs[tid ^ !odd];
			int cmp = v[x0] > v[x1];
			if (cmp ^ odd) x = x0;
			else x = x0;
			xrefs[tid] = x;
		}
		
		if (active)
		{
			if (tid & 1) e = max(v[xrefs[tid-1]], v[xrefs[tid]]);
			else e = min(v[xrefs[tid]], v[xrefs[tid+1]]);
			v[xrefs[tid]] = e;
		}
	}*/
	
//	float3 tmp = colors[tid];
//	colors[cmp[tid]] = tmp;
}

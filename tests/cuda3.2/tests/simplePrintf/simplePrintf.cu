#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <cuPrintf.cu>
#include <shrUtils.h>
#include "cutil_inline.h"

//The macro CUPRINTF is defined for architectures
//with different compute capabilities.
#if __CUDA_ARCH__ < 200 	//Compute capability 1.x architectures
#define CUPRINTF cuPrintf
#else						//Compute capability 2.x architectures
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
								blockIdx.y*gridDim.x+blockIdx.x,\
								threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
								__VA_ARGS__)
#endif

__global__ void testKernel(int val)
{
	CUPRINTF("\tValue is:%d\n", val);
}

int main(int argc, char **argv)
{
	int devID;	
	hipDeviceProp_t props;
	
	//Check which GPU is used
	cutilChooseCudaDevice(argc, argv);
	
	//Get GPU information
	cutilSafeCall(hipGetDevice(&devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
	printf("Device %d: \"%s\" with Compute %d.%d capability\n", 
			devID, props.name, props.major, props.minor);

	//Architectures with compute capability 1.x, function
	//cuPrintf() is used. Otherwise, function printf() is called.
	bool use_cuPrintf = (props.major < 2);

	if (use_cuPrintf)
	{
		//Initializaton, allocate buffers on both host
		//and device for data to be printed.
		cudaPrintfInit();

		printf("cuPrintf() is called. Output:\n\n");
	}
	//Architecture with compute capability 2.x, function
	//printf() is called.
	else
	{
		printf("printf() is called. Output:\n\n");
	}

	//Kernel configuration, where a two-dimensional grid and 
	//three-dimensional blocks are configured.
	dim3 dimGrid(2, 2);
	dim3 dimBlock(2, 2, 2);
	testKernel<<<dimGrid, dimBlock>>>(10);
	hipDeviceSynchronize();

	if (use_cuPrintf)
	{
		//Dump current contents of output buffer to standard 
		//output, and origin (block id and thread id) of each line 
		//of output is enabled(true).
		cudaPrintfDisplay(stdout, true);

		//Free allocated buffers by cudaPrintfInit().
		cudaPrintfEnd();
	}

	printf("\nPASSED\n");

    cutilExit(argc, argv);
    return 0;
}


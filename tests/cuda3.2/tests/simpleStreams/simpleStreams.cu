#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 *
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to 
 * initialize an array to a specific value, after which the array is 
 * copied to the host (CPU) memory.  To increase performance, multiple 
 * kernel/memcopy pairs are launched asynchronously, each pair in its 
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5 
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

const char *sSDKsample = "simpleStreams";

#include <stdio.h>
#include <cutil_inline.h>

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=0;i<num_iterations;i++)
	    g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

int correct_data(int *a, const int n, const int c)
{
    for(int i = 0; i < n; i++)
        if(a[i] != c)
		{
			printf("%d: %d %d\n", i, a[i], c);
            return 0;
		}

    return 1;
}

int main(int argc, char *argv[])
{
    int cuda_device = 0;
    int nstreams = 4;               // number of streams for CUDA calls
    int nreps = 10;                 // number of times each experiment is repeated
    int n = 16 * 1024 * 1024;       // number of ints in the data set
    int nbytes = n * sizeof(int);   // number of data bytes
    dim3 threads, blocks;           // kernel launch configuration
    float elapsed_time, time_memcpy, time_kernel;   // timing variables
    float scale_factor = 1.0f;

	int niterations;	// number of iterations for the loop inside the kernel

    printf("[ %s ]\n", sSDKsample);
    if( cutCheckCmdLineFlag( argc, (const char **)argv, "help") )
    {
        printf("Usage: %s [--use_blocking_sync]\n", sSDKsample);
        printf("\t--use_blocking_sync (optional) enable blocking sync to demontrate\n");
        printf("\t\t low CPU utilization\n");
        exit(0);
    }
    bool use_blocking_sync = false; 
    if( cutCheckCmdLineFlag( argc, (const char **)argv, "use_blocking_sync") )
    {
        printf("Enabling blocking sync, setting reps to 100 to demonstrate steady state\n");
        nreps = 100;
        use_blocking_sync = true;
    }

	printf("\n> ");
	cuda_device = cutilChooseCudaDevice(argc, argv);

    // check the compute capability of the device
    int num_devices=0;
    cutilSafeCall( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
        printf("your system does not have a CUDA capable device\n");
        return 1;
	}
	
    // check if the command-line chosen device ID is within range, exit if not
    if( cuda_device >= num_devices )
	{
        printf("cuda_device=%d is invalid, must choose device ID between 0 and %d\n", cuda_device, num_devices-1);
		cutilExit(argc, argv);
        return 1;
	}

    hipSetDevice( cuda_device );

    if( use_blocking_sync ) {
        // enable use of blocking sync, to reduce CPU usage
        hipSetDeviceFlags( hipDeviceScheduleBlockingSync );
    }

    hipDeviceProp_t deviceProp;
	
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (1 == deviceProp.major) && (deviceProp.minor < 1))
        printf("%s does not have compute capability 1.1 or later\n", deviceProp.name);
    if(deviceProp.major >= 2)
        niterations = 100;
    else
	{		    
	    if(deviceProp.minor > 1)
		niterations = 5;
	    else
		niterations = 1; // reduced workload for compute capability 1.0 and 1.1
	}

#ifdef __DEVICE_EMULATION__
    n = 4096;   // reduced workload for emulation (n should be divisible by 512*nstreams)
#else
    // Anything that is less than 4 SM's will be scaled down in terms of workload
    scale_factor = max((4.0f / (float)deviceProp.multiProcessorCount), 1.0f);
    n = rint( (float)n / scale_factor );
#endif

//	printf("> Device name : %s\n", deviceProp.name );
    printf("> CUDA Capable SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 
    printf("> scale_factor = %1.4f\n", 1.0f/scale_factor);
    printf("> array_size   = %d\n\n", n);

    // allocate host memory
    int c = 5;                      // value to which the array will be initialized
    int *a = 0;                     // pointer to the array data in host memory
    // allocate host memory (pinned is required for achieve asynchronicity)
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) ); 

    // allocate device memory
    int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );
    cutilSafeCall( hipMalloc((void**)&d_c, sizeof(int)) );
    cutilSafeCall( hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice) );

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
        cutilSafeCall( hipStreamCreate(&(streams[i])) );

    // create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
    int eventflags = ( use_blocking_sync ? hipEventBlockingSync: hipEventDefault );

    cutilSafeCall( hipEventCreateWithFlags(&start_event, eventflags) );
    cutilSafeCall( hipEventCreateWithFlags(&stop_event, eventflags) );

    // time memcopy from device
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, streams[0]);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    cutilSafeCall( hipEventElapsedTime(&time_memcpy, start_event, stop_event) );
    printf("memcopy:\t%.2f\n", time_memcpy);
    
    // time kernel
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    cutilSafeCall( hipEventElapsedTime(&time_kernel, start_event, stop_event) );
    printf("kernel:\t\t%.2f\n", time_kernel);

    //////////////////////////////////////////////////////////////////////
    // time non-streamed execution for reference
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        init_array<<<blocks, threads>>>(d_a, d_c, niterations);
        hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    cutilSafeCall( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("non-streamed:\t%.2f (%.2f expected)\n", elapsed_time / nreps, time_kernel + time_memcpy);

    //////////////////////////////////////////////////////////////////////
    // time execution with nstreams streams
    threads=dim3(512,1);
    blocks=dim3(n/(nstreams*threads.x),1);
    memset(a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
    hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        // asynchronously launch nstreams kernels, each operating on its own portion of data
        for(int i = 0; i < nstreams; i++)
            init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);

        // asynchronoously launch nstreams memcopies.  Note that memcopy in stream x will only
        //   commence executing when all previous CUDA calls in stream x have completed
        for(int i = 0; i < nstreams; i++)
            hipMemcpyAsync(a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    cutilSafeCall( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("%d streams:\t%.2f (%.2f expected with compute capability 1.1 or later)\n", nstreams, elapsed_time / nreps, time_kernel + time_memcpy / nstreams);

    // check whether the output is correct
    printf("-------------------------------\n");
    if(correct_data(a, n, c*nreps*niterations))	// each element should have been incremented by c a total of npres*niterations times
        printf("PASSED\n");
    else
        printf("FAILED\n");

    // release resources
    for(int i = 0; i < nstreams; i++)
        hipStreamDestroy(streams[i]);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipHostFree(a);
    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();

    cutilExit(argc, argv);

    return 0;
}

/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 /*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* This sample demonstrates how use texture fetches in CUDA
*
* This sample takes an input PGM image (image_filename) and generates 
* an output PGM image (image_filename_out).  This CUDA kernel performs
* a simple 2D transform (rotation) on the texture coordinates (u,v).
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <simpleTexture_kernel.cu>

char *image_filename = "lena_bw.pgm";
char *ref_filename   = "ref_rotated.pgm";
float angle = 0.5f;    // angle to rotate image by (in radians)

#define MIN_EPSILON_ERROR 5e-3f


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
	{
		cutilDeviceInit(argc, argv);
	}
	else
	{
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	}

    // load image from disk
    float* h_data = NULL;
    unsigned int width, height;
    char* image_path = cutFindFilePath(image_filename, argv[0]);
    if (image_path == 0) {
        printf("Unable to source file file %s\n", image_filename);
        exit(EXIT_FAILURE);
    }
    cutilCheckError( cutLoadPGMf(image_path, &h_data, &width, &height));

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", image_filename, width, height);

    // load reference image from image (output)
    float *h_data_ref = (float*) malloc(size);
    char* ref_path = cutFindFilePath(ref_filename, argv[0]);
    if (ref_path == 0) {
        printf("Unable to find reference file %s\n", ref_filename);
        exit(EXIT_FAILURE);
    }
    cutilCheckError( cutLoadPGMf(ref_path, &h_data_ref, &width, &height));

    // allocate device memory for result
    float* d_data = NULL;
    cutilSafeCall( hipMalloc( (void**) &d_data, size));

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_array;
    cutilSafeCall( hipMallocArray( &cu_array, &channelDesc, width, height )); 
    cutilSafeCall( hipMemcpyToArray( cu_array, 0, 0, h_data, size, hipMemcpyHostToDevice));

    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray( tex, cu_array, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // warmup
    transformKernel<<< dimGrid, dimBlock, 0 >>>( d_data, width, height, angle);

    cutilSafeCall( hipDeviceSynchronize() );
    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    // execute the kernel
    transformKernel<<< dimGrid, dimBlock, 0 >>>( d_data, width, height, angle);

    // check if kernel execution generated an error
    cutilCheckMsg("Kernel execution failed");

    cutilSafeCall( hipDeviceSynchronize() );
    cutilCheckError( cutStopTimer( timer));
    printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));
    printf("%.2f Mpixels/sec\n", (width*height / (cutGetTimerValue( timer) / 1000.0f)) / 1e6);
    cutilCheckError( cutDeleteTimer( timer));

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( size);
    // copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_data, size, hipMemcpyDeviceToHost) );

    // write result to file
    char output_filename[1024];
    strcpy(output_filename, image_path);
    strcpy(output_filename + strlen(image_path) - 4, "_out.pgm");
    cutilCheckError( cutSavePGMf( output_filename, h_odata, width, height));
    printf("Wrote '%s'\n", output_filename);

    // write regression file if necessary
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression"))
    {
        // write file for regression test
        cutilCheckError( cutWriteFilef( "./data/regression.dat", h_odata, width*height, 0.0));
    } 
    else 
    {
        // We need to reload the data from disk, because it is inverted upon output
        cutilCheckError( cutLoadPGMf(output_filename, &h_odata, &width, &height));

        printf("Comparing files\n");
        printf("\toutput:    <%s>\n", output_filename);
        printf("\treference: <%s>\n", ref_path);
        CUTBoolean res = cutComparefe( h_odata, h_data_ref, width*height, MIN_EPSILON_ERROR );
        printf( "%s\n", (1 == res) ? "PASSED" : "FAILED");
    }

    // cleanup memory
//    free(h_data);
//    free(h_data_ref);
//    free(h_odata);

    cutilSafeCall(hipFree(d_data));
    cutilSafeCall(hipFreeArray(cu_array));
    cutFree(image_path);
    cutFree(ref_path);

    // If we are doing the QAtest, we quite without prompting
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "qatest") ||
        cutCheckCmdLineFlag( argc, (const char**) argv, "noprompt"))
    {
        hipDeviceReset();
        exit(0);
    }

    hipDeviceReset();
}

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
// ----------------------------------------------------------------------------------------
// Transpose
//
// This file contains both device and host code for transposing a floating-point
// matrix.  It performs several transpose kernels, which incrementally improve performance
// through coalescing, removing shared memory bank conflicts, and eliminating partition
// camping.  Several of the kernels perform a copy, used to represent the best case
// performance that a transpose can achieve.
//
// Please see the whitepaper in the docs folder of the transpose project for a detailed
// description of this performance study.
// ----------------------------------------------------------------------------------------

// Utilities and system includes
#include <shrUtils.h>
#include <cutil_inline.h>

const char *sSDKsample = "Transpose";

// Each block transposes/copies a tile of TILE_DIM x TILE_DIM elements
// using TILE_DIM x BLOCK_ROWS threads, so that each thread transposes
// TILE_DIM/BLOCK_ROWS elements.  TILE_DIM must be an integral multiple of BLOCK_ROWS

#define TILE_DIM    16
#define BLOCK_ROWS  16

// Modified the amount of workload, so during device emulation it runs in a reasonable amount of time.
// This sample assumes that MATRIX_SIZE_X = MATRIX_SIZE_Y
#ifdef __DEVICE_EMULATION__
    int MATRIX_SIZE_X = 128;
    int MATRIX_SIZE_Y = 128;
    int MUL_FACTOR    = 4;
#else
    int MATRIX_SIZE_X = 1024;
    int MATRIX_SIZE_Y = 1024;
    int MUL_FACTOR    = TILE_DIM;
#endif

#define FLOOR(a,b) (a-(a%b))

// Compute the tile size necessary to illustrate performance cases for SM12+ hardware
int MAX_TILES_SM12 = (FLOOR(MATRIX_SIZE_X,512) * FLOOR(MATRIX_SIZE_Y,512)) / (TILE_DIM*TILE_DIM);   
// Compute the tile size necessary to illustrate performance cases for SM10,SM11 hardware
int MAX_TILES_SM10 = (FLOOR(MATRIX_SIZE_X,384) * FLOOR(MATRIX_SIZE_Y,384)) / (TILE_DIM*TILE_DIM);

// Number of repetitions used for timing.  Two sets of repetitions are performed:
// 1) over kernel launches and 2) inside the kernel over just the loads and stores

#define NUM_REPS  100

// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void copy(float *odata, float* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  
  int index  = xIndex + width*yIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index+i*width] = idata[index+i*width];
    }
  }
}

__global__ void copySharedMem(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  
  int index  = xIndex + width*yIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
	  if (xIndex < width && yIndex < height)
        tile[threadIdx.y][threadIdx.x] = idata[index];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      if (xIndex < height && yIndex < width)
        odata[index] = tile[threadIdx.y][threadIdx.x];
    }
  }
}

// -------------------------------------------------------
// Transposes
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transposeNaive(float *odata, float* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index_in  = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i] = idata[index_in+i*width];
    }
  }
}

// coalesced transpose (with bank conflicts)

__global__ void transposeCoalesced(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

// Coalesced transpose with no bank conflicts

__global__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

// Transpose that effectively reorders execution of thread blocks along diagonals of the 
// matrix (also coalesced and has no bank conflicts)
//
// Here blockIdx.x is interpreted as the distance along a diagonal and blockIdx.y as 
// corresponding to different diagonals
//
// blockIdx_x and blockIdx_y expressions map the diagonal coordinates to the more commonly 
// used cartesian coordinates so that the only changes to the code from the coalesced version 
// are the calculation of the blockIdx_x and blockIdx_y and replacement of blockIdx.x and 
// bloclIdx.y with the subscripted versions in the remaining code

__global__ void transposeDiagonal(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int blockIdx_x, blockIdx_y;

  // do diagonal reordering
  if (width == height) {
    blockIdx_y = blockIdx.x;
    blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
  } else {
    int bid = blockIdx.x + gridDim.x*blockIdx.y;
    blockIdx_y = bid%gridDim.y;
    blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
  }    

  // from here on the code is same as previous kernel except blockIdx_x replaces blockIdx.x
  // and similarly for y

  int xIndex = blockIdx_x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx_y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx_y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx_x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

// --------------------------------------------------------------------
// Partial transposes
// NB: the coarse- and fine-grained routines only perform part of a 
//     transpose and will fail the test against the reference solution
//
//     They are used to assess performance characteristics of different
//     components of a full transpose
// --------------------------------------------------------------------

__global__ void transposeFineGrained(float *odata, float *idata, int width, int height,  int nreps)
{
  __shared__ float block[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index = xIndex + (yIndex)*width;

  for (int r=0; r<nreps; r++) {
    for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
      block[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
    }  
     
    __syncthreads();

    for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
      odata[index+i*height] = block[threadIdx.x][threadIdx.y+i];
    }
  }
}


__global__ void transposeCoarseGrained(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float block[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r<nreps; r++) {
    for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) {
      block[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();

    for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) {
      odata[index_out+i*height] = block[threadIdx.y+i][threadIdx.x];
    }
  }
}


// ---------------------
// host utility routines
// ---------------------

void computeTransposeGold(float* gold, float* idata,
			  const  int size_x, const  int size_y)
{
  for(  int y = 0; y < size_y; ++y) {
    for(  int x = 0; x < size_x; ++x) {
      gold[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}


void getParams(int argc, char **argv, hipDeviceProp_t &deviceProp, int &size_x, int &size_y, int max_tile_dim)
{
    // set matrix size (if (x,y) dim of matrix is not square, then this will have to be modified
  if ( cutGetCmdLineArgumenti(argc, (const char **) argv, "dimx", &size_x) || 
	   cutGetCmdLineArgumenti(argc, (const char **) argv, "dimX", &size_x) ) {
      if (size_x > max_tile_dim) {
          shrLog("> MatrixSize X = %d is greater than the recommended size = %d\n", size_x, max_tile_dim);
      } else {
          shrLog("> MatrixSize X = %d\n", size_x);
      }
  } else {
      size_x = max_tile_dim;
      // If this is SM12 hardware, we want to round down to a multiple of 512
      if (deviceProp.major >= 1 && deviceProp.minor >= 2) {
          size_x = FLOOR(size_x, 512);
      } else { // else for SM10,SM11 we round down to a multiple of 384
          size_x = FLOOR(size_x, 384);
      }
  }

  if ( cutGetCmdLineArgumenti(argc, (const char **) argv, "dimy", &size_y) ||
	   cutGetCmdLineArgumenti(argc, (const char **) argv, "dimY", &size_y) ) {
      if (size_y > max_tile_dim) {
          shrLog("> MatrixSize Y = %d is greater than the recommended size = %d\n", size_y, max_tile_dim);
      } else {
          shrLog("> MatrixSize Y = %d\n", size_y);
      }
  } else {
      size_y = max_tile_dim;
      // If this is SM12 hardware, we want to round down to a multiple of 512
      if (deviceProp.major >= 1 && deviceProp.minor >= 2) {
          size_y = FLOOR(size_y, 512);
      } else { // else for SM10,SM11 we round down to a multiple of 384
          size_y = FLOOR(size_y, 384);
      }
  }
}


void
showHelp()
{
  shrLog("\n> Command line options\n", sSDKsample);
  shrLog("\t-device=n          (where n=0,1,2.... for the GPU device)\n\n");
  shrLog("> The default matrix size can be overridden with these parameters\n");
  shrLog("\t-dimX=row_dim_size (matrix row    dimensions)\n");
  shrLog("\t-dimY=col_dim_size (matrix column dimensions)\n");
}


// ----
// main
// ----

int
main( int argc, char** argv) 
{
  // Start logs
  shrSetLogFileName ("transpose.txt");
  shrLog("%s Starting...\n\n", argv[0]);

  if( cutCheckCmdLineFlag(argc, (const char**)argv, "help") ) {
    showHelp();
    return 0;
  }

  // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
    cutilDeviceInit(argc, argv);
  else
    hipSetDevice( cutGetMaxGflopsDeviceId() );

  int devID;
  hipDeviceProp_t deviceProp;

  // get number of SMs on this GPU
  cutilSafeCall(hipGetDevice(&devID));
  cutilSafeCall(hipGetDeviceProperties(&deviceProp, devID));

  // compute the scaling factor (for GPUs with fewer MPs)
  float scale_factor, total_tiles;
  scale_factor = max((24.0f / (float)deviceProp.multiProcessorCount), 1.0f);

  if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        shrLog("[%s] - there is no device supporting CUDA.\n", sSDKsample);
        hipDeviceReset();
        cutilExit(argc, argv);
  } else {
        shrLog("> Device %d: \"%s\"\n", devID, deviceProp.name);
        shrLog("> SM Capability %d.%d detected:\n", deviceProp.major, deviceProp.minor);
  }

  // Calculate number of tiles we will run for the Matrix Transpose performance tests
  int size_x, size_y, max_matrix_dim, matrix_size_test;

  if (deviceProp.major >= 1 && deviceProp.minor >= 2) {
      matrix_size_test = 512;  // we round down max_matrix_dim for this perf test
      total_tiles = (float)MAX_TILES_SM12 / scale_factor;
  } else {
      matrix_size_test = 384;  // we round down max_matrix_dim for this perf test
      total_tiles = (float)MAX_TILES_SM10 / scale_factor;
  }
  max_matrix_dim = FLOOR((int)(floor(sqrt(total_tiles))* TILE_DIM), matrix_size_test);
 
  shrLog("> CUDA device has %d Multi-Processors\n",   deviceProp.multiProcessorCount);
  shrLog("> SM performance scaling factor = %4.2f\n", scale_factor);

  // Extract parameters if there are any, command line -dimx and -dimy can override 
  // any of these settings
  getParams(argc, argv, deviceProp, size_x, size_y, max_matrix_dim);

  if (size_x != size_y) {
      shrLog("\n[%s] does not support non-square matrices (row_dim_size(%d) != col_dim_size(%d))\nExiting...\n\n", sSDKsample, size_x, size_y);
      shrLog("FAILED...\n\n");
      hipDeviceReset();
      cutilExit(argc, argv);
  }

  if (size_x%TILE_DIM != 0 || size_y%TILE_DIM != 0) {
    shrLog("[%s] Matrix size must be integral multiple of tile size\nExiting...\n\n", sSDKsample);
    shrLog("FAILED\n\n");
    hipDeviceReset();
    cutilExit(argc, argv);
  }

  // kernel pointer and descriptor
  void (*kernel)(float *, float *, int, int, int);
  char *kernelName;

  // execution configuration parameters
  dim3 grid(size_x/TILE_DIM, size_y/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);

  // CUDA events
  hipEvent_t start, stop;

  // size of memory required to store the matrix
  const  int mem_size = sizeof(float) * size_x*size_y;

  // allocate host memory
  float *h_idata = (float*) malloc(mem_size);
  float *h_odata = (float*) malloc(mem_size);
  float *transposeGold = (float *) malloc(mem_size);  
  float *gold;

  // allocate device memory
  float *d_idata, *d_odata;
  cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size) );
  cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size) );

  // initalize host data
  for(  int i = 0; i < (size_x*size_y); ++i)
    h_idata[i] = (float) i;
  
  // copy host data to device
  cutilSafeCall( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );

  // Compute reference transpose solution
  computeTransposeGold(transposeGold, h_idata, size_x, size_y);

  // print out common data for all kernels
  shrLog("\nMatrix size: %dx%d (%dx%d tiles), tile size: %dx%d, block size: %dx%d\n\n", 
	 size_x, size_y, size_x/TILE_DIM, size_y/TILE_DIM, TILE_DIM, TILE_DIM, TILE_DIM, BLOCK_ROWS);

  // initialize events
  cutilSafeCall( hipEventCreate(&start) );
  cutilSafeCall( hipEventCreate(&stop) );

  //
  // loop over different kernels
  //

  CUTBoolean success = CUTTrue;

  for (int k = 0; k<8; k++) {
    // set kernel pointer
    switch (k) {
    case 0:
      kernel = &copy;                           kernelName = "simple copy       "; break;
    case 1:
      kernel = &copySharedMem;                  kernelName = "shared memory copy"; break;
    case 2:
      kernel = &transposeNaive;                 kernelName = "naive             "; break;
    case 3:
      kernel = &transposeCoalesced;             kernelName = "coalesced         "; break;
    case 4:
      kernel = &transposeNoBankConflicts;       kernelName = "optimized         "; break;
    case 5:
      kernel = &transposeCoarseGrained;         kernelName = "coarse-grained    "; break;
    case 6:
      kernel = &transposeFineGrained;           kernelName = "fine-grained      "; break;
    case 7:
      kernel = &transposeDiagonal;              kernelName = "diagonal          "; break;
    }      

    // set reference solution
    if (kernel == &copy || kernel == &copySharedMem) {
      gold = h_idata;
    } else if (kernel == &transposeCoarseGrained || kernel == &transposeFineGrained) {
      gold = h_odata;   // fine- and coarse-grained kernels are not full transposes, so bypass check
    } else {
      gold = transposeGold;
    }

    // warmup to avoid timing startup
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, 1);

    // take measurements for loop over kernel launches
    cutilSafeCall( hipEventRecord(start, 0) );
    for (int i=0; i < NUM_REPS; i++) {
      kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, 1);
    }
    cutilSafeCall( hipEventRecord(stop, 0) );
    cutilSafeCall( hipEventSynchronize(stop) );
    float outerTime;
    cutilSafeCall( hipEventElapsedTime(&outerTime, start, stop) );    

    cutilSafeCall( hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost) );
    CUTBoolean res = cutComparef(gold, h_odata, size_x*size_y);
    if (res == CUTFalse) {
      shrLog("*** %s kernel FAILED ***\n", kernelName);
      success = CUTFalse;
    }

    // take measurements for loop inside kernel
    cutilSafeCall( hipEventRecord(start, 0) );
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, NUM_REPS);
    cutilSafeCall( hipEventRecord(stop, 0) );
    cutilSafeCall( hipEventSynchronize(stop) );
    float innerTime;
    cutilSafeCall( hipEventElapsedTime(&innerTime, start, stop) );    

    cutilSafeCall( hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost) );
    res = cutComparef(gold, h_odata, size_x*size_y);
    if (res == CUTFalse) {
      shrLog("*** %s kernel FAILED ***\n", kernelName);
      success = CUTFalse;
    }
    
    // report effective bandwidths
    float outerBandwidth = 2.0f * 1000.0f * mem_size/(1024*1024*1024)/(outerTime/NUM_REPS);
    float innerBandwidth = 2.0f * 1000.0f * mem_size/(1024*1024*1024)/(innerTime/NUM_REPS);
    
    shrLog("\n");
    shrLogEx(LOGBOTH | MASTER, 0, "transpose-Outer-%s, Throughput = %.4f GB/s, Time = %.5f s, Size = %u fp32 elements, NumDevsUsed = %u, Workgroup = %u\n", 
           kernelName,
           outerBandwidth, 
           outerTime/NUM_REPS, 
           (size_x * size_y), 1, TILE_DIM * BLOCK_ROWS); 
    shrLogEx(LOGBOTH | MASTER, 0, "transpose-Inner-%s, Throughput = %.4f GB/s, Time = %.5f s, Size = %u fp32 elements, NumDevsUsed = %u, Workgroup = %u\n", 
           kernelName,
           innerBandwidth, 
           innerTime/NUM_REPS, 
           (size_x * size_y), 1, TILE_DIM * BLOCK_ROWS);
                       
   }
  
  shrLog("\n%s\n\n", (success == CUTTrue) ? "PASSED" : "FAILED");

  // cleanup
  free(h_idata);
  free(h_odata);
  free(transposeGold);
  cutilSafeCall( hipFree(d_idata) );
  cutilSafeCall( hipFree(d_odata) );

  cutilSafeCall( hipEventDestroy(start) );
  cutilSafeCall( hipEventDestroy(stop) );
  
  hipDeviceReset();
  shrEXIT(argc, (const char**)argv);
  return 0;

}

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 *
 * Quadro and Tesla GPUs with compute capability >= 2.0 can overlap two memcopies 
 * with kernel execution. This sample illustrates the usage of CUDA streams to 
 * achieve overlapping of kernel execution with copying data to and from the device. 
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5 
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

const char *sSDKname = "simpleMultiCopy";

#include <stdio.h>
#include <cutil_inline.h>

#include <shrUtils.h>


__global__ void incKernel(int *g_out, int *g_in, int N, int inner_reps)
{ 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if( idx < N ) {
        for( int i=0; i<inner_reps; ++i ) {        
            g_out[idx] = g_in[idx] + 1;      
        }
    }
}

#define STREAM_COUNT 4


// Uncomment to simulate data source/sink IO times
//#define SIMULATE_IO


int *h_data_source;
int *h_data_sink;

int *h_data_in[STREAM_COUNT];
int *d_data_in[STREAM_COUNT];

int *h_data_out[STREAM_COUNT];
int *d_data_out[STREAM_COUNT];


hipEvent_t cycleDone[STREAM_COUNT];
hipStream_t stream[STREAM_COUNT];

hipEvent_t start, stop;
    
int N = 1 << 22;
int nreps = 10;                 // number of times each experiment is repeated
int inner_reps = 5;

int memsize;

dim3 block(512);
dim3 grid;
    
int thread_blocks;


float processWithStreams(int streams_used);
void init();
bool test();


int main(int argc, char *argv[])
{
    int cuda_device = 0;
    float scale_factor;
    
    printf("[%s]\n", sSDKname);

    if(shrCheckCmdLineFlag(argc, (const char**)argv, "device"))
    {
       cuda_device = cutilDeviceInit(argc, argv);
       if (cuda_device < 0) {
          printf("exiting...\n");
		  cutilExit(argc, argv);
          exit(0);
      }
    }

    hipDeviceProp_t deviceProp;	
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );

    // Anything that is less than 4 SM's will have scaled down workload
    scale_factor = max((4.0f / (float)deviceProp.multiProcessorCount), 1.0f);
    N = (int)( (float)N / scale_factor );

    printf("> Device name: %s\n", deviceProp.name);
    printf("> CUDA Capability %d.%d hardware with %d multi-processors\n", 
		deviceProp.major, deviceProp.minor, 
		deviceProp.multiProcessorCount); 
    printf("> scale_factor = %.2f\n", 1.0f/scale_factor);
    printf("> array_size   = %d\n\n", N);

    memsize = N * sizeof(int);

    thread_blocks = N / block.x;
    
    grid.x = thread_blocks % 65535;
    grid.y = (thread_blocks / 65535 + 1);    
 

    // Allocate resources

    h_data_source = (int*) malloc(memsize);
    h_data_sink = (int*) malloc(memsize);    

    for( int i =0; i<STREAM_COUNT; ++i ) {
        
        CUDA_SAFE_CALL( hipHostAlloc(&h_data_in[i], memsize, 
			hipHostMallocDefault) );
        CUDA_SAFE_CALL( hipMalloc(&d_data_in[i], memsize) );
        
        CUDA_SAFE_CALL( hipHostAlloc(&h_data_out[i], memsize, 
			hipHostMallocDefault) );
        CUDA_SAFE_CALL( hipMalloc(&d_data_out[i], memsize) );

        
        CUDA_SAFE_CALL( hipStreamCreate(&stream[i]) );
        CUDA_SAFE_CALL( hipEventCreate(&cycleDone[i]) ); 
        
        hipEventRecord(cycleDone[i], stream[i]);
    }

    hipEventCreate(&start); hipEventCreate(&stop);

    init();

    // Kernel warmup
    incKernel<<<grid, block>>>(d_data_out[0], d_data_in[0], N, inner_reps);

    
    // Time copies and kernel    
    hipEventRecord(start,0);
    CUDA_SAFE_CALL( hipMemcpyAsync(d_data_in[0], h_data_in[0], memsize, 
		hipMemcpyHostToDevice,0) );
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    float memcpy_h2d_time;    
    hipEventElapsedTime(&memcpy_h2d_time, start, stop);

    
    hipEventRecord(start,0);
    CUDA_SAFE_CALL( hipMemcpyAsync(h_data_out[0], d_data_out[0], memsize, 
		hipMemcpyDeviceToHost, 0) );        
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    float memcpy_d2h_time;    
    hipEventElapsedTime(&memcpy_d2h_time, start, stop);
    
    hipEventRecord(start,0);
    incKernel<<<grid, block,0,0>>>(d_data_out[0], d_data_in[0], N, inner_reps);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    float kernel_time;    
    hipEventElapsedTime(&kernel_time, start, stop);
    
    printf("Measured timings (throughput):\n");
    printf(" Memcpy host to device\t: %f ms (%f GB/s)\n", 
		memcpy_h2d_time, (memsize * 1e-6)/ memcpy_h2d_time );
    printf(" Memcpy device to host\t: %f ms (%f GB/s)\n", 
		memcpy_d2h_time, (memsize * 1e-6)/ memcpy_d2h_time);
    printf(" Kernel\t\t\t: %f ms (%f GB/s)\n", 
		kernel_time, (inner_reps * memsize * 2e-6)/ kernel_time);
	
    int type=0;
    if( deviceProp.major == 2 && strstr(deviceProp.name, "GeForce") == 0 )
        type = 2;
    else if( deviceProp.deviceOverlap )
        type = 1;

    printf("\nTheoretical limits for overlaps (* capability of this device):\n");
    printf(" %s        c <  1.0 \t: %f ms (No overlap, fully serial)\n", 
		type == 0 ? "*" : " ", memcpy_h2d_time + memcpy_d2h_time + kernel_time);
    printf(" %s 1.1 <= c <  2.0 \t: %f ms (Compute overlaps with one memcopy)\n",
		type == 1 ? "*" : " ", max((memcpy_h2d_time + memcpy_d2h_time), kernel_time));
    printf(" %s        c >= 2.0 \t: %f ms (Compute overlaps with two memcopies)\n",
		type == 2 ? "*" : " ", max(max(memcpy_h2d_time,memcpy_d2h_time), kernel_time));

	// Process pipelined work
    float serial_time = processWithStreams(1);  
    float overlap_time = processWithStreams(STREAM_COUNT);  

    printf("\nAverage measured timings over %d repetitions:\n", nreps);
    printf(" Avg. time when execution fully serialized\t: %f ms\n", 
		serial_time / nreps);
    printf(" Avg. time when overlapped using %d streams\t: %f ms\n", 
		STREAM_COUNT, overlap_time / nreps);
    printf(" Avg. latency hidden (serialized - overlapped)\t: %f ms\n", 
		(serial_time - overlap_time) / nreps);

    printf("\nMeasured throughput:\n");    
    printf(" Fully serialized execution\t\t: %f GB/s\n", 
		(nreps * (memsize * 2e-6))/ serial_time);
    printf(" Overlapped using %d streams\t\t: %f GB/s\n", 
		STREAM_COUNT, (nreps * (memsize * 2e-6))/ overlap_time);
    
    // Test result
    
    if(test())	
        printf("\nPASSED\n");
    else
        printf("\nFAILED\n");


    // Free resources

    free( h_data_source );
    free( h_data_sink );

    for( int i =0; i<STREAM_COUNT; ++i ) {
        
        hipHostFree(h_data_in[i]);
        hipFree(d_data_in[i]);

        hipHostFree(h_data_out[i]);
        hipFree(d_data_out[i]);
        
        hipStreamDestroy(stream[i]);
        hipEventDestroy(cycleDone[i]);        
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipDeviceReset();

    cutilExit(argc, argv);

    return 0;
}

float processWithStreams(int streams_used) {
    
    int current_stream = 0;

    float time;

    // Do processing in a loop
    //
    // Note: All memory commands are processed in the order  they are issued, 
    // independent of the stream they are enqueued in. Hence the pattern by 
    // which the copy and kernel commands are enqueued in the stream 
    // has an influence on the achieved overlap.

    hipEventRecord(start, 0);
    for( int i=0; i<nreps; ++i ) {
        int next_stream = (current_stream + 1 ) % streams_used;

#ifdef SIMULATE_IO
        // Store the result
        memcpy(h_data_sink, h_data_out[current_stream],memsize);            

        // Read new input        
        memcpy(h_data_in[next_stream], h_data_source, memsize);
#endif

        // Ensure that processing and copying of the last cycle has finished
        hipEventSynchronize(cycleDone[next_stream]);

        // Process current frame
        incKernel<<<grid, block, 0, stream[current_stream]>>>(
			d_data_out[current_stream],
			d_data_in[current_stream], 
			N, 
			inner_reps);
        
        // Upload next frame
        CUDA_SAFE_CALL( hipMemcpyAsync(
			d_data_in[next_stream], 
			h_data_in[next_stream], 
			memsize, 
			hipMemcpyHostToDevice, 
			stream[next_stream]) );

        // Download current frame
        CUDA_SAFE_CALL( hipMemcpyAsync(
			h_data_out[current_stream], 
			d_data_out[current_stream], 
			memsize, 
			hipMemcpyDeviceToHost, 
			stream[current_stream]) );    
        
        CUDA_SAFE_CALL( hipEventRecord(
			cycleDone[current_stream], 
			stream[current_stream]) );
        
        current_stream = next_stream;
    }
    hipEventRecord(stop, 0);    

    hipDeviceSynchronize();

    hipEventElapsedTime(&time, start, stop);

    return time;

}

void init() {    
    for( int i=0; i<N; ++i) {        
        h_data_source[i] = 0;
    }
    
    for( int i =0; i<STREAM_COUNT; ++i ) {
        memcpy(h_data_in[i], h_data_source, memsize);        
    }
}


bool test() {

    bool passed = true;

    for( int j =0; j<STREAM_COUNT; ++j ) {
        for( int i =0; i<N; ++i ) {
            passed &= (h_data_out[j][i] == 1);
        }
    }

    return passed;
}


#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs.
 *
 * There's one CUDA context per thread. To use multiple CUDA contexts you
 * have to create multiple threads. One for each GPU. For optimal performance,
 * the number of CPU cores should be equal to the number of GPUs in the system.
 *
 * Creating CPU threads has a certain overhead. So, this is only worth when you
 * have a significant amount of work to do per thread. It's also recommended to
 * create a pool of threads and reuse them to avoid this overhead.
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the 
 * application. On the other side, you can still extend your desktop to screens 
 * attached to both GPUs.
 */


////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA SDK sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N){
    const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;
    for(int pos = tid; pos < N; pos += threadN)
        sum += d_Input[pos];

    d_Result[tid] = sum;
}

extern "C" 
void launch_reduceKernel(float *d_Result, float *d_Input, int N, int BLOCK_N, int THREAD_N) 
{
	reduceKernel<<<BLOCK_N, THREAD_N>>>(d_Result, d_Input, N);
}

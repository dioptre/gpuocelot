/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 /*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include "simpleTemplates_kernel.cu"

int g_TotalFailures = 0;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
template <class T>
void runTest( int argc, char** argv, int len);

template<class T>
void
computeGold( T* reference, T* idata, const unsigned int len) 
{
    const T T_len = static_cast<T>( len);
    for( unsigned int i = 0; i < len; ++i) 
    {
        reference[i] = idata[i] * T_len;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    printf("[simpleTemplates]\n");

    printf("> runTest<float,32>\n");
    runTest<float>( argc, argv, 32);
    printf("> runTest<int,64>\n");
    runTest<int>( argc, argv, 64);

    printf("\n[simpleTemplates] -> Test Results: %d Failures\n", g_TotalFailures);
    printf( (g_TotalFailures == 0) ? "PASSED\n" : "FAILED\n" );

    cutilExit(argc, argv);
}

// To completely templatize runTest (below) with cutil, we need to use 
// template specialization to wrap up CUTIL's array comparison and file writing
// functions for different types.  

// Here's the generic wrapper for cutCompare*
template<class T>
class ArrayComparator
{
public:
    CUTBoolean compare( const T* reference, T* data, unsigned int len)
    {
        fprintf(stderr, "Error: no comparison function implemented for this type\n");
        return CUTFalse;
    }
};

// Here's the specialization for ints:
template<>
class ArrayComparator<int>
{
public:
    CUTBoolean compare( const int* reference, int* data, unsigned int len)
    {
        return cutComparei(reference, data, len);
    }
};

// Here's the specialization for floats:
template<>
class ArrayComparator<float>
{
public:
    CUTBoolean compare( const float* reference, float* data, unsigned int len)
    {
        return cutComparef(reference, data, len);
    }
};

// Here's the generic wrapper for cutWriteFile*
template<class T>
class ArrayFileWriter
{
public:
    CUTBoolean write(const char* filename, T* data, unsigned int len, float epsilon)
    {
        fprintf(stderr, "Error: no file write function implemented for this type\n");
        return CUTFalse;
    }
};

// Here's the specialization for ints:
template<>
class ArrayFileWriter<int>
{
public:
    CUTBoolean write(const char* filename, int* data, unsigned int len, float epsilon)
    {
        return cutWriteFilei(filename, data, len, epsilon != 0);
    }
};

// Here's the specialization for floats:
template<>
class ArrayFileWriter<float>
{
public:
    CUTBoolean write(const char* filename, float* data, unsigned int len, float epsilon)
    {
        return cutWriteFilef(filename, data, len, epsilon);
    }
};


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
template<class T>
void
runTest( int argc, char** argv, int len) 
{
    int devID;
    hipDeviceProp_t deviceProps;

	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
	    devID = cutilDeviceInit(argc, argv);
            if (devID < 0) {
               printf("exiting...\n");
               cutilExit(argc, argv);
               exit(0);
            }
	}
	else {
	    devID = cutGetMaxGflopsDeviceId();
	    hipSetDevice( devID );
	}
		
    // get number of SMs on this GPU
    cutilSafeCall(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    unsigned int num_threads = len;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    T* h_idata = (T*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (T) i;
    }

    // allocate device memory
    T* d_idata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // allocate device memory for result
    T* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
    testKernel<T><<< grid, threads, mem_size >>>( d_idata, d_odata);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // allocate mem for the result on host side
    T* h_odata = (T*) malloc( mem_size);
    // copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_odata, sizeof(T) * num_threads,
                                hipMemcpyDeviceToHost) );

    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));

    // compute reference solution
    T* reference = (T*) malloc( mem_size);
    computeGold<T>( reference, h_idata, num_threads);

    ArrayComparator<T> comparator;
    ArrayFileWriter<T> writer;

    // check result
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test
        cutilCheckError( writer.write( "./data/regression.dat",
                                     h_odata, num_threads, 0.0));
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        CUTBoolean res = comparator.compare( reference, h_odata, num_threads);
        printf( "Compare %s\n\n", (1 == res) ? "OK" : "MISMATCH");
        g_TotalFailures += (1 != res);
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));

    hipDeviceReset();
}

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This sample is an implementation of a simple line-of-sight algorithm:
// Given a height map and a ray originating at some observation point,
// it computes all the points along the ray that are visible from the
// observation point.
// It is based on the description made in "Guy E. Blelloch. Vector models
// for data-parallel computing. MIT Press, 1990" and uses the parallel scan
// primitive provided by the CUDPP library (http://www.gpgpu.org/developer/cudpp/).

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, library
#include "cudpp/cudpp.h"

// includes, project
#include "cutil_inline.h"
#include "cutil_math.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, types

// Boolean
typedef unsigned char Bool;
enum {
    False = 0,
    True = 1
};

// 2D height field
struct HeightField {
	int     width;
    float*  height;
};

// Ray
struct Ray {
	float3 origin;
    float2 dir;
    int    length;
    float  oneOverLength;
};

////////////////////////////////////////////////////////////////////////////////
// declaration, variables

// Height field texture reference
texture<float, 2, hipReadModeElementType> g_HeightFieldTex;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
__global__ void computeAngles_kernel(const Ray, float*);
__global__ void computeVisibilities_kernel(const float*, const float*, int, Bool*);
void lineOfSight_gold(const HeightField, const Ray, Bool*);
__device__ __host__ float2 getLocation(const Ray, int);
__device__ __host__ float getAngle(const Ray, float2, float);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a line-of-sight test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv) 
{
    ////////////////////////////////////////////////////////////////////////////
    // Device initialization

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    ////////////////////////////////////////////////////////////////////////////
    // Timer
    
    // Create
    uint timer;
    cutilCheckError(cutCreateTimer(&timer));
    
    // Number of iterations to get accurate timing
#ifdef __DEVICE_EMULATION__
    uint numIterations = 1;
#else
    uint numIterations = 100;
#endif

    ////////////////////////////////////////////////////////////////////////////
    // Height field
    
    HeightField heightField;
    
    // Allocate in host memory
    int2 dim = make_int2(10000, 100);
    heightField.width = dim.x;
    int heightFieldSize = dim.x * dim.y * sizeof(float);
    cutilSafeMalloc(heightField.height = (float*)malloc(heightFieldSize));
    
    // Fill in with an arbitrary sine surface
    for (int x = 0; x < dim.x; ++x)
        for (int y = 0; y < dim.y; ++y) {
            float amp = 0.1f * (x + y);
            float period = 2.0f + amp;
            *(heightField.height + dim.x * y + x) =
                amp * (sinf(sqrtf((float)(x * x + y * y)) * 2.0f * 3.1416f / period) + 1.0f);
        }
    
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* heightFieldArray;
    cutilSafeCall(hipMallocArray(&heightFieldArray, &channelDesc, dim.x, dim.y));
     
    // Initialize device memory
    cutilSafeCall(hipMemcpyToArray(heightFieldArray, 0, 0, heightField.height,
                                     heightFieldSize, hipMemcpyHostToDevice));

    // Set texture parameters
    g_HeightFieldTex.addressMode[0] = hipAddressModeClamp;
    g_HeightFieldTex.addressMode[1] = hipAddressModeClamp;
    g_HeightFieldTex.filterMode = hipFilterModePoint;
    g_HeightFieldTex.normalized = 0;

    // Bind CUDA array to texture reference
    cutilSafeCall(hipBindTextureToArray(g_HeightFieldTex, heightFieldArray,
                                          channelDesc));
                                          
    ////////////////////////////////////////////////////////////////////////////
    // Ray (starts at origin and traverses the height field diagonally)
    
    Ray ray;
    ray.origin = make_float3(0, 0, 2.0f);
    int2 dir = make_int2(dim.x - 1, dim.y - 1); 
    ray.dir = make_float2((float)dir.x, (float)dir.y);
    ray.length = max(abs(dir.x), abs(dir.y));
    ray.oneOverLength = 1.0f / ray.length;

    ////////////////////////////////////////////////////////////////////////////
    // View angles

    // Allocate view angles for each point along the ray
    float* d_angles;
    int raySize = ray.length * sizeof(float);
    cutilSafeCall(hipMalloc((void**)&d_angles, raySize));
    
    // Allocate result of max-scan operation on the array of view angles
    float* d_scannedAngles;
    cutilSafeCall(hipMalloc((void**)&d_scannedAngles, raySize));
    
    ////////////////////////////////////////////////////////////////////////////
    // Visibility results

    // Allocate visibility results for each point along the ray
    Bool* d_visibilities;
    cutilSafeCall(hipMalloc((void**)&d_visibilities, raySize));
    Bool* h_visibilities;
    cutilSafeMalloc(h_visibilities = (Bool*)malloc(raySize));
    Bool* h_visibilitiesRef;
    cutilSafeMalloc(h_visibilitiesRef = (Bool*)malloc(raySize));
    
    ////////////////////////////////////////////////////////////////////////////
    // Reference solution
    lineOfSight_gold(heightField, ray, h_visibilitiesRef);
    
    ////////////////////////////////////////////////////////////////////////////
    // Device solution

    // Execution configuration
    dim3 block(256);
    dim3 grid((uint)ceil(ray.length / (double)block.x));
    
    // Scan configuration
    CUDPPHandle scanPlan;
   
    CUDPPConfiguration config;
    config.algorithm = CUDPP_SCAN;
    config.op = CUDPP_MAX;
    config.datatype = CUDPP_FLOAT;  
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
    cudppPlan(&scanPlan, config, ray.length, 1, 0);
    
    // Compute device solution
    printf("Line of sight\n");
    cutStartTimer(timer);
    for (uint i = 0; i < numIterations; ++i) {
    
        // Compute view angle for each point along the ray
        computeAngles_kernel<<<grid, block>>>(ray, d_angles);
        cutilCheckMsg("Kernel execution failed");
        
        // Perform a max-scan operation on the array of view angles
        cudppScan(scanPlan, d_scannedAngles, d_angles, ray.length);
        cutilCheckMsg("Kernel execution failed");
        
        // Compute visibility results based on the array of view angles
        // and its scanned version
        computeVisibilities_kernel<<<grid, block>>>(d_angles, d_scannedAngles,
                                                    ray.length, d_visibilities);
        cutilCheckMsg("Kernel execution failed");
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    cudppDestroyPlan(scanPlan);
    cutilCheckMsg("Kernel execution failed");

    // Copy visibility results back to the host
    cutilSafeCall(hipMemcpy(h_visibilities, d_visibilities, raySize,
                              hipMemcpyDeviceToHost));
                              
    // Compare device visibility results against reference results
    CUTBoolean res = cutCompareub(h_visibilitiesRef, h_visibilities, ray.length);
    printf("%s \n", (1 == res) ? "PASSED" : "FAILED");
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    cutResetTimer(timer);

    // Cleanup memory
    free(heightField.height);
    free(h_visibilities);
    free(h_visibilitiesRef);
    cutilSafeCall(hipFree(d_angles));
    cutilSafeCall(hipFree(d_scannedAngles));
    cutilSafeCall(hipFree(d_visibilities));
    cutilSafeCall(hipFreeArray(heightFieldArray));

    hipDeviceReset();
}

////////////////////////////////////////////////////////////////////////////////
//! Compute view angles for each point along the ray
//! @param ray         ray
//! @param angles      view angles
////////////////////////////////////////////////////////////////////////////////
__global__ void computeAngles_kernel(const Ray ray, float* angles)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < ray.length) {
        float2 location = getLocation(ray, i + 1);
        float height = tex2D(g_HeightFieldTex, location.x, location.y);
        float angle = getAngle(ray, location, height);
        angles[i] = angle;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute visibility for each point along the ray
//! @param angles          view angles
//! @param scannedAngles   max-scanned view angles
//! @param numAngles       number of view angles
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
__global__ void computeVisibilities_kernel(const float* angles,
                                           const float* scannedAngles,
                                           int numAngles,
                                           Bool* visibilities)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numAngles)
        visibilities[i] = scannedAngles[i] <= angles[i];
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! @param heightField     height field
//! @param ray             ray
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
void lineOfSight_gold(const HeightField heightField, const Ray ray,
                      Bool* visibilities)
{
    float angleMax = asinf(-1.0f);
    for (int i = 0; i < ray.length; ++i) {
        float2 location = getLocation(ray, i + 1);
        float height = *(heightField.height
                       + heightField.width * (int)floorf(location.y)
                       + (int)floorf(location.x));
        float angle = getAngle(ray, location, height);
        if (angle > angleMax) {
            angleMax = angle;
            visibilities[i] = True;
        }
        else
            visibilities[i] = False;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the 2D coordinates of the point located at i steps from the origin
//! of the ray
//! @param ray      ray
//! @param i        integer offset along the ray
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float2 getLocation(const Ray ray, int i)
{
    float step = i * ray.oneOverLength;
    return make_float2(ray.origin.x, ray.origin.y) + step * ray.dir;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the angle of view between a 3D point and the origin of the ray
//! @param ray        ray
//! @param location   2D coordinates of the input point
//! @param height     height of the input point
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float getAngle(const Ray ray, float2 location, float height)
{
    float2 dir = location - make_float2(ray.origin.x, ray.origin.y);
    return atanf((height - ray.origin.z) / length(dir));
}

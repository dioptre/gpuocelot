#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11. 
 *
 */

// Utilities and system includes
#include <shrUtils.h>
#include "cutil_inline.h"

// includes, kernels
#include <matrixMul_kernel.cu>

static char *sSDKsample = "matrixMul";

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	printf("[ %s ]\n", sSDKsample);

    shrSetLogFileName ("matrixMul.txt");
    shrLog("%s Starting...\n\n", argv[0]);

    runTest(argc, argv);

    shrEXIT(argc, (const char**)argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv)
{
    if(shrCheckCmdLineFlag(argc, (const char**)argv, "device"))
    {
        cutilDeviceInit(argc, argv);
    }
    else
    {
        hipSetDevice(cutGetMaxGflopsDeviceId());
    }

    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    cutilSafeCall(hipGetDevice(&devID));
    cutilSafeCall(hipGetDeviceProperties(&props, devID));

    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

	// set seed for rand()
    srand(2006);

    // Optional Command-line multiplier for matrix sizes
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
    int iSizeMultiple = 1;
    shrGetCmdLineArgumenti(argc, (const char**)argv, "sizemult", &iSizeMultiple); 
    iSizeMultiple = CLAMP(iSizeMultiple, 1, 10);

	// For GPUs with fewer # of SM's, we limit the maximum size of the matrix
	if (props.multiProcessorCount <= 4) {
		uiWA = 2 * BLOCK_SIZE * iSizeMultiple;
		uiHA = 4 * BLOCK_SIZE * iSizeMultiple;
		uiWB = 2 * BLOCK_SIZE * iSizeMultiple;
		uiHB = 4 * BLOCK_SIZE * iSizeMultiple;
		uiWC = 2 * BLOCK_SIZE * iSizeMultiple;
		uiHC = 4 * BLOCK_SIZE * iSizeMultiple;
	} else {
		uiWA = WA * iSizeMultiple;
		uiHA = HA * iSizeMultiple;
		uiWB = WB * iSizeMultiple;
		uiHB = HB * iSizeMultiple;
		uiWC = WC * iSizeMultiple;
		uiHC = HC * iSizeMultiple;
	}
    shrLog("\nUsing Matrix Sizes: A(%u x %u), B(%u x %u), C(%u x %u)\n\n", 
            uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);

    // allocate host memory for matrices A and B
    unsigned int size_A = uiWA * uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)malloc(mem_size_A);
    unsigned int size_B = uiWB * uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    unsigned int size_C = uiWC * uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(uiWC / threads.x, uiHC / threads.y);

    // kernel warmup
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
    hipDeviceSynchronize();
    
    // create and start timer
    shrLog("Run Kernels...\n\n");
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    // execute the kernel
    int nIter = 30;
    for (int j = 0; j < nIter; j++) 
		{
            matrixMul<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
        }

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    hipDeviceSynchronize();
    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    double dSeconds = cutGetTimerValue(timer)/((double)nIter * 1000.0);
    double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
    double gflops = 1.0e-9 * dNumOps/dSeconds;

    //Log througput, etc
    shrLogEx(LOGBOTH | MASTER, 0, "matrixMul, Throughput = %.4f GFlop/s, Time = %.5f s, Size = %.0f Ops, NumDevsUsed = %d, Workgroup = %u\n", 
            gflops, dSeconds, dNumOps, 1, threads.x * threads.y);
    cutilCheckError(cutDeleteTimer(timer));

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );

    // compute reference solution
    shrLog("\nCheck against Host computation...\n\n");    
    float* reference = (float*)malloc(mem_size_C);
    computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

    // check result
    shrBOOL res = shrCompareL2fe(reference, h_C, size_C, 1.0e-6f);
    if (res != shrTRUE) 
    {
        printDiff(reference, h_C, uiWC, uiHC, 100, 1.0e-5f);
    }
    shrLog("%s \n\n", (shrTRUE == res) ? "PASSED" : "FAILED");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    shrLog("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;
    for (j = 0; j < height; j++) 
    {
        if (error_count < iListLength)
        {
            shrLog("\n  Row %d:\n", j);
        }
        for (i = 0; i < width; i++) 
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);
            if (fDiff > fListTol) 
            {                
                if (error_count < iListLength)
                {
                    shrLog("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    shrLog(" \n  Total Errors = %d\n\n", error_count);
}

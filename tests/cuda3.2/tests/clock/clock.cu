#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>

#include <cutil_inline.h>

#include "clock_kernel.cu"

// This example shows how to use the clock function to measure the performance of 
// a kernel accurately.
// 
// Blocks are executed in parallel and out of order. Since there's no synchronization
// mechanism between blocks, we measure the clock once for each block. The clock 
// samples are written to device memory.

#define NUM_BLOCKS    64
#define NUM_THREADS   256

// It's interesting to change the number of blocks and the number of threads to 
// understand how to keep the hardware busy.
//
// Here are some numbers I get on my G80:
//    blocks - clocks
//    1 - 3096
//    8 - 3232
//    16 - 3364
//    32 - 4615
//    64 - 9981
//
// With less than 16 blocks some of the multiprocessors of the device are idle. With
// more than 16 you are using all the multiprocessors, but there's only one block per
// multiprocessor and that doesn't allow you to hide the latency of the memory. With
// more than 32 the speed scales linearly.

int main(int argc, char** argv)

{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s

    if ( cutCheckCmdLineFlag(argc, (const char **)argv, "device")) {
        int devID = cutilDeviceInit(argc, argv);
        if (devID < 0) {
           printf("exiting...\n");
           cutilExit(argc, argv);
           exit(0);
        }
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }

    float * dinput = NULL;
    float * doutput = NULL;
    clock_t * dtimer = NULL;

    clock_t timer[NUM_BLOCKS * 2];
    float input[NUM_THREADS * 2];

    for (int i = 0; i < NUM_THREADS * 2; i++)
    {
        input[i] = (float)i;
    }

    cutilSafeCall(hipMalloc((void**)&dinput, sizeof(float) * NUM_THREADS * 2));
    cutilSafeCall(hipMalloc((void**)&doutput, sizeof(float) * NUM_BLOCKS));
    cutilSafeCall(hipMalloc((void**)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

    cutilSafeCall(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2, hipMemcpyHostToDevice));

    timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS>>>(dinput, doutput, dtimer);

    //cutilSafeCall(hipMemcpy(output, doutput, sizeof(float) * NUM_BLOCKS, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2, hipMemcpyDeviceToHost));

    cutilSafeCall(hipFree(dinput));
    cutilSafeCall(hipFree(doutput));
    cutilSafeCall(hipFree(dtimer));

    // This test always passes.
    printf( "PASSED\n");

    // Compute the difference between the last block end and the first block start.
    clock_t minStart = timer[0];
    clock_t maxEnd = timer[NUM_BLOCKS];

    for (int i = 1; i < NUM_BLOCKS; i++)
    {
        minStart = timer[i] < minStart ? timer[i] : minStart;
        maxEnd = timer[NUM_BLOCKS+i] > maxEnd ? timer[NUM_BLOCKS+i] : maxEnd;
    }

    printf("time = %d\n", maxEnd - minStart);

    hipDeviceReset();

    cutilExit(argc, argv);
}

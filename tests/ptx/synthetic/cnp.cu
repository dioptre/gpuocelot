#include "hip/hip_runtime.h"
#include <cnp_runtime_api.h>

__device__ int array[10];

__global__ void nestedCall(int count)
{
	array[count] = count;

	if(count > 0) nestedCall<<<1, 1>>>(count - 1);
}

__global__ void mainKernel()
{
	nestedCall<<<1, 1>>>(10);
}




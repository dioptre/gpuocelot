#include "hip/hip_runtime.h"
/*!
	\file computebound.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\date Nov 5, 2010; March 8, 2011
*/

// C++ includes
#include <stdio.h>
#include <stdlib.h>

// Hydrazine includes
#include <hydrazine/implementation/Timer.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

#ifndef TOOLCHAIN
#define TOOLCHAIN 0
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////

#define Iterations 1536

#define BLOCK1 { p1 = p1 * a + b; p2 = p2 * b + c; p3 = p3 * c + d; p4 = p4 * d + e; }

#define BLOCK2 { p1 = p1 * a + b; q1 = q1 + a; p2 = p2 * b + c; q2 = q2 + b; \
	p3 = p3 * c + d; q3 = q3 + c; p4 = p4 * d + e; q4 = q4 + d; }
	
#define BLOCK3 { p1 = p1 * a; p2 = p2 * b; p3 = p3 * c; p4 = p4 * d; }

#define BLOCK4 { p1 = p1 * a + a; p2 = p2 * b + b; p3 = p3 * c + c; p4 = p4 * d + d; }

#define BLOCK5 { p1 = p1 * a; q1 = q1 + a; p2 = p2 * b; q2 = q2 + b; \
	p3 = p3 * c; q3 = q3 + c; p4 = p4 * d; q4 = q4 + d; }

#define BLOCKN BLOCK4

#define GROUP1 BLOCKN BLOCKN BLOCKN BLOCKN
#define GROUP2 GROUP1 GROUP1 GROUP1 GROUP1
#define GROUP4 GROUP2 GROUP2 GROUP2 GROUP2
#define GROUP8 GROUP4 GROUP4 GROUP4 GROUP4
#define GROUP16 GROUP8 GROUP8 GROUP8 GROUP8

#define MADs 0
#define ADDs 2048
#define MULs 2048
__global__ void FloatComputeBound(
	float *A, const int N,
	const float a, const float b, const float c, const float d, const float e, int store) {
	int tid = threadIdx.x;
	
	float q1 = A[tid] - 1;
	float q2 = A[tid] * 0.75f;
	float q3 = A[tid] * 0.25f;
	float q4 = A[tid] * 0.5f;

	float p1 = A[tid] - 1;
	float p2 = A[tid] * 1.25f;
	float p3 = A[tid] * 0.75f;
	float p4 = A[tid] * 0.5f;

	for (int i = 0; i < Iterations; i++) {
		GROUP8
		GROUP8
	}	

	if (store) {
		A[tid] = p1 - q1 + p2 - q2 + p3 - q3 + p4 - q4;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////

void run(const int M) {
	
	float *A_cpu;
	float *A_gpu;
	
	const size_t N = 576;
	
	const size_t kernelLaunches = 8;
	const size_t K = M * N;
	size_t bytes = sizeof(float)*K;
	
	A_cpu = (float *)malloc(bytes);
	int i;
	for (i = 0; i < K; i++) {
		A_cpu[i] = (float)i / (float)(K - 1);
	}
	hipMalloc((void **)&A_gpu, bytes);
	
	hydrazine::Timer timer;
	
	timer.start();
	
	for (size_t i = 0; i < kernelLaunches; i++) {
		dim3 grid(M,1);
		dim3 block(N, 1, 1);
		FloatComputeBound<<< grid, block >>>(A_gpu, 1, 1.125f, 0.125f, 0.5f, 1.25f, 1.25f, 0);
	}
	
	timer.stop();
	
	double s = timer.seconds();
	double GFLOPs = (MADs + ADDs + MULs) / 1.0e6 * Iterations * K / 1.0e3 * kernelLaunches;
	double GFLOPsPerSec = (GFLOPs / s);
	
	printf("%d\t\t%f\n", M, GFLOPsPerSec );
	
	hipFree(A_gpu);
	free(A_cpu);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main() {
	int i;
	printf("\n%s\n# blocks\tGFLOPs/second\n", (TOOLCHAIN == 1 ? "gcc" : (TOOLCHAIN == 2 ? "llvm" : "")));
	run(8);
	for (i = 8; i <= 33; i+=8) {
		run(i);
	}
	return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////


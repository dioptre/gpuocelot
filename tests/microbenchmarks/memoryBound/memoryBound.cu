#include "hip/hip_runtime.h"
/*!
	\file memoryBound.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief test application measuring achieved device-to-device memory bandwidth 
*/

#define LINESIZE 64
#define BlockSize 4096

#include <hydrazine/implementation/Timer.h>
#include <stdio.h>
#include <assert.h>

#define cudaCheckCall(x) { hipError_t result = x; if (result != hipSuccess) \
	{ printf("Error: %s\n", hipGetErrorString(result)); assert(0); } }

extern "C" __global__ void kernel_MemoryBound(float *dest, float *source, int blockLines) {
	int tid = threadIdx.x;
	
	int blockLineSize = sizeof(float)*LINESIZE;
	int blockSize = blockLineSize * blockLines;
	int blockOffset = blockSize * blockIdx.x;
	
	float *destPtr = dest + tid + blockOffset;
	float *sourcePtr = source + tid + blockOffset;
	
	for (int i = 0; i < blockLines; i++) {
		*destPtr = *sourcePtr;
		destPtr += blockLineSize;
		sourcePtr += blockLineSize;
	}
}

void run(const int N) {
	int blockLines = BlockSize / LINESIZE;
	
	dim3 grid(N / BlockSize, 1);
	dim3 block(LINESIZE, 1);
	
	float *A_gpu_dest, *A_gpu_source;
	
	size_t bytes = sizeof(float)*N;
	cudaCheckCall(hipMalloc((void **)&A_gpu_dest, bytes));
	cudaCheckCall(hipMalloc((void **)&A_gpu_source, bytes));
	
	hydrazine::Timer timer;
	timer.start();
	
	kernel_MemoryBound<<< grid, block >>>(A_gpu_dest, A_gpu_source, blockLines);
	
	timer.stop();
	
	double transferSize = N * 2.0;
	double GBperSec = transferSize / timer.seconds() / 1.0e9;
	printf("Bandwidth: %f\n", GBperSec);
}

int main(int argc, char *arg[]) {

	run((10 << 20));
	
	return 0;
}


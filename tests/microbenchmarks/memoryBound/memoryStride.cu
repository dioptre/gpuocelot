#include "hip/hip_runtime.h"
/*!
	\file memoryStride.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief test application measuring achieved device-to-device memory bandwidth 
*/

#define LINESIZE 64
#define BlockSize 4096

#include <hydrazine/implementation/Timer.h>
#include <stdio.h>
#include <assert.h>

#define cudaCheckCall(x) { hipError_t result = x; if (result != hipSuccess) \
	{ printf("Error: %s\n", hipGetErrorString(result)); assert(0); } }

extern "C" __global__ void kernel_MemoryStride(int *dest, int threadCount, int threadStride) {
	int tid = threadIdx.x;
	
	dest += tid;
	for (int i = 0; i < threadCount; i++) {
		*dest = tid;
		dest += threadStride;
	}
}

double run(const int N, int threads, int stride) {
	int *A_gpu_dest;
	
	size_t bytes = sizeof(int) * N * threads;
	cudaCheckCall(hipMalloc((void **)&A_gpu_dest, bytes));
	
	dim3 grid(1, 1);
	dim3 block(threads, 1, 1);
	
	hydrazine::Timer timer;
	timer.start();
	kernel_MemoryStride<<< grid, block >>>(A_gpu_dest, threads, stride);
	timer.stop();
	
	hipFree(A_gpu_dest);
	
	double transferSize = (double)bytes;
	double GBperSec = transferSize / timer.seconds() / 1.0e9;
	
	return GBperSec;
}

int main(int argc, char *arg[]) {
	const int N = (1 << 14);
	for (int i = 1; i < 512; i++) {
		double bw_unit = run(N, i, 1);
		double bw_inter = run(N, i, i);
		printf("(%d, %f, %f),\n", i, bw_unit, bw_inter);
	}
	return 0;
}


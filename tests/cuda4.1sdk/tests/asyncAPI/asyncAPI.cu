#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 *
 * This sample illustrates the usage of CUDA events for both GPU timing and
 * overlapping CPU and GPU execution.  Events are insterted into a stream
 * of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
 * perform computations while GPU is executing (including DMA memcopies
 * between the host and device).  CPU can query CUDA events to determine
 * whether GPU has completed tasks.
 *
*/

#include <stdio.h>
#include <cutil_inline.h>
#include <shrQATest.h>

__global__ void increment_kernel(int *g_data, int inc_value)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   g_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++)
        if(data[i] != x)
            return 0;
    return 1;
}

int main(int argc, char *argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;

    shrQAStart(argc, argv);

    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        devID = cutilDeviceInit(argc, argv);
        if (devID < 0) {
            printf("No CUDA Capable devices found, exiting...\n");
            shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
        }
    } else {
        devID = cutGetMaxGflopsDeviceId();
        hipSetDevice( devID );
    }

    // get device name 
    cutilSafeCall(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

    // allocate host memory
    int *a = 0;
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) );
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );
    cutilSafeCall( hipMemset(d_a, 255, nbytes) );

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    cutilSafeCall( hipEventCreate(&start) );
    cutilSafeCall( hipEventCreate(&stop)  );
    
    unsigned int timer;
    cutilCheckError(  cutCreateTimer(&timer)  );
    cutilCheckError(  cutResetTimer(timer)    );
    cutilSafeCall( cutilDeviceSynchronize() );
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    cutilCheckError( cutStartTimer(timer) );
        hipEventRecord(start, 0);
        hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
        increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
        hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
        hipEventRecord(stop, 0);
    cutilCheckError( cutStopTimer(timer) );

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;
    while( hipEventQuery(stop) == hipErrorNotReady )
    {
        counter++;
    }
    cutilSafeCall( hipEventElapsedTime(&gpu_time, start, stop) );

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", cutGetTimerValue(timer) );
    printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    bool bFinalResults = (bool)correct_output(a, n, value);
	
    // release resources
    cutilSafeCall( hipEventDestroy(start) );
    cutilSafeCall( hipEventDestroy(stop) );
    cutilSafeCall( hipHostFree(a) );
    cutilSafeCall( hipFree(d_a) );

    cutilDeviceReset();

	shrQAFinishExit(argc, (const char **)argv, (bFinalResults ? QA_PASSED : QA_FAILED));
}

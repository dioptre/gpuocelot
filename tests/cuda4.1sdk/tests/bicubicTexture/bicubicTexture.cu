#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <cutil_math.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include <bicubicTexture_kernel.cuh>

hipArray *d_imageArray = 0;

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    cutilSafeCall( hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight) ); 
    uint size = imageWidth * imageHeight * sizeof(uchar);
    cutilSafeCall( hipMemcpyToArray(d_imageArray, 0, 0, h_data, size, hipMemcpyHostToDevice) );
    cutFree(h_data);

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;    // access with integer texture coordinates

    cutilCheckMsg("initTexture");

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray(tex, d_imageArray) );

    // bind same array to 2nd texture reference with point sampling
    tex2.addressMode[0] = hipAddressModeClamp;
    tex2.addressMode[1] = hipAddressModeClamp;
    tex2.filterMode = hipFilterModePoint;
    tex2.normalized = false;    // access with integer texture coordinates

    cutilSafeCall( hipBindTextureToArray(tex2, d_imageArray) );
}

extern "C"
void freeTexture()
{
    cutilSafeCall(hipFreeArray(d_imageArray));
}


// render image using CUDA
extern "C" 
void render(int width, int height, float tx, float ty, float scale, float cx, float cy, 
            dim3 blockSize, dim3 gridSize, int mode, uchar4 *output)
{
    // call CUDA kernel, writing results to PBO memory
    switch(mode) {
    case MODE_NEAREST:
        tex.filterMode = hipFilterModePoint;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BILINEAR:
        tex.filterMode = hipFilterModeLinear;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BICUBIC:
        tex.filterMode = hipFilterModePoint;
        d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_FAST_BICUBIC:
        tex.filterMode = hipFilterModeLinear;
        d_renderFastBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_CATROM:
        tex.filterMode = hipFilterModePoint;
        d_renderCatRom<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    }
    cutilCheckMsg("kernel failed");
}

#endif

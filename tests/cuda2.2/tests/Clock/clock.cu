#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdio.h>
#include <stdlib.h>

#include <cutil_inline.h>

#include "clock_kernel.cu"

// This example shows how to use the clock function to measure the performance of 
// a kernel accurately.
// 
// Blocks are executed in parallel and out of order. Since there's no synchronization
// mechanism between blocks, we measure the clock once for each block. The clock 
// samples are written to device memory.

#define NUM_BLOCKS    64
#define NUM_THREADS   256

// It's interesting to change the number of blocks and the number of threads to 
// understand how to keep the hardware busy.
//
// Here are some numbers I get on my G80:
//    blocks - clocks
//    1 - 3096
//    8 - 3232
//    16 - 3364
//    32 - 4615
//    64 - 9981
//
// With less than 16 blocks some of the multiprocessors of the device are idle. With
// more than 16 you are using all the multiprocessors, but there's only one block per
// multiprocessor and that doesn't allow you to hide the latency of the memory. With
// more than 32 the speed scales linearly.

int main(int argc, char** argv)

{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s

    if ( cutCheckCmdLineFlag(argc, (const char **)argv, "device"))
	    cutilDeviceInit(argc, argv);
    else
	    hipSetDevice( cutGetMaxGflopsDeviceId() );


    float * dinput = NULL;
    float * doutput = NULL;
    clock_t * dtimer = NULL;

    clock_t timer[NUM_BLOCKS * 2];
    float input[NUM_THREADS * 2];

    for (int i = 0; i < NUM_THREADS * 2; i++)
    {
        input[i] = (float)i;
    }

    cutilSafeCall(hipMalloc((void**)&dinput, sizeof(float) * NUM_THREADS * 2));
    cutilSafeCall(hipMalloc((void**)&doutput, sizeof(float) * NUM_BLOCKS));
    cutilSafeCall(hipMalloc((void**)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

    cutilSafeCall(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2, hipMemcpyHostToDevice));

    timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS>>>(dinput, doutput, dtimer);

    //cutilSafeCall(hipMemcpy(output, doutput, sizeof(float) * NUM_BLOCKS, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2, hipMemcpyDeviceToHost));

    cutilSafeCall(hipFree(dinput));
    cutilSafeCall(hipFree(doutput));
    cutilSafeCall(hipFree(dtimer));

    // This test always passes.
    printf( "Test PASSED\n");

    // Compute the difference between the last block end and the first block start.
    clock_t minStart = timer[0];
    clock_t maxEnd = timer[NUM_BLOCKS];

    for (int i = 1; i < NUM_BLOCKS; i++)
    {
        minStart = timer[i] < minStart ? timer[i] : minStart;
        maxEnd = timer[NUM_BLOCKS+i] > maxEnd ? timer[NUM_BLOCKS+i] : maxEnd;
    }

    printf("time = %d\n", maxEnd - minStart);

    hipDeviceReset();

    cutilExit(argc, argv);
}

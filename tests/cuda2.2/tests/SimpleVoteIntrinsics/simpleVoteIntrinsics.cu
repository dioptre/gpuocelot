/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>


////////////////////////////////////////////////////////////////////////////////
// Global types and parameters
////////////////////////////////////////////////////////////////////////////////
#define VOTE_DATA_GROUP 4

////////////////////////////////////////////////////////////////////////////////
// CUDA Voting Kernel functions
////////////////////////////////////////////////////////////////////////////////
#include "simpleVote_kernel.cu"


// Generate the test pattern for Tests 1 and 2
void genVoteTestPattern(unsigned int *VOTE_PATTERN, int size)
{
	// For testing VOTE.Any (all of these threads will return 0)
	for (int i=0; i < size/4; i++)
		VOTE_PATTERN[i] = 0x00000000;

	// For testing VOTE.Any (1/2 these threads will return 1)
	for (int i=2*size/8; i < 4*size/8; i++)
		VOTE_PATTERN[i] = (i&0x01) ? i : 0;

	// For testing VOTE.all (1/2 of these threads will return 0)
	for (int i=2*size/4; i < 3*size/4; i++)
		VOTE_PATTERN[i] = (i&0x01) ? 0 : i;
	
	// For testing VOTE.all (all of these threads will return 1)
	for (int i=3*size/4; i < 4*size/4; i++)
		VOTE_PATTERN[i] = 0xffffffff;
}

int checkErrors1(unsigned int *h_result, int start, int end, int warp_size, char *voteType)
{
	int i, sum=0;

	for (sum=0, i=start; i < end; i++) {
		sum += h_result[i];
	}
	if (sum > 0) {
		printf("\t<%s>[%d - %d] = ", voteType, start, end-1);
		for (i=start; i < end; i++) {
			printf("%d", h_result[i]);
		}
		printf(" - FAILED!\n", sum);
	}
	return (sum>0);
}

int checkErrors2(unsigned int *h_result, int start, int end, int warp_size, char *voteType)
{
	int i, sum=0;

	for (sum=0, i=start; i < end; i++) {
		sum += h_result[i];
	}
	if (sum!=warp_size) {
		printf("\t<%s>[%d - %d] = ", voteType, start, end-1);
		for (i=start; i < end; i++) {
			printf("%d", h_result[i]);
		}
		printf(" - FAILED!\n");
	}
	return (sum!=warp_size);
}


// Verification code for Kernel #1
int checkResultsVoteAnyKernel1(unsigned int *h_result, int size, int warp_size)
{
	int error_count = 0;

	error_count += checkErrors1(h_result,                             0,   VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.Any");
	error_count += checkErrors2(h_result,   VOTE_DATA_GROUP*warp_size/4, 2*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.Any");
	error_count += checkErrors2(h_result, 2*VOTE_DATA_GROUP*warp_size/4, 3*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.Any");
	error_count += checkErrors2(h_result, 3*VOTE_DATA_GROUP*warp_size/4, 4*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.Any");

	return error_count;
}

// Verification code for Kernel #2
int checkResultsVoteAllKernel2(unsigned int *h_result, int size, int warp_size)
{
	int error_count = 0;

#if 1	// results behave differently because DEVICE_EMULATION has WARP size of 1
	error_count += checkErrors1(h_result,                             0,   VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
	error_count += checkErrors2(h_result,   VOTE_DATA_GROUP*warp_size/4, 2*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
	error_count += checkErrors2(h_result, 2*VOTE_DATA_GROUP*warp_size/4, 3*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
	error_count += checkErrors2(h_result, 3*VOTE_DATA_GROUP*warp_size/4, 4*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
#else
	error_count += checkErrors1(h_result,                             0,   VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
	error_count += checkErrors1(h_result,   VOTE_DATA_GROUP*warp_size/4, 2*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
	error_count += checkErrors1(h_result, 2*VOTE_DATA_GROUP*warp_size/4, 3*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
	error_count += checkErrors2(h_result, 3*VOTE_DATA_GROUP*warp_size/4, 4*VOTE_DATA_GROUP*warp_size/4, warp_size, "Vote.All");
#endif
	return error_count;
}

// Verification code for Kernel #3
int checkResultsVoteAnyKernel3(bool *hinfo, int size)
{
	int i, error_count = 0;

#if 1	// comparison is different due to DEVICE_EMULATION having a WARP size of 1
	for (i = 0; i < size * 3; i++)  {
      // All warps should be all zeros.
	  if (i >= 0 && i < size) {
		  if (hinfo[i] != false) {
			error_count++;
		  }
	  } else if (i >= size && i < 2*size) {
          if (hinfo[i] != true) {
            error_count++;
          }
	  } else if (i >= 2*size && i < 3*size) {
          if (hinfo[i] != true) {
            error_count++;
          }
	  }
	}
#else
	for (i = 0; i < size * 3; i++)  {
      switch(i % 3) {
        case 0:
          // First warp should be all zeros.
          if (hinfo[i] != (i >= size * 1)) {
            error_count++;
          }
          break;
        case 1:
          // First warp and half of second should be all zeros.
          if (hinfo[i] != (i >= size * 3 / 2)) {
            error_count++;
          }
          break;
        case 2:
          // First two warps should be all zeros.
          if (hinfo[i] != (i >= size * 2)) {
            error_count++;
          }
          break;
      }
    }
#endif
	printf((error_count == 0) ? "\tPASSED!\n" : "\tFAILED!\n");
	return error_count;
}

int main(int argc, char **argv)
{
    unsigned int *h_input, *h_result;
    unsigned int *d_input, *d_result;

    bool *dinfo = NULL, *hinfo = NULL;
	int error_count[3] = { 0, 0, 0 };
    
    hipDeviceProp_t deviceProp;
    int dev, warp_size;

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( dev = cutGetMaxGflopsDeviceId() );

    cutilSafeCall(hipChooseDevice(&dev, &deviceProp));
    cutilSafeCall(hipGetDeviceProperties(&deviceProp, 0));

#if 1
	warp_size = 1;
#else
	if ((deviceProp.major > 1 || deviceProp.minor >= 2))
	{
		printf("simpleVoteIntrinsics: Using Device %d: \"%s\"\n", dev, deviceProp.name);
	} else 
	{
		printf("simpleVoteIntrinsics: requires Compute Capability 1.2 or higher\n");
		printf("Aborting test\n");
		printf("TEST PASSED\n");
		cutilExit(argc,argv);
    }
	warp_size = deviceProp.warpSize;
#endif

    h_input  = (unsigned int *)malloc(             VOTE_DATA_GROUP*warp_size * sizeof(unsigned int));
    h_result = (unsigned int *)malloc(             VOTE_DATA_GROUP*warp_size * sizeof(unsigned int));
    cutilSafeCall( hipMalloc((void **)&d_input,  VOTE_DATA_GROUP*warp_size * sizeof(unsigned int)) );
    cutilSafeCall( hipMalloc((void **)&d_result, VOTE_DATA_GROUP*warp_size * sizeof(unsigned int)) );
	genVoteTestPattern(h_input, VOTE_DATA_GROUP*warp_size);
	cutilSafeCall( hipMemcpy(d_input, h_input, VOTE_DATA_GROUP*warp_size * sizeof(unsigned int), hipMemcpyHostToDevice) );

	// Start of Vote Any Test Kernel #1
	printf("[VOTE Kernel Test 1/3]\n");
	printf("\tRunning <<Vote.Any>> kernel1 ...\n");
	{
		cutilSafeCall( hipDeviceSynchronize() );
		dim3 gridBlock(VOTE_DATA_GROUP, 1);
		dim3 threadBlock(warp_size, 1);
		VoteAnyKernel1<<<gridBlock, threadBlock>>>(d_input, d_result, VOTE_DATA_GROUP*warp_size);
		cutilCheckMsg("VoteAnyKernel() execution failed\n");
		cutilSafeCall( hipDeviceSynchronize() );
	}
	cutilSafeCall( hipMemcpy(h_result, d_result, VOTE_DATA_GROUP*warp_size * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	error_count[0] += checkResultsVoteAnyKernel1(h_result, VOTE_DATA_GROUP*warp_size, warp_size);

	// Start of Vote All Test Kernel #2
	printf("\n[VOTE Kernel Test 2/3]\n");
	printf("\tRunning <<Vote.All>> kernel2 ...\n");
	{
		cutilSafeCall( hipDeviceSynchronize() );
		dim3 gridBlock(VOTE_DATA_GROUP, 1);
		dim3 threadBlock(warp_size, 1);
	    VoteAllKernel2<<<gridBlock, threadBlock>>>(d_input, d_result, VOTE_DATA_GROUP*warp_size);
		cutilCheckMsg("VoteAllKernel() execution failed\n");
		cutilSafeCall( hipDeviceSynchronize() );
	}
	cutilSafeCall( hipMemcpy(h_result, d_result, VOTE_DATA_GROUP*warp_size * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	error_count[1] += checkResultsVoteAllKernel2(h_result, VOTE_DATA_GROUP*warp_size, warp_size);

	// Second Vote Kernel Test #3 (both Any/All)
	hinfo = (bool *)calloc(warp_size * 3 * 3, sizeof(bool));
    hipMalloc((void**)&dinfo, warp_size * 3 * 3 * sizeof(bool));
    hipMemcpy(dinfo, hinfo, warp_size * 3 * 3 * sizeof(bool),
               hipMemcpyHostToDevice);

	printf("\n[VOTE Kernel Test 3/3]\n");
	printf("\tRunning <<Vote.Any>> kernel3 ...\n");
	{
		cutilSafeCall( hipDeviceSynchronize() );
		VoteAnyKernel3<<<3, warp_size>>>(dinfo, warp_size);
		cutilSafeCall( hipDeviceSynchronize() );
	}

    hipMemcpy(hinfo, dinfo, warp_size * 3 * 3 * sizeof(bool),
               hipMemcpyDeviceToHost);

	error_count[2] = checkResultsVoteAnyKernel3(hinfo, warp_size * 3);

	printf("\n");
    if (error_count[0] == 0 && 
		error_count[1] == 0 &&
		error_count[2] == 0)
	{
        printf("TEST PASSED\n");
	}
	else {
        printf("TEST FAILED\n");      
	}
 
	// Now free these resources for Test #1,2
    cutilSafeCall( hipFree(d_input) );
    cutilSafeCall( hipFree(d_result) );
    free(h_input);
    free(h_result);

	// Free resources from Test #3
    free(hinfo); hipFree(dinfo);
		
	printf("\tShutting down...\n");
    hipDeviceReset();
	cutilExit(argc, argv);
}

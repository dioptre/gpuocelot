#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.numIterations1 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */


// ----------------------------------------------------------------------------------------
// Transpose
//
// This file contains both device and host code for transposing a floating-point
// matrix.  It performs several transpose kernels, which incrementally improve performance
// through coalescing, removing shared memory bank conflicts, and eliminating partition
// camping.  Several of the kernels perform a copy, used to represent the best case
// performance that a transpose can achieve.
//
// Please see the whitepaper in the docs folder of the transpose project for a detailed
// description of this performance study.
// ----------------------------------------------------------------------------------------

#include <stdio.h>
#include <cutil_inline.h>

// Each block transposes/copies a tile of TILE_DIM x TILE_DIM elements
// using TILE_DIM x BLOCK_ROWS threads, so that each thread transposes
// TILE_DIM/BLOCK_ROWS elements.  TILE_DIM must be an integral multiple of BLOCK_ROWS

#define TILE_DIM 32
#define BLOCK_ROWS 8

// modified so it doesn't take forever to run on emulation mode
#if 1
int MATRIX_SIZE_X = 128;
int MATRIX_SIZE_Y = 128;
int MUL_FACTOR    = 4;
#else
int MATRIX_SIZE_X = 2048;
int MATRIX_SIZE_Y = 2048;
int MUL_FACTOR    = 32;
#endif


// Number of repetitions used for timing.  Two sets of repetitions are performed:
// 1) over kernel launches and 2) inside the kernel over just the loads and stores

#define NUM_REPS  100

// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void copy(float *odata, float* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  
  int index  = xIndex + width*yIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index+i*width] = idata[index+i*width];
    }
  }
}

__global__ void copySharedMem(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  
  int index  = xIndex + width*yIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index+i*width] = tile[threadIdx.y+i][threadIdx.x];
    }
  }
}

// -------------------------------------------------------
// Transposes
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transposeNaive(float *odata, float* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index_in  = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;
  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i] = idata[index_in+i*width];
    }
  }
}

// coalesced transpose (with bank conflicts)

__global__ void transposeCoalesced(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

// Coalesced transpose with no bank conflicts

__global__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

// Transpose that effectively reorders execution of thread blocks along diagonals of the 
// matrix (also coalesced and has no bank conflicts)
//
// Here blockIdx.x is interpreted as the distance along a diagonal and blockIdx.y as 
// corresponding to different diagonals
//
// blockIdx_x and blockIdx_y expressions map the diagonal coordinates to the more commonly 
// used cartesian coordinates so that the only changes to the code from the coalesced version 
// are the calculation of the blockIdx_x and blockIdx_y and replacement of blockIdx.x and 
// bloclIdx.y with the subscripted versions in the remaining code

__global__ void transposeDiagonal(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int blockIdx_x, blockIdx_y;

  // do diagonal reordering
  if (width == height) {
    blockIdx_y = blockIdx.x;
    blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
  } else {
    int bid = blockIdx.x + gridDim.x*blockIdx.y;
    blockIdx_y = bid%gridDim.y;
    blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
  }    

  // from here on the code is same as previous kernel except blockIdx_x replaces blockIdx.x
  // and similarly for y

  int xIndex = blockIdx_x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx_y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx_y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx_x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

// --------------------------------------------------------------------
// Partial transposes
// NB: the coarse- and fine-grained routines only perform part of a 
//     transpose and will fail the test against the reference solution
//
//     They are used to assess performance characteristics of different
//     components of a full transpose
// --------------------------------------------------------------------

__global__ void transposeFineGrained(float *odata, float *idata, int width, int height,  int nreps)
{
  __shared__ float block[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index = xIndex + (yIndex)*width;

  for (int r=0; r<nreps; r++) {
    for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
      block[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
    }  
     
    __syncthreads();

    for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
      odata[index+i*height] = block[threadIdx.x][threadIdx.y+i];
    }
  }
}


__global__ void transposeCoarseGrained(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float block[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r<nreps; r++) {
    for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) {
      block[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();

    for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) {
      odata[index_out+i*height] = block[threadIdx.y+i][threadIdx.x];
    }
  }
}


// ---------------------
// host utility routines
// ---------------------

void computeTransposeGold(float* gold, float* idata,
			  const  int size_x, const  int size_y)
{
  for(  int y = 0; y < size_y; ++y) {
    for(  int x = 0; x < size_x; ++x) {
      gold[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}


// ----
// main
// ----

int
main( int argc, char** argv) 
{
  // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
    cutilDeviceInit(argc, argv);
  else
    hipSetDevice( cutGetMaxGflopsDeviceId() );

  int devID;
  hipDeviceProp_t props;

  // get number of SMs on this GPU
  cutilSafeCall(hipGetDevice(&devID));
  cutilSafeCall(hipGetDeviceProperties(&props, devID));

  int SMcount = 32 / props.multiProcessorCount;

  printf("CUDA device has %d Multi-Processors\n", props.multiProcessorCount);

    // set matrix size
  int size_x = max(MATRIX_SIZE_X / SMcount, TILE_DIM*MUL_FACTOR);
  int size_y = max(MATRIX_SIZE_Y / SMcount, TILE_DIM*MUL_FACTOR); 

  if (size_x%TILE_DIM != 0 || size_y%TILE_DIM != 0) {
    printf("Matrix size must be integral multiple of tile size\n\nTest PASSED\n");
    cutilExit(argc, argv);
  }

  // kernel pointer and descriptor
  void (*kernel)(float *, float *, int, int, int);
  char *kernelName;

  // execution configuration parameters
  dim3 grid(size_x/TILE_DIM, size_y/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);

  // CUDA events
  hipEvent_t start, stop;

  // size of memory required to store the matrix
  const  int mem_size = sizeof(float) * size_x*size_y;


  // allocate host memory
  float *h_idata = (float*) malloc(mem_size);
  float *h_odata = (float*) malloc(mem_size);
  float *transposeGold = (float *) malloc(mem_size);  
  float *gold;

  // allocate device memory
  float *d_idata, *d_odata;
  cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size) );
  cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size) );

  // initalize host data
  for(  int i = 0; i < (size_x*size_y); ++i)
    h_idata[i] = (float) i;
  
  // copy host data to device
  cutilSafeCall( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );

  // Compute reference transpose solution
  computeTransposeGold(transposeGold, h_idata, size_x, size_y);

  // print out common data for all kernels
  printf("\nMatrix size: %dx%d, tile size: %dx%d, block size: %dx%d\n\n", 
	 size_x, size_y, TILE_DIM, TILE_DIM, TILE_DIM, BLOCK_ROWS);
  
  printf("Kernel\t\t\tLoop over kernel\tLoop within kernel\n");
  printf("------\t\t\t----------------\t------------------\n");

  // initialize events
  
  cutilSafeCall( hipEventCreate(&start) );
  cutilSafeCall( hipEventCreate(&stop) );

  //
  // loop over different kernels
  //

  CUTBoolean success = CUTTrue;

  for (int k = 0; k<8; k++) {
    // set kernel pointer
    switch (k) {
    case 0:
      kernel = &copy;                           kernelName = "simple copy           "; break;
    case 1:
      kernel = &copySharedMem;                  kernelName = "shared memory copy    "; break;
    case 2:
      kernel = &transposeNaive;                 kernelName = "naive transpose       "; break;
    case 3:
      kernel = &transposeCoalesced;             kernelName = "coalesced transpose   "; break;
    case 4:
      kernel = &transposeNoBankConflicts;       kernelName = "no bank conflict trans"; break;
    case 5:
      kernel = &transposeCoarseGrained;         kernelName = "coarse-grained        "; break;
    case 6:
      kernel = &transposeFineGrained;           kernelName = "fine-grained          "; break;
    case 7:
      kernel = &transposeDiagonal;              kernelName = "diagonal transpose    "; break;
    }      

    // set reference solution
    if (kernel == &copy || kernel == &copySharedMem) {
      gold = h_idata;
    } else if (kernel == &transposeCoarseGrained || kernel == &transposeFineGrained) {
      gold = h_odata;   // fine- and coarse-grained kernels are not full transposes, so bypass check
    } else {
      gold = transposeGold;
    }

    // warmup to avoid timing startup
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, 1);

    // take measurements for loop over kernel launches
    cutilSafeCall( hipEventRecord(start, 0) );
    for (int i=0; i < NUM_REPS; i++) {
      kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, 1);
    }
    cutilSafeCall( hipEventRecord(stop, 0) );
    cutilSafeCall( hipEventSynchronize(stop) );
    float outerTime;
    cutilSafeCall( hipEventElapsedTime(&outerTime, start, stop) );    

    cutilSafeCall( hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost) );
    CUTBoolean res = cutComparef(gold, h_odata, size_x*size_y);
    if (res == CUTFalse) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = CUTFalse;
    }

    // take measurements for loop inside kernel
    cutilSafeCall( hipEventRecord(start, 0) );
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, NUM_REPS);
    cutilSafeCall( hipEventRecord(stop, 0) );
    cutilSafeCall( hipEventSynchronize(stop) );
    float innerTime;
    cutilSafeCall( hipEventElapsedTime(&innerTime, start, stop) );    

    cutilSafeCall( hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost) );
    res = cutComparef(gold, h_odata, size_x*size_y);
    if (res == CUTFalse) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = CUTFalse;
    }
    
    // report effective bandwidths
    float outerBandwidth = 2.*1000*mem_size/(1024*1024*1024)/(outerTime/NUM_REPS);
    float innerBandwidth = 2.*1000*mem_size/(1024*1024*1024)/(innerTime/NUM_REPS);
    printf("%s\t%5.2f GB/s\t\t%5.2f GB/s\n", kernelName, outerBandwidth, innerBandwidth);
  }
  
  printf("\nTest %s\n", (success == CUTTrue) ? "PASSED" : "FAILED");

  // cleanup

  free(h_idata);
  free(h_odata);
  free(transposeGold);
  cutilSafeCall( hipFree(d_idata) );
  cutilSafeCall( hipFree(d_odata) );

  cutilSafeCall( hipEventDestroy(start) );
  cutilSafeCall( hipEventDestroy(stop) );
  
  hipDeviceReset();
  cutilExit(argc, argv);
  return 0;

}

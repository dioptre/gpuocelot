
#include <hip/hip_runtime.h>

// Work in progress:

// Odd/Even sort.
__device__ void sortColors3(float * v, float3 * colors, int * xrefs)
{
	int tid = threadIdx.x;

	xrefs[tid] = tid;

	float e;
	const bool active = (tid != 0) && (tid != 15);
	
	for (int i = 0; i < 8; i++)
	{
		if (tid & 1) e = min(v[tid], v[tid+1]);
		else e = max(v[tid-1], v[tid]);
		v[tid] = e;

		if (active)
		{
			if (tid & 1) e = max(v[tid-1], v[tid]);
			else e = min(v[tid], v[tid+1]);
			v[tid] = e;
		}
	}
	
	/*for (int i = 0; i < 8; i++)
	{
		int x;
		{
			int odd = tid & 1;
			int x0 = xrefs[tid ^ odd];
			int x1 = xrefs[tid ^ !odd];
			int cmp = v[x0] > v[x1];
			if (cmp ^ odd) x = x0;
			else x = x0;
			xrefs[tid] = x;
		}
		
		if (active)
		{
			if (tid & 1) e = max(v[xrefs[tid-1]], v[xrefs[tid]]);
			else e = min(v[xrefs[tid]], v[xrefs[tid+1]]);
			v[xrefs[tid]] = e;
		}
	}*/
	
//	float3 tmp = colors[tid];
//	colors[cmp[tid]] = tmp;
}

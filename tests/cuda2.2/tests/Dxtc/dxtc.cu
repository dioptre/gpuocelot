#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h> // for FLT_MAX

//#include <cutil_inline.h>
#include <cutil_inline.h>
#include <cutil_math.h>

#include "CudaMath.h"
#include "dds.h"
#include "permutations.h"

// Definitions
#define INPUT_IMAGE "lena_std.ppm"
#define REFERENCE_IMAGE "lena_ref.dds"

#if 1
#define ERROR_THRESHOLD 0.1f
#else
#define ERROR_THRESHOLD 0.02f
#endif

#define NUM_THREADS 64        // Number of threads per block.

#if 1
#define __debugsync() __syncthreads()
#else
#define __debugsync()
#endif


template <class T> 
__device__ inline void swap(T & a, T & b)
{
    T tmp = a;
    a = b;
    b = tmp;
}


//__constant__ float3 kColorMetric = { 0.2126f, 0.7152f, 0.0722f };
__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };


////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(const float * values, int * ranks)
{
#if 1
    if (threadIdx.x == 0)
    {
        for (int tid = 0; tid < 16; tid++)
        {
            int rank = 0;
            for (int i = 0; i < 16; i++)
            {
                rank += (values[i] < values[tid]);
            }
            
            ranks[tid] = rank;
        }

        // Resolve elements with the same index.
        for (int i = 0; i < 15; i++)
        {
            for (int tid = 0; tid < 16; tid++)
            {
                if (tid > i && ranks[tid] == ranks[i]) ++ranks[tid];
            }
        }
    }
#else
    const int tid = threadIdx.x;

    int rank = 0;

    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        rank += (values[i] < values[tid]);
    }
    
    ranks[tid] = rank;

    // Resolve elements with the same index.
    #pragma unroll
    for (int i = 0; i < 15; i++)
    {
        if (tid > i && ranks[tid] == ranks[i]) ++ranks[tid];
    }
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint * image, float3 colors[16], float3 sums[16], int xrefs[16])
{
    const int bid = blockIdx.x;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    float3 tmp;

    if (idx < 16)
    {
        // Read color and copy to shared mem.
        uint c = image[(bid) * 16 + idx];
    
        colors[idx].x = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
        colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
        colors[idx].z = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
        
        // No need to synchronize, 16 < warp size.
#if 1
        } __debugsync();
#endif

        // Sort colors along the best fit line.
        colorSums(colors, sums);
        float3 axis = bestFitLine(colors, sums[0]);
        
        if (idx < 16) {
        dps[idx] = dot(colors[idx], axis);
        
#if 1
        } __debugsync(); if (idx < 16) {
#endif
        
        sortColors(dps, xrefs);
        
        tmp = colors[idx];

#if 1
        } __debugsync(); if (idx < 16) {
#endif

        colors[xrefs[idx]] = tmp;
    }
}



////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand(float3 v, ushort * w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);

    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f; // approximate integer bit expansion.
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}


__constant__ float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const int prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const int prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

#define USE_TABLES 1

////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
static __device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = 9*color_sum - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) * (1.0f / 3.0f);
        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    // alpha2, beta2, alphabeta and factor could be precomputed for each permutation, but it's faster to recompute them.
    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (1.0f / 9.0f) * dot(e, kColorMetric);
}

static __device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = 4*color_sum - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (1.0f / 4.0f) * dot(e, kColorMetric);
}



__device__ void evalAllPermutations(const float3 * colors, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors, float3 color_sum)
{
    const int idx = threadIdx.x;
    
    float bestError = FLT_MAX;
    
    __shared__ uint s_permutations[160];

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 992) break;
        
        ushort start, end;
        uint permutation = permutations[pidx];
        if (pidx < 160) s_permutations[pidx] = permutation;
                
        float error = evalPermutation4(colors, permutation, &start, &end, color_sum);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;    // Flip indices.
    }

    for(int i = 0; i < 3; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 160) break;
        
        ushort start, end;
        uint permutation = s_permutations[pidx];
        float error = evalPermutation3(colors, permutation, &start, &end, color_sum);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
            
            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
    }

    errors[idx] = bestError;
}


////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float * errors)
{
    const int idx = threadIdx.x;

    __shared__ int indices[NUM_THREADS];
    indices[idx] = idx;

#if 1
    for(int d = NUM_THREADS/2; d > 0; d >>= 1)
    {
        __syncthreads();
        
        if (idx < d)
        {
            float err0 = errors[idx];
            float err1 = errors[idx + d];
            
            if (err1 < err0) {
                errors[idx] = err1;
                indices[idx] = indices[idx + d];
            }
        }
    }

#else
    for(int d = NUM_THREADS/2; d > 32; d >>= 1)
    {
        __syncthreads();
        
        if (idx < d)
        {
            float err0 = errors[idx];
            float err1 = errors[idx + d];
            
            if (err1 < err0) {
                errors[idx] = err1;
                indices[idx] = indices[idx + d];
            }
        }
    }

    __syncthreads();

    // unroll last 6 iterations
    if (idx < 32)
    {
        if (errors[idx + 32] < errors[idx]) {
            errors[idx] = errors[idx + 32];
            indices[idx] = indices[idx + 32];
        }
        if (errors[idx + 16] < errors[idx]) {
            errors[idx] = errors[idx + 16];
            indices[idx] = indices[idx + 16];
        }
        if (errors[idx + 8] < errors[idx]) {
            errors[idx] = errors[idx + 8];
            indices[idx] = indices[idx + 8];
        }
        if (errors[idx + 4] < errors[idx]) {
            errors[idx] = errors[idx + 4];
            indices[idx] = indices[idx + 4];
        }
        if (errors[idx + 2] < errors[idx]) {
            errors[idx] = errors[idx + 2];
            indices[idx] = indices[idx + 2];
        }
        if (errors[idx + 1] < errors[idx]) {
            errors[idx] = errors[idx + 1];
            indices[idx] = indices[idx + 1];
        }
    }
#endif

    __syncthreads();

    return indices[0];
}


////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result)
{
    const int bid = blockIdx.x;

    if (start == end)
    {
        permutation = 0;
    }
    
    // Reorder permutation.
    uint indices = 0;
    for(int i = 0; i < 16; i++)
    {
        int ref = xrefs[i];
        indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
    }
    
    // Write endpoints.
    result[bid].x = (end << 16) | start;
    
    // Write palette indices.
    result[bid].y = indices;
}


////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compress(const uint * permutations, const uint * image, uint2 * result)
{
    const int idx = threadIdx.x;
    
    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int xrefs[16];
    
    loadColorBlock(image, colors, sums, xrefs);

    __syncthreads();

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutations(colors, permutations, bestStart, bestEnd, bestPermutation, errors, sums[0]);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);
    
    __syncthreads();
    
    // Only write the result of the winner thread.
    if (idx == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
    }
}


// Helper structs and functions to validate the output of the compressor.
// We cannot simply do a bitwise compare, because different compilers produce different
// results for different targets due to floating point arithmetic.

union Color32 {
    struct {
        unsigned char b, g, r, a;
    };
    unsigned int u;
};

union Color16 {
    struct {
        unsigned short b : 5;
        unsigned short g : 6;
        unsigned short r : 5;
    };
    unsigned short u;
};

struct BlockDXT1
{
    Color16 col0;
    Color16 col1;
    union {
        unsigned char row[4];
        unsigned int indices;
    };
    
    void decompress(Color32 colors[16]) const;
};

void BlockDXT1::decompress(Color32 * colors) const
{
    Color32 palette[4];
    
    // Does bit expansion before interpolation.
    palette[0].b = (col0.b << 3) | (col0.b >> 2);
    palette[0].g = (col0.g << 2) | (col0.g >> 4);
    palette[0].r = (col0.r << 3) | (col0.r >> 2);
    palette[0].a = 0xFF;
    
    palette[1].r = (col1.r << 3) | (col1.r >> 2);
    palette[1].g = (col1.g << 2) | (col1.g >> 4);
    palette[1].b = (col1.b << 3) | (col1.b >> 2);
    palette[1].a = 0xFF;
    
    if( col0.u > col1.u ) {
        // Four-color block: derive the other two colors.
        palette[2].r = (2 * palette[0].r + palette[1].r) / 3;
        palette[2].g = (2 * palette[0].g + palette[1].g) / 3;
        palette[2].b = (2 * palette[0].b + palette[1].b) / 3;
        palette[2].a = 0xFF;
        
        palette[3].r = (2 * palette[1].r + palette[0].r) / 3;
        palette[3].g = (2 * palette[1].g + palette[0].g) / 3;
        palette[3].b = (2 * palette[1].b + palette[0].b) / 3;
        palette[3].a = 0xFF;
    }
    else {
        // Three-color block: derive the other color.
        palette[2].r = (palette[0].r + palette[1].r) / 2;
        palette[2].g = (palette[0].g + palette[1].g) / 2;
        palette[2].b = (palette[0].b + palette[1].b) / 2;
        palette[2].a = 0xFF;

        palette[3].r = 0x00;
        palette[3].g = 0x00;
        palette[3].b = 0x00;
        palette[3].a = 0x00;
    }

    for (int i = 0; i < 16; i++)
    {
        colors[i] = palette[(indices >> (2*i)) & 0x3];
    }
}

static int compareColors(const Color32 * b0, const Color32 * b1)
{
    int sum = 0;
    for (int i = 0; i < 16; i++)
    {
        int r = (b0[i].r - b1[i].r);
        int g = (b0[i].g - b1[i].g);
        int b = (b0[i].b - b1[i].b);
        sum += r*r + g*g + b*b;
    }
    return sum;
}

static int compareBlock(const BlockDXT1 * b0, const BlockDXT1 * b1)
{
    Color32 colors0[16];
    Color32 colors1[16];

    if (memcmp(b0, b1, sizeof(BlockDXT1)) == 0)
    {
        return 0;
    }
    else
    {
        b0->decompress(colors0);
        b1->decompress(colors1);
    
        return compareColors(colors0, colors1);
    }
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    // Load input image.
    unsigned char * data = NULL;
    uint W, H;

    char* image_path = cutFindFilePath(INPUT_IMAGE, argv[0]);
    if (image_path == 0) {
        printf("Error, unable to find source image\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    if (!cutLoadPPM4ub( image_path, &data, &W, &H)) {
        printf("Error, unable to open source image\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    uint w, h;
#if 1
    // Reduce the image size so that it doesn't take so long on emulation.
    w = W >> 4;
    h = H >> 4;
#else
    w = W >> 4;
    h = H >> 4;
#endif

    // Allocate input image.
    const uint memSize = w * h * 4;
    cutilCondition( 0 != memSize );
    
    uint * image = (uint *) malloc(memSize);

    // Convert linear image to block linear. 
    for(uint by = 0; by < h/4; by++) {
        for(uint bx = 0; bx < w/4; bx++) {
            for (int i = 0; i < 16; i++) {
                const int x = i & 3;
                const int y = i / 4;
                image[(by * w/4 + bx) * 16 + i] = 
                    ((uint *)data)[(by * 4 + y) * 4 * (W/4) + bx * 4 + x];
            }
        }
    }

    // copy into global mem
    uint * d_data = NULL;
    cutilSafeCall( hipMalloc((void**) &d_data, memSize) );

    // Result
    uint * d_result = NULL;
    const uint compressedSize = (w / 4) * (h / 4) * 8;
    cutilSafeCall( hipMalloc((void**) &d_result, compressedSize) );
    uint * result = (uint *)malloc(compressedSize);

    // Compute permutations.
    uint permutations[1024];
    computePermutations(permutations);

    // Upload permutations.
    uint * d_permutations = NULL;    
    cutilSafeCall( hipMalloc((void**) &d_permutations, 1024 * sizeof(uint)) );
    cutilSafeCall( hipMemcpy(d_permutations, permutations, 1024 * sizeof(uint), 
                   hipMemcpyHostToDevice) );


    uint timer;
    cutilCheckError(cutCreateTimer(&timer));

    cutilSafeCall(hipDeviceSynchronize()); 
    cutilCheckError(cutStartTimer(timer));
    
    // Upload image.
    cutilSafeCall( hipMemcpy(d_data, image, memSize, hipMemcpyHostToDevice) );

    uint blocks = ((w + 3) / 4) * ((h + 3) / 4);
    compress<<<blocks, NUM_THREADS>>>(d_permutations, d_data, (uint2 *)d_result);

    cutilCheckMsg("compress");

    cutilSafeCall(hipMemcpy(result, d_result, compressedSize, hipMemcpyDeviceToHost));

    cutilSafeCall(hipDeviceSynchronize());
    cutilCheckError(cutStopTimer(timer));

    float time = cutGetTimerValue(timer);
    printf("Time %f msec\n", time);

    char output_filename[1024];
    strcpy(output_filename, image_path);
    strcpy(output_filename + strlen(image_path) - 3, "dds");

    // Write DDS file.
    FILE * fp = fopen(output_filename, "wb");
    if (fp == 0) {
        printf("Error, unable to open output image\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    DDSHeader header;
    header.fourcc = FOURCC_DDS;
    header.size = 124;
    header.flags  = (DDSD_WIDTH|DDSD_HEIGHT|DDSD_CAPS|DDSD_PIXELFORMAT|DDSD_LINEARSIZE);
    header.height = h;
    header.width = w;
    header.pitch = compressedSize;
    header.depth = 0;
    header.mipmapcount = 0;
    memset(header.reserved, 0, sizeof(header.reserved));
    header.pf.size = 32;
    header.pf.flags = DDPF_FOURCC;
    header.pf.fourcc = FOURCC_DXT1;
    header.pf.bitcount = 0;
    header.pf.rmask = 0;
    header.pf.gmask = 0;
    header.pf.bmask = 0;
    header.pf.amask = 0;
    header.caps.caps1 = DDSCAPS_TEXTURE;
    header.caps.caps2 = 0;
    header.caps.caps3 = 0;
    header.caps.caps4 = 0;
    header.notused = 0;

    fwrite(&header, sizeof(DDSHeader), 1, fp);
    fwrite(result, compressedSize, 1, fp);

    fclose(fp);

    // Make sure the generated image is correct.
    const char* reference_image_path = cutFindFilePath(REFERENCE_IMAGE, argv[0]);
    if (reference_image_path == 0) {
        printf("Error, unable to find reference image\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    fp = fopen(reference_image_path, "rb");
    if (fp == 0) {
        printf("Error, unable to open reference image\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    fseek(fp, sizeof(DDSHeader), SEEK_SET);

    uint referenceSize = (W / 4) * (H / 4) * 8;
    uint * reference = (uint *)malloc(referenceSize);
    fread(reference, referenceSize, 1, fp);

    fclose(fp);

    float rms = 0;

    for (uint y = 0; y < h; y += 4)
    {
        for (uint x = 0; x < w; x += 4)
        {
            uint referenceBlockIdx = ((y/4) * (W/4) + (x/4));
            uint resultBlockIdx = ((y/4) * (w/4) + (x/4));

            int cmp = compareBlock(((BlockDXT1 *)result) + resultBlockIdx, ((BlockDXT1 *)reference) + referenceBlockIdx);
            if (cmp != 0.0f) {
                printf("Error at (%d, %d):\t%f rms\n", x/4, y/4, float(cmp)/16/3);
            }
            rms += cmp;
        }
    }
    rms /= w * h * 3;

    printf("RMS(reference, result) = %f\n", rms);

    if (rms <= ERROR_THRESHOLD)
    {
        printf("Test PASSED\n");
    }
    else
    {
        printf("Test FAILED\n");
    }

    // Free allocated memory.
    cutFree(image_path);
    cutFree(data);
    free(image);
    cutilSafeCall(hipFree(d_permutations));
    cutilSafeCall(hipFree(d_data));
    cutilSafeCall(hipFree(d_result));
    free(result);

    cutilCheckError(cutDeleteTimer(timer));

    hipDeviceReset();

    cutilExit(argc, argv);
}


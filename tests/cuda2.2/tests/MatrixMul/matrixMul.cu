/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <tests/MatrixMul/matrixMul_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // create and start timer
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    cutilCheckError(cutDeleteTimer(timer));

    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);

    // check result
    CUTBoolean res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}


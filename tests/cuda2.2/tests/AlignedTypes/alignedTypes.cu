#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This is a simple test showing huge access speed gap
 * between aligned and misaligned structures
 * (those having/missing __align__ keyword).
 * It measures per-element copy throughput for 
 * aligned and misaligned structures on 
 * big chunks of data.
 */



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil_inline.h>



////////////////////////////////////////////////////////////////////////////////
// Misaligned types
////////////////////////////////////////////////////////////////////////////////
typedef unsigned char uint8;

typedef unsigned short int uint16;

typedef struct{
    unsigned char r, g, b, a;
} RGBA8_misaligned;

typedef struct{
    unsigned int l, a;
} LA32_misaligned;

typedef struct{
    unsigned int r, g, b;
} RGB32_misaligned;

typedef struct{
    unsigned int r, g, b, a;
} RGBA32_misaligned;



////////////////////////////////////////////////////////////////////////////////
// Aligned types
////////////////////////////////////////////////////////////////////////////////
typedef struct __align__(4){
    unsigned char r, g, b, a;
} RGBA8;

typedef unsigned int I32;

typedef struct __align__(8){
    unsigned int l, a;
} LA32;

typedef struct __align__(16){
    unsigned int r, g, b;
} RGB32;

typedef struct __align__(16){
    unsigned int r, g, b, a;
} RGBA32;


////////////////////////////////////////////////////////////////////////////////
// Because G80 class hardware natively supports global memory operations
// only with data elements of 4, 8 and 16 bytes, if structure size
// exceeds 16 bytes, it can't be efficiently read or written,
// since more than one global memory non-coalescable load/store instructions
// will be generated, even if __align__ option is supplied.
// "Structure of arrays" storage strategy offers best performance 
// in general case. See section 5.1.2 of the Programming Guide.
////////////////////////////////////////////////////////////////////////////////
typedef struct __align__(16){
    RGBA32 c1, c2;
} RGBA32_2;



////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}



////////////////////////////////////////////////////////////////////////////////
// Simple CUDA kernel.
// Copy is carried out on per-element basis,
// so it's not per-byte in case of padded structures.
////////////////////////////////////////////////////////////////////////////////
template<class TData> __global__ void testKernel(
    TData *d_odata,
    TData *d_idata,
    int numElements
){
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for(int pos = tid; pos < numElements; pos += numThreads)
        d_odata[pos] = d_idata[pos];
}



////////////////////////////////////////////////////////////////////////////////
// Validation routine for simple copy kernel.
// We must know "packed" size of TData (number_of_fields * sizeof(simple_type))
// and compare only these "packed" parts of the structure, 
// containig actual user data. The compiler behavior with padding bytes
// is undefined, since padding is merely a placeholder 
// and doesn't contain any user data.
////////////////////////////////////////////////////////////////////////////////
template<class TData> int testCPU(
    TData *h_odata,
    TData *h_idata,
    int numElements,
    int packedElementSize
){
    for(int pos = 0; pos < numElements; pos++){
        TData src = h_idata[pos];
        TData dst = h_odata[pos];
        for(int i = 0; i < packedElementSize; i++)
            if( ((char *)&src)[i] != ((char *)&dst)[i] ) return 0;
    }

    return 1;
}



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//Memory chunk size in bytes. Reused for test
#if 1
    const int       MEM_SIZE = 4000000;
    const int NUM_ITERATIONS = 1;
#else
    const int       MEM_SIZE = 50000000;
    const int NUM_ITERATIONS = 32;
#endif

//GPU input and output data
unsigned char *d_idata, *d_odata;
//CPU input data and instance of GPU output data
unsigned char *h_idataCPU, *h_odataGPU;
unsigned int hTimer;


template<class TData> void runTest(int packedElementSize, int memory_size){
    const int totalMemSizeAligned = iAlignDown(memory_size, sizeof(TData));
    const int         numElements = iDivDown(memory_size, sizeof(TData));

    //Clean output buffer before current test
    cutilSafeCall( hipMemset(d_odata, 0, memory_size) );
    //Run test
    cutilSafeCall( hipDeviceSynchronize() );
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );
    for(int i = 0; i < NUM_ITERATIONS; i++){
        testKernel<TData><<<64, 256>>>(
            (TData *)d_odata,
            (TData *)d_idata,
            numElements
        );
        cutilCheckMsg("testKernel() execution failed\n");
    }
    cutilSafeCall( hipDeviceSynchronize() );
    cutilCheckError( cutStopTimer(hTimer) );
    double gpuTime = cutGetTimerValue(hTimer) / NUM_ITERATIONS;
    printf(
        "Avg. time: %f ms / Copy throughput: %f GB/s.\n", gpuTime,
        (double)totalMemSizeAligned / (gpuTime * 0.001 * 1073741824.0)
    );

    //Read back GPU results and run validation
    cutilSafeCall( hipMemcpy(h_odataGPU, d_odata, memory_size, hipMemcpyDeviceToHost) );
    int flag = testCPU(
        (TData *)h_odataGPU,
        (TData *)h_idataCPU,
        numElements,
        packedElementSize
    );
    printf(flag ? "TEST PASSED\n" : "TEST FAILED\n" );
}



int main(int argc, char **argv){
    int i;

    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );
		
    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    cutilSafeCall(hipGetDevice(&devID));
    cutilSafeCall(hipGetDeviceProperties(&props, devID));

    int SMscaling  = max((16 / props.multiProcessorCount), 1);
    int MemorySize = MEM_SIZE/SMscaling;

    printf("CUDA device has %d Multi-Processors\n", props.multiProcessorCount);
    printf(" Memory Size = %d\n", MemorySize);

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Allocating memory...\n");
        h_idataCPU = (unsigned char *)malloc(MemorySize);
        h_odataGPU = (unsigned char *)malloc(MemorySize);
        cutilSafeCall(hipMalloc((void **)&d_idata, MemorySize));
        cutilSafeCall(hipMalloc((void **)&d_odata, MemorySize));

    printf("Generating host input data array...\n");
        for(i = 0; i < MemorySize; i++)
            h_idataCPU[i] = (i & 0xFF) + 1;
    printf("Uploading input data to GPU memory...\n");
        cutilSafeCall(hipMemcpy(d_idata, h_idataCPU, MemorySize, hipMemcpyHostToDevice) );


    printf("Testing misaligned types...\n");
        printf("uint8...\n");
        runTest<uint8>(1, MemorySize);

        printf("uint16...\n");
        runTest<uint16>(2, MemorySize);

        printf("RGBA8_misaligned...\n");
        runTest<RGBA8_misaligned>(4, MemorySize);

        printf("LA32_misaligned...\n");
        runTest<LA32_misaligned>(8, MemorySize);

        printf("RGB32_misaligned...\n");
        runTest<RGB32_misaligned>(12, MemorySize);

        printf("RGBA32_misaligned...\n");
        runTest<RGBA32_misaligned>(16, MemorySize);

    printf("Testing aligned types...\n");
        printf("RGBA8...\n");
        runTest<RGBA8>(4, MemorySize);

        printf("I32...\n");
        runTest<I32>(4, MemorySize);

        printf("LA32...\n");
        runTest<LA32>(8, MemorySize);

        printf("RGB32...\n");
        runTest<RGB32>(12, MemorySize);

        printf("RGBA32...\n");
        runTest<RGBA32>(16, MemorySize);

        printf("RGBA32_2...\n");
        runTest<RGBA32_2>(32, MemorySize);


    printf("Shutting down...\n");
        cutilSafeCall(hipFree(d_idata));
        cutilSafeCall(hipFree(d_odata));
        free(h_odataGPU);
        free(h_idataCPU);

    cutilCheckError( cutDeleteTimer(hTimer) );
    cutilExit(argc, argv);

    hipDeviceReset();
}

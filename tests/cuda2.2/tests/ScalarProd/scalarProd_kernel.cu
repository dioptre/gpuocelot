
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */



///////////////////////////////////////////////////////////////////////////////
// On G80-class hardware 24-bit multiplication takes 4 clocks per warp
// (the same as for floating point  multiplication and addition),
// whereas full 32-bit multiplication takes 16 clocks per warp.
// So if integer multiplication operands are  guaranteed to fit into 24 bits
// (always lie withtin [-8M, 8M - 1] range in signed case),
// explicit 24-bit multiplication is preferred for performance.
///////////////////////////////////////////////////////////////////////////////
#define IMUL(a, b) __mul24(a, b)



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
// Parameters restrictions:
// 1) ElementN is strongly preferred to be a multiple of warp size to 
//    meet alignment constraints of memory coalescing.
// 2) ACCUM_N must be a power of two.
///////////////////////////////////////////////////////////////////////////////
#define ACCUM_N 1024
__global__ void scalarProdGPU(
    float *d_C,
    float *d_A,
    float *d_B,
    int vectorN,
    int elementN
){
    //Accumulators cache
    __shared__ float accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for(int vec = blockIdx.x; vec < vectorN; vec += gridDim.x){
        int vectorBase = IMUL(elementN, vec);
        int vectorEnd  = vectorBase + elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for(int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x){
            float sum = 0;

            for(int pos = vectorBase + iAccum; pos < vectorEnd; pos += ACCUM_N)
                sum += d_A[pos] * d_B[pos];

            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for(int stride = ACCUM_N / 2; stride > 0; stride >>= 1){
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
                accumResult[iAccum] += accumResult[stride + iAccum];
        }
		
		__syncthreads();
		
        if(threadIdx.x == 0) d_C[vec] = accumResult[0];
    }
}

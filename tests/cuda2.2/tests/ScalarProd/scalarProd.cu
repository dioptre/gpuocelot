#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This sample calculates scalar products of a 
 * given set of input vector pairs
 */



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProdCPU(
    float *h_C,
    float *h_A,
    float *h_B,
    int vectorN,
    int elementN
);



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cu"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

//Total number of input vector pairs; arbitrary
const int VECTOR_N = 256;
//Number of elements per vector; arbitrary, 
//but strongly preferred to be a multiple of warp size
//to meet memory coalescing constraints
const int ELEMENT_N = 4096;
//Total number of data elements
const int    DATA_N = VECTOR_N * ELEMENT_N;

const int   DATA_SZ = DATA_N * sizeof(float);
const int RESULT_SZ = VECTOR_N  * sizeof(float);



///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
    float *d_A, *d_B, *d_C;
    double delta, ref, sum_delta, sum_ref, L1norm;
    unsigned int hTimer;
    int i;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data...\n");
        printf("...allocating CPU memory.\n");
        h_A     = (float *)malloc(DATA_SZ);
        h_B     = (float *)malloc(DATA_SZ);
        h_C_CPU = (float *)malloc(RESULT_SZ);
        h_C_GPU = (float *)malloc(RESULT_SZ);

        printf("...allocating GPU memory.\n");
        cutilSafeCall( hipMalloc((void **)&d_A, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_B, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_C, RESULT_SZ) );

        printf("...generating input data in CPU mem.\n");
        srand(123);
        //Generating input data on CPU
        for(i = 0; i < DATA_N; i++){
            h_A[i] = RandFloat(0.0f, 1.0f);
            h_B[i] = RandFloat(0.0f, 1.0f);
        }

        printf("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing 
        cutilSafeCall( hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice) );
    printf("Data init done.\n");


    printf("Executing GPU kernel...\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
        cutilCheckMsg("scalarProdGPU() execution failed\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
    printf("GPU time: %f msecs.\n", cutGetTimerValue(hTimer));

    printf("Reading back GPU result...\n");
        //Read back GPU results to compare them to CPU results
        cutilSafeCall( hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost) );


    printf("Checking GPU results...\n");
        printf("..running CPU scalar product calculation\n");
        scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

        printf("...comparing the results\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        for(i = 0; i < VECTOR_N; i++){
            delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
            ref   = h_C_CPU[i];
            sum_delta += delta;
            sum_ref   += ref;
        }
        L1norm = sum_delta / sum_ref;
    printf("L1 error: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");


    printf("Shutting down...\n");
        cutilSafeCall( hipFree(d_C) );
        cutilSafeCall( hipFree(d_B)   );
        cutilSafeCall( hipFree(d_A)   );
        free(h_C_GPU);
        free(h_C_CPU);
        free(h_B);
        free(h_A);
        cutilCheckError( cutDeleteTimer(hTimer) );

    hipDeviceReset();

    cutilExit(argc, argv);

}

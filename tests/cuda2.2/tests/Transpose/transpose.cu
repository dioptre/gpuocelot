/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.numIterations1 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */
 
/* Matrix transpose with Cuda 
 * Host code.

 * This example transposes arbitrary-size matrices.  It compares a naive
 * transpose kernel that suffers from non-coalesced writes, to an optimized
 * transpose with fully coalesced memory access and no bank conflicts.  On 
 * a G80 GPU, the optimized transpose can be more than 10x faster for large
 * matrices.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include "transpose_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
extern "C" void computeGold( float* reference, float* idata, 
                         const unsigned int size_x, const unsigned int size_y );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // size of the matrix
#if 1
    const unsigned int size_x = 32;
    const unsigned int size_y = 128;
#else
    const unsigned int size_x = 256;
    const unsigned int size_y = 4096;
#endif
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * size_x * size_y;
    
    unsigned int timer;
    cutCreateTimer(&timer);

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

    // allocate host memory
    float* h_idata = (float*) malloc(mem_size);
    // initalize the memory
    srand(15235911);
    for( unsigned int i = 0; i < (size_x * size_y); ++i) 
    {
        h_idata[i] = (float) i;    // rand(); 
    }

    // allocate device memory
    float* d_idata;
    float* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 grid(size_x / BLOCK_DIM, size_y / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    // warmup so we don't time CUDA startup
    transpose_naive<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    transpose<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);

	// synchronize here, so we make sure that we don't count any time from the asynchronize kernel launches.
	hipDeviceSynchronize();

    int numIterations = 1;

    printf("Transposing a %d by %d matrix of floats...\n", size_x, size_y);

    // execute the kernel
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        transpose_naive<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float naiveTime = cutGetTimerValue(timer);

    // execute the kernel
    
    cutResetTimer(timer);
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        transpose<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float optimizedTime = cutGetTimerValue(timer);

    printf("Naive transpose average time:     %0.3f ms\n", naiveTime / numIterations);
    printf("Optimized transpose average time: %0.3f ms\n\n", optimizedTime / numIterations);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // copy result from device to    host
    float* h_odata = (float*) malloc(mem_size);
    cutilSafeCall( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

    // compute reference solution
    float* reference = (float*) malloc( mem_size);

    computeGold( reference, h_idata, size_x, size_y);

    // check result
    CUTBoolean res = cutComparef( reference, h_odata, size_x * size_y);
    printf(    "Test %s\n", (1    == res)    ? "PASSED" : "FAILED");

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));
    cutilCheckError( cutDeleteTimer(timer));

    hipDeviceReset();
}

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>

#include "SobelFilter_kernels.h"

#define SV 0.003921f
#define IV 255.f

// Texture reference for reading image
texture<unsigned char, 2> tex;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

#define Radius 1

#ifdef FIXED_BLOCKWIDTH
#define BlockWidth 80
#define SharedPitch 384
#endif

__device__ short __abs(short a) {
	return ((a) < 0 ? -(a) : a);
}

__device__ unsigned char
ComputeSobel(unsigned char ul, // upper left
             unsigned char um, // upper middle
             unsigned char ur, // upper right
             unsigned char ml, // middle left
             unsigned char mm, // middle (unused)
             unsigned char mr, // middle right
             unsigned char ll, // lower left
             unsigned char lm, // lower middle
             unsigned char lr, // lower right
             float fScale )
{
    short Horz = ur + 2*mr + lr - ul - 2*ml - ll;
    short Vert = ul + 2*um + ur - ll - 2*lm - lr;
    short Sum = (short) (fScale*(__abs(Horz)+__abs(Vert)));
    if ( Sum < 0 ) return 0; else if ( Sum > 0xff ) return 0xff;
    return (unsigned char) Sum;
}

__global__ void 
SobelShared( uchar4 *pSobelOriginal, unsigned short SobelPitch, 
#ifndef FIXED_BLOCKWIDTH
             short BlockWidth, short SharedPitch,
#endif
             short w, short h, float fScale )
{ 
    short u = 4*blockIdx.x*BlockWidth;
    short v = blockIdx.y*blockDim.y + threadIdx.y;
    short ib;

    int SharedIdx = threadIdx.y * SharedPitch;

    for ( ib = threadIdx.x; ib < BlockWidth+2*Radius; ib += blockDim.x ) {
        LocalBlock[SharedIdx+4*ib+0] = tex2D( tex, 
            (float) (u+4*ib-Radius+0), (float) (v-Radius) );
        LocalBlock[SharedIdx+4*ib+1] = tex2D( tex, 
            (float) (u+4*ib-Radius+1), (float) (v-Radius) );
        LocalBlock[SharedIdx+4*ib+2] = tex2D( tex, 
            (float) (u+4*ib-Radius+2), (float) (v-Radius) );
        LocalBlock[SharedIdx+4*ib+3] = tex2D( tex, 
            (float) (u+4*ib-Radius+3), (float) (v-Radius) );
    }
    if ( threadIdx.y < Radius*2 ) {
        //
        // copy trailing Radius*2 rows of pixels into shared
        //
        SharedIdx = (blockDim.y+threadIdx.y) * SharedPitch;
        for ( ib = threadIdx.x; ib < BlockWidth+2*Radius; ib += blockDim.x ) {
            LocalBlock[SharedIdx+4*ib+0] = tex2D( tex, 
                (float) (u+4*ib-Radius+0), (float) (v+blockDim.y-Radius) );
            LocalBlock[SharedIdx+4*ib+1] = tex2D( tex, 
                (float) (u+4*ib-Radius+1), (float) (v+blockDim.y-Radius) );
            LocalBlock[SharedIdx+4*ib+2] = tex2D( tex, 
                (float) (u+4*ib-Radius+2), (float) (v+blockDim.y-Radius) );
            LocalBlock[SharedIdx+4*ib+3] = tex2D( tex, 
                (float) (u+4*ib-Radius+3), (float) (v+blockDim.y-Radius) );
        }
    }

    __syncthreads();

    u >>= 2;    // index as uchar4 from here
    uchar4 *pSobel = (uchar4 *) (((char *) pSobelOriginal)+v*SobelPitch);
    SharedIdx = threadIdx.y * SharedPitch;

    for ( ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x ) {

        unsigned char pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+0];
        unsigned char pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+1];
        unsigned char pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+2];
        unsigned char pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+0];
        unsigned char pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+1];
        unsigned char pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+2];
        unsigned char pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+0];
        unsigned char pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+1];
        unsigned char pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+2];

        uchar4 out;

        out.x = ComputeSobel(pix00, pix01, pix02, 
                             pix10, pix11, pix12, 
                             pix20, pix21, pix22, fScale );

        pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+3];
        pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+3];
        pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+3];
        out.y = ComputeSobel(pix01, pix02, pix00, 
                             pix11, pix12, pix10, 
                             pix21, pix22, pix20, fScale );

        pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+4];
        pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+4];
        pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+4];
        out.z = ComputeSobel( pix02, pix00, pix01, 
                              pix12, pix10, pix11, 
                              pix22, pix20, pix21, fScale );

        pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+5];
        pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+5];
        pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+5];
        out.w = ComputeSobel( pix00, pix01, pix02, 
                              pix10, pix11, pix12, 
                              pix20, pix21, pix22, fScale );
        if ( u+ib < w/4 && v < h ) {
            pSobel[u+ib] = out;
        }
    }

    __syncthreads();
}

__global__ void 
SobelCopyImage( Pixel *pSobelOriginal, unsigned int Pitch, 
                int w, int h, float fscale )
{ 
    unsigned char *pSobel = 
      (unsigned char *) (((char *) pSobelOriginal)+blockIdx.x*Pitch);
    for ( int i = threadIdx.x; i < w; i += blockDim.x ) {
        pSobel[i] = min( max((tex2D( tex, (float) i, (float) blockIdx.x ) * fscale), 0.f), 255.f);
    }
}

__global__ void 
SobelTex( Pixel *pSobelOriginal, unsigned int Pitch, 
          int w, int h, float fScale )
{ 
    unsigned char *pSobel = 
      (unsigned char *) (((char *) pSobelOriginal)+blockIdx.x*Pitch);
    for ( int i = threadIdx.x; i < w; i += blockDim.x ) {
        unsigned char pix00 = tex2D( tex, (float) i-1, (float) blockIdx.x-1 );
        unsigned char pix01 = tex2D( tex, (float) i+0, (float) blockIdx.x-1 );
        unsigned char pix02 = tex2D( tex, (float) i+1, (float) blockIdx.x-1 );
        unsigned char pix10 = tex2D( tex, (float) i-1, (float) blockIdx.x+0 );
        unsigned char pix11 = tex2D( tex, (float) i+0, (float) blockIdx.x+0 );
        unsigned char pix12 = tex2D( tex, (float) i+1, (float) blockIdx.x+0 );
        unsigned char pix20 = tex2D( tex, (float) i-1, (float) blockIdx.x+1 );
        unsigned char pix21 = tex2D( tex, (float) i+0, (float) blockIdx.x+1 );
        unsigned char pix22 = tex2D( tex, (float) i+1, (float) blockIdx.x+1 );
        pSobel[i] = ComputeSobel(pix00, pix01, pix02, 
                                 pix10, pix11, pix12,
                                 pix20, pix21, pix22, fScale );
    }
}

extern "C" void setupTexture(int iw, int ih, Pixel *data, int Bpp)
{
    hipChannelFormatDesc desc;
    
    if (Bpp == 1) {
        desc = hipCreateChannelDesc<unsigned char>();
    } else {
        desc = hipCreateChannelDesc<uchar4>();
    }

    cutilSafeCall(hipMallocArray(&array, &desc, iw, ih));
    cutilSafeCall(hipMemcpyToArray(array, 0, 0, data, Bpp*sizeof(Pixel)*iw*ih, hipMemcpyHostToDevice));
}

extern "C" void deleteTexture(void)
{
    cutilSafeCall(hipFreeArray(array));
}


// Wrapper for the __global__ call that sets up the texture and threads
extern "C" void sobelFilter(Pixel *odata, int iw, int ih, enum SobelDisplayMode mode, float fScale) 
{
    cutilSafeCall(hipBindTextureToArray(tex, array));

    switch ( mode ) {
        case  SOBELDISPLAY_IMAGE: 
            SobelCopyImage<<<ih, 384>>>(odata, iw, iw, ih, fScale );
            break;
        case SOBELDISPLAY_SOBELTEX:
            SobelTex<<<ih, 384>>>(odata, iw, iw, ih, fScale );
            break;
        case SOBELDISPLAY_SOBELSHARED:
        {
            dim3 threads(16,4);
#ifndef FIXED_BLOCKWIDTH
	          int BlockWidth = 80; // must be divisible by 16 for coalescing
#endif
        		dim3 blocks = dim3(iw/(4*BlockWidth)+(0!=iw%(4*BlockWidth)),
                               ih/threads.y+(0!=ih%threads.y));
        		int SharedPitch = ~0x3f&(4*(BlockWidth+2*Radius)+0x3f);
        		int sharedMem = SharedPitch*(threads.y+2*Radius);

        		// for the shared kernel, width must be divisible by 4
        		iw &= ~3;

        		SobelShared<<<blocks, threads, sharedMem>>>((uchar4 *) odata, 
                                                        iw, 
#ifndef FIXED_BLOCKWIDTH
                                                        BlockWidth, SharedPitch,
#endif
                                                		    iw, ih, fScale );
        }
        break;
    }

    cutilSafeCall(hipUnbindTexture(tex));
}

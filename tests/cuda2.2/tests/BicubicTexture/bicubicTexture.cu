#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_inline.h>
#include <cutil_math.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include <tests/BicubicTexture/bicubicTexture_kernel.cuh>

hipArray *d_imageArray = 0;

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    cutilSafeCall( hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight) ); 
    uint size = imageWidth * imageHeight * sizeof(uchar);
    cutilSafeCall( hipMemcpyToArray(d_imageArray, 0, 0, h_data, size, hipMemcpyHostToDevice) );
    cutFree(h_data);

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;    // access with integer texture coordinates

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray(tex, d_imageArray, channelDesc) );
}

extern "C"
void freeTexture()
{
    cutilSafeCall(hipFreeArray(d_imageArray));
}


// render image using CUDA
extern "C" 
void render(int width, int height, float tx, float ty, float scale, float cx, float cy, 
            dim3 blockSize, dim3 gridSize, int mode, uchar4 *output)
{
    // call CUDA kernel, writing results to PBO memory
    switch(mode) {
    case MODE_NEAREST:
        tex.filterMode = hipFilterModePoint;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BILINEAR:
        tex.filterMode = hipFilterModeLinear;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BICUBIC:
        tex.filterMode = hipFilterModePoint;
        d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_FAST_BICUBIC:
        tex.filterMode = hipFilterModeLinear;
        d_renderFastBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    }
    cutilCheckMsg("kernel failed");
}

#endif

/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
    return (a + (b - 1)) / b;
}


// complex math functions
__device__
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__
float2 complex_add(float2 a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

// generate wave heightfield at time t based on initial heightfield and dispersion relationship
__global__ void generateSpectrumKernel(float2* h0, float2 *ht, unsigned int width, unsigned int height, float t, float patchSize)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;
    
    // calculate coordinates
    float2 k;
    k.x = HIP_PI_F * x / (float) patchSize;
    k.y = 2.0f * HIP_PI_F * y / (float) patchSize;

    // calculate dispersion w(k)
    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);

    if ((x < width) && (y < height)) {
		float2 h0_k = h0[i];
		float2 h0_mk = h0[(((height-1)-y)*width)+x];

		float2 h_tilda = complex_add( complex_mult(h0_k, complex_exp(w * t)),
		                              complex_mult(conjugate(h0_mk), 
		                              complex_exp(-w * t)) );

		// output frequency-space complex values
        ht[i] = h_tilda;
    }
}


// generate slope by partial differences in spatial domain
__global__ void calculateSlopeKernel(float* h, float2 *slopeOut, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float2 slope;
    if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)) {
        slope.x = h[i+1] - h[i-1];
        slope.y = h[i+width] - h[i-width];
    } else {
        slope = make_float2(0.0f, 0.0f);
    }
    slopeOut[i] = slope;
}

extern "C" 
void cudaGenerateSpectrumKernel(float2* d_h0, float2 *d_ht, 
                                unsigned int width, unsigned int height, 
                                float animTime, float patchSize)
{
    dim3 block(8, 8, 1);
    dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
    generateSpectrumKernel<<<grid, block>>>(d_h0, d_ht, width, height, animTime, patchSize);
}

extern "C"
void cudaCalculateSlopeKernel(  float* hptr, float2 *slopeOut, 
                                unsigned int width, unsigned int height)
{
    dim3 block(8, 8, 1);
    dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
    calculateSlopeKernel<<<grid2, block>>>(hptr, slopeOut, width, height);
}

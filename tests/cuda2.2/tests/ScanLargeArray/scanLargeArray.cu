/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil_inline.h>

// includes, kernels
#include <tests/ScanLargeArray/scan.cuh>  // defines prescanArray()

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                     const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

#if 0
    unsigned int num_test_iterations = 100;
    unsigned int num_elements = 1000000; // can support large, non-power-of-2 arrays!
#else
    unsigned int num_test_iterations = 1;
    unsigned int num_elements = 10000; // can support large, non-power-of-2 arrays!
#endif
    
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&num_test_iterations);

    unsigned int mem_size = sizeof( float) * num_elements;
    
    unsigned int timerGPU, timerCPU;
    cutilCheckError(cutCreateTimer(&timerCPU));
    cutilCheckError(cutCreateTimer(&timerGPU));

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( mem_size);
      
    // initialize the input data on the host
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_data[i] = 1.0f;//(int)(10 * rand()/32768.f);
    }

    // compute reference solution
    float* reference = (float*) malloc( mem_size); 
    cutStartTimer(timerCPU);
    for (unsigned int i = 0; i < num_test_iterations; i++)
    {
        computeGold( reference, h_data, num_elements);
    }
    cutStopTimer(timerCPU);

    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;

    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));
    
    // copy host memory to device input array
    cutilSafeCall( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );
    // initialize all the other device arrays to be safe
    cutilSafeCall( hipMemcpy( d_odata, h_data, mem_size, hipMemcpyHostToDevice) );

    printf("Running parallel prefix sum (prescan) of %d elements\n", num_elements);
    printf("This version is work efficient (O(n) adds)\n");
    printf("and has very few shared memory bank conflicts\n\n");

    preallocBlockSums(num_elements);

    // run once to remove startup overhead
    prescanArray(d_odata, d_idata, num_elements);


    // Run the prescan
    cutStartTimer(timerGPU);
    for (unsigned int i = 0; i < num_test_iterations; i++)
    {
        //printf("prescanArray\n");
        prescanArray(d_odata, d_idata, num_elements);
    }
    cutStopTimer(timerGPU);

    deallocBlockSums();    

    // copy result from device to host
    cutilSafeCall(hipMemcpy( h_data, d_odata, sizeof(float) * num_elements, 
                               hipMemcpyDeviceToHost));

    // If this is a regression test write the results to a file
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test 
        cutWriteFilef( "./data/result.dat", h_data, num_elements, 0.0);
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        unsigned int result_regtest = cutComparef( reference, h_data, num_elements);
        printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
        printf( "Average GPU execution time: %f ms\n", cutGetTimerValue(timerGPU) / num_test_iterations);
        printf( "CPU execution time:         %f ms\n", cutGetTimerValue(timerCPU) / num_test_iterations);
    }

    printf("\nCheck out the CUDA Data Parallel Primitives Library for more on scan.\n");
    printf("http://www.gpgpu.org/developer/cudpp\n");

    // cleanup memory
    cutDeleteTimer(timerCPU);
    cutDeleteTimer(timerGPU);
    free( h_data);
    free( reference);
    hipFree( d_odata);
    hipFree( d_idata);

    hipDeviceReset();
}

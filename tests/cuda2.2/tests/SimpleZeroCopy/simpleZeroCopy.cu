#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>

/* Add two vectors on the GPU */

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = a[idx] + b[idx];
}

int main(int argc, char **argv)
{
  int n, nelem, idev, deviceCount;
  char *device = NULL;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;                      // Pinned memory allocated on the CPU
  float *d_a, *d_b, *d_c;                // Device pointers for mapped memory  
  float errorNorm, refNorm, ref, diff;
  hipDeviceProp_t deviceProp;

  if(cutCheckCmdLineFlag(argc, (const char **)argv, "help"))
  {
    printf("Usage:  simpleZeroCopy [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    cutilExit(argc, argv);    
  }

  /* Get the device selected by the user or default to 0, and then set it. */

  if(cutGetCmdLineArgumentstr(argc, (const char**)argv, "device", &device))
  {
    hipGetDeviceCount(&deviceCount);
    idev = atoi(device);
    if(idev >= deviceCount || idev < 0)
    {
      fprintf(stderr, "Invalid device number %d, using default device 0.\n",
              idev);
      idev = 0;
    }
  }
  else
  {
    idev = 0;
  }

  cutilSafeCall(hipSetDevice(idev));

  /* Verify the selected device supports mapped memory and set the device
     flags for mapping host memory. */

  cutilSafeCall(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
  if(!deviceProp.canMapHostMemory)
  {
    fprintf(stderr, "Device %d cannot map host memory!\n", idev);
    printf("Test PASSED");
    cutilExit(argc, argv);
  }
  cutilSafeCall(hipSetDeviceFlags(hipDeviceMapHost));
#else
  fprintf(stderr, "This CUDART version does not support <hipDeviceProp_t.canMapHostMemory> field\n");
  printf("Test PASSED");
  cutilExit(argc, argv);
#endif

  /* Allocate mapped CPU memory. */

  nelem = 1048576;
  bytes = nelem*sizeof(float);
#if CUDART_VERSION >= 2020
  flags = hipHostMallocMapped;
  cutilSafeCall(hipHostAlloc((void **)&a, bytes, flags));
  cutilSafeCall(hipHostAlloc((void **)&b, bytes, flags));
  cutilSafeCall(hipHostAlloc((void **)&c, bytes, flags));
#endif
  /* Initialize the vectors. */

  for(n = 0; n < nelem; n++)
  {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  /* Get the device pointers for the pinned CPU memory mapped into the GPU
     memory space. */

#if CUDART_VERSION >= 2020
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

  /* Call the GPU kernel using the device pointers for the mapped memory. */ 

  printf("Adding vectors using mapped CPU memory...\n");
  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem/(float)block.x));
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);  
  cutilSafeCall(hipDeviceSynchronize());
  cutilCheckMsg("vectorAddGPU() execution failed");

  /* Compare the results */

  printf("Checking the results...\n");
  errorNorm = 0.f;
  refNorm = 0.f;
  for(n = 0; n < nelem; n++)
  {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff*diff;
    refNorm += ref*ref;
  }
  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);

  /* Memory clean up */

  printf("Releasing CPU memory...\n");
#if CUDART_VERSION >= 2020
  cutilSafeCall(hipHostFree(a));
  cutilSafeCall(hipHostFree(b));
  cutilSafeCall(hipHostFree(c));
#endif

  hipDeviceReset();

  printf("Test %s\n", (errorNorm/refNorm < 1.e-6f) ? "PASSED" : "FAILED");

  cutilExit(argc, argv);
}

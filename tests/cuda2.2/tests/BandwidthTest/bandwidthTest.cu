/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* 
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth 
 * for pageable and pinned memory, and device to host copy bandwidth for pageable 
 * and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>
#include <hip/hip_runtime.h>

// defines, project
#define MEMCOPY_ITERATIONS  10
#define DEFAULT_SIZE        ( 32 * ( 1 << 20 ) )    //32 M
#define DEFAULT_INCREMENT   (1 << 22)               //4 M
#define CACHE_CLEAR_SIZE    (1 << 24)               //16 M

//shmoo mode defines
#define SHMOO_MEMSIZE_MAX     (1 << 26)         //64 M
#define SHMOO_MEMSIZE_START   (1 << 10)         //1 KB
#define SHMOO_INCREMENT_1KB   (1 << 10)         //1 KB
#define SHMOO_INCREMENT_2KB   (1 << 11)         //2 KB
#define SHMOO_INCREMENT_10KB  (10 * (1 << 10))  //10KB
#define SHMOO_INCREMENT_100KB (100 * (1 << 10)) //100 KB
#define SHMOO_INCREMENT_1MB   (1 << 20)         //1 MB
#define SHMOO_INCREMENT_2MB   (1 << 21)         //2 MB
#define SHMOO_INCREMENT_4MB   (1 << 22)         //4 MB
#define SHMOO_LIMIT_20KB      (20 * (1 << 10))  //20 KB
#define SHMOO_LIMIT_50KB      (50 * (1 << 10))  //50 KB
#define SHMOO_LIMIT_100KB     (100 * (1 << 10)) //100 KB
#define SHMOO_LIMIT_1MB       (1 << 20)         //1 MB
#define SHMOO_LIMIT_16MB      (1 << 24)         //16 MB
#define SHMOO_LIMIT_32MB      (1 << 25)         //32 MB

//enums, project
enum testMode { QUICK_MODE, RANGE_MODE, SHMOO_MODE };
enum memcpyKind { DEVICE_TO_HOST, HOST_TO_DEVICE, DEVICE_TO_DEVICE };
enum printMode { USER_READABLE, CSV };
enum memoryMode { PINNED, PAGEABLE };

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(const int argc, const char **argv);
void testBandwidth( unsigned int start, unsigned int end, unsigned int increment, 
                   testMode mode, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthRange(unsigned int start, unsigned int end, unsigned int increment, 
                        memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthShmoo(memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc);
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testDeviceToDeviceTransfer(unsigned int memSize);
void printResultsReadable(unsigned int *memSizes, float *bandwidths, unsigned int count);
void printResultsCSV(unsigned int *memSizes, float *bandwidths, unsigned int count);
void printHelp(void);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv) 
{
    runTest(argc, (const char**)argv);
    cutilExit(argc, argv);
}

///////////////////////////////////////////////////////////////////////////////
//Parse args, run the appropriate tests
///////////////////////////////////////////////////////////////////////////////
void runTest(const int argc, const char **argv)
{
    int start = DEFAULT_SIZE;
    int end = DEFAULT_SIZE;
    int startDevice = 0;
    int endDevice = 0;
    int increment = DEFAULT_INCREMENT;
    testMode mode = QUICK_MODE;
    bool htod = false;
    bool dtoh = false;
    bool dtod = false;
    bool wc = false;
    char *modeStr;
    char *device = NULL;
    printMode printmode = USER_READABLE;
    char *memModeStr = NULL;
    memoryMode memMode = PAGEABLE;

    //process command line args
    if(cutCheckCmdLineFlag( argc, argv, "help"))
    {
        printHelp();
        return;
    }

    if(cutCheckCmdLineFlag( argc, argv, "csv"))
    {
        printmode = CSV;
    }

    if( cutGetCmdLineArgumentstr(argc, argv, "memory", &memModeStr) )
    {
        if( strcmp(memModeStr, "pageable") == 0 )
        {
            memMode = PAGEABLE;
        }
        else if( strcmp(memModeStr, "pinned") == 0)
        {
            memMode = PINNED;
        }
        else
        {
            printf("Invalid memory mode - valid modes are pageable or pinned\n");
            printf("See --help for more information\n");
            return;
        }
    }
    else
    {
        //default - pageable memory
        memMode = PAGEABLE;
    }

    if( cutGetCmdLineArgumentstr(argc, argv, "device", &device) )
    {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if( deviceCount == 0 )
        {
            printf("!!!!!No devices found!!!!!\n");
            return;
        } 
        if( strcmp (device, "all") == 0 )
        {
            printf ("\n!!!!!Cumulative Bandwidth to be computed from all the devices !!!!!!\n\n");
            startDevice = 0;
            endDevice = deviceCount-1;
        }
        else
        {
            startDevice = endDevice = atoi(device);
            if( startDevice >= deviceCount || startDevice < 0)
            {
                printf("\n!!!!!Invalid GPU number %d given hence default gpu %d will be used !!!!!\n", startDevice,0);
                startDevice = endDevice = 0;
            }
        }
    }
    printf("Running on......\n");
    for( int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, currentDevice);
        printf ("      device %d:%s\n", currentDevice,deviceProp.name);
    }

    if( cutGetCmdLineArgumentstr(argc, argv, "mode", &modeStr) )
    {
        //figure out the mode
        if( strcmp(modeStr, "quick") == 0 )
        {
            mode = QUICK_MODE;
        }
        else if( strcmp(modeStr, "shmoo") == 0 )
        {
            mode = SHMOO_MODE;
        }
        else if( strcmp(modeStr, "range") == 0 )
        {
            mode = RANGE_MODE;
        }
        else
        {
            printf("Invalid mode - valid modes are quick, range, or shmoo\n");
            printf("See --help for more information\n");
            return;
        }
    }
    else
    {
        //default mode - quick
        mode = QUICK_MODE;
    }
    
    if(cutCheckCmdLineFlag( argc, argv, "htod"))
        htod = true;
    if(cutCheckCmdLineFlag( argc, argv, "dtoh"))
        dtoh = true;
    if(cutCheckCmdLineFlag( argc, argv, "dtod"))
        dtod = true;
#if CUDART_VERSION >= 2020
    if(cutCheckCmdLineFlag( argc, argv, "wc"))
        wc = true;
#endif
    if(cutCheckCmdLineFlag( argc, argv, "cputiming"))
        bDontUseGPUTiming = true;

    if( !htod && !dtoh && !dtod )
    {
        //default:  All
        htod = true;
        dtoh = true;
        dtod = true;
    }

    if( RANGE_MODE == mode )
    {
        if( cutGetCmdLineArgumenti( argc, argv, "start", &start) )
        {
            if( start <= 0 )
            {
                printf("Illegal argument - start must be greater than zero\n");
                return;
            }   
        }
        else
        {
            printf("Must specify a starting size in range mode\n");
            printf("See --help for more information\n");
            return;
        }

        if( cutGetCmdLineArgumenti( argc, argv, "end", &end) )
        {
            if( end <= 0 )
            {
                printf("Illegal argument - end must be greater than zero\n");
                return;
            }

            if( start > end )
            {
                printf("Illegal argument - start is greater than end\n");
                return;
            }
        }
        else
        {
            printf("Must specify an end size in range mode.\n");
            printf("See --help for more information\n");
            return;
        }


        if( cutGetCmdLineArgumenti( argc, argv, "increment", &increment) )
        {
            if( increment <= 0 )
            {
                printf("Illegal argument - increment must be greater than zero\n");
                return;
            }
        }
        else
        {
            printf("Must specify an increment in user mode\n");
            printf("See --help for more information\n");
            return;
        }
    }
   
    if( htod )
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment, 
                       mode, HOST_TO_DEVICE, printmode, memMode, startDevice, endDevice, wc);
    }                       
    if( dtoh )
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                       mode, DEVICE_TO_HOST, printmode, memMode, startDevice, endDevice, wc);
    }                       
    if( dtod )
    {
        testBandwidth((unsigned int)start, (unsigned int)end, (unsigned int)increment,
                      mode, DEVICE_TO_DEVICE, printmode, memMode, startDevice, endDevice, wc);
    }                       

    printf("&&&& Test PASSED\n");

    cutFree( memModeStr); 

    return;
}

///////////////////////////////////////////////////////////////////////////////
//  Run a bandwidth test
///////////////////////////////////////////////////////////////////////////////
void
testBandwidth(unsigned int start, unsigned int end, unsigned int increment, 
              testMode mode, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    switch( mode )
    {
    case QUICK_MODE:
        printf("Quick Mode\n");
        testBandwidthQuick( DEFAULT_SIZE, kind, printmode, memMode, startDevice, endDevice, wc );
        break;
    case RANGE_MODE:
        printf("Range Mode\n");
        testBandwidthRange(start, end, increment, kind, printmode, memMode, startDevice, endDevice, wc);
        break;
    case SHMOO_MODE: 
        printf("Shmoo Mode\n");
        testBandwidthShmoo(kind, printmode, memMode, startDevice, endDevice, wc);
        break;
    default:  
        printf("Invalid testing mode\n");
        break;
    }

}

//////////////////////////////////////////////////////////////////////
//  Run a quick mode bandwidth test
//////////////////////////////////////////////////////////////////////
void
testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    testBandwidthRange(size, size, DEFAULT_INCREMENT, kind, printmode, memMode, startDevice, endDevice, wc);
}

///////////////////////////////////////////////////////////////////////
//  Run a range mode bandwidth test
//////////////////////////////////////////////////////////////////////
void
testBandwidthRange(unsigned int start, unsigned int end, unsigned int increment, 
                   memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    //count the number of copies we're going to run
    unsigned int count = 1 + ((end - start) / increment);
    
    unsigned int *memSizes = ( unsigned int * )malloc( count * sizeof( unsigned int ) );
    float *bandwidths = ( float * ) malloc( count * sizeof(float) );

    //print information for use
    switch(kind)
    {
    case DEVICE_TO_HOST:    printf("Device to Host Bandwidth for ");
        break;
    case HOST_TO_DEVICE:    printf("Host to Device Bandwidth for ");
        break;
    case DEVICE_TO_DEVICE:  printf("Device to Device Bandwidth\n");
        break;
    }
    if( DEVICE_TO_DEVICE != kind )
    {   switch(memMode)
        {
        case PAGEABLE:  printf("Pageable memory\n");
            break;
        case PINNED:    printf("Pinned memory\n");
						if (wc) printf("Write-Combined memory enabled\n");
            break;
        }
    }

    // Before calculating the cumulative bandwidth, initialize bandwidths array to NULL
    for (int i = 0; i < count; i++)
        bandwidths[i] = 0.0f;

    // Use the device asked by the user
    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipSetDevice(currentDevice);
	    //run each of the copies
	    for(unsigned int i = 0; i < count; i++)
	    {

			memSizes[i] = start + i * increment;
	        switch(kind)
	        {
	        case DEVICE_TO_HOST:    bandwidths[i] = testDeviceToHostTransfer( memSizes[i], memMode, wc );
	            break;
	        case HOST_TO_DEVICE:    bandwidths[i] = testHostToDeviceTransfer( memSizes[i], memMode, wc );
	            break;
	        case DEVICE_TO_DEVICE:  bandwidths[i] = testDeviceToDeviceTransfer( memSizes[i] );
	            break;
	        }
	        printf(".");
		}
		hipDeviceReset();
    } // Complete the bandwidth computation on all the devices
    printf("\n");

    //print results
    if(printmode == CSV)
    {
        printResultsCSV(memSizes, bandwidths, count);
    }
    else
    {
        printResultsReadable(memSizes, bandwidths, count);
    }

    //clean up
    free(memSizes);
    free(bandwidths);
}

//////////////////////////////////////////////////////////////////////////////
// Intense shmoo mode - covers a large range of values with varying increments
//////////////////////////////////////////////////////////////////////////////
void
testBandwidthShmoo(memcpyKind kind, printMode printmode, memoryMode memMode, int startDevice, int endDevice, bool wc)
{
    //print info for user
    switch(kind)
    {
    case DEVICE_TO_HOST:    printf("Device to Host Bandwidth for ");
        break;
    case HOST_TO_DEVICE:    printf("Host to Device Bandwidth for ");
        break;
    case DEVICE_TO_DEVICE:  printf("Device to Device Bandwidth\n");
        break;
    }
    if( DEVICE_TO_DEVICE != kind )
    {   switch(memMode)
        {
        case PAGEABLE:  printf("Pageable memory\n");
            break;
        case PINNED:    printf("Pinned memory\n");
						if (wc) printf("Write-Combined memory enabled\n");
            break;
        }
    }

    //count the number of copies to make
    unsigned int count = 1 + (SHMOO_LIMIT_20KB  / SHMOO_INCREMENT_1KB)
                        + ((SHMOO_LIMIT_50KB - SHMOO_LIMIT_20KB) / SHMOO_INCREMENT_2KB)
                        + ((SHMOO_LIMIT_100KB - SHMOO_LIMIT_50KB) / SHMOO_INCREMENT_10KB)
                        + ((SHMOO_LIMIT_1MB - SHMOO_LIMIT_100KB) / SHMOO_INCREMENT_100KB)
                        + ((SHMOO_LIMIT_16MB - SHMOO_LIMIT_1MB) / SHMOO_INCREMENT_1MB)
                        + ((SHMOO_LIMIT_32MB - SHMOO_LIMIT_16MB) / SHMOO_INCREMENT_2MB)
                        + ((SHMOO_MEMSIZE_MAX - SHMOO_LIMIT_32MB) / SHMOO_INCREMENT_4MB);

    unsigned int *memSizes = ( unsigned int * )malloc( count * sizeof( unsigned int ) );
    float *bandwidths = ( float * ) malloc( count * sizeof(float) );


    // Before calculating the cumulative bandwidth, initialize bandwidths array to NULL
    for (int i = 0; i < count; i++)
        bandwidths[i] = 0.0f;
   
    // Use the device asked by the user
    for (int currentDevice = startDevice; currentDevice <= endDevice; currentDevice++)
    {
        hipSetDevice(currentDevice);
        //Run the shmoo
        int iteration = 0;
        unsigned int memSize = 0;
        while( memSize <= SHMOO_MEMSIZE_MAX )
        {
            if( memSize < SHMOO_LIMIT_20KB )
            {
                memSize += SHMOO_INCREMENT_1KB;
            }
            else if( memSize < SHMOO_LIMIT_50KB )
            {
                memSize += SHMOO_INCREMENT_2KB;
            }else if( memSize < SHMOO_LIMIT_100KB )
            {
                memSize += SHMOO_INCREMENT_10KB;
            }else if( memSize < SHMOO_LIMIT_1MB )
            {
                memSize += SHMOO_INCREMENT_100KB;
            }else if( memSize < SHMOO_LIMIT_16MB )
            {
                memSize += SHMOO_INCREMENT_1MB;
            }else if( memSize < SHMOO_LIMIT_32MB )
            {
                memSize += SHMOO_INCREMENT_2MB;
            }else 
            {
                memSize += SHMOO_INCREMENT_4MB;
            }

            memSizes[iteration] = memSize;
            switch(kind)
            {
            case DEVICE_TO_HOST:    bandwidths[iteration] += testDeviceToHostTransfer( memSizes[iteration], memMode, wc );
                break;
            case HOST_TO_DEVICE:    bandwidths[iteration] += testHostToDeviceTransfer( memSizes[iteration], memMode, wc );
                break;
            case DEVICE_TO_DEVICE:  bandwidths[iteration] += testDeviceToDeviceTransfer( memSizes[iteration] );
                break;
            }
            iteration++;
            printf(".");
       }
    } // Complete the bandwidth computation on all the devices

    printf("\n");

    //print results
    if( CSV == printmode)
    {
        printResultsCSV(memSizes, bandwidths, count);
    }
    else
    {
        printResultsReadable(memSizes, bandwidths, count);
    }

    //clean up
    free(memSizes);
    free(bandwidths);
}

///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    unsigned int timer = 0;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    unsigned char *h_idata = NULL;
    unsigned char *h_odata = NULL;
    hipEvent_t start, stop;

    cutilCheckError( cutCreateTimer( &timer ) );
    cutilSafeCall  ( hipEventCreate( &start ) );
    cutilSafeCall  ( hipEventCreate( &stop ) );
    
    //allocate host memory
    if( PINNED == memMode )
    {
        //pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
		cutilSafeCall( hipHostAlloc( (void**)&h_idata, memSize, (wc) ? hipHostMallocWriteCombined : 0 ) );
        cutilSafeCall( hipHostAlloc( (void**)&h_odata, memSize, (wc) ? hipHostMallocWriteCombined : 0 ) );
#else
		cutilSafeCall( hipHostMalloc( (void**)&h_idata, memSize ) );
        cutilSafeCall( hipHostMalloc( (void**)&h_odata, memSize ) );
#endif
    }
    else
    {
        //pageable memory mode - use malloc
        h_idata = (unsigned char *)malloc( memSize );
        h_odata = (unsigned char *)malloc( memSize );
    }
    //initialize the memory
    for(unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_idata[i] = (unsigned char) (i & 0xff);
    }

    // allocate device memory
    unsigned char* d_idata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, memSize));

    //initialize the device memory
    cutilSafeCall( hipMemcpy( d_idata, h_idata, memSize,
                                hipMemcpyHostToDevice) );

    //copy data from GPU to Host
    cutilCheckError( cutStartTimer( timer));
    cutilSafeCall( hipEventRecord( start, 0 ) );
    if( PINNED == memMode )
    {
        for( unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++ )
        {
            cutilSafeCall( hipMemcpyAsync( h_odata, d_idata, memSize,
                                    hipMemcpyDeviceToHost, 0) );
        }
    }
    else
    {
        for( unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++ )
        {
            cutilSafeCall( hipMemcpy( h_odata, d_idata, memSize,
                                    hipMemcpyDeviceToHost) );
        }
    }
    cutilSafeCall( hipEventRecord( stop, 0 ) );

    // make sure GPU has finished copying
    cutilSafeCall( hipDeviceSynchronize() );
    //get the the total elapsed time in ms
    cutilCheckError( cutStopTimer( timer));
    cutilSafeCall( hipEventElapsedTime( &elapsedTimeInMs, start, stop ) );
    if( PINNED != memMode || bDontUseGPUTiming )
    {
        elapsedTimeInMs = cutGetTimerValue( timer);
    }
    
    //calculate bandwidth in MB/s
    bandwidthInMBs = (1e3f * memSize * (float)MEMCOPY_ITERATIONS) / 
                                        (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    cutilSafeCall( hipEventDestroy(stop) );
    cutilSafeCall( hipEventDestroy(start) );
    cutilCheckError( cutDeleteTimer( timer));
    if( PINNED == memMode )
    {
        cutilSafeCall( hipHostFree(h_idata) );
        cutilSafeCall( hipHostFree(h_odata) );
    }
    else
    {
        free(h_idata);
        free(h_odata);
    }
    cutilSafeCall(hipFree(d_idata));
    
    return bandwidthInMBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    unsigned int timer = 0;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;
    cutilCheckError( cutCreateTimer( &timer ) );
    cutilSafeCall( hipEventCreate( &start ) );
    cutilSafeCall( hipEventCreate( &stop ) );

    //allocate host memory
    unsigned char *h_odata = NULL;
    if( PINNED == memMode )
    {
#if CUDART_VERSION >= 2020
        //pinned memory mode - use special function to get OS-pinned memory
        cutilSafeCall( hipHostAlloc( (void**)&h_odata, memSize, (wc) ? hipHostMallocWriteCombined : 0 ) );
#else
        //pinned memory mode - use special function to get OS-pinned memory
        cutilSafeCall( hipHostMalloc( (void**)&h_odata, memSize ) );
#endif
    }
    else
    {
        //pageable memory mode - use malloc
        h_odata = (unsigned char *)malloc( memSize );
    }
    unsigned char *h_cacheClear1 = (unsigned char *)malloc( CACHE_CLEAR_SIZE );
    unsigned char *h_cacheClear2 = (unsigned char *)malloc( CACHE_CLEAR_SIZE );
    //initialize the memory
    for(unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_odata[i] = (unsigned char) (i & 0xff);
    }
    for(unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++)
    {
        h_cacheClear1[i] = (unsigned char) (i & 0xff);
        h_cacheClear2[i] = (unsigned char) (0xff - (i & 0xff));
    }

    //allocate device memory
    unsigned char* d_idata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, memSize));

    cutilCheckError( cutStartTimer( timer));
    cutilSafeCall( hipEventRecord( start, 0 ) );
    //copy host memory to device memory
    if( PINNED == memMode )
    {
        for(unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            cutilSafeCall( hipMemcpyAsync( d_idata, h_odata, memSize,
                                    hipMemcpyHostToDevice, 0) );
        }
    }
    else {
        for(unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++)
        {
            cutilSafeCall( hipMemcpy( d_idata, h_odata, memSize,
                                    hipMemcpyHostToDevice) );
        }
    }

    cutilSafeCall( hipEventRecord( stop, 0 ) );
    cutilSafeCall( hipDeviceSynchronize() );
    //total elapsed time in ms
    cutilCheckError( cutStopTimer( timer));
    cutilSafeCall( hipEventElapsedTime( &elapsedTimeInMs, start, stop ) );
    if ( PINNED != memMode || bDontUseGPUTiming )
    {
        elapsedTimeInMs = cutGetTimerValue( timer);
    }
    cutilCheckError( cutResetTimer( timer));
        
    //calculate bandwidth in MB/s
    bandwidthInMBs = (1e3f * memSize * (float)MEMCOPY_ITERATIONS) / 
                                        (elapsedTimeInMs * (float)(1 << 20));

    //clean up memory
    cutilSafeCall( hipEventDestroy(stop) );
    cutilSafeCall( hipEventDestroy(start) );
    cutilCheckError( cutDeleteTimer( timer));
    if( PINNED == memMode )
    {
        cutilSafeCall( hipHostFree(h_odata) );
    }
    else
    {
        free(h_odata);
    }
    free(h_cacheClear1);
    free(h_cacheClear2);
    cutilSafeCall(hipFree(d_idata));

    return bandwidthInMBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float
testDeviceToDeviceTransfer(unsigned int memSize)
{
    unsigned int timer = 0;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;

    cutilCheckError( cutCreateTimer( &timer ) );
    cutilSafeCall( hipEventCreate( &start ) );
    cutilSafeCall( hipEventCreate( &stop ) );

    //allocate host memory
    unsigned char *h_idata = (unsigned char *)malloc( memSize );
    //initialize the host memory
    for(unsigned int i = 0; i < memSize/sizeof(unsigned char); i++)
    {
        h_idata[i] = (unsigned char) (i & 0xff);
    }

    //allocate device memory
    unsigned char *d_idata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, memSize));
    unsigned char *d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, memSize));

    //initialize memory
    cutilSafeCall( hipMemcpy( d_idata, h_idata, memSize,
                                hipMemcpyHostToDevice) );

    //run the memcopy
    cutilCheckError( cutStartTimer( timer));
    cutilSafeCall( hipEventRecord( start, 0 ) );
    for( unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++ )
    {
        cutilSafeCall( hipMemcpy( d_odata, d_idata, memSize,
                                hipMemcpyDeviceToDevice) );
    }
    cutilSafeCall( hipEventRecord( stop, 0 ) );
  
    //Since device to device memory copies are non-blocking,
    //hipDeviceSynchronize() is required in order to get
    //proper timing.
    cutilSafeCall( hipDeviceSynchronize() );

    //get the the total elapsed time in ms
    cutilCheckError( cutStopTimer( timer));
    cutilSafeCall( hipEventElapsedTime( &elapsedTimeInMs, start, stop ) );
    if ( bDontUseGPUTiming )
    {
        elapsedTimeInMs = cutGetTimerValue( timer);
    }
    
    //calculate bandwidth in MB/s
    bandwidthInMBs = 2.0f * (1e3f * memSize * (float)MEMCOPY_ITERATIONS) / 
                                        (elapsedTimeInMs * (float)(1 << 20));
    
    //clean up memory
    cutilCheckError( cutDeleteTimer( timer));
    free(h_idata);
    cutilSafeCall(hipEventDestroy(stop));
    cutilSafeCall(hipEventDestroy(start));
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));

    return bandwidthInMBs;
}

/////////////////////////////////////////////////////////
//print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, float *bandwidths, unsigned int count)
{
    printf("Transfer Size (Bytes)\tBandwidth(MB/s)\n");
    for(unsigned int i = 0; i < count; i++)
    {
        printf("%9u\t\t%.1f\n", memSizes[i], bandwidths[i]);
    }
    printf("\n");
    fflush(stdout);
}

///////////////////////////////////////////////////////////////////////////
//print results in CSV format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, float *bandwidths, unsigned int count)
{
    printf("Transfer size (Bytes),");
    for(unsigned int i = 0; i < count; i++)
    {
        printf("%u,", memSizes[i]);
    }
    printf("\n");

    printf("Bandwidth (MB/s),");
    for(unsigned int i = 0; i < count; i++)
    {
        printf("%.1f,", bandwidths[i]);
    }
    printf("\n\n");
    fflush(stdout);
}

///////////////////////////////////////////////////////////////////////////
//Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void)
{
    printf("Usage:  bandwidthTest [OPTION]...\n");
    printf("Test the bandwidth for device to host, host to device, and device to device transfers\n");
    printf("\n");
    printf("Example:  measure the bandwidth of device to host pinned memory copies in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments\n");
    printf("./bandwidthTest --memory=pinned --mode=range --start=1024 --end=102400 --increment=1024 --dtoh\n");

    printf("\n");
    printf("Options:\n");
    printf("--help\tDisplay this help menu\n");
    printf("--csv\tPrint results as a CSV\n");
    printf("--device=[deviceno]\tSpecify the device device to be used\n");
    printf("  all - compute cumulative bandwidth on all the devices\n");
    printf("  0,1,2,...,n - Specify any particular device to be used\n");
    printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
    printf("  pageable - pageable memory\n");
    printf("  pinned   - non-pageable system memory\n");
    printf("--mode=[MODE]\tSpecify the mode to use\n");
    printf("  quick - performs a quick measurement\n");
    printf("  range - measures a user-specified range of values\n");
    printf("  shmoo - performs an intense shmoo of a large range of values\n");

    printf("--htod\tMeasure host to device transfers\n");   
    printf("--dtoh\tMeasure device to host transfers\n");
    printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
	printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
    printf("--cputiming\tForce CPU-based timing always\n");
    
    printf("Range mode options\n");
    printf("--start=[SIZE]\tStarting transfer size in bytes\n");
    printf("--end=[SIZE]\tEnding transfer size in bytes\n");
    printf("--increment=[SIZE]\tIncrement size in bytes\n");
}

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <cutil_inline.h>
#include <cutil_math.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

texture<uchar,  3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>     transferTex; // 1D transfer function texture

typedef struct {
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray {
	float3 o;	// origin
	float3 d;	// direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
         float density, float brightness,
         float transferOffset, float transferScale)
{
    int maxSteps = 500;
    float tstep = 0.01f;
    float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from back to front, accumulating color
    float4 sum = make_float4(0.0f);;
    float t = tfar;
	for(int i=0; i<maxSteps; i++) {		
        float3 pos = eyeRay.o + eyeRay.d*t;
        pos = pos*0.5f+0.5f;    // map position to [0, 1] coordinates

        // read from 3D texture
        float sample = tex3D(tex, pos.x, pos.y, pos.z);

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);

        // accumulate result
        sum = lerp(sum, col, col.w*density);

        t -= tstep;
        if (t < tnear) break;
    }
    sum *= brightness;

    if ((x < imageW) && (y < imageH)) {
        // write output color
        uint i = __umul24(y, imageW) + x;
        d_output[i] = rgbaFloatToInt(sum);
    }
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void initCuda(uchar *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    cutilSafeCall( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );  

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

    // create transfer function texture
    float4 transferFunc[] = {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };

    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray* d_transferFuncArray;
    cutilSafeCall(hipMallocArray( &d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1)); 
    cutilSafeCall(hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc2));
}

extern "C" 
void freeCudaBuffers()
{
    cutilSafeCall(hipFreeArray(d_volumeArray));
    cutilSafeCall(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH, 
				   float density, float brightness, float transferOffset, float transferScale)
{
	d_render<<<gridSize, blockSize>>>( d_output, imageW, imageH, density, 
										brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix) );
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_

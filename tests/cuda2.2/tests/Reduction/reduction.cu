#include "hip/hip_runtime.h"
/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.  This source code is a "commercial item" as
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer software" and "commercial computer software
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*/

/*
    Parallel reduction

    This sample shows how to perform a reduction operation on an array of values
    to produce a single value.

    Reductions are a very common computation in parallel algorithms.  Any time
    an array of values needs to be reduced to a single value using a binary 
    associative operator, a reduction can be used.  Example applications include
    statistics computaions such as mean and standard deviation, and image 
    processing applications such as finding the total luminance of an
    image.

    This code performs sum reductions, but any associative operator such as
    min() or max() could also be used.

    It assumes the input size is a power of 2.

    COMMAND LINE ARGUMENTS

    "--shmoo":         Test performance for 1 to 32M elements with each of the 7 different kernels
    "--n=<N>":         Specify the number of elements to reduce (default 1048576)
    "--threads=<N>":   Specify the number of threads per block (default 128)
    "--kernel=<N>":    Specify which kernel to run (0-6, default 6)
    "--maxblocks=<N>": Specify the maximum number of thread blocks to launch (kernel 6 only, default 64)
    "--cpufinal":      Read back the per-block results and do final sum of block sums on CPU (default false)
    "--cputhresh=<N>": The threshold of number of blocks sums below which to perform a CPU final reduction (default 1)
    
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

#include <tests/Reduction/reduction.h>

enum ReduceType
{
    REDUCE_INT,
    REDUCE_FLOAT,
    REDUCE_DOUBLE
};

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
template <class T>
void runTest( int argc, char** argv, ReduceType datatype);

#ifdef WIN32
#define strcasecmp strcmpi
#endif

extern "C"
bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    char *typeChoice;
    cutGetCmdLineArgumentstr( argc, (const char**) argv, "type", &typeChoice);

    if (0 == typeChoice)
    {
        typeChoice = (char*)malloc(4 * sizeof(char));
        strcpy(typeChoice, "int");
    }

    ReduceType datatype = REDUCE_INT;

    if (!strcasecmp(typeChoice, "float"))
        datatype = REDUCE_FLOAT;
    else if (!strcasecmp(typeChoice, "double"))
        datatype = REDUCE_DOUBLE;
    else
        datatype = REDUCE_INT;

    printf("Reducing array of type %s.\n", typeChoice);

    hipDeviceProp_t deviceProp;
    deviceProp.major = 1;
    deviceProp.minor = 0;
    int desiredMinorRevision = 0;

    if (datatype == REDUCE_DOUBLE)
    {
        deviceProp.minor = 3;
        desiredMinorRevision = 3;
    }

    int dev;

    cutilSafeCallNoSync(hipChooseDevice(&dev, &deviceProp));
    cutilSafeCallNoSync(hipGetDeviceProperties(&deviceProp, dev));

    if(deviceProp.major > 1 || deviceProp.minor >= desiredMinorRevision)
    {
        printf("Using Device %d: \"%s\"\n", dev, deviceProp.name);
        cutilSafeCallNoSync(hipSetDevice(dev));
    }
    else if (desiredMinorRevision == 3)
    {
        printf("There is no device supporting compute capability %d.%d.\n\n",
            1, desiredMinorRevision);
        printf("TEST PASSED");
        hipDeviceReset();
        cutilExit(argc, argv);
    }

    switch (datatype)
    {
    default:
    case REDUCE_INT:
        runTest<int>( argc, argv, datatype);
        break;
    case REDUCE_FLOAT:
        runTest<float>( argc, argv, datatype);
        break;
    case REDUCE_DOUBLE:
        runTest<double>( argc, argv, datatype);
        break;
    }
    
    hipDeviceReset();
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Compute sum reduction on CPU
//! We use Kahan summation for an accurate sum of large arrays.
//! http://en.wikipedia.org/wiki/Kahan_summation_algorithm
//! 
//! @param data       pointer to input data
//! @param size       number of input data elements
////////////////////////////////////////////////////////////////////////////////
template<class T>
T reduceCPU(T *data, int size)
{
    T sum = data[0];
    T c = (T)0.0;              
    for (int i = 1; i < size; i++)
    {
        T y = data[i] - c;  
        T t = sum + y;      
        c = (t - sum) - y;  
        sum = t;            
    }
    return sum;
}

unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel 
// 6, we observe the maximum specified number of blocks, because each thread in 
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
    
    
    
    if (whichKernel < 3)
    {
        threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
        blocks = (n + threads - 1) / threads;
    }
    else
    {
        threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }
        

    if (whichKernel == 6)
        blocks = min(maxBlocks, blocks);
}

////////////////////////////////////////////////////////////////////////////////
// This function performs a reduction of the input data multiple times and 
// measures the average reduction time.
////////////////////////////////////////////////////////////////////////////////
template <class T>
T benchmarkReduce(int  n, 
                  int  numThreads,
                  int  numBlocks,
                  int  maxThreads,
                  int  maxBlocks,
                  int  whichKernel, 
                  int  testIterations,
                  bool cpuFinalReduction,
                  int  cpuFinalThreshold,
                  unsigned int timer,
                  bool useSM13,
                  T* h_odata,
                  T* d_idata, 
                  T* d_odata)
{
    T gpu_result = 0;
    bool needReadBack = true;

    for (int i = 0; i < testIterations; ++i)
    {
        gpu_result = 0;

        hipDeviceSynchronize();
        cutilCheckError( cutStartTimer( timer));

        // execute the kernel
        if (useSM13)
            reduce_sm13<T>(n, numThreads, numBlocks, whichKernel, d_idata, d_odata);
        else
            reduce_sm10<T>(n, numThreads, numBlocks, whichKernel, d_idata, d_odata);

        // check if kernel execution generated an error
        cutilCheckMsg("Kernel execution failed");

        if (cpuFinalReduction)
        {
            // sum partial sums from each block on CPU        
            // copy result from device to host
            cutilSafeCallNoSync( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(T), hipMemcpyDeviceToHost) );

            for(int i=0; i<numBlocks; i++) 
            {
                gpu_result += h_odata[i];
            }

            needReadBack = false;
        }
        else
        {
            // sum partial block sums on GPU
            int s=numBlocks;
            int kernel = (whichKernel == 6) ? 5 : whichKernel;
            while(s > cpuFinalThreshold) 
            {
                int threads = 0, blocks = 0;
                getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);
                
                if (useSM13)
                    reduce_sm13<T>(s, threads, blocks, kernel, d_odata, d_odata);
                else
                    reduce_sm10<T>(s, threads, blocks, kernel, d_odata, d_odata);
                
                if (kernel < 3)
                    s = (s + threads - 1) / threads;
                else
                    s = (s + (threads*2-1)) / (threads*2);
            }
            
            if (s > 1)
            {
                // copy result from device to host
                cutilSafeCallNoSync( hipMemcpy( h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost) );

                for(int i=0; i < s; i++) 
                {
                    gpu_result += h_odata[i];
                }

                needReadBack = false;
            }
        }

        hipDeviceSynchronize();
        cutilCheckError( cutStopTimer(timer) );      
    }

    if (needReadBack)
    {
        // copy final sum from device to host
        cutilSafeCallNoSync( hipMemcpy( &gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost) );
    }

    return gpu_result;
}

////////////////////////////////////////////////////////////////////////////////
// This function calls benchmarkReduce multple times for a range of array sizes
// and prints a report in CSV (comma-separated value) format that can be used for
// generating a "shmoo" plot showing the performance for each kernel variation
// over a wide range of input sizes.
////////////////////////////////////////////////////////////////////////////////
template <class T>
void shmoo(int minN, int maxN, int maxThreads, int maxBlocks, ReduceType datatype)
{ 
    bool useSM13 = (datatype == REDUCE_DOUBLE); 

    // create random input data on CPU
    unsigned int bytes = maxN * sizeof(T);

    T *h_idata = (T*) malloc(bytes);

    for(int i = 0; i < maxN; i++) {
        // Keep the numbers small so we don't get truncation error in the sum
        if (datatype == REDUCE_INT)
            h_idata[i] = (T)(rand() & 0xFF);
        else
            h_idata[i] = (rand() & 0xFF) / (T)RAND_MAX;
    }

    int maxNumBlocks = maxN / maxThreads;

    // allocate mem for the result on host side
    T* h_odata = (T*) malloc(maxNumBlocks*sizeof(T));

    // allocate device memory and data
    T* d_idata = NULL;
    T* d_odata = NULL;

    cutilSafeCallNoSync( hipMalloc((void**) &d_idata, bytes) );
    cutilSafeCallNoSync( hipMalloc((void**) &d_odata, maxNumBlocks*sizeof(T)) );

    // copy data directly to device memory
    cutilSafeCallNoSync( hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice) );
    cutilSafeCallNoSync( hipMemcpy(d_odata, h_idata, maxNumBlocks*sizeof(T), hipMemcpyHostToDevice) );

    // warm-up
#if 0
    for (int kernel = 0; kernel < 7; kernel++)
    {
        if (useSM13)
            reduce_sm13<T>(maxN, maxThreads, maxNumBlocks, kernel, d_idata, d_odata);
        else
            reduce_sm10<T>(maxN, maxThreads, maxNumBlocks, kernel, d_idata, d_odata);
    }
    int testIterations = 100;
#else
    int testIterations = 1;
#endif

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    
    // print headers
    printf("Time in milliseconds for various numbers of elements for each kernel\n");
    printf("\n\n");
    printf("Kernel");
    for (int i = minN; i <= maxN; i *= 2)
    {
        printf(", %d", i);
    }
   
    for (int kernel = 0; kernel < 7; kernel++)
    {
        printf("\n");
        printf("%d", kernel);
        for (int i = minN; i <= maxN; i *= 2)
        {
            cutResetTimer(timer);
            int numBlocks = 0;
            int numThreads = 0;
            getNumBlocksAndThreads(kernel, i, maxBlocks, maxThreads, numBlocks, numThreads);
            
            
            benchmarkReduce(i, numThreads, numBlocks, maxThreads, maxBlocks, kernel, 
                            testIterations, false, 1, timer, useSM13, h_odata, d_idata, d_odata);

            float reduceTime = cutGetAverageTimerValue(timer);
            printf(", %f", reduceTime);
        }
        
    }

    // cleanup
    cutilCheckError(cutDeleteTimer(timer));
    free(h_idata);
    free(h_odata);

    cutilSafeCallNoSync(hipFree(d_idata));
    cutilSafeCallNoSync(hipFree(d_odata));    
}

////////////////////////////////////////////////////////////////////////////////
// The main function whihc runs the reduction test.
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
runTest( int argc, char** argv, ReduceType datatype) 
{
    int size = 1<<20;    // number of elements to reduce
    int maxThreads = 128;  // number of threads per block
    int whichKernel = 6;
    int maxBlocks = 64;
    bool cpuFinalReduction = false;
    int cpuFinalThreshold = 1;

    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", &size);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "threads", &maxThreads);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "kernel", &whichKernel);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "maxblocks", &maxBlocks);
    
    printf("%d elements\n", size);
    printf("%d threads (max)\n", maxThreads);

    cpuFinalReduction = (cutCheckCmdLineFlag( argc, (const char**) argv, "cpufinal") == CUTTrue);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "cputhresh", &cpuFinalThreshold);

    bool runShmoo = (cutCheckCmdLineFlag(argc, (const char**) argv, "shmoo") == CUTTrue);

    if (runShmoo)
    {
        shmoo<T>(1, 33554432, maxThreads, maxBlocks, datatype);
    }
    else
    {

        // create random input data on CPU
        unsigned int bytes = size * sizeof(T);

        T *h_idata = (T *) malloc(bytes);

        for(int i=0; i<size; i++) 
        {
            // Keep the numbers small so we don't get truncation error in the sum
            if (datatype == REDUCE_INT)
                h_idata[i] = (T)(rand() & 0xFF);
            else
                h_idata[i] = (rand() & 0xFF) / (T)RAND_MAX;
        }

        int numBlocks = 0;
        int numThreads = 0;
        getNumBlocksAndThreads(whichKernel, size, maxBlocks, maxThreads, numBlocks, numThreads);
        if (numBlocks == 1) cpuFinalThreshold = 1;

        // allocate mem for the result on host side
        T* h_odata = (T*) malloc(numBlocks*sizeof(T));

        printf("%d blocks\n", numBlocks);

        // allocate device memory and data
        T* d_idata = NULL;
        T* d_odata = NULL;

        cutilSafeCallNoSync( hipMalloc((void**) &d_idata, bytes) );
        cutilSafeCallNoSync( hipMalloc((void**) &d_odata, numBlocks*sizeof(T)) );

        // copy data directly to device memory
        cutilSafeCallNoSync( hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice) );
        cutilSafeCallNoSync( hipMemcpy(d_odata, h_idata, numBlocks*sizeof(T), hipMemcpyHostToDevice) );

    #if 0
        // warm-up
        if (datatype == REDUCE_DOUBLE)
            reduce_sm13<T>(size, numThreads, numBlocks, whichKernel, d_idata, d_odata);
        else
            reduce_sm10<T>(size, numThreads, numBlocks, whichKernel, d_idata, d_odata);
        
        int testIterations = 100;
    #else
        int testIterations = 1;
    #endif

        unsigned int timer = 0;
        cutilCheckError( cutCreateTimer( &timer));
        
        T gpu_result = 0;

        gpu_result = benchmarkReduce<T>(size, numThreads, numBlocks, maxThreads, maxBlocks,
                                        whichKernel, testIterations, cpuFinalReduction, 
                                        cpuFinalThreshold, timer, datatype == REDUCE_DOUBLE,
                                        h_odata, d_idata, d_odata);

        float reduceTime = cutGetAverageTimerValue(timer);
        printf("Average time: %f ms\n", reduceTime);
        printf("Bandwidth:    %f GB/s\n\n", (size * sizeof(int)) / (reduceTime * 1.0e6));

        // compute reference solution
        T cpu_result = reduceCPU<T>(h_idata, size);

        if (datatype == REDUCE_INT)
        {
            printf("GPU result = %d\n", gpu_result);
            printf("CPU result = %d\n", cpu_result);

            printf("TEST %s\n", (gpu_result == cpu_result) ? "PASSED" : "FAILED");
        }
        else
        {
            printf("GPU result = %0.12f\n", gpu_result);
            printf("CPU result = %0.12f\n", cpu_result);

            double threshold = (datatype == REDUCE_FLOAT) ? 1e-8 * size : 1e-12;
            double diff = abs((double)gpu_result - (double)cpu_result);
            printf("TEST %s\n", (diff < threshold) ? "PASSED" : "FAILED");
        }

        
      
        // cleanup
        cutilCheckError( cutDeleteTimer(timer) );
        free(h_idata);
        free(h_odata);

        cutilSafeCallNoSync(hipFree(d_idata));
        cutilSafeCallNoSync(hipFree(d_odata));
    }
}

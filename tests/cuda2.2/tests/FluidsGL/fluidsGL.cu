#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <GL/glew.h>
#include <hipfft/hipfft.h>
#include <cutil_inline.h>
#include <cutil_gl_inline.h>
#include <cuda_gl_interop.h>
//#include <rendercheck_gl.h>


#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include "fluidsGL_kernels.cuh"

#define MAX_EPSILON_ERROR 1.0f

// Define the files that are to be save and the reference images for validation
const char *sOriginal[] =
{
    "fluidsGL.ppm",
    NULL
};

const char *sReference[] =
{
    "ref_fluidsGL.ppm",
    NULL
};

// CUDA example code that implements the frequency space version of 
// Jos Stam's paper 'Stable Fluids' in 2D. This application uses the 
// CUDA FFT library (CUFFT) to perform velocity diffusion and to 
// force non-divergence in the velocity field at each time step. It uses 
// CUDA-OpenGL interoperability to update the particle field directly
// instead of doing a copy to system memory before drawing. Texture is
// used for automatic bilinear interpolation at the velocity advection step. 

#if 1
#define DIM    64        // Square size of solver domain
#else
#define DIM    512       // Square size of solver domani
#endif
#define DS    (DIM*DIM)  // Total domain size
#define CPADW (DIM/2+1)  // Padded width for real->complex in-place FFT
#define RPADW (2*(DIM/2+1))  // Padded width for real->complex in-place FFT
#define PDS   (DIM*CPADW) // Padded total domain size

#define DT     0.09f     // Delta T for interative solver
#define VIS    0.0025f   // Viscosity constant
#define FORCE (5.8f*DIM) // Force scale factor 
#define FR     4         // Force update radius

#define TILEX 64 // Tile width
#define TILEY 64 // Tile height
#define TIDSX 64 // Tids in X
#define TIDSY 4  // Tids in Y

void cleanup(void);       

// CUFFT plan handle
static hipfftHandle planr2c;
static hipfftHandle planc2r;
static cData *vxfield = NULL;
static cData *vyfield = NULL;

cData *hvfield = NULL;
cData *dvfield = NULL;
static int wWidth = max(512,DIM);
static int wHeight = max(512,DIM);

static int clicked = 0;
static int fpsCount = 0;
static int fpsLimit = 1;
unsigned int timer;

// Particle data
GLuint vbo = 0;                 // OpenGL vertex buffer object
static cData *particles = NULL; // particle positions in host memory
static int lastx = 0, lasty = 0;

// Texture pitch
size_t tPitch = 0; // Now this is compatible with gcc in 64-bit

bool				  g_bQAReadback     = false;
bool				  g_bQAAddTestForce = true;
int					  g_iFrameToCompare = 4;
int                   frame = 0;
int                   g_TotalErrors     = 0;

// CheckFBO/BackBuffer class objects
//CheckRender       *g_CheckRender = NULL;

void autoTest();


void addForces(cData *v, int dx, int dy, int spx, int spy, float fx, float fy, int r) { 

    dim3 tids(2*r+1, 2*r+1);
    
    addForces_k<<<1, tids>>>(v, dx, dy, spx, spy, fx, fy, r, tPitch);
    cutilCheckMsg("addForces_k failed.");
}

void advectVelocity(cData *v, float *vx, float *vy,
                    int dx, int pdx, int dy, float dt) { 
    
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));

    dim3 tids(TIDSX, TIDSY);

    updateTexture(v, DIM*sizeof(cData), DIM, tPitch);
    advectVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, dt, TILEY/TIDSY);

    cutilCheckMsg("advectVelocity_k failed.");
}

void diffuseProject(cData *vx, cData *vy, int dx, int dy, float dt,
                    float visc) { 
    // Forward FFT
    hipfftExecR2C(planr2c, (hipfftReal*)vx, (hipfftComplex*)vx); 
    hipfftExecR2C(planr2c, (hipfftReal*)vy, (hipfftComplex*)vy);

    uint3 grid = make_uint3((dx/TILEX)+(!(dx%TILEX)?0:1), 
                            (dy/TILEY)+(!(dy%TILEY)?0:1), 1);

    uint3 tids = make_uint3(TIDSX, TIDSY, 1);
    
    diffuseProject_k<<<grid, tids>>>(vx, vy, dx, dy, dt, visc, TILEY/TIDSY);
    cutilCheckMsg("diffuseProject_k failed.");

    // Inverse FFT
    hipfftExecC2R(planc2r, (hipfftComplex*)vx, (hipfftReal*)vx); 
    hipfftExecC2R(planc2r, (hipfftComplex*)vy, (hipfftReal*)vy);
}

void updateVelocity(cData *v, float *vx, float *vy, 
                    int dx, int pdx, int dy) { 

    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));

    dim3 tids(TIDSX, TIDSY);

    updateVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, TILEY/TIDSY, tPitch);
    cutilCheckMsg("updateVelocity_k failed.");
}

void advectParticles(GLuint buffer, cData *v, int dx, int dy, float dt) {
    
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));

    dim3 tids(TIDSX, TIDSY);

    cData *p;
    cudaGLMapBufferObject((void**)&p, buffer);
    cutilCheckMsg("cudaGLMapBufferObject failed");
   
    advectParticles_k<<<grid, tids>>>(p, v, dx, dy, dt, TILEY/TIDSY, tPitch);
    cutilCheckMsg("advectParticles_k failed.");
    
    cudaGLUnmapBufferObject(buffer);
    cutilCheckMsg("cudaGLUnmapBufferObject failed");
}

void display(void) {  
   cutilCheckError(cutStartTimer(timer));  
    
   // simulate fluid
   advectVelocity(dvfield, (float*)vxfield, (float*)vyfield, DIM, RPADW, DIM, DT);
   diffuseProject(vxfield, vyfield, CPADW, DIM, DT, VIS);
   updateVelocity(dvfield, (float*)vxfield, (float*)vyfield, DIM, RPADW, DIM);
   advectParticles(vbo, dvfield, DIM, DIM, DT);
   
   // render points from vertex buffer
   glClear(GL_COLOR_BUFFER_BIT);
   glColor4f(0,1,0,0.5f); glPointSize(1);
   glEnable(GL_POINT_SMOOTH);
   glEnable(GL_BLEND);
   glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
   glEnableClientState(GL_VERTEX_ARRAY);    
   glDisable(GL_DEPTH_TEST);
   glDisable(GL_CULL_FACE); 
   glBindBufferARB(GL_ARRAY_BUFFER_ARB, vbo);
   glVertexPointer(2, GL_FLOAT, 0, NULL);
   glDrawArrays(GL_POINTS, 0, DS);
   glBindBufferARB(GL_ARRAY_BUFFER_ARB, 0);
   glDisableClientState(GL_VERTEX_ARRAY); 
   glDisableClientState(GL_TEXTURE_COORD_ARRAY); 
   glDisable(GL_TEXTURE_2D);

   // Finish timing before swap buffers to avoid refresh sync
   cutilCheckError(cutStopTimer(timer));  
   glutSwapBuffers();
    
   fpsCount++;
   if (fpsCount == fpsLimit) {
       char fps[256];
       float ifps = 1.f / (cutGetAverageTimerValue(timer) / 1000.f);
       sprintf(fps, "Cuda/GL Stable Fluids (%d x %d): %3.1f fps", DIM, DIM, ifps);  
       glutSetWindowTitle(fps);
       fpsCount = 0; 
       fpsLimit = (int)max(ifps, 1.f);
       cutilCheckError(cutResetTimer(timer));  
    }

    glutPostRedisplay();
    
    if( frame < g_iFrameToCompare )
    {
    	++frame;
    }
    else
    {
    	printf( "TEST PASSED\n" );
    	exit (0);  	
    }
}

void autoTest() 
{
	for(int count=0;count<g_iFrameToCompare;count++)
	{
		// add in a little force so the automated testing is interesing.
		if(g_bQAReadback && g_bQAAddTestForce) 
		{
			int x = wWidth/(count+1); int y = wHeight/(count+1);
			float fx = (x / (float)wWidth);        
			float fy = (y / (float)wHeight);
			int nx = (int)(fx * DIM);        
			int ny = (int)(fy * DIM);   

			int ddx = 35;
			int ddy = 35;
			fx = ddx / (float)wWidth;
			fy = ddy / (float)wHeight;
			int spy = ny-FR;
			int spx = nx-FR;

            addForces(dvfield, DIM, DIM, spx, spy, FORCE * DT * fx, FORCE * DT * fy, FR);
            lastx = x; lasty = y;
			//g_bQAAddTestForce = false; // only add it once
		}
        display();
    }

	// compare to offical reference image, printing PASS or FAIL.
    /*
    printf("> (Frame %d) Readback BackBuffer\n", 100);
    g_CheckRender->readback( wWidth, wHeight, NULL );
    g_CheckRender->savePPM(sOriginal[0], true, NULL);
    if (!g_CheckRender->PPMvsPPM(sOriginal[0], sReference[0], MAX_EPSILON_ERROR)) {
        g_TotalErrors++;
    }
    */
}


void idle(void) {
    glutPostRedisplay();
}

void initParticles(cData *p, int dx, int dy) {
    int i, j;
    for (i = 0; i < dy; i++) {
        for (j = 0; j < dx; j++) {
            p[i*dx+j].x = ((j+0.5)/dx) + 
                          (rand() / (float)RAND_MAX - 0.5f) / dx;
            p[i*dx+j].y = ((i+0.5)/dy) + 
                          (rand() / (float)RAND_MAX - 0.5f) / dy;
        }
    }
}

void keyboard( unsigned char key, int x, int y) {
    switch( key) {
        case 27:
        exit (0);
        case 'r':
            memset(hvfield, 0, sizeof(cData) * DS);
            hipMemcpy(dvfield, hvfield, sizeof(cData) * DS, 
                       hipMemcpyHostToDevice);

            initParticles(particles, DIM, DIM);

            cudaGLUnregisterBufferObject(vbo);
            cutilCheckMsg("cudaGLUnregisterBufferObject failed");
    
            glBindBufferARB(GL_ARRAY_BUFFER_ARB, vbo);
            glBufferDataARB(GL_ARRAY_BUFFER_ARB, sizeof(cData) * DS, 
                            particles, GL_DYNAMIC_DRAW_ARB);
            glBindBufferARB(GL_ARRAY_BUFFER_ARB, 0);

            cudaGLRegisterBufferObject(vbo);
            cutilCheckMsg("cudaGLRegisterBufferObject failed");
            break;
        default: break;
    }
}

void click(int button, int updown, int x, int y) {
    lastx = x; lasty = y;
    clicked = !clicked;
}

void motion (int x, int y) {
    // Convert motion coordinates to domain
    float fx = (lastx / (float)wWidth);        
    float fy = (lasty / (float)wHeight);
    int nx = (int)(fx * DIM);        
    int ny = (int)(fy * DIM);   
    
    if (clicked && nx < DIM-FR && nx > FR-1 && ny < DIM-FR && ny > FR-1) {
        int ddx = x - lastx;
        int ddy = y - lasty;
        fx = ddx / (float)wWidth;
        fy = ddy / (float)wHeight;
        int spy = ny-FR;
        int spx = nx-FR;
        addForces(dvfield, DIM, DIM, spx, spy, FORCE * DT * fx, FORCE * DT * fy, FR);
        lastx = x; lasty = y;
    } 
    glutPostRedisplay();
}

void reshape(int x, int y) {
    wWidth = x; wHeight = y;
    glViewport(0, 0, x, y);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, 1, 1, 0, 0, 1); 
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutPostRedisplay();
}

void cleanup(void) {
    cudaGLUnregisterBufferObject(vbo);
    cutilCheckMsg("cudaGLUnregisterBufferObject failed");

    unbindTexture();
    deleteTexture();

    // Free all host and device resources
    free(hvfield); free(particles); 
    hipFree(dvfield); 
    hipFree(vxfield); hipFree(vyfield);
    hipfftDestroy(planr2c);
    hipfftDestroy(planc2r);

    glBindBufferARB(GL_ARRAY_BUFFER_ARB, 0);
    glDeleteBuffersARB(1, &vbo);
    
    cutilCheckError(cutDeleteTimer(timer));  
}

int initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(wWidth, wHeight);
    glutCreateWindow("Compute Stable Fluids");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(click);
    glutMotionFunc(motion);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    glewInit();
    if (! glewIsSupported(
        "GL_ARB_vertex_buffer_object"
		)) {
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush( stderr);
        return CUTFalse;
    }
    return CUTTrue;
}


int main(int argc, char** argv) 
{
    // First initialize OpenGL context, so we can properly set the GL for CUDA.
    // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    if (CUTFalse == initGL(argc, argv)) {
        return CUTFalse;
    }

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilGLDeviceInit(argc, argv);
    else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }

	// automatied build testing harness
    if (cutCheckCmdLineFlag(argc, (const char **)argv, "qatest") ||
		cutCheckCmdLineFlag(argc, (const char **)argv, "noprompt"))
    {
        g_bQAReadback = true;
    }

    // Allocate and initialize host data
    GLint bsize;

    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutResetTimer(timer));  
    
    hvfield = (cData*)malloc(sizeof(cData) * DS);
    memset(hvfield, 0, sizeof(cData) * DS);
  
    // Allocate and initialize device data
    hipMallocPitch((void**)&dvfield, &tPitch, sizeof(cData)*DIM, DIM);
    
    hipMemcpy(dvfield, hvfield, sizeof(cData) * DS, 
               hipMemcpyHostToDevice); 
    // Temporary complex velocity field data     
    hipMalloc((void**)&vxfield, sizeof(cData) * PDS);
    hipMalloc((void**)&vyfield, sizeof(cData) * PDS);
    
    setupTexture(DIM, DIM);
    bindTexture();
    
    // Create particle array
    particles = (cData*)malloc(sizeof(cData) * DS);
    memset(particles, 0, sizeof(cData) * DS);   
    
    initParticles(particles, DIM, DIM); 

    // Create CUFFT transform plan configuration
    hipfftPlan2d(&planr2c, DIM, DIM, HIPFFT_R2C);
    hipfftPlan2d(&planc2r, DIM, DIM, HIPFFT_C2R);
#if 0    
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(wWidth, wHeight);
    glutCreateWindow("Compute Stable Fluids");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(click);
    glutMotionFunc(motion);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    glewInit();
    if (! glewIsSupported(
        "GL_ARB_vertex_buffer_object"
		)) {
        fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush( stderr);
        return CUTFalse;
    }
#endif
    glGenBuffersARB(1, &vbo);
    glBindBufferARB(GL_ARRAY_BUFFER_ARB, vbo);
    glBufferDataARB(GL_ARRAY_BUFFER_ARB, sizeof(cData) * DS, 
                    particles, GL_DYNAMIC_DRAW_ARB);

    glGetBufferParameterivARB(GL_ARRAY_BUFFER_ARB, GL_BUFFER_SIZE_ARB, &bsize); 
    if (bsize != (sizeof(cData) * DS))
        goto EXTERR;
    glBindBufferARB(GL_ARRAY_BUFFER_ARB, 0);

    cudaGLRegisterBufferObject(vbo);
    cutilCheckMsg("cudaGLRegisterBufferObject failed");

    if (g_bQAReadback)
    {
/*      g_CheckRender = new CheckBackBuffer(wWidth, wHeight, 4);
        g_CheckRender->setPixelFormat(GL_RGBA);
        g_CheckRender->setExecPath(argv[0]);
        g_CheckRender->EnableQAReadback(true);

        autoTest();
*/
    } else {
        atexit(cleanup); 
        glutMainLoop();
    }

    hipDeviceReset();
    return 0;

EXTERR:
    printf("Failed to initialize GL extensions.\n");

    hipDeviceReset();
    return 1;
}

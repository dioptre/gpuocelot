#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Determine eigenvalues for small symmetric, tridiagonal matrix */

#ifndef _BISECT_KERNEL_SMALL_H_
#define _BISECT_KERNEL_SMALL_H_

// includes, project
#include "config.h"
#include "util.h"

// additional kernel
#include "bisect_util.cu"


////////////////////////////////////////////////////////////////////////////////
//! Bisection to find eigenvalues of a real, symmetric, and tridiagonal matrix
//! @param  g_d  diagonal elements in global memory
//! @param  g_s  superdiagonal elements in global elements (stored so that the 
//!              element *(g_s - 1) can be accessed an equals 0
//! @param  n   size of matrix
//! @param  lg  lower bound of input interval (e.g. Gerschgorin interval)
//! @param  ug  upper bound of input interval (e.g. Gerschgorin interval)
//! @param  lg_eig_count  number of eigenvalues that are smaller than \a lg
//! @param  lu_eig_count  number of eigenvalues that are smaller than \a lu
//! @param  epsilon  desired accuracy of eigenvalues to compute
////////////////////////////////////////////////////////////////////////////////
__global__
void
bisectKernel( float* g_d, float* g_s, const unsigned int n,
              float* g_left, float* g_right, 
              unsigned int* g_left_count, unsigned int* g_right_count,
              const float lg, const float ug,
              const unsigned int lg_eig_count, const unsigned int ug_eig_count,
              float epsilon
             ) 
{
  // intervals (store left and right because the subdivision tree is in general 
  // not dense
  __shared__  float  s_left[MAX_THREADS_BLOCK_SMALL_MATRIX];
  __shared__  float  s_right[MAX_THREADS_BLOCK_SMALL_MATRIX];

  // number of eigenvalues that are smaller than s_left / s_right 
  // (correspondence is realized via indices)
  __shared__  unsigned int  s_left_count[MAX_THREADS_BLOCK_SMALL_MATRIX];
  __shared__  unsigned int  s_right_count[MAX_THREADS_BLOCK_SMALL_MATRIX];
  
  // helper for stream compaction
  __shared__  unsigned int  
    s_compaction_list[MAX_THREADS_BLOCK_SMALL_MATRIX + 1];

  // state variables for whole block
  // if 0 then compaction of second chunk of child intervals is not necessary
  // (because all intervals had exactly one non-dead child)
  __shared__  unsigned int compact_second_chunk;
  __shared__  unsigned int all_threads_converged;

  // number of currently active threads
  __shared__  unsigned int num_threads_active;

  // number of threads to use for stream compaction
  __shared__  unsigned int num_threads_compaction;

  // helper for exclusive scan
  unsigned int* s_compaction_list_exc = s_compaction_list + 1;


  // variables for currently processed interval
  // left and right limit of active interval
  float  left = 0.0f;
  float  right = 0.0f;
  unsigned int left_count = 0;
  unsigned int right_count = 0;
  // midpoint of active interval
  float  mid = 0.0f;
  // number of eigenvalues smaller then mid
  unsigned int mid_count = 0;
  // affected from compaction
  unsigned int  is_active_second = 0;

  s_compaction_list[threadIdx.x] = 0;
  s_left[threadIdx.x] = 0;
  s_right[threadIdx.x] = 0;
  s_left_count[threadIdx.x] = 0;
  s_right_count[threadIdx.x] = 0;

  __syncthreads();

  // set up initial configuration
  if( 0 == threadIdx.x) {
    s_left[0] = lg;
    s_right[0] = ug;
    s_left_count[0] = lg_eig_count;
    s_right_count[0] = ug_eig_count;

    compact_second_chunk = 0;
    num_threads_active = 1;

    num_threads_compaction = 1;
  }

  // for all active threads read intervals from the last level 
  // the number of (worst case) active threads per level l is 2^l
  while( true) {

    all_threads_converged = 1;
    __syncthreads();

    is_active_second = 0;
    subdivideActiveInterval( threadIdx.x, 
                             s_left, s_right, s_left_count, s_right_count,
                             num_threads_active,
                             left, right, left_count, right_count,
                             mid, all_threads_converged);

    __syncthreads();

    // check if done
    if( 1 == all_threads_converged) {
      break;
    }

    __syncthreads();

    // compute number of eigenvalues smaller than mid
    // use all threads for reading the necessary matrix data from global 
    // memory
    // use s_left and s_right as scratch space for diagonal and
    // superdiagonal of matrix
    mid_count = computeNumSmallerEigenvals( g_d, g_s, n, mid, 
                                            threadIdx.x, num_threads_active,
                                            s_left, s_right,
                                            (left == right) );

#if 0
    if(    ((mid_count < left_count) || (mid_count > right_count)) 
        && (left != right) ) {
      printf( "%f / %f / %f :: %i / %i / %i\n",
              left, mid, right, left_count, mid_count, right_count);
      cutilCondition( 0);
    }
#endif

    __syncthreads();

    // store intervals 
    // for all threads store the first child interval in a continuous chunk of
    // memory, and the second child interval -- if it exists -- in a second
    // chunk; it is likely that all threads reach convergence up to 
    // \a epsilon at the same level; furthermore, for higher level most / all
    // threads will have only one child, storing the first child compactly will
    // (first) avoid to perform a compaction step on the first chunk, (second)
    // make it for higher levels (when all threads / intervals have 
    // exactly one child)  unnecessary to perform a compaction of the second 
    // chunk
    if( threadIdx.x < num_threads_active) {

      if(left != right) {

        // store intervals
        storeNonEmptyIntervals( threadIdx.x, num_threads_active,
                                s_left, s_right, s_left_count, s_right_count,
                                left, mid, right, 
                                left_count, mid_count, right_count,
                                epsilon, compact_second_chunk,
                                s_compaction_list_exc,
                                is_active_second ); 
      }
      else {

        storeIntervalConverged( s_left, s_right, s_left_count, s_right_count,
                                left, mid, right, 
                                left_count, mid_count, right_count, 
                                s_compaction_list_exc, compact_second_chunk,
                                num_threads_active,
                                is_active_second );
      }
    }

    // necessary so that compact_second_chunk is up-to-date
    __syncthreads();

    // perform compaction of chunk where second children are stored
    // scan of (num_threads_active / 2) elements, thus at most
    // (num_threads_active / 4) threads are needed
    if( compact_second_chunk > 0) {

      createIndicesCompaction( s_compaction_list_exc, num_threads_compaction);

      compactIntervals( s_left, s_right, s_left_count, s_right_count,
                        mid, right, mid_count, right_count,
                        s_compaction_list, num_threads_active,
                        is_active_second );
    } 

    __syncthreads();

    if( 0 == threadIdx.x) {

      // update number of active threads with result of reduction
      num_threads_active += s_compaction_list[num_threads_active];

      num_threads_compaction = ceilPow2( num_threads_active);

      compact_second_chunk = 0;
    }

    __syncthreads();

  }

  __syncthreads();

  // write resulting intervals to global mem
  // for all threads write if they have been converged to an eigenvalue to 
  // a separate array
 
  // at most n valid intervals
  if( threadIdx.x < n) {

    // intervals converged so left and right limit are identical
    g_left[threadIdx.x]  = s_left[threadIdx.x]; 
    // left count is sufficient to have global order
    g_left_count[threadIdx.x]  = s_left_count[threadIdx.x]; 
  }

}

#endif // #ifndef _BISECT_KERNEL_SMALL_H_


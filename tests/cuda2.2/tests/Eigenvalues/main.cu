#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Computation of eigenvalues of symmetric, tridiagonal matrix using
 * bisection. 
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include "cutil_inline.h"
#include "config.h"
#include "structs.h"
#include "matlab.h"
#include "util.h"
#include "gerschgorin.h"

#include "bisect_small.cuh"
#include "bisect_large.cuh"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize the input data to the algorithm
//! @param input  handles to the input data
//! @param exec_path  path where executable is run (argv[0])
//! @param mat_size  size of the matrix
//! @param user_defined  1 if the matrix size has been requested by the user, 
//!                      0 if the default size
////////////////////////////////////////////////////////////////////////////////
void
initInputData( InputData& input, char* exec_path, 
               const unsigned int mat_size, const unsigned int user_defined) 
{   
    // allocate memory
    input.a = (float*) malloc( sizeof(float) * mat_size);
    input.b = (float*) malloc( sizeof(float) * mat_size);

    if( 1 == user_defined) {
    
        // initialize diagonal and superdiagonal entries with random values
        srand( 278217421);
        // srand( clock());
        for( unsigned int i = 0; i < mat_size; ++i) {
            input.a[i] = (float) (2.0 * (((double)rand() 
                                / (double) RAND_MAX) - 0.5));
            input.b[i] = (float) (2.0 * (((double)rand() 
                                / (double) RAND_MAX) - 0.5));
        } 
    
        // the first element of s is used as padding on the device (thus the 
        // whole vector is copied to the device but the kernels are launched
        // with (s+1) as start address
        input.b[0] = 0.0f;
    }
    else {

        // read default matrix
        unsigned int input_data_size = mat_size;
        char* diag_path = cutFindFilePath( "diagonal.dat", exec_path);
        cutilCondition( 0 != diag_path);
        cutilCheckError( cutReadFilef( diag_path, &(input.a), &input_data_size));

        char* sdiag_path = cutFindFilePath( "superdiagonal.dat", exec_path);
        cutilCondition( 0 != sdiag_path);
        cutilCheckError( cutReadFilef( sdiag_path, &(input.b), &input_data_size,
                                   1));

        cutFree( diag_path);
        cutFree( sdiag_path);
    }

    // allocate device memory for input
    cutilSafeCall( hipMalloc( (void**) &(input.g_a)    , sizeof(float) * mat_size));
    cutilSafeCall( hipMalloc( (void**) &(input.g_b_raw), sizeof(float) * mat_size));

    // copy data to device
    cutilSafeCall( hipMemcpy( input.g_a    , input.a, sizeof(float) * mat_size, hipMemcpyHostToDevice ));
    cutilSafeCall( hipMemcpy( input.g_b_raw, input.b, sizeof(float) * mat_size, hipMemcpyHostToDevice ));

    input.g_b = input.g_b_raw + 1;
}

////////////////////////////////////////////////////////////////////////////////
//! Clean up input data, in particular allocated memory
//! @param input  handles to the input data
////////////////////////////////////////////////////////////////////////////////
void
cleanupInputData( InputData& input) {

    freePtr( input.a);
    freePtr( input.b);

    cutilSafeCall( hipFree( input.g_a));
    input.g_a = NULL;
    cutilSafeCall( hipFree( input.g_b_raw));
    input.g_b_raw = NULL;
    input.g_b = NULL;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a specific matrix size has to be used
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param matrix_size  size of matrix, updated if specific size specified on 
//!                     command line
////////////////////////////////////////////////////////////////////////////////
void 
getMatrixSize( int argc, char** argv,
               unsigned int& mat_size, unsigned int& user_defined) 
{
    int temp = -1;
    cutGetCmdLineArgumenti( argc, (const char**) argv, "matrix-size", &temp);
    if( temp > 0) {
    
        mat_size = (unsigned int) temp;
        // data type short is used in the kernel
        cutilCondition( mat_size < (1 << 16));

        user_defined = 1;
    }

    printf( "Matrix size: %i x %i\n", mat_size, mat_size);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a specific precision of the eigenvalue has to be obtained
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param iters_timing  numbers of iterations for timing, updated if a 
//!                      specific number is specified on the command line
////////////////////////////////////////////////////////////////////////////////
void
getPrecision( int argc, char** argv, float& precision) {

    float temp = -1.0f;
    cutGetCmdLineArgumentf( argc, (const char**) argv, "precision", &temp);
    if( temp > 0.0f) {
        precision = temp;
    }

    printf( "Precision: %f\n", precision);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a particular number of iterations for timings has to be used
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param  iters_timing  number of timing iterations, updated if user 
//!                       specific value
////////////////////////////////////////////////////////////////////////////////
void
getItersTiming( int argc, char** argv, unsigned int& iters_timing) {

    int temp = -1;
    cutGetCmdLineArgumenti( argc, (const char**) argv, "iters-timing", &temp);
    if( temp > 0) {
        iters_timing = temp;
    }

    printf( "Iterations to be timed: %i\n", iters_timing);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a particular filename has to be used for the file where the result
//! is stored
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param  filename  filename of result file, updated if user specified
//!                   filename
////////////////////////////////////////////////////////////////////////////////
void
getResultFilename( int argc, char** argv, char*& filename) {

    char* temp = NULL;
    cutGetCmdLineArgumentstr( argc, (const char**) argv, "filename-result",
                              &temp);
    if( NULL != temp) {

        filename = (char*) malloc( sizeof(char) * strlen( temp));
        strcpy( filename, temp);

        cutFree( temp);
    }

    printf( "Result filename: '%s'\n", filename);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) {

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    unsigned int timer = 0;
    unsigned int timer_total = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutCreateTimer( &timer_total));

    // default
    unsigned int mat_size = 2048;
    // flag if the matrix size is due to explicit user request
    unsigned int user_defined = 0;
    // desired precision of eigenvalues
    float  precision = 0.00001f;
#if 1
    unsigned int iters_timing = 1;
#else
    unsigned int iters_timing = 1;
#endif
    char*  result_file = "eigenvalues.dat";
    
    // check if there is a command line request for the matrix size
    getMatrixSize( argc, argv, mat_size, user_defined);

    // check if user requested specific precision
    getPrecision( argc, argv, precision);

    // check if user requested specific number of iterations for timing
    getItersTiming( argc, argv, iters_timing);

    // file name for result file
    getResultFilename( argc, argv, result_file);

    // set up input 
    InputData input;
    initInputData( input, argv[0], mat_size, user_defined);

    // compute Gerschgorin interval
    float lg = FLT_MAX;
    float ug = -FLT_MAX;
    computeGerschgorin( input.a, input.b+1, mat_size, lg, ug);
    printf( "Gerschgorin interval: %f / %f\n", lg, ug);

    // two kernels, for small matrices a lot of overhead can be avoided
    if( mat_size <= MAX_SMALL_MATRIX) {

        // initialize memory for result
        ResultDataSmall result;
        initResultSmallMatrix( result, mat_size);

        // run the kernel
        computeEigenvaluesSmallMatrix( input, result, mat_size, lg, ug, 
                                       precision, iters_timing);

        // get the result from the device and do some sanity checks,
        // save the result
        processResultSmallMatrix( input, result, mat_size, result_file);

        // clean up
        cleanupResultSmallMatrix( result);
    }
    else {

        // initialize memory for result
        ResultDataLarge  result;
        initResultDataLargeMatrix( result, mat_size);

        // run the kernel
        computeEigenvaluesLargeMatrix( input, result, mat_size, 
                                       precision, lg, ug, 
                                       iters_timing );

        // get the result from the device and do some sanity checks
        // save the result if user specified matrix size
        processResultDataLargeMatrix( input, result, mat_size, result_file,
                                      user_defined, argv[0]);

        // cleanup
        cleanupResultDataLargeMatrix(result);
    }

    cleanupInputData( input);

    hipDeviceReset();
}


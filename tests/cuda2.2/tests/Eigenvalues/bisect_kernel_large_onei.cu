#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Determine eigenvalues for large matrices for intervals that contained after
 * the first step one eigenvalue
 */

#ifndef _BISECT_KERNEL_LARGE_ONEI_H_
#define _BISECT_KERNEL_LARGE_ONEI_H_

// includes, project
#include "config.h"
#include "util.h"

// additional kernel
#include "bisect_util.cu"

////////////////////////////////////////////////////////////////////////////////
//! Determine eigenvalues for large matrices for intervals that after
//! the first step contained one eigenvalue
//! @param  g_d  diagonal elements of symmetric, tridiagonal matrix
//! @param  g_s  superdiagonal elements of symmetric, tridiagonal matrix
//! @param  n    matrix size
//! @param  num_intervals  total number of intervals containing one eigenvalue 
//!                         after the first step
//! @param g_left  left interval limits
//! @param g_right  right interval limits
//! @param g_pos  index of interval / number of intervals that are smaller than
//!               right interval limit
//! @param  precision  desired precision of eigenvalues
////////////////////////////////////////////////////////////////////////////////
__global__
void
bisectKernelLarge_OneIntervals( float* g_d, float* g_s, const unsigned int n,
                                unsigned int num_intervals,
                                float* g_left, float* g_right,
                                unsigned int* g_pos,
                                float  precision )
{
  const unsigned int gtid = (blockDim.x * blockIdx.x) + threadIdx.x;

  __shared__  float  s_left_scratch[MAX_THREADS_BLOCK];
  __shared__  float  s_right_scratch[MAX_THREADS_BLOCK];

  // active interval of thread
  // left and right limit of current interval
  float left, right;
  // number of threads smaller than the right limit (also corresponds to the
  // global index of the eigenvalues contained in the active interval)
  unsigned int right_count;
  // flag if current thread converged
  unsigned int converged = 0;
  // midpoint when current interval is subdivided
  float mid = 0.0f;
  // number of eigenvalues less than mid
  unsigned int mid_count = 0;

  // read data from global memory
  if( gtid < num_intervals) {
    left = g_left[gtid];
    right = g_right[gtid];
    right_count = g_pos[gtid];
  }


  // flag to determine if all threads converged to eigenvalue
  __shared__  unsigned int  converged_all_threads;

  // initialized shared flag
  if( 0 == threadIdx.x) {
    converged_all_threads = 0;
  }
  __syncthreads();

  // process until all threads converged to an eigenvalue
  // while( 0 == converged_all_threads) {
  while( true) {

    converged_all_threads = 1;

    // update midpoint for all active threads
    if((gtid < num_intervals) && ( 0 == converged)) {

      mid = computeMidpoint( left, right);
    }

    // find number of eigenvalues that are smaller than midpoint
    mid_count = computeNumSmallerEigenvalsLarge( g_d, g_s, n,
                                                 mid, gtid, num_intervals,
                                                 s_left_scratch,
                                                 s_right_scratch,
                                                 converged );

     __syncthreads();

     // for all active threads
     if( (gtid < num_intervals) && ( 0 == converged)) {

       // udpate intervals -- always one child interval survives
       if( right_count == mid_count) {     
         right = mid;      
       }
       else {
         left = mid;
       }

       // check for convergence
       float t0 = right - left;
       float t1 = max( abs(right), abs(left)) * precision;
       if( t0 < min( precision, t1)) {

         float lambda = computeMidpoint( left, right);
         left = lambda;
         right = lambda;

         converged = 1;
       }
       else {
         converged_all_threads = 0;
       }
     }

     __syncthreads();

     if( 1 == converged_all_threads) {
       break;
     }

     __syncthreads();
  }

  // write data back to global memory
  __syncthreads();
  if( gtid < num_intervals) {
    // intervals converged so left and right interval limit are both identical
    // and identical to the eigenvalue
    g_left[gtid] = left;
  }

}

#endif // #ifndef _BISECT_KERNEL_LARGE_ONEI_H_

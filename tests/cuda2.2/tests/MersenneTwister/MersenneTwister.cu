#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This sample implements Mersenne Twister random number generator 
 * and Cartesian Box-Muller transformation on the GPU.
 * See supplied whitepaper for more explanations.
 */



#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>
#include "MersenneTwister.h"



///////////////////////////////////////////////////////////////////////////////
// Common host and device function 
///////////////////////////////////////////////////////////////////////////////
//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}



///////////////////////////////////////////////////////////////////////////////
// Reference MT front-end and Box-Muller transform
///////////////////////////////////////////////////////////////////////////////
extern "C" void initMTRef(const char *fname);
extern "C" void RandomRef(float *h_Random, int NPerRng, unsigned int seed);
extern "C" void BoxMullerRef(float *h_Random, int NPerRng);



///////////////////////////////////////////////////////////////////////////////
// Fast GPU random number generator and Box-Muller transform
///////////////////////////////////////////////////////////////////////////////
#include "MersenneTwister_kernel.cuh"



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////
const int    PATH_N = 24000000;
const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

const unsigned int SEED = 777;



#define DO_BOXMULLER
///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    float
        *d_Rand;

    float
        *h_RandCPU,
        *h_RandGPU;

    double
        rCPU, rGPU, delta, sum_delta, max_delta, sum_ref, L1norm, gpuTime;

    int i, j;
    unsigned int hTimer;


    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data for %i samples...\n", PATH_N);
        h_RandCPU  = (float *)malloc(RAND_N * sizeof(float));
        h_RandGPU  = (float *)malloc(RAND_N * sizeof(float));
        cutilSafeCall( hipMalloc((void **)&d_Rand, RAND_N * sizeof(float)) );

    printf("Loading CPU and GPU twisters configurations...\n");
        const char *raw_path = cutFindFilePath("MersenneTwister.raw", argv[0]);
        const char *dat_path = cutFindFilePath("MersenneTwister.dat", argv[0]);
        initMTRef(raw_path);
        loadMTGPU(dat_path);
        seedMTGPU(SEED);

    printf("Generating random numbers on GPU...\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        RandomGPU<<<32, 128>>>(d_Rand, N_PER_RNG);
        cutilCheckMsg("RandomGPU() execution failed\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer);
    printf("Generated samples : %i \n", RAND_N);
    printf("RandomGPU() time  : %f \n", gpuTime);
    printf("Samples per second: %E \n", RAND_N / (gpuTime * 0.001));


#ifdef DO_BOXMULLER
    printf("Applying Box-Muller transformation on GPU...\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        BoxMullerGPU<<<32, 128>>>(d_Rand, N_PER_RNG);
        cutilCheckMsg("BoxMullerGPU() execution failed\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer);
    printf("Transformed samples : %i \n", RAND_N);
    printf("BoxMullerGPU() time : %f \n", gpuTime);
    printf("Samples per second  : %E \n", RAND_N / (gpuTime * 0.001));
#endif

    printf("Reading back the results...\n");
        cutilSafeCall( hipMemcpy(h_RandGPU, d_Rand, RAND_N * sizeof(float), hipMemcpyDeviceToHost) );


    printf("Checking GPU results...\n");
        printf("...generating random numbers on CPU using reference generator\n");
        RandomRef(h_RandCPU, N_PER_RNG, SEED);

        #ifdef DO_BOXMULLER
            printf("...applying Box-Muller transformation on CPU\n");
            BoxMullerRef(h_RandCPU, N_PER_RNG);
        #endif

        printf("...comparing the results\n");
        max_delta = 0;
        sum_delta = 0;
        sum_ref   = 0;
        for(i = 0; i < MT_RNG_COUNT; i++)
            for(j = 0; j < N_PER_RNG; j++){
                rCPU = h_RandCPU[i * N_PER_RNG + j];
                rGPU = h_RandGPU[i + j * MT_RNG_COUNT];
                delta = fabs(rCPU - rGPU);
                sum_delta += delta;
                sum_ref   += fabs(rCPU);
                if(delta >= max_delta) max_delta = delta;
            }
    L1norm = (float)(sum_delta / sum_ref);
    printf("Max absolute error: %E\n", max_delta);
    printf("L1 norm: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");

    printf("Shutting down...\n");
        cutilSafeCall( hipFree(d_Rand) );
        free(h_RandGPU);
        free(h_RandCPU);

    cutilCheckError( cutDeleteTimer( hTimer) );

    hipDeviceReset();

    cutilExit(argc, argv);
}

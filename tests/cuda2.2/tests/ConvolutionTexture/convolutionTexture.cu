#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  Users and possessors of this source code 
 * are hereby granted a nonexclusive, royalty-free license to use this code 
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein. 
 *
 * Any use of this source code in individual and commercial software must 
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* 
 * This sample implements the same algorithm as the convolutionSeparable
 * CUDA SDK sample, but without using the shared memory at all.
 * Instead, it uses textures in exactly the same way an OpenGL-based
 * implementation would do. 
 * Refer to the "Performance" section of convolutionSeparable whitepaper.
 */



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil_inline.h>



////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}



////////////////////////////////////////////////////////////////////////////////
// Reference CPU convolution
////////////////////////////////////////////////////////////////////////////////
extern "C" void convolutionRowCPU(
    float *h_Result,
    float *h_DataA,
    float *h_Kernel,
    int dataW,
    int dataH,
    int kernelR
);

extern "C" void convolutionColumnCPU(
    float *h_Result,
    float *h_DataA,
    float *h_Kernel,
    int dataW,
    int dataH,
    int kernelR
);



////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
#define UNROLL_INNER
#include "convolutionTexture_kernel.cu"



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
#if 1
//Reduce problem size to have reasonable emulation time
const int     DATA_W = iAlignUp(256, 16);
const int     DATA_H = 256;
#else
const int     DATA_W = iAlignUp(3072, 16);
const int     DATA_H = 3072;
#endif

const int KERNEL_SIZE = KERNEL_W * sizeof(float);
const int   DATA_SIZE = DATA_W   * DATA_H   * sizeof(float);



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    float
        *h_Kernel,
        *h_DataA,
        *h_DataB,
        *h_ResultGPU;

    hipArray
        *a_Data;

    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();

    float
        *d_Result;

    double
        gpuTime, L1norm, rCPU, rGPU, sum_delta, sum_ref;

    int i, x, y;

    unsigned int hTimer;


    // use command-line specified CUDA device, otherwise use device with highest Gflops/s

    if ( cutCheckCmdLineFlag(argc, (const char **)argv, "device"))
	    cutilDeviceInit(argc, argv);
    else
    	hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data...\n");
        h_Kernel    = (float *)malloc(KERNEL_SIZE);
        h_DataA     = (float *)malloc(DATA_SIZE);
        h_DataB     = (float *)malloc(DATA_SIZE);
        h_ResultGPU = (float *)malloc(DATA_SIZE);
        cutilSafeCall( hipMallocArray(&a_Data, &floatTex, DATA_W, DATA_H) );
        cutilSafeCall( hipMalloc((void **)&d_Result, DATA_SIZE) );

        float kernelSum = 0;
        for(i = 0; i < KERNEL_W; i++){
            float dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
            h_Kernel[i] = expf(- dist * dist / 2);
            kernelSum += h_Kernel[i];
        }
        for(i = 0; i < KERNEL_W; i++)
            h_Kernel[i] /= kernelSum;

        srand(2007);
        for(i = 0; i < DATA_W * DATA_H; i++)
            h_DataA[i] = (float)rand() / (float)RAND_MAX;

        cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, KERNEL_SIZE) );
        cutilSafeCall( hipMemcpyToArray(a_Data, 0, 0, h_DataA, DATA_SIZE, hipMemcpyHostToDevice) );
        cutilSafeCall( hipBindTextureToArray(texData, a_Data) );

        //Block width should be a multiple of maximum coalesced write size
        //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
        dim3 threadBlock(16, 12);
        dim3 blockGrid(iDivUp(DATA_W, threadBlock.x), iDivUp(DATA_H, threadBlock.y));


    printf("convolutionRowGPU()\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        convolutionRowGPU<<<blockGrid, threadBlock>>>(
            d_Result,
            DATA_W,
            DATA_H
        );
        cutilCheckMsg("convolutionRowGPU() execution failed\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer);
    printf("...convolutionRowGPU() time: %f msecs; //%f Mpix/s\n", gpuTime, DATA_W * DATA_H * 1e-6 / (0.001 * gpuTime));


    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    printf("Copying convolutionRowGPU() output back to a_Data...\n");
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        cutilSafeCall( hipMemcpyToArray(a_Data, 0, 0, d_Result, DATA_SIZE, hipMemcpyDeviceToDevice) );
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer);
    printf("...hipMemcpyToArray() time: %f msecs; //%f Mpix/s\n", gpuTime, DATA_W * DATA_H * 1e-6 / (0.001 * gpuTime));


    printf("convolutionColumnGPU()...\n");
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        convolutionColumnGPU<<<blockGrid, threadBlock>>>(
            d_Result,
            DATA_W,
            DATA_H
        );
        cutilCheckMsg("convolutionColumnGPU() execution failed\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer);
    printf("...convolutionColumnGPU() time: %f msecs; //%f Mpix/s\n", gpuTime, DATA_W * DATA_H * 1e-6 / (0.001 * gpuTime));


    printf("Reading back GPU results...\n");
        cutilSafeCall( hipMemcpy(h_ResultGPU, d_Result, DATA_SIZE, hipMemcpyDeviceToHost) );


    printf("Checking GPU results...\n");
        printf("...convolutionRowCPU()\n");
        convolutionRowCPU(
            h_DataB,
            h_DataA,
            h_Kernel,
            DATA_W,
            DATA_H,
            KERNEL_RADIUS
        );

        printf("...convolutionColumnCPU()\n");
        convolutionColumnCPU(
            h_DataA,
            h_DataB,
            h_Kernel,
            DATA_W,
            DATA_H,
            KERNEL_RADIUS
        );

        printf("...comparing the results\n");
        sum_delta = 0;
        sum_ref   = 0;
        for(y = 0; y < DATA_H; y++)
            for(x = 0; x < DATA_W; x++){
                rCPU = h_DataA[y * DATA_W + x];
                rGPU = h_ResultGPU[y * DATA_W  + x];
                sum_delta += fabs(rCPU - rGPU);
                sum_ref   += fabs(rCPU);
            }
        L1norm = sum_delta / sum_ref;
        printf("L1 norm: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");


    printf("Shutting down...\n");
        cutilSafeCall( hipUnbindTexture(texData) );
        cutilSafeCall( hipFree(d_Result)   );
        cutilSafeCall( hipFreeArray(a_Data)   );
        free(h_ResultGPU);
        free(h_DataB);
        free(h_DataA);
        free(h_Kernel);

    cutilExit(argc, argv);

    hipDeviceReset();
}

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  Users and possessors of this source code 
 * are hereby granted a nonexclusive, royalty-free license to use this code 
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein. 
 *
 * Any use of this source code in individual and commercial software must 
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */



//Fast integer multiplication macro
#define IMUL(a, b) __mul24(a, b)



//Input data texture reference
texture<float, 2, hipReadModeElementType> texData;



////////////////////////////////////////////////////////////////////////////////
// Kernel configuration
////////////////////////////////////////////////////////////////////////////////
#define KERNEL_RADIUS 8
#define KERNEL_W      (2 * KERNEL_RADIUS + 1)
__device__ __constant__ float d_Kernel[KERNEL_W];



////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
template<int i> __device__ float convolutionRow(float x, float y){
    return 
        tex2D(texData, x + KERNEL_RADIUS - i, y) * d_Kernel[i]
        + convolutionRow<i - 1>(x, y);
}

template<> __device__ float convolutionRow<-1>(float x, float y){
    return 0;
}

template<int i> __device__ float convolutionColumn(float x, float y){
    return 
        tex2D(texData, x, y + KERNEL_RADIUS - i) * d_Kernel[i]
        + convolutionColumn<i - 1>(x, y);
}

template<> __device__ float convolutionColumn<-1>(float x, float y){
    return 0;
}



////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(
    float *d_Result,
    int dataW,
    int dataH
){
    const   int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    const   int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if(ix < dataW && iy < dataH){
        float sum = 0;

#ifdef UNROLL_INNER
        sum = convolutionRow<2 * KERNEL_RADIUS>(x, y);
#else
        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += tex2D(texData, x + k, y) * d_Kernel[KERNEL_RADIUS - k];
#endif

        d_Result[IMUL(iy, dataW) + ix] = sum;
    }
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnGPU(
    float *d_Result,
    int dataW,
    int dataH
){
    const   int ix = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    const   int iy = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if(ix < dataW && iy < dataH){
        float sum = 0;

#ifdef UNROLL_INNER
        sum =  convolutionColumn<2 * KERNEL_RADIUS>(x, y);
#else
        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += tex2D(texData, x, y + k) * d_Kernel[KERNEL_RADIUS - k];
#endif

        d_Result[IMUL(iy, dataW) + ix] = sum;
    }
}

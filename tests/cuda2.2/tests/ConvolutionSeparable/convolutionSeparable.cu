#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  Users and possessors of this source code 
 * are hereby granted a nonexclusive, royalty-free license to use this code 
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein. 
 *
 * Any use of this source code in individual and commercial software must 
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This sample implements a separable convolution filter 
 * of a 2D signal with a gaussian kernel.
 */



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_inline.h>



////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}



////////////////////////////////////////////////////////////////////////////////
// Reference CPU convolution
////////////////////////////////////////////////////////////////////////////////
extern "C" void convolutionRowCPU(
    float *h_Result,
    float *h_Data,
    float *h_Kernel,
    int dataW,
    int dataH,
    int kernelR
);

extern "C" void convolutionColumnCPU(
    float *h_Result,
    float *h_Data,
    float *h_Kernel,
    int dataW,
    int dataH,
    int kernelR
);



////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
//Global macro, controlling innermost convolution loop unrolling
#define UNROLL_INNER
#include <tests/ConvolutionSeparable/convolutionSeparable_kernel.cu>



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//Image width should be aligned to maximum coalesced read/write size
//for best global memory performance in both row and column filter.
#if 1
//Reduce problem size to have reasonable emulation time
const int      DATA_W = iAlignUp(256, 16);
const int      DATA_H = 256;
#else
const int      DATA_W = iAlignUp(3072, 16);
const int      DATA_H = 3072;
#endif
const int   DATA_SIZE = DATA_W * DATA_H * sizeof(float);
const int KERNEL_SIZE = KERNEL_W * sizeof(float);



//Carry out dummy calculations before main computation loop
//in order to "warm up" the hardware/driver
#define WARMUP
////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    float
        *h_Kernel,
        *h_DataA,
        *h_DataB,
        *h_ResultGPU;

    float
        *d_DataA,
        *d_DataB,
        *d_Temp;

    double
        sum_delta, sum_ref, L1norm, gpuTime;

    int i;

    unsigned int hTimer;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s

    if ( cutCheckCmdLineFlag(argc, (const char **)argv, "device"))
	    cutilDeviceInit(argc, argv);
    else
	    hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError(cutCreateTimer(&hTimer));

    printf("%i x %i\n", DATA_W, DATA_H);
    printf("Initializing data...\n");
        h_Kernel    = (float *)malloc(KERNEL_SIZE);
        h_DataA     = (float *)malloc(DATA_SIZE);
        h_DataB     = (float *)malloc(DATA_SIZE);
        h_ResultGPU = (float *)malloc(DATA_SIZE);
        cutilSafeCall( hipMalloc( (void **)&d_DataA, DATA_SIZE) );
        cutilSafeCall( hipMalloc( (void **)&d_DataB, DATA_SIZE) );
        cutilSafeCall( hipMalloc( (void **)&d_Temp , DATA_SIZE) );

        float kernelSum = 0;
        for(i = 0; i < KERNEL_W; i++){
            float dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
            h_Kernel[i] = expf(- dist * dist / 2);
            kernelSum += h_Kernel[i];
        }
        for(i = 0; i < KERNEL_W; i++)
            h_Kernel[i] /= kernelSum;

        srand(2007);
        for(i = 0; i < DATA_W * DATA_H; i++)
            h_DataA[i] = (float)rand() / (float)RAND_MAX;

        cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, KERNEL_SIZE) );
        cutilSafeCall( hipMemcpy(d_DataA, h_DataA, DATA_SIZE, hipMemcpyHostToDevice) );


    dim3 blockGridRows(iDivUp(DATA_W, ROW_TILE_W), DATA_H);
    dim3 blockGridColumns(iDivUp(DATA_W, COLUMN_TILE_W), iDivUp(DATA_H, COLUMN_TILE_H));
    dim3 threadBlockRows(KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS);
    dim3 threadBlockColumns(COLUMN_TILE_W, 8);


#ifdef WARMUP
    printf("Warm up...\n");
    convolutionRowGPU<<<blockGridRows, threadBlockRows>>>(
        d_Temp,
        d_DataA,
        DATA_W,
        DATA_H
    );
    cutilCheckMsg("convolutionRowGPU() execution failed\n");

    convolutionColumnGPU<<<blockGridColumns, threadBlockColumns>>>(
        d_Temp,
        d_DataA,
        DATA_W,
        DATA_H,
        COLUMN_TILE_W * threadBlockColumns.y,
        DATA_W * threadBlockColumns.y
    );
    cutilCheckMsg("convolutionColumnGPU() execution failed\n");
    cutilSafeCall( hipDeviceSynchronize() );
#endif


    printf("GPU convolution...\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        convolutionRowGPU<<<blockGridRows, threadBlockRows>>>(
            d_DataB,
            d_DataA,
            DATA_W,
            DATA_H
        );
        cutilCheckMsg("convolutionRowGPU() execution failed\n");

        convolutionColumnGPU<<<blockGridColumns, threadBlockColumns>>>(
            d_DataA,
            d_DataB,
            DATA_W,
            DATA_H,
            COLUMN_TILE_W * threadBlockColumns.y,
            DATA_W * threadBlockColumns.y
        );
        cutilCheckMsg("convolutionColumnGPU() execution failed\n");
    cutilSafeCall( hipDeviceSynchronize() );
    cutilCheckError(cutStopTimer(hTimer));
    gpuTime = cutGetTimerValue(hTimer);
    printf("GPU convolution time : %f msec //%f Mpixels/sec\n", gpuTime, 1e-6 * DATA_W * DATA_H / (gpuTime * 0.001));

    printf("Reading back GPU results...\n");
        cutilSafeCall( hipMemcpy(h_ResultGPU, d_DataA, DATA_SIZE, hipMemcpyDeviceToHost) );


    printf("Checking the results...\n");
        printf("...running convolutionRowCPU()\n");
        convolutionRowCPU(
            h_DataB,
            h_DataA,
            h_Kernel,
            DATA_W,
            DATA_H,
            KERNEL_RADIUS
        );

        printf("...running convolutionColumnCPU()\n");
        convolutionColumnCPU(
            h_DataA,
            h_DataB,
            h_Kernel,
            DATA_W,
            DATA_H,
            KERNEL_RADIUS
        );

        printf("...comparing the results\n");
        sum_delta = 0;
        sum_ref   = 0;
        for(i = 0; i < DATA_W * DATA_H; i++){
            sum_delta += fabs(h_DataA[i] - h_ResultGPU[i]);
            sum_ref   += fabs(h_DataA[i]);
        }
        L1norm = sum_delta / sum_ref;
        printf("L1 norm: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");

    printf("Shutting down...\n");
        cutilSafeCall( hipFree(d_Temp ) );
        cutilSafeCall( hipFree(d_DataB) );
        cutilSafeCall( hipFree(d_DataA) );
        free(h_ResultGPU);
        free(h_DataB);
        free(h_DataA);
        free(h_Kernel);

    cutilCheckError(cutDeleteTimer(hTimer));

    hipDeviceReset();

    cutilExit(argc, argv);
}

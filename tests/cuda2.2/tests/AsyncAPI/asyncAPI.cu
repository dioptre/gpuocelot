#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 *
 *
 * This sample illustrates the usage of CUDA events for both GPU timing and
 * overlapping CPU and GPU execution.  Events are insterted into a stream
 * of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
 * perform computations while GPU is executing (including DMA memcopies
 * between the host and device).  CPU can query CUDA events to determine
 * whether GPU has completed tasks.
 *
*/

#include <stdio.h>
#include <cutil_inline.h>

__global__ void increment_kernel(int *g_data, int inc_value)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   g_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++)
        if(data[i] != x)
            return 0;
    return 1;
}

int main(int argc, char *argv[])
{
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
	    cutilDeviceInit(argc, argv);
	else
	    hipSetDevice( cutGetMaxGflopsDeviceId());

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

#if 1
    n = 1024;   // reduce workload for emulation
#endif
    
    // allocate host memory
    int *a = 0;
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) );
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );
    cutilSafeCall( hipMemset(d_a, 255, nbytes) );

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    cutilSafeCall( hipEventCreate(&start) );
    cutilSafeCall( hipEventCreate(&stop)  );
    
    unsigned int timer;
    cutilCheckError(  cutCreateTimer(&timer)  );
    cutilCheckError(  cutResetTimer(timer)    );
    cutilSafeCall( hipDeviceSynchronize() );
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    cutilCheckError( cutStartTimer(timer) );
        hipEventRecord(start, 0);
        hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
        increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
        hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
        hipEventRecord(stop, 0);
    cutilCheckError( cutStopTimer(timer) );

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;
    while( hipEventQuery(stop) == hipErrorNotReady )
    {
        counter++;
    }
    cutilSafeCall( hipEventElapsedTime(&gpu_time, start, stop) );

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", cutGetTimerValue(timer) );
    printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    printf("--------------------------------------------------------------\n");
    if( correct_output(a, n, value) )
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // release resources
    cutilSafeCall( hipEventDestroy(start) );
    cutilSafeCall( hipEventDestroy(stop) );
    cutilSafeCall( hipHostFree(a) );
    cutilSafeCall( hipFree(d_a) );

    hipDeviceReset();

    cutilExit(argc, argv);

    return 0;
}

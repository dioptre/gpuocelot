#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdio.h>
#include <stdlib.h>

#include <cutil_inline.h>

#include "bitonic_kernel.cu"

//
// A sorting network is a sorting algorith, where the sequence of comparisons
// is not data-dependent. That makes them suitable for parallel implementations.
//
// Bitonic sort is one of the fastest sorting networks, consisting of o(n log^2 n)
// comparators. It has a simple implemention and it's very efficient when sorting 
// a small number of elements:
//
// http://citeseer.ist.psu.edu/blelloch98experimental.html
//
// This implementation is based on:
//
// http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
//

int main(int argc, char** argv)
{
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

    int values[NUM];

    for(int i = 0; i < NUM; i++)
    {
        values[i] = rand();
    }

    int * dvalues;
    cutilSafeCall(hipMalloc((void**)&dvalues, sizeof(int) * NUM));
    cutilSafeCall(hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice));

    bitonicSort<<<1, NUM, sizeof(int) * NUM>>>(dvalues);

    // check for any errors
    cutilCheckMsg("Kernel execution failed");

    cutilSafeCall(hipMemcpy(values, dvalues, sizeof(int) * NUM, hipMemcpyDeviceToHost));

    cutilSafeCall(hipFree(dvalues));

    bool passed = true;
    for(int i = 1; i < NUM; i++)
    {
        if (values[i-1] > values[i])
        {
            passed = false;
        }
    }

    printf( "Test %s\n", passed ? "PASSED" : "FAILED");

    hipDeviceReset();

    cutilExit(argc, argv);
}

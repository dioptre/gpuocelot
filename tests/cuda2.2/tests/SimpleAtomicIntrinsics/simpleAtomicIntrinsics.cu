/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* A simple program demonstrating trivial use of global memory atomic 
   device functions (atomic*() functions).
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "cutil_inline.h"

// includes, kernels
#include "simpleAtomicIntrinsics_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
int computeGold( int* gpuData, const int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char **argv)
{
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    int dev;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( dev = cutGetMaxGflopsDeviceId() );

    cutilSafeCall( hipChooseDevice(&dev, &deviceProp) );
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, dev) );

    if(deviceProp.major > 1 || deviceProp.minor > 0)
    {
        printf("Using Device %d: \"%s\"\n", dev, deviceProp.name);
    }
    else
    {
        printf("There is no device supporting CUDA compute capability 1.1.\n");
        printf("TEST PASSED");
        hipDeviceReset();
        cutilExit(argc, argv);
    }

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    unsigned int numThreads = 256;
    unsigned int numBlocks = 64;
    unsigned int numData = 11;
    unsigned int memSize = sizeof(int) * numData;

    //allocate mem for the result on host side
    int *h_odata = (int *)malloc(memSize);

    //initalize the memory
    for(unsigned int i = 0; i < numData; i++)
        h_odata[i] = 0;

    //To make the AND and XOR tests generate something other than 0...
    h_odata[8] = h_odata[10] = 0xff; 

    // allocate device memory for result
    int *d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, memSize));
    // copy host memory to device to initialize to zers
    cutilSafeCall( hipMemcpy( d_odata, h_odata, memSize, hipMemcpyHostToDevice) );


    // execute the kernel
    testKernel<<<numBlocks, numThreads>>>(d_odata);
    cutilCheckMsg("Kernel execution failed");
    //Copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_odata, memSize, hipMemcpyDeviceToHost) );

    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));

    //compute reference solution
    if(computeGold(h_odata, numThreads * numBlocks))
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // cleanup memory
    free(h_odata);
    cutilSafeCall(hipFree(d_odata));

    hipDeviceReset();
}

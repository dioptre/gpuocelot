#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <tests/Scan/scan_naive_kernel.cu>
#include <tests/Scan/scan_workefficient_kernel.cu>
#include <tests/Scan/scan_best_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                      const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    unsigned int num_elements = 512;
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);

    unsigned int timer;
    cutilCheckError( cutCreateTimer(&timer));
    
    const unsigned int num_threads = num_elements / 2;
    const unsigned int mem_size = sizeof( float) * num_elements;

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extra_space = num_elements / NUM_BANKS;

#ifdef ZERO_BANK_CONFLICTS
    extra_space += extra_space / NUM_BANKS;
#endif

    const unsigned int shared_mem_size = sizeof(float) * 
        (num_elements + extra_space);

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( mem_size);
      
    // initialize the input data on the host to be integer values
    // between 0 and 1000
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
    }

    // compute reference solution
    float* reference = (float*) malloc( mem_size);  
    computeGold( reference, h_data, num_elements);

    // allocate device memory input and output arrays
    float* d_idata;
    float* d_odata[3];
    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &(d_odata[0]), mem_size));
    cutilSafeCall( hipMalloc( (void**) &(d_odata[1]), mem_size));
    cutilSafeCall( hipMalloc( (void**) &(d_odata[2]), mem_size));

    // copy host memory to device input array
    cutilSafeCall( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );

    // setup execution parameters
    // Note that these scans only support a single thread-block worth of data,
    // but we invoke them here on many blocks so that we can accurately compare
    // performance
#if 0
    dim3  grid(256, 1, 1);  
#else
    dim3  grid(1, 1, 1); // only one run block in device emu mode or it will be too slow
#endif
    dim3  threads(num_threads*2, 1, 1);

    // make sure there are no CUDA errors before we start
    cutilCheckMsg("Kernel execution failed");

    printf("Running parallel prefix sum (scan) of %d elements\n", num_elements);
    printf("Comparing 3 versions:\n\n");
  
    // execute the kernels
    unsigned int numIterations = 100;

    printf("1. scan_naive -- not work efficient (O(n log n) adds).\n");
    cutStartTimer(timer);
    for (unsigned int i = 0; i < numIterations; ++i)
    {
        scan_naive<<< grid, threads, 2 * shared_mem_size >>>
            (d_odata[0], d_idata, num_elements);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    cutResetTimer(timer);

    threads.x /= 2;
    
    printf("2. scan_workefficient -- Work efficient (O(n) adds), but many bank conflicts.\n");
    cutStartTimer(timer);
    for (unsigned int i = 0; i < numIterations; ++i)
    {
        scan_workefficient<<< grid, threads, shared_mem_size >>>
            (d_odata[1], d_idata, num_elements);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    cutResetTimer(timer);

    printf("3. scan_best -- work efficient with very few bank conflicts.\n");
    cutStartTimer(timer);
    for (unsigned int i = 0; i < numIterations; ++i)
    {
        scan_best<<< grid, threads, shared_mem_size >>>
            (d_odata[2], d_idata, num_elements);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    cutResetTimer(timer);

    // check for any errors
    cutilCheckMsg("Kernel execution failed");

    for (int i = 0; i < 3; ++i) // check all 3 results
    {
        // copy result from device to host
        cutilSafeCall(hipMemcpy( h_data, d_odata[i], sizeof(float) * num_elements, 
                                   hipMemcpyDeviceToHost));

        // If this is a regression test write the results to a file
        if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
        {
            // write file for regression test 
            cutWriteFilef( "./data/result.dat", h_data, num_elements, 0.0);
        }
        else 
        {
            // custom output handling when no regression test running
            // in this case check if the result is equivalent to the expected soluion
            
            // We can use an epsilon of 0 since values are integral and in a range 
            // that can be exactly represented
            float epsilon = 0.0f;
            unsigned int result_regtest = cutComparefe( reference, h_data, num_elements, epsilon);
            char* names[] = {"scan_naive", "scan_workefficient", "scan_best"};
            printf( "%s: Test %s\n", names[i], (1 == result_regtest) ? "PASSED" : "FAILED");
        }
    }

    printf("\nCheck out the CUDA Data Parallel Primitives Library for more on scan.\n");
    printf("http://www.gpgpu.org/developer/cudpp\n");

    // cleanup memory
    free( h_data);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata[0]));
    cutilSafeCall(hipFree(d_odata[1]));
    cutilSafeCall(hipFree(d_odata[2]));
    cutilCheckError(cutDeleteTimer(timer));

    hipDeviceReset();
}

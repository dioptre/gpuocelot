#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  Users and possessors of this source code 
 * are hereby granted a nonexclusive, royalty-free license to use this code 
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein. 
 *
 * Any use of this source code in individual and commercial software must 
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This sample demonstrates how 2D convolutions 
 * with very large kernel sizes 
 * can be efficiently implemented 
 * using FFT transformations.
 */



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include <cutil_inline.h>



typedef float2 Complex;



////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}



////////////////////////////////////////////////////////////////////////////////
// Reference straightfroward CPU convolution
////////////////////////////////////////////////////////////////////////////////
extern "C" void convolutionCPU(
    Complex *h_Result,
    Complex *h_Data,
    Complex *h_Kernel,
    int dataW,
    int dataH,
    int kernelW,
    int kernelH,
    int kernelX,
    int kernelY
);



////////////////////////////////////////////////////////////////////////////////
// Padding kernels
////////////////////////////////////////////////////////////////////////////////
#include "convolutionFFT2D_kernel.cuh"



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
int calculateFFTsize(int dataSize){
    //Highest non-zero bit position of dataSize
    int hiBit;
    //Neares lower and higher powers of two numbers for dataSize
    unsigned int lowPOT, hiPOT;

    //Align data size to a multiple of half-warp
    //in order to have each line starting at properly aligned addresses
    //for coalesced global memory writes in padKernel() and padData()
    dataSize = iAlignUp(dataSize, 16);

    //Find highest non-zero bit
    for(hiBit = 31; hiBit >= 0; hiBit--)
        if(dataSize & (1U << hiBit)) break;

    //No need to align, if already power of two
    lowPOT = 1U << hiBit;
    if(lowPOT == dataSize) return dataSize;

    //Align to a nearest higher power of two, if the size is small enough,
    //else align only to a nearest higher multiple of 512,
    //in order to save computation and memory bandwidth
    hiPOT = 1U << (hiBit + 1);
    if(hiPOT <= 1024)
        return hiPOT;
    else 
        return iAlignUp(dataSize, 512);
}



//Kernel dimensions
const int KERNEL_W = 7;
const int KERNEL_H = 7;

//Kernel center position
const int KERNEL_X = 1;
const int KERNEL_Y = 6;

//Width and height of padding for "clamp to border" addressing mode
const int PADDING_W = KERNEL_W - 1;
const int PADDING_H = KERNEL_H - 1;

//Input data dimension
#if 1
const int DATA_W = 200;
const int DATA_H = 200;
#else
const int DATA_W = 1000;
const int DATA_H = 1000;
#endif

//Derive FFT size from data and kernel dimensions
const int FFT_W = calculateFFTsize(DATA_W + PADDING_W);
const int FFT_H = calculateFFTsize(DATA_H + PADDING_H);

const int    FFT_SIZE = FFT_W    * FFT_H    * sizeof(Complex);
const int KERNEL_SIZE = KERNEL_W * KERNEL_H * sizeof(Complex);
const int   DATA_SIZE = DATA_W   * DATA_H   * sizeof(Complex);



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    Complex
        *h_Kernel,
        *h_Data,
        *h_ResultCPU,
        *h_ResultGPU;

    hipArray
        *a_Kernel,
        *a_Data;

    hipChannelFormatDesc float2tex 
        = hipCreateChannelDesc<float2>();

    Complex
        *d_PaddedKernel,
        *d_PaddedData;

    hipfftHandle FFTplan;

    Complex
        rCPU, rGPU;

    double
        max_delta_ref, delta, ref, sum_delta2, sum_ref2, L2norm;

    int i, x, y;

    unsigned int hTimer;


    // use command-line specified CUDA device, otherwise use device with highest Gflops/s

    if ( cutCheckCmdLineFlag(argc, (const char **)argv, "device"))
	    cutilDeviceInit(argc, argv);
    else
    	hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Input data size           : %i x %i\n", DATA_W,             DATA_H            );
    printf("Convolution kernel size   : %i x %i\n", KERNEL_W,           KERNEL_H          );
    printf("Padded image size         : %i x %i\n", DATA_W + PADDING_W, DATA_H + PADDING_H);
    printf("Aligned padded image size : %i x %i\n", FFT_W,              FFT_H             );

    printf("Allocating memory...\n");
        h_Kernel       = (Complex *)malloc(KERNEL_SIZE);
        h_Data         = (Complex *)malloc(DATA_SIZE);
        h_ResultCPU    = (Complex *)malloc(DATA_SIZE);
        h_ResultGPU    = (Complex *)malloc(FFT_SIZE);
        cutilSafeCall( hipMallocArray(&a_Kernel, &float2tex, KERNEL_W, KERNEL_H) );
        cutilSafeCall( hipMallocArray(&a_Data,   &float2tex,   DATA_W,   DATA_H) );
        cutilSafeCall( hipMalloc((void **)&d_PaddedKernel, FFT_SIZE) );
        cutilSafeCall( hipMalloc((void **)&d_PaddedData,   FFT_SIZE) );

    printf("Generating random input data...\n");
        srand(2007);
        for(i = 0; i < (KERNEL_W * KERNEL_H); i++){
            h_Kernel[i].x = (float)rand() / (float)RAND_MAX;
            h_Kernel[i].y = 0;
        }
        for(i = 0; i < (DATA_W * DATA_H); i++){
            h_Data[i].x = (float)rand() / (float)RAND_MAX;
            h_Data[i].y = 0;
        }

    printf("Creating FFT plan for %i x %i...\n", FFT_W, FFT_H);
        cufftSafeCall( hipfftPlan2d(&FFTplan, FFT_H, FFT_W, HIPFFT_C2C) );

    printf("Uploading to GPU and padding convolution kernel and input data...\n");
        printf("...initializing padded kernel and data storage with zeroes...\n");
        cutilSafeCall( hipMemset(d_PaddedKernel, 0, FFT_SIZE) );
        cutilSafeCall( hipMemset(d_PaddedData,   0, FFT_SIZE) );
        printf("...copying input data and convolution kernel from host to CUDA arrays\n");
        cutilSafeCall( hipMemcpyToArray(a_Kernel, 0, 0, h_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpyToArray(a_Data,   0, 0, h_Data,   DATA_SIZE,   hipMemcpyHostToDevice) );
        printf("...binding CUDA arrays to texture references\n");
        cutilSafeCall( hipBindTextureToArray(texKernel, a_Kernel) );
        cutilSafeCall( hipBindTextureToArray(texData,   a_Data)   );

        //Block width should be a multiple of maximum coalesced write size 
        //for coalesced memory writes in padKernel() and padData()
        dim3 threadBlock(16, 12);
        dim3 kernelBlockGrid(iDivUp(KERNEL_W, threadBlock.x), iDivUp(KERNEL_H, threadBlock.y));
        dim3 dataBlockGrid(iDivUp(FFT_W, threadBlock.x), iDivUp(FFT_H, threadBlock.y));

        printf("...padding convolution kernel\n");
        padKernel<<<kernelBlockGrid, threadBlock>>>(
            d_PaddedKernel,
            FFT_W,
            FFT_H,
            KERNEL_W,
            KERNEL_H,
            KERNEL_X,
            KERNEL_Y
        );
        cutilCheckMsg("padKernel() execution failed\n");

        printf("...padding input data array\n");
        padData<<<dataBlockGrid, threadBlock>>>(
            d_PaddedData,
            FFT_W,
            FFT_H,
            DATA_W,
            DATA_H,
            KERNEL_W,
            KERNEL_H,
            KERNEL_X,
            KERNEL_Y
        );
        cutilCheckMsg("padData() execution failed\n");

    //Not including kernel transformation into time measurement,
    //since convolution kernel is not changed very frequently
    printf("Transforming convolution kernel...\n");
        cufftSafeCall( hipfftExecC2C(FFTplan, (hipfftComplex *)d_PaddedKernel, (hipfftComplex *)d_PaddedKernel, HIPFFT_FORWARD) );

    printf("Running GPU FFT convolution...\n");
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        cufftSafeCall( hipfftExecC2C(FFTplan, (hipfftComplex *)d_PaddedData,   (hipfftComplex *)d_PaddedData,   HIPFFT_FORWARD) );
        modulateAndNormalize<<<16, 128>>>(
            d_PaddedData,
            d_PaddedKernel,
            FFT_W * FFT_H
        );
        cutilCheckMsg("modulateAndNormalize() execution failed\n");
        cufftSafeCall( hipfftExecC2C(FFTplan, (hipfftComplex *)d_PaddedData,   (hipfftComplex *)d_PaddedData,   HIPFFT_BACKWARD) );
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        double gpuTime = cutGetTimerValue(hTimer);
    printf("GPU time: %f msecs. //%f MPix/s\n", gpuTime, DATA_W * DATA_H * 1e-6 / (gpuTime * 0.001) );

    printf("Reading back GPU FFT results...\n");
        cutilSafeCall( hipMemcpy(h_ResultGPU, d_PaddedData, FFT_SIZE, hipMemcpyDeviceToHost) );


    printf("Checking GPU results...\n");
        printf("...running reference CPU convolution\n");
        convolutionCPU(
            h_ResultCPU,
            h_Data,
            h_Kernel,
            DATA_W,
            DATA_H,
            KERNEL_W,
            KERNEL_H,
            KERNEL_X,
            KERNEL_Y
        );

        printf("...comparing the results\n");
        sum_delta2 = 0;
        sum_ref2   = 0;
        max_delta_ref = 0;
        for(y = 0; y < DATA_H; y++)
            for(x = 0; x < DATA_W; x++){
                rCPU = h_ResultCPU[y * DATA_W + x];
                rGPU = h_ResultGPU[y * FFT_W  + x];
                delta = (rCPU.x - rGPU.x) * (rCPU.x - rGPU.x) + (rCPU.y - rGPU.y) * (rCPU.y - rGPU.y);
                ref   = rCPU.x * rCPU.x + rCPU.y * rCPU.y;
                if((delta / ref) > max_delta_ref) max_delta_ref = delta / ref;
                sum_delta2 += delta;
                sum_ref2   += ref;
            }
        L2norm = sqrt(sum_delta2 / sum_ref2);
    printf("Max delta / CPU value %E\n", sqrt(max_delta_ref));
    printf("L2 norm: %E\n", L2norm);
    printf((L2norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");


    printf("Shutting down...\n");
        cutilSafeCall( hipUnbindTexture(texData) );
        cutilSafeCall( hipUnbindTexture(texKernel) );
        cufftSafeCall( hipfftDestroy(FFTplan) );
        cutilSafeCall( hipFree(d_PaddedData)   );
        cutilSafeCall( hipFree(d_PaddedKernel) );
        cutilSafeCall( hipFreeArray(a_Data)   );
        cutilSafeCall( hipFreeArray(a_Kernel) );
        free(h_ResultGPU);
        free(h_ResultCPU);
        free(h_Data);
        free(h_Kernel);

    hipDeviceReset();

    cutilExit(argc, argv);
}

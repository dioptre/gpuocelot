
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs.
 *
 * There's one CUDA context per thread. To use multiple CUDA contexts you
 * have to create multiple threads. One for each GPU. For optimal performance,
 * the number of CPU cores should be equal to the number of GPUs in the system.
 *
 * Creating CPU threads has a certain overhead. So, this is only worth when you
 * have a significant amount of work to do per thread. It's also recommended to
 * create a pool of threads and reuse them to avoid this overhead.
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the 
 * application. On the other side, you can still extend your desktop to screens 
 * attached to both GPUs.
 */


////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA SDK sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N){
    const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;
    for(int pos = tid; pos < N; pos += threadN)
        sum += d_Input[pos];

    d_Result[tid] = sum;
}

extern "C" 
void launch_reduceKernel(float *d_Result, float *d_Input, int N, int BLOCK_N, int THREAD_N) 
{
	reduceKernel<<<BLOCK_N, THREAD_N>>>(d_Result, d_Input, N);
}

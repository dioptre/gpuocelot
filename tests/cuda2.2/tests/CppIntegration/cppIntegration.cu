#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include "cppIntegration_kernel.cu"
//
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void
computeGold(char* reference, char* idata, const unsigned int len);
extern "C" void
computeGold2(int2* reference, int2* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" void
runTest(const int argc, const char** argv, char* data, int2* data_int2, unsigned int len)
{

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, (char**)argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    const unsigned int num_threads = len / 4;
    cutilCondition(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    char* d_data;
    cutilSafeCall(hipMalloc((void**) &d_data, mem_size));
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_data, data, mem_size,
                            hipMemcpyHostToDevice) );
    // allocate device memory for int2 version
    int2* d_data_int2;
    cutilSafeCall(hipMalloc((void**) &d_data_int2, mem_size_int2));
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                            hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    kernel<<< grid, threads >>>((int*) d_data);
    kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // compute reference solutions
    char* reference = (char*) malloc(mem_size);
    computeGold(reference, data, len);
    int2* reference2 = (int2*) malloc(mem_size_int2);
    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    cutilSafeCall(hipMemcpy(data, d_data, mem_size,
                            hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                            hipMemcpyDeviceToHost));

    // check result
    bool success = true;
    for(unsigned int i = 0; i < len; i++ )
    {
        if( reference[i] != data[i] || 
	    reference2[i].x != data_int2[i].x || 
	    reference2[i].y != data_int2[i].y)
            success = false;
    }
    printf("Test %s\n", success ? "PASSED" : "FAILED");

    // cleanup memory
    cutilSafeCall(hipFree(d_data));
    cutilSafeCall(hipFree(d_data_int2));
    free(reference);
    free(reference2);

    hipDeviceReset();
}

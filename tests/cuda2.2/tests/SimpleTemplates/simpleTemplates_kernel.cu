#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* This sample is a templatized version of the template project.
 * It also shows how to correctly templatize dynamically allocated shared
 * memory arrays.
 * Device code.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "sharedmem.cuh"


////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template<class T>
__global__ void
testKernel( T* g_idata, T* g_odata) 
{
  // Shared mem size is determined by the host app at run time
  SharedMemory<T> smem;
  T* sdata = smem.getPointer();

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  // use the bank checker macro to check for bank conflicts during host
  // emulation
  sdata[tid] = g_idata[tid];
  __syncthreads();

  // perform some computations
  sdata[tid] = (T) num_threads * sdata[tid];
  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

#endif // #ifndef _TEMPLATE_KERNEL_H_

/*
* Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.  This source code is a "commercial item" as
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer software" and "commercial computer software
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*/

/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include "simpleTemplates_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
template <class T>
void runTest( int argc, char** argv, int len);

template<class T>
void
computeGold( T* reference, T* idata, const unsigned int len) 
{
    const T T_len = static_cast<T>( len);
    for( unsigned int i = 0; i < len; ++i) 
    {
        reference[i] = idata[i] * T_len;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest<float>( argc, argv, 32);
    runTest<int>( argc, argv, 64);

    cutilExit(argc, argv);
}

// To completely templatize runTest (below) with cutil, we need to use 
// template specialization to wrap up CUTIL's array comparison and file writing
// functions for different types.  

// Here's the generic wrapper for cutCompare*
template<class T>
class ArrayComparator
{
public:
    CUTBoolean compare( const T* reference, T* data, unsigned int len)
    {
        fprintf(stderr, "Error: no comparison function implemented for this type\n");
        return CUTFalse;
    }
};

// Here's the specialization for ints:
template<>
class ArrayComparator<int>
{
public:
    CUTBoolean compare( const int* reference, int* data, unsigned int len)
    {
        return cutComparei(reference, data, len);
    }
};

// Here's the specialization for floats:
template<>
class ArrayComparator<float>
{
public:
    CUTBoolean compare( const float* reference, float* data, unsigned int len)
    {
        return cutComparef(reference, data, len);
    }
};

// Here's the generic wrapper for cutWriteFile*
template<class T>
class ArrayFileWriter
{
public:
    CUTBoolean write(const char* filename, T* data, unsigned int len, float epsilon)
    {
        fprintf(stderr, "Error: no file write function implemented for this type\n");
        return CUTFalse;
    }
};

// Here's the specialization for ints:
template<>
class ArrayFileWriter<int>
{
public:
    CUTBoolean write(const char* filename, int* data, unsigned int len, float epsilon)
    {
        return cutWriteFilei(filename, data, len, epsilon != 0);
    }
};

// Here's the specialization for floats:
template<>
class ArrayFileWriter<float>
{
public:
    CUTBoolean write(const char* filename, float* data, unsigned int len, float epsilon)
    {
        return cutWriteFilef(filename, data, len, epsilon);
    }
};


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
template<class T>
void
runTest( int argc, char** argv, int len) 
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
	{
		cutilDeviceInit(argc, argv);
	}
	else
	{
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	}

    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    unsigned int num_threads = len;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    T* h_idata = (T*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (T) i;
    }

    // allocate device memory
    T* d_idata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // allocate device memory for result
    T* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
    testKernel<T><<< grid, threads, mem_size >>>( d_idata, d_odata);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // allocate mem for the result on host side
    T* h_odata = (T*) malloc( mem_size);
    // copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_odata, sizeof(T) * num_threads,
                                hipMemcpyDeviceToHost) );

    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));

    // compute reference solution
    T* reference = (T*) malloc( mem_size);
    computeGold<T>( reference, h_idata, num_threads);

    ArrayComparator<T> comparator;
    ArrayFileWriter<T> writer;

    // check result
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test
        cutilCheckError( writer.write( "./data/regression.dat",
                                     h_odata, num_threads, 0.0));
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        CUTBoolean res = comparator.compare( reference, h_odata, num_threads);
        printf( "Test %s\n", (1 == res) ? "PASSED" : "FAILED");
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));

    hipDeviceReset();
}

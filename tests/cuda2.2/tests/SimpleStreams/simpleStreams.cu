#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 *
 *
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to 
 * initialize an array to a specific value, after which the array is 
 * copied to the host (CPU) memory.  To increase performance, multiple 
 * kernel/memcopy pairs are launched asynchronously, each pair in its 
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5 
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

#include <stdio.h>
#include <cutil_inline.h>

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=0;i<num_iterations;i++)
	    g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

int correct_data(int *a, const int n, const int c)
{
    for(int i = 0; i < n; i++)
        if(a[i] != c)
		{
			printf("%d: %d %d\n", i, a[i], c);
            return 0;
		}

    return 1;
}

int main(int argc, char *argv[])
{
    int cuda_device = 0;
    int nstreams = 4;               // number of streams for CUDA calls
    int nreps = 10;                 // number of times each experiment is repeated
    int n = 16 * 1024 * 1024;       // number of ints in the data set
    int nbytes = n * sizeof(int);   // number of data bytes
    dim3 threads, blocks;           // kernel launch configuration
    float elapsed_time, time_memcpy, time_kernel;   // timing variables

	int niterations;	// number of iterations for the loop inside the kernel

    if( argc > 1 )
        cuda_device = atoi( argv[1] );

#if 1
    n = 4096;   // reduced workload for emulation (n should be divisible by 512*nstreams)
#endif

    // check the compute capability of the device
    int num_devices=0;
    cutilSafeCall( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
        printf("your system does not have a CUDA capable device\n");
        return 1;
	}
	
    // check if the command-line chosen device ID is within range, exit if not
    if( cuda_device >= num_devices )
	{
        printf("choose device ID between 0 and %d\n", num_devices-1);
        return 1;
	}

    hipSetDevice( cuda_device );

    hipDeviceProp_t device_properties;
	
    cutilSafeCall( hipGetDeviceProperties(&device_properties, cuda_device) );
    if( (1 == device_properties.major) && (device_properties.minor < 1))
        printf("%s does not have compute capability 1.1 or later\n\n", device_properties.name);
    if(device_properties.minor > 1)
        niterations = 5;
    else
        niterations = 1;    // reduced workload for compute capability 1.0 and 1.1

	printf("running on: %s\n\n", device_properties.name );

    // allocate host memory
    int c = 5;                      // value to which the array will be initialized
    int *a = 0;                     // pointer to the array data in host memory
    // allocate host memory (pinned is required for achieve asynchronicity)
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) ); 

    // allocate device memory
    int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );
    cutilSafeCall( hipMalloc((void**)&d_c, sizeof(int)) );
    cutilSafeCall( hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice) );

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
        cutilSafeCall( hipStreamCreate(&(streams[i])) );

    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    cutilSafeCall( hipEventCreate(&start_event) );
    cutilSafeCall( hipEventCreate(&stop_event) );

    // time memcopy from device
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, streams[0]);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    cutilSafeCall( hipEventElapsedTime(&time_memcpy, start_event, stop_event) );
    printf("memcopy:\t%.2f\n", time_memcpy);
    
    // time kernel
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    cutilSafeCall( hipEventElapsedTime(&time_kernel, start_event, stop_event) );
    printf("kernel:\t\t%.2f\n", time_kernel);

    //////////////////////////////////////////////////////////////////////
    // time non-streamed execution for reference
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        init_array<<<blocks, threads>>>(d_a, d_c, niterations);
        hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    cutilSafeCall( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("non-streamed:\t%.2f (%.2f expected)\n", elapsed_time / nreps, time_kernel + time_memcpy);

    //////////////////////////////////////////////////////////////////////
    // time execution with nstreams streams
    threads=dim3(512,1);
    blocks=dim3(n/(nstreams*threads.x),1);
    memset(a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
    hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        // asynchronously launch nstreams kernels, each operating on its own portion of data
        for(int i = 0; i < nstreams; i++)
            init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);

        // asynchronoously launch nstreams memcopies.  Note that memcopy in stream x will only
        //   commence executing when all previous CUDA calls in stream x have completed
        for(int i = 0; i < nstreams; i++)
            hipMemcpyAsync(a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    cutilSafeCall( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("%d streams:\t%.2f (%.2f expected with compute capability 1.1 or later)\n", nstreams, elapsed_time / nreps, time_kernel + time_memcpy / nstreams);

    // check whether the output is correct
    printf("-------------------------------\n");
    if(correct_data(a, n, c*nreps*niterations))	// each element should have been incremented by c a total of npres*niterations times
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // release resources
    for(int i = 0; i < nstreams; i++)
        hipStreamDestroy(streams[i]);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipHostFree(a);
    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();

    cutilExit(argc, argv);

    return 0;
}

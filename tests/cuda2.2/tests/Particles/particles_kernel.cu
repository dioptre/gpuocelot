#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* 
 * Device code.
 */

#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include <stdio.h>
#include <math.h>
#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"

#if USE_TEX
// textures for particle position and velocity
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;

texture<uint2, 1, hipReadModeElementType> particleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;

texture<uint, 1, hipReadModeElementType> gridCountersTex;
texture<uint, 1, hipReadModeElementType> gridCellsTex;
#endif

__constant__ SimParams params;

// integrate particle attributes
__global__ void
integrate(float4* newPos, float4* newVel, 
          float4* oldPos, float4* oldVel, 
          float deltaTime)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	float4 pos4 = oldPos[index];
    float4 vel4 = oldVel[index];
    float3 pos = make_float3(pos4);
    float3 vel = make_float3(vel4);

    vel += params.gravity * deltaTime;
    vel *= params.globalDamping;

    // new position = old position + velocity * deltaTime
    pos += vel * deltaTime;

    // bounce off cube sides
    if (pos.x > 1.0f - params.particleRadius) { pos.x = 1.0f - params.particleRadius; vel.x *= params.boundaryDamping; }
    if (pos.x < -1.0f + params.particleRadius) { pos.x = -1.0f + params.particleRadius; vel.x *= params.boundaryDamping;}
    if (pos.y > 1.0f - params.particleRadius) { pos.y = 1.0f - params.particleRadius; vel.y *= params.boundaryDamping; }
    if (pos.y < -1.0f + params.particleRadius) { pos.y = -1.0f + params.particleRadius; vel.y *= params.boundaryDamping;}
    if (pos.z > 1.0f - params.particleRadius) { pos.z = 1.0f - params.particleRadius; vel.z *= params.boundaryDamping; }
    if (pos.z < -1.0f + params.particleRadius) { pos.z = -1.0f + params.particleRadius; vel.z *= params.boundaryDamping;}

    // store new position and velocity
    newPos[index] = make_float4(pos, pos4.w);
    newVel[index] = make_float4(vel, vel4.w);
}

// calculate position in uniform grid
__device__ int3 calcGridPos(float4 p)
{
    int3 gridPos;
    gridPos.x = floor((p.x - params.worldOrigin.x) / params.cellSize.x);
    gridPos.y = floor((p.y - params.worldOrigin.y) / params.cellSize.y);
    gridPos.z = floor((p.z - params.worldOrigin.z) / params.cellSize.z);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = max(0, min(gridPos.x, params.gridSize.x-1));
    gridPos.y = max(0, min(gridPos.y, params.gridSize.y-1));
    gridPos.z = max(0, min(gridPos.z, params.gridSize.z-1));
    return __mul24(__mul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __mul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

// add particle to cell using atomics
__device__ void addParticleToCell(int3 gridPos,
                                  uint index,
                                  uint* gridCounters,
                                  uint* gridCells)
{
    // calculate grid hash
    uint gridHash = calcGridHash(gridPos);

    // increment cell counter using atomics
#if defined CUDA_NO_SM_11_ATOMIC_INTRINSICS
    int counter = 0;
#else
    int counter = atomicAdd(&gridCounters[gridHash], 1); // returns previous value
    counter = min(counter, params.maxParticlesPerCell-1);
#endif

    // write particle index into this cell (very uncoalesced!)
    gridCells[gridHash*params.maxParticlesPerCell + counter] = index;
}


// update uniform grid
__global__ void
updateGridD(float4* pos,
            uint*   gridCounters,
            uint*   gridCells)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    float4 p = pos[index];

    // get address in grid
    int3 gridPos = calcGridPos(p);

    addParticleToCell(gridPos, index, gridCounters, gridCells);
}

// calculate grid hash value for each particle
__global__ void
calcHashD(float4* pos,
          uint2*  particleHash)
{
    int index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    float4 p = pos[index];

    // get address in grid
    int3 gridPos = calcGridPos(p);
    uint gridHash = calcGridHash(gridPos);

    // store grid hash and particle index
    particleHash[index] = make_uint2(gridHash, index);
}

// rearrange particle data into sorted order, and find the start of each cell in the
// sorted hash array
__global__ void
reorderDataAndFindCellStartD(uint2*  particleHash,  // particle id sorted by hash
				             float4* oldPos,
							 float4* oldVel,
							 float4* sortedPos, 
							 float4* sortedVel,
							 uint*   cellStart)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

    uint2 sortedData = particleHash[index];

	// Load hash data into shared memory so that we can look 
	// at neighboring particle's hash value without loading
	// two hash values per thread
	__shared__ uint sharedHash[257];
	sharedHash[threadIdx.x+1] = sortedData.x;
	if (index > 0 && threadIdx.x == 0)
	{
		// first thread in block must load neighbor particle hash
		volatile uint2 prevData = particleHash[index-1];
		sharedHash[0] = prevData.x;
	}

	__syncthreads();
	if (index == 0 || sortedData.x != sharedHash[threadIdx.x])
	{
		cellStart[sortedData.x] = index;
	}

	// Now use the sorted index to reorder the pos and vel data
	float4 pos = FETCH(oldPos, sortedData.y);       // macro does either global read or texture fetch
    float4 vel = FETCH(oldVel, sortedData.y);       // see particles_kernel.cuh

	__syncthreads();

    sortedPos[index] = pos;
    sortedVel[index] = vel;

}

// collide two spheres using DEM method
__device__ float3 collideSpheres(float4 posA, float4 posB,
                                 float4 velA, float4 velB,
                                 float radiusA, float radiusB,
                                 float attraction)
{
	// calculate relative position
    float3 relPos;
    relPos.x = posB.x - posA.x;
    relPos.y = posB.y - posA.y;
    relPos.z = posB.z - posA.z;

    float dist = length(relPos);
    float collideDist = radiusA + radiusB;

    float3 force = make_float3(0.0f);
    if (dist < collideDist) {
        float3 norm = relPos / dist;

		// relative velocity
        float3 relVel;
        relVel.x = velB.x - velA.x;
        relVel.y = velB.y - velA.y;
        relVel.z = velB.z - velA.z;

        // relative tangential velocity
        float3 tanVel = relVel - (dot(relVel, norm) * norm);

        // spring force
        force = -params.spring*(collideDist - dist) * norm;
        // dashpot (damping) force
        force += params.damping*relVel;
        // tangential shear force
        force += params.shear*tanVel;
		// attraction
        force += attraction*relPos;
    }

    return force;
}


// collide particle with all particles in a given cell
// version using grid built with atomics
__device__
float3 collideCell(int3 gridPos,
                   uint index,
                   float4 pos,
                   float4 vel,
                   float4* oldPos, 
                   float4* oldVel,
                   uint*   gridCounters,
                   uint*   gridCells)
{
    float3 force = make_float3(0.0f);

    if ((gridPos.x < 0) || (gridPos.x > params.gridSize.x-1) ||
        (gridPos.y < 0) || (gridPos.y > params.gridSize.y-1) ||
        (gridPos.z < 0) || (gridPos.z > params.gridSize.z-1)) {
        return force;
    }

    uint gridHash = calcGridHash(gridPos);
    
    // iterate over particles in this cell
    uint particlesInCell = FETCH(gridCounters, gridHash);
    particlesInCell = min(particlesInCell, params.maxParticlesPerCell-1);

    for(uint i=0; i<particlesInCell; i++) {
        uint index2 = FETCH(gridCells, gridHash*params.maxParticlesPerCell + i);

        if (index2 != index) {              // check not colliding with self
	        float4 pos2 = FETCH(oldPos, index2);
            float4 vel2 = FETCH(oldVel, index2);

            // collide two spheres
            float3 projVec = collideSpheres(pos, pos2, vel, vel2, params.particleRadius, params.particleRadius, params.attraction);
            force += projVec;
        }
    }

    return force;
}


// version using sorted grid
__device__
float3 collideCell2(int3   gridPos,
                   uint    index,
                   float4  pos,
                   float4  vel,
                   float4* oldPos, 
                   float4* oldVel,
                   uint2*  particleHash,
                   uint*   cellStart)
{
    float3 force = make_float3(0.0f);

    if ((gridPos.x < 0) || (gridPos.x > params.gridSize.x-1) ||
        (gridPos.y < 0) || (gridPos.y > params.gridSize.y-1) ||
        (gridPos.z < 0) || (gridPos.z > params.gridSize.z-1)) {
        return force;
    }

    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint bucketStart = FETCH(cellStart, gridHash);
    if (bucketStart == 0xffffffff)
        return force;   // cell empty
 
    // iterate over particles in this cell
    for(uint i=0; i<params.maxParticlesPerCell; i++) {
        uint index2 = bucketStart + i;
        uint2 cellData = FETCH(particleHash, index2);
        if (cellData.x != gridHash) break;   // no longer in same bucket

        if (index2 != index) {              // check not colliding with self
	        float4 pos2 = FETCH(oldPos, index2);
            float4 vel2 = FETCH(oldVel, index2);

            // collide two spheres
            float3 projVec = collideSpheres(pos, pos2, vel, vel2, params.particleRadius, params.particleRadius, params.attraction);
            force += projVec;
        }
    }

    return force;
}


__global__ void
collideD(float4* newPos, float4* newVel, 
         float4* oldPos, float4* oldVel, 
#if USE_SORT
         uint2*  particleHash,
         uint*   cellStart
#else
         uint*   gridCounters,
         uint*   gridCells
#endif
         )
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

    // read particle data from sorted arrays
	float4 pos = FETCH(oldPos, index);
    float4 vel = FETCH(oldVel, index);

    // get address in grid
    int3 gridPos = calcGridPos(pos);

    float3 force = make_float3(0.0f);

    // examine only neighbouring cells
    for(int z=-1; z<=1; z++) {
        for(int y=-1; y<=1; y++) {
            for(int x=-1; x<=1; x++) {
#if USE_SORT
                force += collideCell2(gridPos + make_int3(x, y, z), index, pos, vel, oldPos, oldVel, particleHash, cellStart);
#else
                force += collideCell(gridPos + make_int3(x, y, z), index, pos, vel, oldPos, oldVel, gridCounters, gridCells);
#endif
            }
        }
    }

    float3 projVec = collideSpheres(pos, params.colliderPos, vel, make_float4(0.0f, 0.0f, 0.0f, 0.0f), params.particleRadius, params.colliderRadius, 0.0f);
    force += projVec;

#if USE_SORT
    // write new velocity back to original unsorted location
    volatile uint2 sortedData = particleHash[index];
    newVel[sortedData.y] = vel + make_float4(force, 0.0f);
#else
    newVel[index] = vel + make_float4(force, 0.0f);
#endif
}

#endif

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>

#include "particles_kernel.cu"
#include "radixsort.cu"

extern "C"
{

void cudaInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );
}

void allocateArray(void **devPtr, size_t size)
{
    cutilSafeCall(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    cutilSafeCall(hipFree(devPtr));
}

void threadSync()
{
    cutilSafeCall(hipDeviceSynchronize());
}

void copyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{   
    if (vbo)
        cutilSafeCall(cudaGLMapBufferObject((void**)&device, vbo));

    cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    
    if (vbo)
        cutilSafeCall(cudaGLUnmapBufferObject(vbo));
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo)
{
    cutilSafeCall(cudaGLRegisterBufferObject(vbo));
}

void unregisterGLBufferObject(uint vbo)
{
    cutilSafeCall(cudaGLUnregisterBufferObject(vbo));
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void 
integrateSystem(uint vboOldPos, uint vboNewPos, 
                float* oldVel, float* newVel, 
                float deltaTime,
                int numParticles)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    float *oldPos, *newPos;
    cutilSafeCall(cudaGLMapBufferObject((void**)&oldPos, vboOldPos));
    cutilSafeCall(cudaGLMapBufferObject((void**)&newPos, vboNewPos));

    // execute the kernel
    integrate<<< numBlocks, numThreads >>>((float4*)newPos, (float4*)newVel,
                                           (float4*)oldPos, (float4*)oldVel,
                                           deltaTime);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("integrate kernel execution failed");

    cutilSafeCall(cudaGLUnmapBufferObject(vboOldPos));
    cutilSafeCall(cudaGLUnmapBufferObject(vboNewPos));
}

void 
updateGrid(uint    vboPos, 
           uint*   gridCounters,
           uint*   gridCells,
           uint    numParticles,
           uint    numCells)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    float *pos;
    cutilSafeCall(cudaGLMapBufferObject((void**)&pos, vboPos));

    cutilSafeCall(hipMemset(gridCounters, 0, numCells*sizeof(uint)));

    // execute the kernel
    updateGridD<<< numBlocks, numThreads >>>((float4 *) pos,
                                             gridCounters,
                                             gridCells);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

    cutilSafeCall(cudaGLUnmapBufferObject(vboPos));
}


void 
calcHash(uint    vboPos, 
         uint*   particleHash,
         int     numParticles)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    float *pos;
    cutilSafeCall(cudaGLMapBufferObject((void**)&pos, vboPos));

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>((float4 *) pos,
                                           (uint2 *) particleHash);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

    cutilSafeCall(cudaGLUnmapBufferObject(vboPos));
}

void 
reorderDataAndFindCellStart(uint*  particleHash,
							uint   vboOldPos,
							float* oldVel,
							float* sortedPos,
							float* sortedVel,
							uint*  cellStart,
							uint   numParticles,
							uint   numCells)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

	cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

    float *oldPos;
    cutilSafeCall(cudaGLMapBufferObject((void**)&oldPos, vboOldPos));

#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
#endif

    reorderDataAndFindCellStartD<<< numBlocks, numThreads >>>(
		(uint2 *)  particleHash,
        (float4 *) oldPos,
        (float4 *) oldVel,
        (float4 *) sortedPos,
        (float4 *) sortedVel,
        (uint *)   cellStart);
    cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
#endif

    cutilSafeCall(cudaGLUnmapBufferObject(vboOldPos));
}

void
collide(uint   vboOldPos, uint vboNewPos,
        float* sortedPos, float* sortedVel,
        float* oldVel, float* newVel,
        uint*  gridCounters,
        uint*  gridCells,
        uint*  particleHash,
        uint*  cellStart,
        uint   numParticles,
        uint   numCells,
        uint   maxParticlesPerCell)
{
    float4 *oldPos, *newPos;
    cutilSafeCall(cudaGLMapBufferObject((void**)&oldPos, vboOldPos));
    cutilSafeCall(cudaGLMapBufferObject((void**)&newPos, vboNewPos));

#if USE_TEX

#if USE_SORT
    // use sorted arrays
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));

    cutilSafeCall(hipBindTexture(0, particleHashTex, particleHash, numParticles*sizeof(uint2)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
#else

    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));

    cutilSafeCall(hipBindTexture(0, gridCountersTex, gridCounters,numCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, gridCellsTex, gridCells, numCells*maxParticlesPerCell*sizeof(uint)));
#endif

#endif

    // thread per particle
    int numThreads, numBlocks;
    computeGridSize(numParticles, BLOCKDIM, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float4*)newPos, (float4*)newVel,
#if USE_SORT
                                          (float4*)sortedPos, (float4*)sortedVel,
                                          (uint2 *) particleHash,
                                          cellStart
#else
                                          (float4*)oldPos, (float4*)oldVel,
                                          gridCounters,
                                          gridCells
#endif
                                          );

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

    cutilSafeCall(cudaGLUnmapBufferObject(vboNewPos));
    cutilSafeCall(cudaGLUnmapBufferObject(vboOldPos));

#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));

#if USE_SORT
    cutilSafeCall(hipUnbindTexture(particleHashTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
#else
    cutilSafeCall(hipUnbindTexture(gridCountersTex));
    cutilSafeCall(hipUnbindTexture(gridCellsTex));
#endif
#endif
}

}   // extern "C"

#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  Users and possessors of this source code 
 * are hereby granted a nonexclusive, royalty-free license to use this code 
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein. 
 *
 * Any use of this source code in individual and commercial software must 
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This sample implements 64-bin histogram calculation
 * of arbitrary-sized 8-bit data array
 */



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
//#include <cutil_inline.h>
#include <cutil_inline.h>



////////////////////////////////////////////////////////////////////////////////
// Reference CPU histogram
////////////////////////////////////////////////////////////////////////////////
extern "C" void histogram64CPU(
    unsigned int *h_Result,
    unsigned int *h_Data,
    int dataN
);



////////////////////////////////////////////////////////////////////////////////
// GPU histogram
////////////////////////////////////////////////////////////////////////////////
#include "histogram64_kernel.cuh"



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//Input data size
const int         DATA_N = 1000000;
const int      DATA_SIZE = DATA_N * sizeof(unsigned char);


//Repeat computations multiple times to improve timing precision
const int NUM_ITERATIONS = 1;

int main(int argc, char **argv){
    unsigned char
        *h_Data;

    unsigned int
        *h_HistogramCPU, *h_HistogramGPU;

    unsigned char
        *d_Data;

    unsigned int
        *d_Histogram;

    int i, sum, delta, iter;
    double timerValue;

    unsigned int hTimer;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError(cutCreateTimer(&hTimer));

    printf("Initializing data...\n");
        printf("...allocating CPU memory.\n");
        h_Data         = (unsigned char *)malloc(DATA_SIZE);
        h_HistogramCPU = (unsigned int *)malloc(HISTOGRAM_SIZE);
        h_HistogramGPU = (unsigned int *)malloc(HISTOGRAM_SIZE);

        printf("...generating input data\n");
        srand(2007);
        for(i = 0; i < DATA_N; i++) 
            h_Data[i] = rand() % 256;

        printf("...allocating GPU memory and copying input data\n");
        cutilSafeCall( hipMalloc((void **)&d_Data, DATA_SIZE  ) );
        cutilSafeCall( hipMalloc((void **)&d_Histogram, HISTOGRAM_SIZE  ) );
        cutilSafeCall( hipMemcpy(d_Data, h_Data, DATA_SIZE, hipMemcpyHostToDevice) );


    printf("Running GPU histogram (%i iterations)...\n", NUM_ITERATIONS);
        initHistogram64GPU();
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        for(iter = 0; iter < NUM_ITERATIONS; iter++);
            histogram64GPU(d_Histogram, (unsigned int *)d_Data, DATA_N);
        cutilSafeCall( hipDeviceSynchronize() );

        cutilCheckError(  cutStopTimer(hTimer));
        timerValue = cutGetTimerValue(hTimer) / NUM_ITERATIONS;
        closeHistogram64GPU();
    printf("histogram64GPU() time (average) : %f msec //%f MB/sec\n", timerValue, DATA_SIZE / (1e+6 * timerValue * 0.001));

    printf("Reading back GPU results...\n");
        cutilSafeCall( hipMemcpy(h_HistogramGPU, d_Histogram, HISTOGRAM_SIZE, hipMemcpyDeviceToHost) );

    printf("Comparing the results...\n");
        printf("...histogramCPU()\n");
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        histogram64CPU(
            h_HistogramCPU,
            (unsigned int *)h_Data,
            DATA_N / 4
        );
        cutilCheckError(cutStopTimer(hTimer));
        timerValue = cutGetTimerValue(hTimer);
    printf("histogram64CPU() time : %f msec //%f MB/sec\n", timerValue, DATA_SIZE / (1048576.0 * timerValue * 0.001));

        sum   = 0;
        delta = 0;
        for(i = 0; i < BIN_COUNT; i++){
            sum   += h_HistogramGPU[i];
            delta += abs((int)h_HistogramGPU[i] - (int)h_HistogramCPU[i]);
        }

    printf("Total sum of histogram elements: %i\n", sum);
    printf("Sum of absolute differences: %i\n", delta);
    printf((delta == 0) ? "TEST PASSED\n" : "TEST FAILED\n");

    printf("Shutting down...\n");
        cutilCheckError(cutDeleteTimer(hTimer));
        cutilSafeCall( hipFree(d_Histogram) );
        cutilSafeCall( hipFree(d_Data) );
        free(h_HistogramGPU);
        free(h_HistogramCPU);
        free(h_Data);

    hipDeviceReset();

    cutilExit(argc, argv);
}

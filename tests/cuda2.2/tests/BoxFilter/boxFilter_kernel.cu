#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _BOXFILTER_KERNEL_H_
#define _BOXFILTER_KERNEL_H_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil_inline.h>
#include <cutil_math.h>

texture<float, 2> tex;
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;

hipArray* d_array, *d_tempArray;

/*
    Perform a fast box filter using the sliding window method.

    As the kernel moves from left to right, we add in the contribution of the new
    sample on the right, and subtract the value of the exiting sample on the left.
    This only requires 2 adds and a mul per output value, independent of the filter radius.
    The box filter is separable, so to perform a 2D box filter we perform the filter in
    the x direction, followed by the same filter in the y direction.
    Applying multiple iterations of the box filter converges towards a Gaussian blur.
    Using CUDA, rows or columns of the image are processed in parallel.
    This version duplicates edge pixels.

    Note that the x (row) pass suffers from uncoalesced global memory reads,
    since each thread is reading from a different row. For this reason it is
    better to use texture lookups for the x pass.
    The y (column) pass is perfectly coalesced.

    Parameters
    id - pointer to input data in global memory
    od - pointer to output data in global memory
    w  - image width
    h  - image height
    r  - filter radius

    e.g. for r = 2, w = 8:

    0 1 2 3 4 5 6 7
    x - -
    - x - -
    - - x - -
      - - x - -
        - - x - -
          - - x - -
            - - x -
              - - x
*/

// process row
__device__ void
d_boxfilter_x(float *id, float *od, int w, int h, int r)
{
    float scale = 1.0f / (2*r+1);

    float t;
    // do left edge
    t = id[0] * r;
    for (int x = 0; x < r+1; x++) {
        t += id[x];
    }
    od[0] = t * scale;

    for(int x = 1; x < r+1; x++) {
        t += id[x+r];
        t -= id[0];
        od[x] = t * scale;
    }
    
    // main loop
    for(int x = r+1; x < w-r; x++) {
        t += id[x+r];
        t -= id[x-r-1];
        od[x] = t * scale;
    }

    // do right edge
    for (int x = w-r; x < w; x++) {
        t += id[w-1];
        t -= id[x-r-1];
        od[x] = t * scale;
    }
}

// process column
__device__ void
d_boxfilter_y(float *id, float *od, int w, int h, int r)
{
    float scale = 1.0f / (2*r+1);
    
    float t;
    // do left edge
    t = id[0] * r;
    for (int y = 0; y < r+1; y++) {
        t += id[y*w];
    }
    od[0] = t * scale;

    for(int y = 1; y < r+1; y++) {
        t += id[(y+r)*w];
        t -= id[0];
        od[y*w] = t * scale;
    }
    
    // main loop
    for(int y = r+1; y < h-r; y++) {
        t += id[(y+r)*w];
        t -= id[((y-r)*w)-w];
        od[y*w] = t * scale;
    }

    // do right edge
    for (int y = h-r; y < h; y++) {
        t += id[(h-1)*w];
        t -= id[((y-r)*w)-w];
        od[y*w] = t * scale;
    }
}

__global__ void
d_boxfilter_x_global(float *id, float *od, int w, int h, int r)
{
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_x(&id[y*w], &od[y*w], w, h, r);
}

__global__ void
d_boxfilter_y_global(float *id, float *od, int w, int h, int r)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	d_boxfilter_y(&id[x], &od[x], w, h, r);
}

// texture version
// texture fetches automatically clamp to edge of image
__global__ void
d_boxfilter_x_tex(float *od, int w, int h, int r)
{
    float scale = 1.0f / (2*r+1);
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;
    for(int x=-r; x<=r; x++) {
        t += tex2D(tex, x, y);
    }
    od[y*w] = t * scale;

    for(int x=1; x<w; x++) {
        t += tex2D(tex, x + r, y);
        t -= tex2D(tex, x - r - 1, y);
        od[y*w+x] = t * scale;
    }
}

__global__ void
d_boxfilter_y_tex(float *od, int w, int h, int r)
{
    float scale = 1.0f / (2*r+1);
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;
    for(int y=-r; y<=r; y++) {
        t += tex2D(tex, x, y);
    }
    od[x] = t * scale;

    for(int y=1; y<h; y++) {
        t += tex2D(tex, x, y + r);
        t -= tex2D(tex, x, y - r - 1);
        od[y*w+x] = t * scale;
    }
}

// RGBA version
// reads from 32-bit uint array holding 8-bit RGBA

// convert floating point rgba color to 32-bit integer
__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ float4 rgbaIntToFloat(uint c)
{
    float4 rgba;
    rgba.x = (c & 0xff) / 255.0f;
    rgba.y = ((c>>8) & 0xff) / 255.0f;
    rgba.z = ((c>>16) & 0xff) / 255.0f;
    rgba.w = ((c>>24) & 0xff) / 255.0f;
    return rgba;
}

// row pass using texture lookups
__global__ void
d_boxfilter_rgba_x(uint *od, int w, int h, int r)
{
    float scale = 1.0f / (2*r+1);
	unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    // as long as address is always less than height, we do work
    if (y < h) {
        float4 t = make_float4(0.0f);
        for(int x=-r; x<=r; x++) {
            t += tex2D(rgbaTex, x, y);
        }
        od[y*w] = rgbaFloatToInt(t * scale);

        for(int x=1; x<w; x++) {
            t += tex2D(rgbaTex, x + r, y);
            t -= tex2D(rgbaTex, x - r - 1, y);
            od[y*w+x] = rgbaFloatToInt(t * scale);
        }
    }
}

// column pass using coalesced global memory reads
__global__ void
d_boxfilter_rgba_y(uint *id, uint *od, int w, int h, int r)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    id = &id[x];
    od = &od[x];

    float scale = 1.0f / (2*r+1);
    
    float4 t;
    // do left edge
    t = rgbaIntToFloat(id[0]) * r;
    for (int y = 0; y < r+1; y++) {
        t += rgbaIntToFloat(id[y*w]);
    }
    od[0] = rgbaFloatToInt(t * scale);

    for(int y = 1; y < r+1; y++) {
        t += rgbaIntToFloat(id[(y+r)*w]);
        t -= rgbaIntToFloat(id[0]);
        od[y*w] = rgbaFloatToInt(t * scale);
    }
    
    // main loop
    for(int y = r+1; y < h-r; y++) {
        t += rgbaIntToFloat(id[(y+r)*w]);
        t -= rgbaIntToFloat(id[((y-r)*w)-w]);
        od[y*w] = rgbaFloatToInt(t * scale);
    }

    // do right edge
    for (int y = h-r; y < h; y++) {
        t += rgbaIntToFloat(id[(h-1)*w]);
        t -= rgbaIntToFloat(id[((y-r)*w)-w]);
        od[y*w] = rgbaFloatToInt(t * scale);
    }
}

extern "C" 
void initTexture(int width, int height, void *pImage)
{
    int size = width * height * sizeof(unsigned int);

    // copy image data to array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    cutilSafeCall( hipMallocArray  ( &d_array, &channelDesc, width, height )); 
    cutilSafeCall( hipMemcpyToArray( d_array, 0, 0, pImage, size, hipMemcpyHostToDevice));

    cutilSafeCall( hipMallocArray  ( &d_tempArray,   &channelDesc, width, height )); 

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = true;

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray(tex, d_array, channelDesc) );
}

extern "C"
void freeTextures()
{
    cutilSafeCall(hipFreeArray(d_array));
    cutilSafeCall(hipFreeArray(d_tempArray));

}


/*
    Perform 2D box filter on image using CUDA

    Parameters:
    d_src  - pointer to input image in device memory
    d_temp - pointer to temporary storage in device memory
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    radius - filter radius
    iterations - number of iterations

*/
extern "C" 
void boxFilter(float *d_src, float *d_temp, float *d_dest, int width, int height, 
               int radius, int iterations, int nthreads)
{
    cutilSafeCall( hipBindTextureToArray(tex, d_array) );

    for(int i=0; i<iterations; i++) {
        // use texture for horizontal pass
        d_boxfilter_x_tex<<< height / nthreads, nthreads, 0 >>>( d_temp, width, height, radius);
        d_boxfilter_y_global<<< width / nthreads, nthreads, 0 >>>( d_temp, d_dest, width, height, radius);
        if (iterations > 1) {
            // copy result back from global memory to array
            cutilSafeCall( hipMemcpyToArray( d_tempArray, 0, 0, d_dest, width * height * sizeof(float), hipMemcpyDeviceToDevice));
            cutilSafeCall( hipBindTextureToArray(tex, d_tempArray) );
        }
    }
}

// RGBA version
extern "C" 
void boxFilterRGBA(uint *d_src, uint *d_temp, uint *d_dest, int width, int height, 
                   int radius, int iterations, int nthreads)
{
    cutilSafeCall( hipBindTextureToArray(rgbaTex, d_array) );

    for(int i=0; i<iterations; i++) {
        // use texture for horizontal pass
        d_boxfilter_rgba_x<<< height / nthreads, nthreads, 0 >>>( d_temp, width, height, radius);
        d_boxfilter_rgba_y<<< width / nthreads, nthreads, 0 >>>( d_temp, d_dest, width, height, radius);
        if (iterations > 1) {
            // copy result back from global memory to array
            cutilSafeCall( hipMemcpyToArray( d_tempArray, 0, 0, d_dest, width * height * sizeof(float), hipMemcpyDeviceToDevice));
            cutilSafeCall( hipBindTextureToArray(rgbaTex, d_tempArray) );
        }
    }
}


#endif // #ifndef _BOXFILTER_KERNEL_H_

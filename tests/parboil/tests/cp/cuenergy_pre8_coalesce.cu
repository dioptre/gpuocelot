#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>

#include "cuenergy.h"

#if UNROLLX != 8
# error "UNROLLX must be 8"
#endif

#if BLOCKSIZEX != 16
# error "BLOCKSIZEX must be 16"
#endif

// Max constant buffer size is 64KB, minus whatever
// the CUDA runtime and compiler are using that we don't know about.
// At 16 bytes for atom, for this program 4070 atoms is about the max
// we can store in the constant buffer.
__constant__ float4 atominfo[MAXATOMS];

// This kernel calculates coulombic potential at each grid point and
// stores the results in the output array.

__global__ void cenergy(int numatoms, float gridspacing, float * energygrid) {
  unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) * UNROLLX
                         + threadIdx.x;
  unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int outaddr = (__umul24(gridDim.x, blockDim.x) * UNROLLX) * yindex
                         + xindex;

  float coory = gridspacing * yindex;
  float coorx = gridspacing * xindex;

  float energyvalx1=0.0f;
  float energyvalx2=0.0f;
  float energyvalx3=0.0f;
  float energyvalx4=0.0f;
  float energyvalx5=0.0f;
  float energyvalx6=0.0f;
  float energyvalx7=0.0f;
  float energyvalx8=0.0f;

  float gridspacing_u = gridspacing * BLOCKSIZEX;

  int atomid;
  for (atomid=0; atomid<numatoms; atomid++) {
    float dy = coory - atominfo[atomid].y;
    float dyz2 = (dy * dy) + atominfo[atomid].z;

    float dx1 = coorx - atominfo[atomid].x;
    float dx2 = dx1 + gridspacing_u;
    float dx3 = dx2 + gridspacing_u;
    float dx4 = dx3 + gridspacing_u;
    float dx5 = dx4 + gridspacing_u;
    float dx6 = dx5 + gridspacing_u;
    float dx7 = dx6 + gridspacing_u;
    float dx8 = dx7 + gridspacing_u;

    energyvalx1 += atominfo[atomid].w * (1.0f / sqrtf(dx1*dx1 + dyz2));
    energyvalx2 += atominfo[atomid].w * (1.0f / sqrtf(dx2*dx2 + dyz2));
    energyvalx3 += atominfo[atomid].w * (1.0f / sqrtf(dx3*dx3 + dyz2));
    energyvalx4 += atominfo[atomid].w * (1.0f / sqrtf(dx4*dx4 + dyz2));
    energyvalx5 += atominfo[atomid].w * (1.0f / sqrtf(dx5*dx5 + dyz2));
    energyvalx6 += atominfo[atomid].w * (1.0f / sqrtf(dx6*dx6 + dyz2));
    energyvalx7 += atominfo[atomid].w * (1.0f / sqrtf(dx7*dx7 + dyz2));
    energyvalx8 += atominfo[atomid].w * (1.0f / sqrtf(dx8*dx8 + dyz2));
  }

  energygrid[outaddr]   += energyvalx1;
  energygrid[outaddr+1*BLOCKSIZEX] += energyvalx2;
  energygrid[outaddr+2*BLOCKSIZEX] += energyvalx3;
  energygrid[outaddr+3*BLOCKSIZEX] += energyvalx4;
  energygrid[outaddr+4*BLOCKSIZEX] += energyvalx5;
  energygrid[outaddr+5*BLOCKSIZEX] += energyvalx6;
  energygrid[outaddr+6*BLOCKSIZEX] += energyvalx7;
  energygrid[outaddr+7*BLOCKSIZEX] += energyvalx8;
}

// This function copies atoms from the CPU to the GPU and
// precalculates (z^2) for each atom.

int copyatomstoconstbuf(float *atoms, int count, float zplane) {
  if (count > MAXATOMS) {
    printf("Atom count exceeds constant buffer storage capacity\n");
    return -1;
  }

  float atompre[4*MAXATOMS];
  int i;
  for (i=0; i<count*4; i+=4) {
    atompre[i    ] = atoms[i    ];
    atompre[i + 1] = atoms[i + 1];
    float dz = zplane - atoms[i + 2];
    atompre[i + 2]  = dz*dz;
    atompre[i + 3] = atoms[i + 3];
  }

  hipMemcpyToSymbol(HIP_SYMBOL(atominfo), atompre, count * 4 * sizeof(float), 0);
  CUERR // check and clear any existing errors

  return 0;
}


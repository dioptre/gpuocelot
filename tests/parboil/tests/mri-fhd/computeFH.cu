/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/


#include <hip/hip_runtime.h>
#include <cstdio>

#define PI   3.1415926535897932384626433832795029
#define PIx2 6.2831853071795864769252867665590058

/* Adjustable parameters */
#define KERNEL_RHO_PHI_THREADS_PER_BLOCK 512
#define KERNEL_FH_THREADS_PER_BLOCK 256
#define KERNEL_FH_K_ELEMS_PER_GRID 512

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))

#define CUDA_ERRCK							\
  {hipError_t err;							\
    if ((err = hipGetLastError()) != hipSuccess) {			\
      fprintf(stderr, "CUDA error on line %d: %s\n", __LINE__, hipGetErrorString(err)); \
      exit(-1);								\
    }									\
  }

struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float RhoPhiR;
  float RhoPhiI;
};

__constant__ __device__ kValues c[KERNEL_FH_K_ELEMS_PER_GRID];

void createDataStructs(int numK, int numX, 
                       float*& realRhoPhi, float*& imagRhoPhi, 
                       float*& outR, float*& outI)
{
  realRhoPhi = (float* ) calloc(numK, sizeof(float));
  imagRhoPhi = (float* ) calloc(numK, sizeof(float));
  outR = (float*) calloc (numX, sizeof (float));
  outI = (float*) calloc (numX, sizeof (float));
}

__global__ void
ComputeRhoPhiGPU(int numK,
                 float* phiR, float* phiI, 
                 float* dR, float* dI, 
                 float* realRhoPhi, float* imagRhoPhi)
{
  int indexK = blockIdx.x*KERNEL_RHO_PHI_THREADS_PER_BLOCK + threadIdx.x;
  if (indexK < numK) {
    float rPhiR = phiR[indexK];
    float rPhiI = phiI[indexK];
    float rDR = dR[indexK];
    float rDI = dI[indexK];
    realRhoPhi[indexK] = rPhiR * rDR + rPhiI * rDI;
    imagRhoPhi[indexK] = rPhiR * rDI - rPhiI * rDR;
  }
}

__global__ void
ComputeFH_GPU(int numK, int kGlobalIndex,
              float* x, float* y, float* z, 
              float* outR, float* outI)
{
  float sX;
  float sY;
  float sZ;
  float sOutR;
  float sOutI;

  // Determine the element of the X arrays computed by this thread
  int xIndex = blockIdx.x*KERNEL_FH_THREADS_PER_BLOCK + threadIdx.x;

  sX = x[xIndex];
  sY = y[xIndex];
  sZ = z[xIndex];
  sOutR = outR[xIndex];
  sOutI = outI[xIndex];

  // Loop over all elements of K in constant mem to compute a partial value
  // for X.
  int kIndex = 0;
  int kCnt = numK - kGlobalIndex;
  if (kCnt < KERNEL_FH_K_ELEMS_PER_GRID) {
    for (kIndex = 0;
	 (kIndex < (kCnt % 4)) && (kGlobalIndex < numK);
	 kIndex++, kGlobalIndex++) {
      float expArg = PIx2 *
	(c[kIndex].Kx * sX + c[kIndex].Ky * sY + c[kIndex].Kz * sZ);
      float cosArg = cos(expArg);
      float sinArg = sin(expArg);
      sOutR += c[kIndex].RhoPhiR * cosArg - c[kIndex].RhoPhiI * sinArg;
      sOutI += c[kIndex].RhoPhiI * cosArg + c[kIndex].RhoPhiR * sinArg;
    }
  }

  for (;
       (kIndex < KERNEL_FH_K_ELEMS_PER_GRID) && (kGlobalIndex < numK);
       kIndex += 4, kGlobalIndex += 4) {
    float expArg = PIx2 *
      (c[kIndex].Kx * sX + c[kIndex].Ky * sY + c[kIndex].Kz * sZ);
    float cosArg = cos(expArg);
    float sinArg = sin(expArg);
    sOutR += c[kIndex].RhoPhiR * cosArg - c[kIndex].RhoPhiI * sinArg;
    sOutI += c[kIndex].RhoPhiI * cosArg + c[kIndex].RhoPhiR * sinArg;
    
    int kIndex1 = kIndex + 1;
    float expArg1 = PIx2 *
      (c[kIndex1].Kx * sX + c[kIndex1].Ky * sY + c[kIndex1].Kz * sZ);
    float cosArg1 = cos(expArg1);
    float sinArg1 = sin(expArg1);
    sOutR += c[kIndex1].RhoPhiR * cosArg1 - c[kIndex1].RhoPhiI * sinArg1;
    sOutI += c[kIndex1].RhoPhiI * cosArg1 + c[kIndex1].RhoPhiR * sinArg1;

    int kIndex2 = kIndex + 2;
    float expArg2 = PIx2 *
      (c[kIndex2].Kx * sX + c[kIndex2].Ky * sY + c[kIndex2].Kz * sZ);
    float cosArg2 = cos(expArg2);
    float sinArg2 = sin(expArg2);
    sOutR += c[kIndex2].RhoPhiR * cosArg2 - c[kIndex2].RhoPhiI * sinArg2;
    sOutI += c[kIndex2].RhoPhiI * cosArg2 + c[kIndex2].RhoPhiR * sinArg2;

    int kIndex3 = kIndex + 3;
    float expArg3 = PIx2 *
      (c[kIndex3].Kx * sX + c[kIndex3].Ky * sY + c[kIndex3].Kz * sZ);
    float cosArg3 = cos(expArg3);
    float sinArg3 = sin(expArg3);
    sOutR += c[kIndex3].RhoPhiR * cosArg3 - c[kIndex3].RhoPhiI * sinArg3;
    sOutI += c[kIndex3].RhoPhiI * cosArg3 + c[kIndex3].RhoPhiR * sinArg3;    
  }

  outR[xIndex] = sOutR;
  outI[xIndex] = sOutI;
}

void computeRhoPhi_GPU(int numK, 
                       float* phiR_d, float* phiI_d, float* dR_d, float* dI_d,
                       float* realRhoPhi_d, float* imagRhoPhi_d)
{
  int rhoPhiBlocks = numK / KERNEL_RHO_PHI_THREADS_PER_BLOCK;

  if (numK % KERNEL_RHO_PHI_THREADS_PER_BLOCK)
    rhoPhiBlocks++;

  dim3 DimRhoPhiBlock(KERNEL_RHO_PHI_THREADS_PER_BLOCK, 1);
  dim3 DimRhoPhiGrid(rhoPhiBlocks, 1);
  printf("Launch RhoPhi Kernel on GPU: Blocks (%d, %d), Threads Per Block %d\n",
         rhoPhiBlocks, 1, KERNEL_RHO_PHI_THREADS_PER_BLOCK);

  ComputeRhoPhiGPU <<< DimRhoPhiGrid, DimRhoPhiBlock >>> 
      (numK, phiR_d, phiI_d, dR_d, dI_d, realRhoPhi_d, imagRhoPhi_d);
}

void computeFH_GPU(int numK, int numX, 
                   float* x_d, float* y_d, float* z_d,
                   kValues* kVals,
                   float* outR_d, float* outI_d)
{
  int FHGrids = numK / KERNEL_FH_K_ELEMS_PER_GRID;
  if (numK % KERNEL_FH_K_ELEMS_PER_GRID)
    FHGrids++;
  int FHBlocks = numX / KERNEL_FH_THREADS_PER_BLOCK;
  if (numX % KERNEL_FH_THREADS_PER_BLOCK)
    FHBlocks++;
  dim3 DimFHBlock(KERNEL_FH_THREADS_PER_BLOCK, 1);
  dim3 DimFHGrid(FHBlocks, 1);

  printf("Launch GPU Kernel: Grids %d, Blocks Per Grid (%d, %d), Threads Per Block (%d, %d), K Elems Per Thread %d\n",
         FHGrids, DimFHGrid.x, DimFHGrid.y, DimFHBlock.x, DimFHBlock.y, KERNEL_FH_K_ELEMS_PER_GRID);

  for (int FHGrid = 0; FHGrid < FHGrids; FHGrid++) {
    // Put the tile of K values into constant mem
    int FHGridBase = FHGrid * KERNEL_FH_K_ELEMS_PER_GRID;

    kValues* kValsTile = kVals + FHGridBase;
    int numElems = MIN(KERNEL_FH_K_ELEMS_PER_GRID, numK - FHGridBase);
	printf("Copying %d bytes to constant memory\n", numElems * sizeof(kValues));
    hipMemcpyToSymbol(HIP_SYMBOL(c), kValsTile, numElems * sizeof(kValues), 0);
    CUDA_ERRCK;

    ComputeFH_GPU <<< DimFHGrid, DimFHBlock >>>
      (numK, FHGridBase, x_d, y_d, z_d, outR_d, outI_d);
    CUDA_ERRCK;
  }
}

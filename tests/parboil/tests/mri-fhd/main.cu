/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
 * CUDA code for creating the FHD data structure for fast convolution-based 
 * Hessian multiplication for arbitrary k-space trajectories.
 * 
 * recommended g++ options:
 *   -O3 -lm -ffast-math -funroll-all-loops
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis 
 *     function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>

#include <parboil.h>

#include "file.h"
#include "computeFH.cu"

static void
setupMemoryGPU(int num, int size, float*& dev_ptr, float*& host_ptr)
{
  hipMalloc ((void **) &dev_ptr, num * size);
  CUDA_ERRCK;
  hipMemcpy (dev_ptr, host_ptr, num * size, hipMemcpyHostToDevice);
  CUDA_ERRCK;
}

static void
cleanupMemoryGPU(int num, int size, float *& dev_ptr, float * host_ptr)
{
  hipMemcpy (host_ptr, dev_ptr, num * size, hipMemcpyDeviceToHost);
  CUDA_ERRCK;
  hipFree(dev_ptr);
  CUDA_ERRCK;
}

int
main (int argc, char *argv[])
{
  int numX, numK;		/* Number of X and K values */
  int original_numK;		/* Number of K values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */
  float *phiR, *phiI;		/* Phi values (complex) */
  float *dR, *dI;		/* D values (complex) */
  float *realRhoPhi, *imagRhoPhi;  /* RhoPhi values (complex) */
  float *outI, *outR;		/* Output signal (complex) */
  kValues* kVals;		/* Copy of X and RhoPhi.  Its
				 * data layout has better cache
				 * performance. */

  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);

  /* Read command line */
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
    }

  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData(params->inpFiles[0],
	    &original_numK, &numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI,
	    &dR, &dI);

  /* Reduce the number of k-space samples if a number is given
   * on the command line */
  if (argc < 2)
    numK = original_numK;
  else
    {
      int inputK;
      char *end;
      inputK = strtol(argv[1], &end, 10);
      if (end == argv[1])
	{
	  fprintf(stderr, "Expecting an integer parameter\n");
	  exit(-1);
	}

      numK = MIN(inputK, original_numK);
    }

  printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  /* Create CPU data structures */
  createDataStructs(numK, numX, realRhoPhi, imagRhoPhi, outR, outI);
  kVals = (kValues*)calloc(numK, sizeof (kValues));

  /* GPU section 1 (precompute Rho, Phi)*/
  {
    /* Mirror several data structures on the device */
    float *phiR_d, *phiI_d;
    float *dR_d, *dI_d;
    float *realRhoPhi_d, *imagRhoPhi_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    setupMemoryGPU(numK, sizeof(float), phiR_d, phiR);
    setupMemoryGPU(numK, sizeof(float), phiI_d, phiI);
    setupMemoryGPU(numK, sizeof(float), dR_d, dR);
    printf( "First dR %f, %x, %x\n", *dR, dR, dR_d );
    setupMemoryGPU(numK, sizeof(float), dI_d, dI);
    hipMalloc((void **)&realRhoPhi_d, numK * sizeof(float));
    CUDA_ERRCK;
    hipMalloc((void **)&imagRhoPhi_d, numK * sizeof(float));
    CUDA_ERRCK;

    if (params->synchronizeGpu) hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_GPU);

    /* Pre-compute the values of rhoPhi on the GPU */
    computeRhoPhi_GPU(numK, phiR_d, phiI_d, dR_d, dI_d, 
		      realRhoPhi_d, imagRhoPhi_d);

    if (params->synchronizeGpu) hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    cleanupMemoryGPU(numK, sizeof(float), realRhoPhi_d, realRhoPhi);
    cleanupMemoryGPU(numK, sizeof(float), imagRhoPhi_d, imagRhoPhi);
    hipFree(phiR_d);
    hipFree(phiI_d);
    hipFree(dR_d);
    hipFree(dI_d);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  /* Fill in kVals values */
  for (int k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].RhoPhiR = realRhoPhi[k];
    kVals[k].RhoPhiI = imagRhoPhi[k];
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COPY);

  /* GPU section 2 (compute FH)*/
  {
    float *x_d, *y_d, *z_d;
    float *outI_d, *outR_d;

    /* Mirror several data structures on the device */
    setupMemoryGPU(numX, sizeof(float), x_d, x);
    setupMemoryGPU(numX, sizeof(float), y_d, y);
    setupMemoryGPU(numX, sizeof(float), z_d, z);

    // Zero out initial values of outR and outI.
    // GPU veiws these arrays as initialized (cleared) accumulators.
    hipMalloc((void **)&outR_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset(outR_d, 0, numX * sizeof(float));
    CUDA_ERRCK;
    hipMalloc((void **)&outI_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset(outI_d, 0, numX * sizeof(float));
    CUDA_ERRCK;

    if (params->synchronizeGpu) hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_GPU);

    /* Compute FH on the GPU (main computation) */
    computeFH_GPU(numK, numX, x_d, y_d, z_d, kVals, outR_d, outI_d);

    if (params->synchronizeGpu) hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    /* Release memory on GPU */
    cleanupMemoryGPU(numX, sizeof(float), outR_d, outR);
    cleanupMemoryGPU(numX, sizeof(float), outI_d, outI);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  if (params->outFile)
    {
      /* Write result to file */
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      outputData(params->outFile, outR, outI, numX);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }

  if( compareData("data/mriFhdOut.bin", outR, outI, numX, 1) )
  {
    printf("TEST PASSED\n");
  }
  else
  {
  	printf("TEST FAILED\n");
  }

  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (dR);
  free (dI);
  free (realRhoPhi);
  free (imagRhoPhi);
  free (kVals);
  free (outR);
  free (outI);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);

  return 0;
}

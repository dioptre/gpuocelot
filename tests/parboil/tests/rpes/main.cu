#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#include <stdio.h>
#include <string.h>
#include <parboil.h>
#include "shell.h"

#include "crys_kernel.cu"

#define CUDA_ERRCK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  exit (-1); }}

uint4* d_Block_Work;
uint2* d_FinalReduce;
float *d_Output, *d_ReductionSum;
hipArray *d_Coors, *d_Sprms, *d_Wghts;

float4* Coors;
float2* Sprms;

uint4* Block_Work;

int totNumBlocks;
int MaxBlocks;

Atom *ComputeAtom, *BasisAtom;
Shell* ComputeShell;
float Coor[BLOCK_SIZE];
float Alpha[BLOCK_SIZE];
float Coeff[BLOCK_SIZE];
float Wghts[TABLESIZE];

float *ReductionSum;
int totReductionElements;

uint2 *FinalReduce;

void AllocateDataOnDevice(int, int, int, int, int, int);
void RunKernel(int, struct pb_TimerSet *, struct pb_Parameters *);

void CalcOnHost(int);
Atom* ReadBasisAtoms(int&, struct pb_Parameters *);
int TotalNumOfShells(char*, int, int&);
void PopulateShells(char*, int);
void PopulateHostData(int, int, int);
int NumOfIntegrals(int);
void DistributeBlockWork(int);
void FreeAllData(struct pb_TimerSet *);
void PopulateWeights();

double root1(double X);

int main(int argc, char* argv[])
{
  struct pb_TimerSet timers;
  struct pb_Parameters *params;

  pb_InitializeTimerSet( &timers );
  params = pb_ReadParameters( &argc, argv );

  totNumBlocks = 0;
  MaxBlocks = 0;

  int numBasisAtoms;
  pb_SwitchToTimer( &timers, pb_TimerID_IO );
  BasisAtom = ReadBasisAtoms(numBasisAtoms, params);
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  
  int totNumAtoms;
  int totNumShells = TotalNumOfShells(params->inpFiles[0], numBasisAtoms, 
  				      totNumAtoms);
  totReductionElements = totNumShells * (totNumShells + 1) * 
    (totNumShells + 2) * (totNumShells + 3) / 24;
  ComputeAtom = (Atom*)malloc(totNumAtoms * sizeof(Atom));
  ComputeShell = (Shell*)malloc(totNumShells * sizeof(Shell));

  pb_SwitchToTimer( &timers, pb_TimerID_IO );
  PopulateShells(params->inpFiles[0], numBasisAtoms);
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  //	all shells are ready now

  //	prepare host data
  int totBasisShells = 0;
  for(int i = 0; i < numBasisAtoms; i ++) 
    for(int j = 0; j < BasisAtom[i].numShells; j ++)
      totBasisShells += BasisAtom[i].AtomShell[j].numPrimitives;

  Coors = (float4*)malloc(totNumAtoms * sizeof(float4));
  Sprms = (float2*)malloc(totBasisShells * sizeof(float2));
  PopulateHostData(totNumAtoms, totNumShells, numBasisAtoms);

  //	distribute the work now
  FinalReduce = (uint2*)malloc(totReductionElements * sizeof(uint2));
  int numIntegrals = NumOfIntegrals(totNumShells);
  pb_SwitchToTimer( &timers, pb_TimerID_IO );
  printf("Total # of integrals to compute: %d\n", numIntegrals);
  printf("Total # of blocks allocated: %d\n", totNumBlocks);
  printf("Final array size: %d\n", totReductionElements);
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  Block_Work = (uint4*)malloc(totNumBlocks * sizeof(uint4));
  DistributeBlockWork(totNumShells);

  int d_output_mem = totNumBlocks * sizeof(float);
  int d_work_mem = totNumBlocks * sizeof(uint4);
  int reduction_mem = totReductionElements * sizeof(float);
  int final_mem = totReductionElements * sizeof(uint2);

  ReductionSum = (float*)malloc(reduction_mem);

  //	prepare device data
  pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  AllocateDataOnDevice(d_output_mem, d_work_mem, reduction_mem, 
		       final_mem, totNumAtoms, totBasisShells);
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  int d_total_mem = d_output_mem + d_work_mem + reduction_mem + final_mem;
  printf("%.2lf MB allocated\n", (double)d_total_mem / 1048576);
  printf("maxblocks = %d\n", MaxBlocks);

  //	okay, now ready to do something useful
  RunKernel(numIntegrals, &timers, params);

  //	loading data back to the host
  pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  hipMemcpy(ReductionSum, d_ReductionSum, 
	     reduction_mem, hipMemcpyDeviceToHost);
  CUDA_ERRCK
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  FreeAllData( &timers );
  if(params->outFile)
    {
      pb_SwitchToTimer( &timers, pb_TimerID_IO );
      FILE *file = fopen(params->outFile, "w");
      printf("totReductionElements %d\n", totReductionElements);
      // only printing the first 20000 
      for(int i = 0; i < 20000; i ++)
        {
          fprintf(file, "%d\t%e\n", i, ReductionSum[i]);
        }
      fclose(file);
    }
    
  if( pb_compareFiles(params->outFile, "data/rpes.out", 1) )
  {
    printf("TEST PASSED\n");
  }
  else
  {
  	printf("TEST FAILED\n");
  }

  pb_SwitchToTimer( &timers, pb_TimerID_NONE );
  pb_PrintTimerSet( &timers );
  pb_FreeParameters( params );

  return 0;
}

void RunKernel(int numIntegrals, struct pb_TimerSet *timers,
               struct pb_Parameters *params)
{
  int runs = (int)(ceil(1.0 * totNumBlocks / GRID_SIZE));
  printf("%d computation cycles will be performed...\n", runs);
  int RemainingBlocks = totNumBlocks;
  int StartBlock = 0;

  for(int run = 0; run < runs; run ++)
    {
      int numBlocks = min(GRID_SIZE, RemainingBlocks);
      dim3 grid(numBlocks, 1, 1);
      dim3 block(BLOCK_SIZE, 1, 1);

      pb_SwitchToTimer( timers, pb_TimerID_GPU );
      ComputeX <<< grid, block >>> (d_Block_Work, d_Output, StartBlock);
      CUDA_ERRCK
      if (params->synchronizeGpu) hipDeviceSynchronize();
      pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );

      RemainingBlocks -= GRID_SIZE;
      StartBlock += numBlocks;
    }

  runs = (int)(ceil(1.0 * totReductionElements / GRID_SIZE));
  printf("done.\n\n%d reduction cycles will be performed...\n", runs);
  int RemainReduction = totReductionElements;
  int Offset = 0;
  for(int run = 0; run < runs; run ++)
    {
      int numBlocks = min(GRID_SIZE, RemainReduction);
      dim3 grid(numBlocks, 1, 1);
      dim3 block(MaxBlocks, 1, 1);

      pb_SwitchToTimer( timers, pb_TimerID_GPU );
      DoReduction <<< grid, block >>> (d_ReductionSum, d_Output, 
				       d_FinalReduce, MaxBlocks, Offset);
      CUDA_ERRCK
      if (params->synchronizeGpu) hipDeviceSynchronize();
      pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );

      RemainReduction -= GRID_SIZE;
      Offset += numBlocks;
    }
}

void AllocateDataOnDevice(int d_output_mem, int d_work_mem, 
			  int reduction_mem, int final_mem,
			  int numCoors, int numSprms)
{
  hipMalloc((void**)&d_ReductionSum, reduction_mem);
  CUDA_ERRCK
  hipMalloc((void**)&d_Output, d_output_mem);
  CUDA_ERRCK
  hipMalloc((void**)&d_Block_Work, d_work_mem);
  CUDA_ERRCK
  hipMalloc((void**)&d_FinalReduce, final_mem);
  CUDA_ERRCK
  hipMallocArray(&d_Coors, &texCoors.channelDesc, 
		  numCoors, 1);
  CUDA_ERRCK
  hipMallocArray(&d_Sprms, &texSprms.channelDesc, 
		  numSprms, 1);
  CUDA_ERRCK
  hipMallocArray(&d_Wghts, &texWghts.channelDesc, 
		  1 << LOG_TABLE_WIDTH, 2);
  CUDA_ERRCK
  
  hipMemcpy(d_Block_Work, Block_Work, d_work_mem, 
	     hipMemcpyHostToDevice);
  CUDA_ERRCK
  hipMemcpy(d_FinalReduce, FinalReduce, final_mem, 
	     hipMemcpyHostToDevice);
  CUDA_ERRCK

  hipMemcpyToArray(d_Coors, 0, 0, (void*)Coors, 
	            numCoors * sizeof(float4), 
		    hipMemcpyHostToDevice);
  CUDA_ERRCK
  hipMemcpyToArray(d_Sprms, 0, 0, (void*)Sprms, 
		    numSprms * sizeof(float2), 
		    hipMemcpyHostToDevice);
  CUDA_ERRCK
  hipMemcpyToArray(d_Wghts, 0, 0, (void*)Wghts, 
		    TABLESIZE * sizeof(float), 
		    hipMemcpyHostToDevice);
  CUDA_ERRCK

  texWghts.filterMode = hipFilterModeLinear;
  
  hipBindTextureToArray(texCoors, d_Coors, texCoors.channelDesc);
  CUDA_ERRCK
  hipBindTextureToArray(texSprms, d_Sprms, texSprms.channelDesc);
  CUDA_ERRCK
  hipBindTextureToArray(texWghts, d_Wghts, texWghts.channelDesc);
  CUDA_ERRCK
}

void FreeAllData( struct pb_TimerSet *timers )
{
  pb_SwitchToTimer( timers, pb_TimerID_COPY );
  hipFree((void*)d_FinalReduce);
  CUDA_ERRCK
  hipFree((void*)d_Block_Work);
  CUDA_ERRCK
  hipFree((void*)d_Output);
  CUDA_ERRCK
  hipFree((void*)d_ReductionSum);
  CUDA_ERRCK
  hipFreeArray(d_Coors);
  CUDA_ERRCK
  hipFreeArray(d_Wghts);
  CUDA_ERRCK
  hipFreeArray(d_Sprms);
  CUDA_ERRCK
  pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );
  
  free ((void*)Block_Work);
  free ((void*)FinalReduce);
  
  free ((void*)ComputeAtom);
  free ((void*)BasisAtom);
  free ((void*)ComputeShell);
}

Atom* ReadBasisAtoms(int& numBasisAtoms, struct pb_Parameters *params)
{
  FILE* basis = fopen(params->inpFiles[1], "r");
  if(!basis)
    {
      printf("Unable to open file %s\n", params->inpFiles[1]);
      exit(0);
    }
  int numAtoms = 0, numShells = 0;
  fscanf(basis, "%*s %*s %d", &numAtoms);
  fscanf(basis, "%*s %*s %d", &numShells);
  printf("\n>>>>>>> STARTED BASIS SET OUTPUT <<<<<<<\n");
  printf("\n# OF KNOWN ATOMS:  %d\n", numAtoms);
  printf("# OF KNOWN SHELLS: %d\n\n", numShells);
  numBasisAtoms = numAtoms;
  
  Atom* BasisAtom = (Atom*)malloc(numAtoms * sizeof(Atom));
  
  for(int atom = 0; atom < numAtoms; atom ++)
    {
      char type[4];
      char buff[4];
      fscanf(basis, "%*s %s", type);
      fscanf(basis, "%*s %d", &numShells);
      BasisAtom[atom].numShells = numShells;
      strcpy(BasisAtom[atom].Type, type);
      printf("\nAtom %s (%d shells)\n", BasisAtom[atom].Type, 
	     BasisAtom[atom].numShells);
      
      for(int shell = 0; shell < numShells; shell ++)
	{
	  int numPrimitives = 0;
	  fscanf(basis, "%*s %*d %*s %d", &numPrimitives);
	  BasisAtom[atom].AtomShell[shell].numPrimitives = numPrimitives;
	  sprintf(buff, "%d", shell + 1);
	  strcpy(BasisAtom[atom].AtomShell[shell].Type    , type);
	  strcpy(BasisAtom[atom].AtomShell[shell].Type + 1, buff);
	  printf("\tShell %s: %d primitives\n", 
		 BasisAtom[atom].AtomShell[shell].Type,
		 BasisAtom[atom].AtomShell[shell].numPrimitives);
	  for(int prim = 0; prim < numPrimitives; prim ++)
	    {
	      fscanf(basis, "%*s %*s %*s %f %f", 
		     &BasisAtom[atom].AtomShell[shell].Alpha[prim], 
		     &BasisAtom[atom].AtomShell[shell].Coeff[prim]);
	      printf("\t\tprimitive %d: %10.2f    %5.2f\n", prim + 1, 
		     BasisAtom[atom].AtomShell[shell].Alpha[prim], 
		     BasisAtom[atom].AtomShell[shell].Coeff[prim]);
	    }
	  printf("\n");
	}
    }
  printf(">>>>>>>> DONE BASIS SET OUTPUT <<<<<<<<\n\n\n");
  fclose(basis);
  return BasisAtom;
}

int TotalNumOfShells(char* fname, int numBasisAtoms, int& totNumAtoms)
{
  FILE* inp = fopen(fname, "r");
  if(!inp)
    {
      printf("Unable to open %s\n", fname);
      exit(0);
    }
  int numShells = 0;
  fscanf(inp, "%*s %d", &totNumAtoms);
  
  for(int atom = 0; atom < totNumAtoms; atom ++)
    {
      char type[8];
      fscanf(inp, "%s %*s %*s %*s", type);
      
      int notfound = 1;
      for(int batom = 0; batom < numBasisAtoms; batom ++)
	{
	  if(!strcmp(BasisAtom[batom].Type, type))
	    {
	      numShells += BasisAtom[batom].numShells;
	      notfound = 0;
	      break;
	    }
	}
      if(notfound)
	{
	  printf("Unable to find atom \'%s\' in the basis set\n", type);
	  exit(0);
	}
    }

  fclose(inp);
  return numShells;
}

void PopulateShells(char* fname, int numBasisAtoms)
{
  FILE* inp = fopen(fname, "r");
  if(!inp)
    {
      printf("Unable to open %s\n", fname);
      exit(0);
    }
  int numAtoms = 0, currentShell = 0;
  fscanf(inp, "%*s %d", &numAtoms);
  
  for(int atom = 0; atom < numAtoms; atom ++)
    {
      fscanf(inp, "%s %f %f %f", &ComputeAtom[atom].Type, 
	     &ComputeAtom[atom].X,
	     &ComputeAtom[atom].Y, &ComputeAtom[atom].Z);
      
      int currentInList = 0;
      for(int batom = 0; batom < numBasisAtoms; batom ++)
	{
	  if(!strcmp(BasisAtom[batom].Type, ComputeAtom[atom].Type))
	    {
	      for(int shell = 0; shell < BasisAtom[batom].numShells; 
		  shell ++)
		{
		  ComputeShell[currentShell] = 
		    BasisAtom[batom].AtomShell[shell];
		  ComputeShell[currentShell].myAtom = atom;
		  
		  //	this part populates inList
		  for(int prim = 0; prim < 
			BasisAtom[batom].AtomShell[shell].numPrimitives; 
		      prim ++)
		    ComputeShell[currentShell].inList[prim] = 
		      currentInList ++;

		  currentShell ++;
		}
	      break;
	    }
	  //	this part populates inList
	  else
	    {
	      for(int shell = 0; shell < BasisAtom[batom].numShells; 
		  shell ++)
		currentInList += 
		  BasisAtom[batom].AtomShell[shell].numPrimitives;
	    }
	}
    }
  fclose(inp);
}

void PopulateHostData(int totNumAtoms, int totNumShells, int numBasisAtoms)
{
  PopulateWeights();
  for(int atom = 0; atom < totNumAtoms; atom ++)
    {
      Coors[atom].x = ComputeAtom[atom].X;
      Coors[atom].y = ComputeAtom[atom].Y;
      Coors[atom].z = ComputeAtom[atom].Z;
    }

  int currentPos = 0;
  for(int batom = 0; batom < numBasisAtoms; batom ++)
    {
      for(int shell = 0; shell < BasisAtom[batom].numShells; shell ++)
	{
	  for(int prim = 0; prim < 
		BasisAtom[batom].AtomShell[shell].numPrimitives; 
	      prim ++)
	    {
	      Sprms[currentPos].x = 
		BasisAtom[batom].AtomShell[shell].Alpha[prim];
	      Sprms[currentPos].y = 
		BasisAtom[batom].AtomShell[shell].Coeff[prim];
	      currentPos ++;
	    }
	}
    }
}

int NumOfIntegrals(int totNumShells)
{
  int numIntegrals = 0;
  int firstRedElement = 0;
  int redElement = 0;
  for(int shell1 = 0; shell1 < totNumShells; shell1 ++)
    for(int shell2 = shell1; shell2 < totNumShells; shell2 ++)
      for(int shell3 = shell2; shell3 < totNumShells; shell3 ++)
	for(int shell4 = shell3; shell4 < totNumShells; shell4 ++)
	  {
	    int integrals = ComputeShell[shell1].numPrimitives * 
	      ComputeShell[shell2].numPrimitives * 
	      ComputeShell[shell3].numPrimitives * 
	      ComputeShell[shell4].numPrimitives;
	    numIntegrals += integrals;
	    
	    int blocks = (int)ceil(1.0 * integrals / BLOCK_SIZE);
	    totNumBlocks += blocks;
	    if(blocks > MaxBlocks)
	      MaxBlocks = blocks;
	    FinalReduce[redElement].x = firstRedElement;
	    FinalReduce[redElement].y = blocks;
	    firstRedElement += blocks;
	    redElement ++;
	  }
  return numIntegrals;
}

void DistributeBlockWork(int totNumShells)
{
  int numElements = 0;
  int StartBlock = 0;
  for(int shell1 = 0; shell1 < totNumShells; shell1 ++)
    {
      for(int shell2 = shell1; shell2 < totNumShells; shell2 ++)
	{
	  for(int shell3 = shell2; shell3 < totNumShells; shell3 ++)
	    {
	      for(int shell4 = shell3; shell4 < totNumShells; shell4 ++)
		{
		  int integrals = ComputeShell[shell1].numPrimitives * 
		    ComputeShell[shell2].numPrimitives * 
		    ComputeShell[shell3].numPrimitives * 
		    ComputeShell[shell4].numPrimitives;

		  int blocks = (int)ceil(1.0 * integrals / BLOCK_SIZE);
		  StartBlock = numElements;
		  
		  for(int block = 0; block < blocks; block ++)
		    {
		      int a4 = ComputeShell[shell4].numPrimitives;
		      int a3 = ComputeShell[shell3].numPrimitives;
		      int a2 = ComputeShell[shell2].numPrimitives;
		      int a1 = ComputeShell[shell1].numPrimitives;
		      
		      int offset4 = ComputeShell[shell4].inList[0];
		      int offset3 = ComputeShell[shell3].inList[0];
		      int offset2 = ComputeShell[shell2].inList[0];
		      int offset1 = ComputeShell[shell1].inList[0];
		      
		      Block_Work[numElements].y = 
			(ComputeShell[shell1].myAtom << 24) | 
			(ComputeShell[shell2].myAtom << 16) | 
			(ComputeShell[shell3].myAtom << 8 ) | 
			(ComputeShell[shell4].myAtom      ) ;
		      
		      Block_Work[numElements].z = 
			(offset1 << 24) | 
			(offset2 << 16) | 
			(offset3 << 8 ) | 
			(offset4) ;
		      
		      Block_Work[numElements].x = 
			(a1         << 12) |
			(a2         <<  8) |
			(a3         <<  4) |
			(a4              ) ;
		      
		      Block_Work[numElements].w = StartBlock;
		      
		      numElements ++;
		    }
		}
	    }
	}
    }
}

double root1(double X)
{
  double PIE4;
  double WW1 = 0.0;
  double F1,E,Y,inv;
  
  PIE4 = 7.85398163397448E-01;
  
  if (X < 3.0e-7)
    {
      WW1 = 1.0 - 0.333333333 * X;
    } 
  else if (X < 1.0) 
    {
      F1 = ((((((((-8.36313918003957E-08*X+1.21222603512827E-06 )*X-
		  1.15662609053481E-05 )*X+9.25197374512647E-05 )*X-
		6.40994113129432E-04 )*X+3.78787044215009E-03 )*X-
	      1.85185172458485E-02 )*X+7.14285713298222E-02 )*X-
	    1.99999999997023E-01 )*X+3.33333333333318E-01;
      WW1 = (X+X)*F1 + exp(-X);
    } 
  else if (X < 3.0) 
    {
      Y = X-2.0;
      F1 = ((((((((((-1.61702782425558E-10*Y+1.96215250865776E-09 )*Y-
		    2.14234468198419E-08 )*Y+2.17216556336318E-07 )*Y-
		  1.98850171329371E-06 )*Y+1.62429321438911E-05 )*Y-
		1.16740298039895E-04 )*Y+7.24888732052332E-04 )*Y-
	      3.79490003707156E-03 )*Y+1.61723488664661E-02 )*Y-
	    5.29428148329736E-02 )*Y+1.15702180856167E-01;
      WW1 = (X+X)*F1+exp(-X);
      
    } 
  else if (X < 5.0)
    {
      Y = X-4.0;
      F1 = ((((((((((-2.62453564772299E-11*Y+3.24031041623823E-10 )*Y-
		    3.614965656163E-09)*Y+3.760256799971E-08)*Y-
		  3.553558319675E-07)*Y+3.022556449731E-06)*Y-
		2.290098979647E-05)*Y+1.526537461148E-04)*Y-
	      8.81947375894379E-04)*Y+4.33207949514611E-03 )*Y-
	    1.75257821619926E-02 )*Y+5.28406320615584E-02;
      WW1 = (X+X)*F1+exp(-X);
      
    } 
  else if (X < 10.0) 
    {
      E = exp(-X);
      inv = 1 / X;
      WW1 = (((((( 4.6897511375022E-01*inv-6.9955602298985E-01)*inv +
		 5.3689283271887E-01)*inv-3.2883030418398E-01)*inv +
	       2.4645596956002E-01)*inv-4.9984072848436E-01)*inv -
	     3.1501078774085E-06)*E + sqrt(PIE4*inv);
      
    } 
  else if (X < 15.0) 
    {
      E = exp(-X);
      inv = 1 / X;
      WW1 = (((-1.8784686463512E-01*inv+2.2991849164985E-01)*inv -
	      4.9893752514047E-01)*inv-2.1916512131607E-05)*E \
	+ sqrt(PIE4*inv);
      
    } 
  else if (X < 33.0) 
    {
      E = exp(-X);
      inv = 1 / X;
      WW1 = (( 1.9623264149430E-01*inv-4.9695241464490E-01)*inv -
	     6.0156581186481E-05)*E + sqrt(PIE4*inv);

    } 
  else 
    {
      inv = 1 / X;
      WW1 = sqrt(PIE4*inv);
    }
  
  return WW1;
}

void PopulateWeights()
{
  for(int i = 0; i < TABLESIZE; i ++)
    {
      float X = (float)(1.0 * i * W_MAX_SIZE / (TABLESIZE - 1));
      Wghts[i] = root1(X);
    }
}

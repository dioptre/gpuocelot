#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#define BLOCK_SIZE 64
#define LOG_BLOCK_SIZE 6
#define GRID_SIZE 65535
#define TABLESIZE 2
#define LOG_TABLE_WIDTH 2
#define TABLE_HEIGHT (TABLESIZE >> LOG_TABLE_WIDTH)
#define W_MAX_SIZE 10

__global__ void ComputeX(uint4*, float*, int);

__device__ float product1D(float, float, float, float, float);
__device__ float dist2(float3, float3);
__device__ float dist2(float4, float4);
__device__ float3 ReducePair(float4, float4, float2, float2, float);
__device__ float ComputeI(float, float, float, float, 
			  float, float, float, float);
__device__ float Root(float X);

__device__ void GetBases(int, int, int, int&, int&, int&, int&, int&);
__device__ void GetAtoms(int, int&, int&, int&, int&);
__device__ void GetOffsets(int, int&, int&, int&, int&);
__device__ void GetNs(int, int, int, int, int, int&, int&, int&, int&);
__device__ void KahanSum(float&, float, float&, float&, float&);

texture<float4, 1, hipReadModeElementType> texCoors;
texture<float2, 1, hipReadModeElementType> texSprms;
texture<float , 1, hipReadModeElementType> texWghts;

__global__ 
void ComputeX(uint4* d_Work, float* d_Output, int Offset)
{
  __shared__ float Data[BLOCK_SIZE];
  __shared__ uint4 s_Work;
  
  int blid = blockIdx.x;
  int thid = threadIdx.x;
  
  int myWorkIndex = ((Offset + blid) << LOG_BLOCK_SIZE) + thid;
  if(!thid)
    s_Work = d_Work[Offset + blid];
  __syncthreads();
  uint4 myWork = s_Work;
  
  int localthid, a1, a2, a3, a4;
  GetBases(myWork.x, myWorkIndex, myWork.w,
	   localthid, a1, a2, a3, a4);
  
  float Result = 0.0f;
  int Maxthid = __mul24(a1, __mul24(a2, __mul24(a3, a4)));
  if(localthid < Maxthid)
    {
      int n1, n2, n3, n4;
      int off1, off2, off3, off4;
      int atom1, atom2, atom3, atom4;
      
      GetAtoms(myWork.y, atom1, atom2, atom3, atom4);
      GetNs(localthid, a1, a2, a3, a4, n1, n2, n3, n4);
      GetOffsets(myWork.z, off1, off2, off3, off4);
      n1 += off1;
      n2 += off2;
      n3 += off3;
      n4 += off4;
      
      float4 Atom1 = tex1D(texCoors, (float)atom1);
      float4 Atom2 = tex1D(texCoors, (float)atom2);
      float4 Atom3 = tex1D(texCoors, (float)atom3);
      float4 Atom4 = tex1D(texCoors, (float)atom4);
      float2 Param1 = tex1D(texSprms, (float)n1);
      float2 Param2 = tex1D(texSprms, (float)n2);
      float2 Param3 = tex1D(texSprms, (float)n3);
      float2 Param4 = tex1D(texSprms, (float)n4);
      
      float R12 = dist2(Atom1, Atom2);
      float R34 = dist2(Atom3, Atom4);
      float sum12  = Param1.x + Param2.x;
      float sum34  = Param3.x + Param4.x;
      float prod12 = Param1.x * Param2.x;
      float prod34 = Param3.x * Param4.x;
      float preexp = __fdividef(prod12, sum12) * R12 + 
	__fdividef(prod34, sum34) * R34;
      float preintegral = __fdividef(Param1.y * Param2.y * Param3.y * 
				     Param4.y * __expf(- preexp), sum12 * 
				     sum34) * (1 / sqrtf(sum12 + sum34));

      if(preintegral * preintegral > 1.0e-23f)
	{
	  float3 Atomp = ReducePair(Atom1, Atom2, Param1, Param2, sum12);
	  float3 Atomq = ReducePair(Atom3, Atom4, Param3, Param4, sum34);
	  
	  float rpq2 = dist2(Atomp, Atomq);
	  float rho = __fdividef(sum12 * sum34, sum12 + sum34);
	  float weight = Root(rpq2 * rho);
	  Result = 34.98683666f * preintegral * weight;
	}
    }
  Data[thid] = Result;
  __syncthreads();

  for(unsigned int s = blockDim.x>>1; s > 0;  s = s>>1)
    {
      if (thid < s)
        Data[thid] += Data[thid+s];
      __syncthreads();
    }

  if(!thid)
    d_Output[Offset + blid] = Data[0];
}

__global__ 
void DoReduction(float* d_ReductionSum, float* d_Output, 
			    uint2* d_FinalReduce, int block_size, int Offset)
{
  int firstElement;
  int offset;
  __shared__ float Result[BLOCK_SIZE];
  
  int blid = blockIdx.x;
  int thid = threadIdx.x;
  
  uint2 myWork = d_FinalReduce[blid + Offset];
  firstElement = myWork.x;
  offset = myWork.y;
  
  if(thid < offset)
    Result[thid] = d_Output[firstElement + thid];
  else
    Result[thid] = 0.0f;
  __syncthreads();

  for(unsigned int s = blockDim.x>>1; s > 0;  s = s>>1)
    {
      if (thid < s)
        Result[thid] += Result[thid+s];
      __syncthreads();
    }

  if(!thid)
    d_ReductionSum[blid + Offset] = Result[0];
}

__device__ 
float product1D(float alpha_a, float coor_a, float alpha_b, 
			   float coor_b, float sum_ab)
{
  return __fdividef(alpha_a * coor_a + alpha_b * coor_b, sum_ab);
}

__device__ 
float dist2(float3 Atom1, float3 Atom2)
{
  float dx = Atom1.x - Atom2.x;
  float dy = Atom1.y - Atom2.y;
  float dz = Atom1.z - Atom2.z;
  return dx * dx + dy * dy + dz * dz;
}

__device__ 
float dist2(float4 Atom1, float4 Atom2)
{
  float dx = Atom1.x - Atom2.x;
  float dy = Atom1.y - Atom2.y;
  float dz = Atom1.z - Atom2.z;
  return dx * dx + dy * dy + dz * dz;
}

__device__ 
float3 ReducePair(float4 Atom1, float4 Atom2, float2 Param1, 
		  float2 Param2, float sum_12)
{
  float3 Atomp;
  Atomp.x = product1D(Param1.x, Atom1.x, Param2.x, Atom2.x, sum_12);
  Atomp.y = product1D(Param1.x, Atom1.y, Param2.x, Atom2.y, sum_12);
  Atomp.z = product1D(Param1.x, Atom1.z, Param2.x, Atom2.z, sum_12);
  
  return Atomp;
}

__device__ 
float Root(float X)
{
  float rPIE4;
  float WW1 = 0.0f;
  float F1,E,Y,inv;
  
  rPIE4 = 1.273239545f;
  if (X < 3.0e-7f)
    {
      WW1 = 1.0f - 0.333333333f * X;
    } 
  else if (X < 1.0f) 
    {
      F1 = ((((((((-8.36313918003957E-08f*X+1.21222603512827E-06f )*X-
		  1.15662609053481E-05f )*X+9.25197374512647E-05f )*X-
		6.40994113129432E-04f )*X+3.78787044215009E-03f )*X-
	      1.85185172458485E-02f )*X+7.14285713298222E-02f )*X-
	    1.99999999997023E-01f )*X+3.33333333333318E-01f;
      WW1 = (X+X)*F1 + __expf(-X);
    } 
  else if (X < 3.0f) 
    {
      Y = X-2.0f;
      F1 = ((((((((((-1.61702782425558E-10f*Y+1.96215250865776E-09f )*Y-
		    2.14234468198419E-08f )*Y+2.17216556336318E-07f )*Y-
		  1.98850171329371E-06f )*Y+1.62429321438911E-05f )*Y-
		1.16740298039895E-04f )*Y+7.24888732052332E-04f )*Y-
	      3.79490003707156E-03f )*Y+1.61723488664661E-02f )*Y-
	    5.29428148329736E-02f )*Y+1.15702180856167E-01f;
      WW1 = (X+X)*F1+__expf(-X);
    } 
  else if (X < 5.0f)
    {
      Y = X-4.0f;
      F1 = ((((((((((-2.62453564772299E-11f*Y+3.24031041623823E-10f )*Y-
		    3.614965656163E-09f)*Y+3.760256799971E-08f)*Y-
		  3.553558319675E-07f)*Y+3.022556449731E-06f)*Y-
		2.290098979647E-05f)*Y+1.526537461148E-04f)*Y-
	      8.81947375894379E-04f)*Y+4.33207949514611E-03f )*Y-
	    1.75257821619926E-02f )*Y+5.28406320615584E-02f;
      WW1 = (X+X)*F1+__expf(-X);
    } 
  else if (X < 10.0f) 
    {
      E = __expf(-X);
      inv = 1 / X;
      WW1 = (((((( 4.6897511375022E-01f*inv-6.9955602298985E-01f)*inv +
		 5.3689283271887E-01f)*inv-3.2883030418398E-01f)*inv +
	       2.4645596956002E-01f)*inv-4.9984072848436E-01f)*inv -
	     3.1501078774085E-06f)*E + 1 / sqrtf(rPIE4 * X);
    } 
  else if (X < 15.0f) 
    {
      E = __expf(-X);
      inv = 1 / X;
      WW1 = (((-1.8784686463512E-01f*inv+2.2991849164985E-01f)*inv -
	      4.9893752514047E-01f)*inv-2.1916512131607E-05f)*E \
	+ 1 / sqrtf(rPIE4 * X);
    } 
  else if (X < 33.0f) 
    {
      E = __expf(-X);
      inv = 1 / X;
      WW1 = (( 1.9623264149430E-01f*inv-4.9695241464490E-01f)*inv -
	     6.0156581186481E-05f)*E + 1 / sqrtf(rPIE4 * X);
    } 
  else 
    {
      WW1 = 1 / sqrtf(rPIE4 * X);
    }
  return WW1;
}

__device__ 
void GetBases(int Work, int thid, int Shift, int& localthid, 
	      int& a1, int& a2, int& a3, int& a4)
{
  a1         = (Work & (unsigned int)61440)   >> 12;
  a2         = (Work & (unsigned int)3840 )   >> 8;
  a3         = (Work & (unsigned int)240  )   >> 4;
  a4         = (Work & (unsigned int)15   );
  localthid = thid - (Shift << LOG_BLOCK_SIZE);
}

__device__ 
void GetNs(int localthid, int a1, int a2, int a3, int a4, 
	   int& n1, int& n2, int& n3, int& n4)
{
  int prod = __mul24(a2, __mul24(a3, a4));
  int res = localthid;
  n1 = __fdividef(res, prod);
  res -= __mul24(n1, prod);

  prod = __mul24(a3, a4);
  n2 = __fdividef(res, prod);
  res -= __mul24(n2, prod);

  prod = a4;
  n3 = __fdividef(res, prod);
  res -= __mul24(n3, prod);

  n4 = res;
}

__device__ 
void GetAtoms(int Work, int& atom1, int& atom2, int& atom3, int& atom4)
{
  atom1 = (Work & (unsigned int)4278190080) >> 24;
  atom2 = (Work & (unsigned int)16711680)   >> 16;
  atom3 = (Work & (unsigned int)65280)      >> 8;
  atom4 = (Work & (unsigned int)255);
}

__device__ 
void GetOffsets(int Work, int& off1, int& off2, int& off3, int& off4)
{
  off1 = (Work & (unsigned int)4278190080) >> 24;
  off2 = (Work & (unsigned int)16711680)   >> 16;
  off3 = (Work & (unsigned int)65280)      >> 8;
  off4 = (Work & (unsigned int)255);
}

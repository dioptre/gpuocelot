/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <parboil.h>

#define CUDA_ERRCK \
  {hipError_t err = hipGetLastError(); \
    if (err) fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
  }

/*
// Place and Transition are implicitly included in the code
// as the grid is a fixed one
typedef struct {
        float mark;
} Place;

typedef struct {
        int from1, from2;
        int to1, to2;
} Transition;

// this starts from row 0 and col 0
P(r,c)    -> T(r,c)   -> P(r,c+1)  ->
  |            |            |
 \/           \/           \/
T(r+1,c-1)-> P(r+1,c) -> T(r+1,c)  ->
  |            |            |
 \/           \/           \/
P(r+2,c)  -> T(r+2,c) -> P(r+2,c+1)->
  |            |            |
 \/           \/           \/
T(r+3,c-1)-> P(r+3,c) -> T(r+3,c)->
  |            |            |
 \/           \/           \/

*/

#include "rand_gen.cuh"
#include "petri_kernel.cuh"

static int N, s, t, N2, NSQUARE2;
uint32 host_mt[MERS_N];


void* AllocateDeviceMemory(int size);
void CopyFromDeviceMemory(void* h_p, void* d_p, int size);
void CopyFromHostMemory(void* d_p, void* h_p, int size);
void FreeDeviceMemory(void* mem);
void PetrinetOnDevice(struct pb_TimerSet *timers, 
                      struct pb_Parameters *params);
void compute_statistics();

float results[4];
float* h_vars;
int* h_maxs;

int main(int argc, char** argv) 
{
  struct pb_TimerSet timers;
  struct pb_Parameters *params;

  pb_InitializeTimerSet( &timers );
  params = pb_ReadParameters( &argc, argv );

  if (argc<4) 
    {
      printf("Usage: petri n s t\n"
	     "n: the place-transition grid is 2nX2n\n"
	     "s: the maximum steps in a trajectory\n"
	     "t: number of trajectories\n");
      return -1;
    }

  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  N = atoi(argv[1]);
  if (N<1)
    return -1;
  s = atoi(argv[2]);
  if (s<1)
    return -1;

  t = atoi(argv[3]);
  if (t<1)
    return -1;

  N2 = N+N;
  NSQUARE2 = N*N2;
  
  h_vars = (float*)malloc(t*sizeof(float));
  h_maxs = (int*)malloc(t*sizeof(int));
  
  // compute the simulation on the GPU
  PetrinetOnDevice( &timers, params );
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  compute_statistics();

  free(h_vars);
  free(h_maxs);
    
  pb_SwitchToTimer( &timers, pb_TimerID_IO );

  printf("petri N=%d s=%d t=%d\n", N, s, t);
  printf("mean_vars: %f    var_vars: %f\n", results[0], results[1]);
  printf("mean_maxs: %f    var_maxs: %f\n", results[2], results[3]);

  if(params->outFile)
    {
      FILE *out;
      out = fopen(params->outFile, "w");
      fprintf(out, "%f %f %f %f\n", results[0], results[1], results[2],
              results[3]);
      fclose(out);
    }

  if( pb_compareFiles(params->outFile, "data/pns.out", 1) )
  {
    printf("TEST PASSED\n");
  }
  else
  {
  	printf("TEST FAILED\n");
  }

  pb_SwitchToTimer ( &timers, pb_TimerID_NONE );
  pb_PrintTimerSet( &timers );
  pb_FreeParameters( params );

  return 0;
}

void compute_statistics() 
{
  float sum = 0;
  float sum_vars = 0;
  float sum_max = 0;
  float sum_max_vars = 0;
  int i;
  for (i=0; i<t; i++) 
    {
      sum += h_vars[i];
      sum_vars += h_vars[i]*h_vars[i];
      sum_max += h_maxs[i];
      sum_max_vars += h_maxs[i]*h_maxs[i];
    }
  results[0] = sum/t;
  results[1] = sum_vars/t - results[0]*results[0];
  results[2] = sum_max/t;
  results[3] = sum_max_vars/t - results[2]*results[2];
}

void PetrinetOnDevice(struct pb_TimerSet *timers, 
                      struct pb_Parameters *params)
{
  // Allocate memory
  int i;
  int unit_size = NSQUARE2*(sizeof(int)+sizeof(char))+
    sizeof(float)+sizeof(int);
  int block_num = MAX_DEVICE_MEM/unit_size;
  int *p_hmaxs;
  float *p_hvars;
  int* g_places;
  float* g_vars;
  int* g_maxs;
  
  pb_SwitchToTimer( timers, pb_TimerID_COPY );
  g_places = (int*)AllocateDeviceMemory((unit_size- sizeof(float)-
					      sizeof(int))*block_num);
  CUDA_ERRCK
  g_vars = (float*)AllocateDeviceMemory(block_num*sizeof(float));
  CUDA_ERRCK
  g_maxs = (int*)AllocateDeviceMemory(block_num*sizeof(int));
  CUDA_ERRCK

  // Setup the execution configuration
  dim3  grid(block_num);  // number of blocks
  dim3  threads(256);  // each block has 256 threads

  p_hmaxs = h_maxs;
  p_hvars = h_vars;

  // Launch the device computation threads!
  for (i = 0; i<t-block_num; i+=block_num) 
    {
      if (params->synchronizeGpu) hipDeviceSynchronize();
      pb_SwitchToTimer( timers, pb_TimerID_GPU );
      PetrinetKernel<<< grid, threads>>>
	(g_places, g_vars, g_maxs, N, s, 5489*(i+1));
      CUDA_ERRCK

      if (params->synchronizeGpu) hipDeviceSynchronize();
      pb_SwitchToTimer( timers, pb_TimerID_COPY );
      CopyFromDeviceMemory(p_hmaxs, g_maxs, block_num*sizeof(int));
      CUDA_ERRCK
      CopyFromDeviceMemory(p_hvars, g_vars, block_num*sizeof(float));
      CUDA_ERRCK
      if (params->synchronizeGpu) hipDeviceSynchronize();

      pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );
      p_hmaxs += block_num;
      p_hvars += block_num;
    }
	
  dim3 grid1(t-i);
  pb_SwitchToTimer( timers, pb_TimerID_GPU );
  PetrinetKernel<<< grid1, threads>>>
    (g_places, g_vars, g_maxs, N, s, time(NULL));
  CUDA_ERRCK

  // Read result from the device
  if (params->synchronizeGpu) hipDeviceSynchronize();
  pb_SwitchToTimer( timers, pb_TimerID_COPY );
  CopyFromDeviceMemory(p_hmaxs, g_maxs, (t-i)*sizeof(int));
  CUDA_ERRCK
  CopyFromDeviceMemory(p_hvars, g_vars, (t-i)*sizeof(float));
  CUDA_ERRCK

  // Free device matrices
  FreeDeviceMemory(g_places);
  CUDA_ERRCK
  FreeDeviceMemory(g_vars);
  CUDA_ERRCK
  FreeDeviceMemory(g_maxs);
  CUDA_ERRCK
}

// Allocate a device matrix of same size as M.
void* AllocateDeviceMemory(int size)
{
  int* mem;
  hipMalloc((void**)&mem, size);
  return mem;
}

// Copy device memory to host memory
void CopyFromDeviceMemory(void* h_p, void* d_p, int size)
{
  hipMemcpy(h_p, d_p, size, hipMemcpyDeviceToHost);
}

// Copy device memory from host memory
void CopyFromHostMemory(void* d_p, void* h_p, int size)
{
  hipMemcpy(d_p, h_p, size, hipMemcpyHostToDevice);
}

// Free a device matrix.
void FreeDeviceMemory(void* mem)
{
  if (mem!=NULL)
    hipFree(mem);
}


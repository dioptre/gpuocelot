#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
  Implementing Breadth first search on CUDA using algorithm given in DAC'10
  paper "An Effective GPU Implementation of Breadth-First Search"

  Copyright (c) 2010 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Author: Lijiuan Luo (lluo3@uiuc.edu)
*/
#ifndef _KERNEL_H_
#define _KERNEL_H_
/*
Define colors for BFS
1) the definition of White, gray and black comes from the text book "Introduction to Algorithms"
2) For path search problems, people may choose to use different colors to record the found paths.
Therefore we reserve numbers (0-16677216) for this purpose. Only nodes with colors bigger than
UP_LIMIT are free to visit 
3) We define two gray shades to differentiate between the new frontier nodes and the old frontier nodes that
 have not been marked BLACK 
*/

#define UP_LIMIT 16677216//2^24
#define WHITE 16677217
#define GRAY 16677218
#define GRAY0 16677219
#define GRAY1 16677220
#define BLACK 16677221

//Distribute computation
//tries to distribute the computation among all the SMs. However, it does not seem to help improve the
//performance.
//#define DIS_COMP

/***
The maximum size of each  w-queue (row-major order)
FIXME
This should be chosen more carefully to avoid  bank conflict.
A better implementation will be to arrange w-queues in column-major order, but the program will be less readable.
****/
#define LOCAL_MEM 400 

texture<Node> g_graph_node_ref;
texture<Edge> g_graph_edge_ref;

volatile __device__ int count = 0;
volatile __device__ int no_of_nodes_vol = 0;
volatile __device__ int stay_vol = 0;
//GPU synchronization.
//implementing the algorithm proposed in 
//S. Xiao and W. Feng, "Inter-block GPU communication via fast barrier
//synchronization," Technical Report TR-09-19, Dept. of Computer Science, VT
//NOTE the algorithm originally given in the report is inaccurate
__device__ void start_global_barrier(int fold){
	//This synchronization is missing in the report
	__syncthreads();

	if(threadIdx.x == 0){
        atomicAdd((int*)&count, 1);
		while( count < NUM_SM*fold){
		    ;
		}
    }
    __syncthreads();
    
}
//-------------------------------------------------
//This is the version for one-block situation. The propagation idea is basically the same as
//BFS_kernel.
//The major differences are:
// 1) This kernel can propagate though multiple BFS levels (while loop) using __synchThreads() between levels 
// 2) the intermediate queues are stored in shared memory (next_wf) 
//\param q1: the current frontier queue when the kernel is launched
//\param q2: the new frontier queue when the  kernel returns
//--------------------------------------------------
__global__ void
BFS_in_GPU_kernel( int * q1, int * q2, Node* g_graph_nodes, Edge* g_graph_edges, int* g_color, int * g_cost, int no_of_nodes, int * tail, int gray_shade, int k ) 
{
	__shared__ int local_q_tail[NUM_BIN];
	__shared__ int local_q[NUM_BIN][LOCAL_MEM]; 
	__shared__ int prefix_q[NUM_BIN];
	__shared__ int thread_n_q[NUM_BIN];
	
	//next/new wave front
	__shared__ int next_wf[MAX_THREADS_PER_BLOCK];
	__shared__ int  tot_sum;
	if(threadIdx.x == 0)	
		tot_sum = 0;//total number of new frontier nodes
	while(1){//propage through multiple BFS levels until the wavfront overgrows one-block limit
		if(threadIdx.x < NUM_BIN){
			local_q_tail[threadIdx.x] = 0;
			thread_n_q[threadIdx.x] = blockDim.x>>EXP;
			if((blockDim.x&MOD_OP) > threadIdx.x){
				thread_n_q[threadIdx.x]++;
			}
		}
		__syncthreads();
		int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
		if( tid<no_of_nodes)
		{
			int pid;
			if(tot_sum == 0)//this is the first BFS level of current kernel call
				pid = q1[tid];  
			else
				pid = next_wf[tid];//read the current frontier info from last level's propagation
			g_color[pid] = BLACK;
			int cur_cost = g_cost[pid];
			int q_i = threadIdx.x&MOD_OP; 
			Node cur_node = tex1Dfetch(g_graph_node_ref,pid);
			for(int i=cur_node.x; i<cur_node.y + cur_node.x; i++) {
				Edge cur_edge = tex1Dfetch(g_graph_edge_ref,i);
				int id = cur_edge.x;
				int cost = cur_edge.y;
				cost += cur_cost;
				int orig_cost = atomicMin(&g_cost[id],cost);
				if(orig_cost > cost){
					int old_color = atomicExch(&g_color[id],gray_shade);
					if(old_color != gray_shade) {
						//push to the queue
						int index = atomicAdd(&local_q_tail[q_i],1);
						local_q[q_i][index] = id;
					}
				}
			}
		}
		__syncthreads();
		if(threadIdx.x == 0){
			prefix_q[0] = 0;
			for(int i = 1; i < NUM_BIN; i++){
				prefix_q[i] = prefix_q[i-1]+local_q_tail[i-1];
			}
			tot_sum = prefix_q[NUM_BIN-1] + local_q_tail[NUM_BIN-1];
			*tail = tot_sum;
		}
		__syncthreads();

		int q_i = threadIdx.x&MOD_OP;
		int local_shift = threadIdx.x>>EXP;
		if(tot_sum == 0)//the new frontier becomes empty; BFS is over
			return;
		if(tot_sum <= MAX_THREADS_PER_BLOCK){//the new frontier is still within one-block limit;
			//stay in current kernel
			while (local_shift < local_q_tail[q_i]){
				next_wf[prefix_q[q_i]+local_shift] = local_q[q_i][local_shift];
				local_shift += thread_n_q[q_i];
			}
			__syncthreads();
			no_of_nodes = tot_sum;
			if(threadIdx.x == 0){
				if(gray_shade == GRAY0)
					gray_shade = GRAY1;
				else
					gray_shade = GRAY0;
			}
		}
		else{//the new frontier outgrows one-block limit; terminate current kernel
			while(local_shift < local_q_tail[q_i]){
				q2[prefix_q[q_i]+local_shift] = local_q[q_i][local_shift];
				local_shift += thread_n_q[q_i];
			}
			return;
		}
	}//while
	
}	
//----------------------------------------------------------------
//This BFS kernel propagates through multiple levels using global synchronization 
//The basic propagation idea is the same as "BFS_kernel"
//The major differences are:
// 1) propagate through multiple levels by using GPU global sync ("start_global_barrier")
// 2) use q1 and q2 alternately for the intermediate queues
//\param q1: the current frontier when the kernel is called
//\param q2: possibly the new frontier when the kernel returns depending on how many levels of propagation
//           has been done in current kernel; the new frontier could also be stored in q1
//\param switch_k: whether or not to adjust the "k" value on the host side
//                Normally on the host side, when "k" is even, q1 is the current frontier; when "k" is
//                odd, q2 is the current frontier; since this kernel can propagate through multiple levels,
//                the k value may need to be adjusted when this kernel returns.
//\param max_nodes_per_block: the maximum frontier node assigned to a block. It is only useful when "DIS_COMP"
//                is enabled
//\param global_kt: the total number of global synchronizations, 
//                   or the number of times to call "start_global_barrier" 
//--------------------------------------------------------------
__global__ void
BFS_kernel_multi_blk_inGPU( int *  q1, int *  q2, Node* g_graph_nodes, Edge* g_graph_edges, int* g_color, int * g_cost, int *no_of_nodes, int * tail, int gray_shade, int k,   
	 int * switch_k, int * max_nodes_per_block, int * global_kt ) 
{
	__shared__ int local_q_tail[NUM_BIN];
	__shared__ int local_q[NUM_BIN][LOCAL_MEM]; 
	__shared__ int prefix_q[NUM_BIN];
	__shared__ int thread_n_q[NUM_BIN];
	__shared__ int shift;
	__shared__ int no_of_nodes_sm;
	__shared__ int odd_time;// the odd level of propagation within current kernel
	if(threadIdx.x == 0){
		odd_time = 1;//true;
        if(blockIdx.x == 0)
            no_of_nodes_vol = *no_of_nodes;
	}
	int kt = *global_kt;// the total count of GPU global synchronization 
	while (1){//propagate through multiple levels
		if(threadIdx.x < NUM_BIN){
			local_q_tail[threadIdx.x] = 0;
			thread_n_q[threadIdx.x] = blockDim.x>>EXP;
			if((blockDim.x&MOD_OP) > threadIdx.x){
				thread_n_q[threadIdx.x]++;
			}
		}
		if(threadIdx.x == 0)
			no_of_nodes_sm =  no_of_nodes_vol; 
		__syncthreads();

		#ifdef DIS_COMP
		int tid = blockIdx.x*(*max_nodes_per_block) + threadIdx.x;
		#else
		int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
		#endif
		if( tid<no_of_nodes_sm)
		{
			int pid;
			if(odd_time == 1)
				pid = atomicOr((int*)&q1[tid], 0);  
			else
				pid = atomicOr((int*)&q2[tid], 0);
			g_color[pid] = BLACK;
			int cur_cost = atomicOr((int*)&g_cost[pid], 0);
			int q_i = threadIdx.x&MOD_OP; 
			Node cur_node = tex1Dfetch(g_graph_node_ref,pid);
			for(int i=cur_node.x; i<cur_node.y + cur_node.x; i++) {
				Edge cur_edge = tex1Dfetch(g_graph_edge_ref,i);
				int id = cur_edge.x;
				int cost = cur_edge.y;
				cost += cur_cost;
				int orig_cost = atomicMin(&g_cost[id],cost);
				if(orig_cost > cost){
					if(g_color[id] > UP_LIMIT){
						int old_color = atomicExch(&g_color[id],gray_shade);
						if(old_color != gray_shade)
							{
							//push to the queue
							int index = atomicAdd(&local_q_tail[q_i],1);
							local_q[q_i][index] = id;
						}
					}
				}
			}
		}
		__syncthreads();
		if(threadIdx.x == 0){
			prefix_q[0] = 0;
			for(int i = 1; i < NUM_BIN; i++){
				prefix_q[i] = prefix_q[i-1] + local_q_tail[i-1];
			}
			int tot_sum = prefix_q[NUM_BIN-1] + local_q_tail[NUM_BIN-1];
			shift = atomicAdd(tail,tot_sum);
		}
		__syncthreads();

		int q_i = threadIdx.x&MOD_OP;
		int local_shift = threadIdx.x>>EXP;
		while (local_shift < local_q_tail[q_i]){
			if(odd_time)
				q2[shift+prefix_q[q_i]+local_shift] = local_q[q_i][local_shift];
			else
				q1[shift+prefix_q[q_i]+local_shift] = local_q[q_i][local_shift];
			local_shift += thread_n_q[q_i];
		}
		if(threadIdx.x == 0){
			odd_time = (odd_time+1)%2;
			if(gray_shade == GRAY0)
				gray_shade = GRAY1;
			else
				gray_shade = GRAY0;
		}

		//synchronize among all the blks
		start_global_barrier(kt+1);
		if(blockIdx.x == 0 && threadIdx.x == 0){
            stay_vol = 0;
			if(*tail< NUM_SM*MAX_THREADS_PER_BLOCK && *tail > MAX_THREADS_PER_BLOCK){
                stay_vol = 1;
				no_of_nodes_vol = *tail;
			
				//this is only useful when DIS_COMP is enabled
				*max_nodes_per_block = ceil(float(*no_of_nodes)/NUM_SM);
				*tail = 0;
			}
		}
		start_global_barrier(kt+2);
		kt+= 2;
        if(stay_vol == 0)
        {
            if(blockIdx.x == 0 && threadIdx.x == 0)
            {
                *global_kt = kt;
                *switch_k = (odd_time+1)%2;
                *no_of_nodes = no_of_nodes_vol;
            }
            return;
        }
	}
}

/*****************************************************************************
This is the  most general version of BFS kernel, i.e. no assumption about #block in the grid  
\param q1: the array to hold the current frontier
\param q2: the array to hold the new frontier
\param g_graph_nodes: the nodes in the input graph
\param g_graph_edges: the edges i nthe input graph
\param g_color: the colors of nodes
\param g_cost: the costs of nodes
\param no_of_nodes: the number of nodes in the current frontier
\param tail: pointer to the location of the tail of the new frontier. *tail is the size of the new frontier 
\param gray_shade: the shade of the gray in current BFS propagation. See GRAY0, GRAY1 macro definitions for more details
\param k: the level of current propagation in the BFS tree. k= 0 for the first propagation.
***********************************************************************/
__global__ void
BFS_kernel( int * q1, int * q2, Node* g_graph_nodes, Edge* g_graph_edges, int* g_color, int * g_cost, int no_of_nodes, int * tail, int gray_shade, int k ) 
{
	__shared__ int local_q_tail[NUM_BIN];//the tails of each local warp-level queue
	__shared__ int local_q[NUM_BIN][LOCAL_MEM];//the local warp-level queues 
	__shared__ int prefix_q[NUM_BIN];//the number of elementss in the w-queues ahead of
					//current w-queue, a.k.a prefix sum
	__shared__ int thread_n_q[NUM_BIN];//#thread which writes into the current w-queue
	__shared__ int shift;

	if(threadIdx.x < NUM_BIN){
		local_q_tail[threadIdx.x] = 0;//initialize the tail of w-queue
		thread_n_q[threadIdx.x] = blockDim.x>>EXP;//#thread/NUM_BIN
		if((blockDim.x&MOD_OP) > threadIdx.x){//#thread%NUM_BIN > threadIdx.x
			thread_n_q[threadIdx.x]++;
		}
	}
	__syncthreads();

	//first, propagate and add the new frontier elements into w-queues
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	if( tid<no_of_nodes)
	{
		int pid = q1[tid]; //the current frontier node, or the parent node of the new frontier nodes 
		g_color[pid] = BLACK;
		int cur_cost = g_cost[pid];
		int q_i = threadIdx.x&MOD_OP; //the id of the queue which new frontier nodes will be pushed
				//into
		Node cur_node = tex1Dfetch(g_graph_node_ref,pid);
		for(int i=cur_node.x; i<cur_node.y + cur_node.x; i++)//visit each neighbor of the
				//current frontier node.
			{
			Edge cur_edge = tex1Dfetch(g_graph_edge_ref,i);
			int id = cur_edge.x;
			int cost = cur_edge.y;
			cost += cur_cost;
			int orig_cost = atomicMin(&g_cost[id],cost);
			if(orig_cost > cost){//the node should be visited
				if(g_color[id] > UP_LIMIT){
					int old_color = atomicExch(&g_color[id],gray_shade);
					//this guarantees that only one thread will push this node
					//into a queue
					if(old_color != gray_shade) {

						//atomic operation guarantees the correctness
						//even if multiple warps are executing simultaneously
						int index = atomicAdd(&local_q_tail[q_i],1);
						local_q[q_i][index] = id;
					}
				}
			}
		}
	}
	__syncthreads();

	if(threadIdx.x == 0){
		//now calculate the prefix sum
		prefix_q[0] = 0;
		for(int i = 1; i < NUM_BIN; i++){
			//the prefix sum of one queue is equal to the prefix sum of its predecessor queue
			//plus the number of elements in the predecessor queue
			prefix_q[i] = prefix_q[i-1]+local_q_tail[i-1];
		}
		//the total number of elements in the block-level queue is the prefix sum of the last w-queue
		//plus the number of elements in the last w-queue
		int tot_sum = prefix_q[NUM_BIN-1] + local_q_tail[NUM_BIN-1];

		//the offset or "shift" of the block-level queue within the grid-level queue
		//is determined by atomic operation
		shift = atomicAdd(tail,tot_sum);
	}
	__syncthreads();

	//now copy the elements from w-queues into grid-level queues.
	//Note that we have bypassed the copy to/from block-level queues for efficiency reason

	int q_i = threadIdx.x&MOD_OP;//w-queue index
	int local_shift = threadIdx.x>>EXP;//shift within a w-queue

	//loop unrolling was originally used for better performance, but removed for better readability
	while(local_shift < local_q_tail[q_i]){
		q2[shift+prefix_q[q_i]+local_shift] = local_q[q_i][local_shift];
		local_shift+= thread_n_q[q_i];//multiple threads are copying elements at the same time,
			//so we shift by multiple elements for next iteration  
	}
	//FIXME, the above implementation has bad coalescing. Better implementation should make
	//consecutive threads write into consecutive locations
}
#endif 

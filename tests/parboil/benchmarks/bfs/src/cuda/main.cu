#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
  Implementing Breadth first search on CUDA using algorithm given in DAC'10
  paper "An Effective GPU Implementation of Breadth-First Search"

  Copyright (c) 2010 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Author: Lijiuan Luo (lluo3@uiuc.edu)
  Revised for Parboil 2 Benchmark Suite by: Geng Daniel Liu (gengliu2@illinois.edu)
*/

#ifdef _GLIBCXX_USE_INT128
#undef _GLIBCXX_USE_INT128
#endif 

#ifdef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_ATOMIC_BUILTINS
#endif


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <parboil.h>
#include <deque>
#include <iostream>

#define MAX_THREADS_PER_BLOCK 512
#define NUM_SM 15 //the number of Streaming Multiprocessors; 15 for Fermi architecture 30 for G280 at the moment of this document
#define NUM_BIN 8 //the number of duplicated frontiers used in BFS_kernel_multi_blk_inGPU
#define EXP 3 // EXP = log(NUM_BIN), assuming NUM_BIN is still power of 2 in the future architecture
	//using EXP and shifting can speed up division operation 
#define MOD_OP 7 // This variable is also related with NUM_BIN; may change in the future architecture;
	//using MOD_OP and "bitwise and" can speed up mod operation
#define INF 2147483647//2^31-1
int no_of_nodes; //the number of nodes in the graph
int edge_list_size;//the number of edges in the graph
FILE *fp;

typedef int2 Node;
typedef int2 Edge;

#include "kernel.cu"
//Somehow "hipMemset" does not work. So I use hipMemcpy of constant variables for initialization
const int h_top = 1;
const int zero = 0;

void runGPU(int argc, char** argv);
////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    //printf(" device 1 \n");
	no_of_nodes=0;
	edge_list_size=0;
    hipSetDevice(0);
	runGPU(argc,argv);

}
bool  BFS_GPU( Node * d_graph_nodes,Edge * d_graph_edges,
	int * d_color, int * d_cost, int * d_q1, int * d_q2, int * tail, int source, int & global_kt){
	int num_of_blocks; 
	int num_of_threads_per_block;
    

	(hipMemcpy(tail,&h_top,sizeof(int),hipMemcpyHostToDevice));
	(hipMemcpy(&d_cost[source],&zero,sizeof(int),hipMemcpyHostToDevice));

	( hipMemcpy( &d_q1[0], &source, sizeof(int), hipMemcpyHostToDevice) );
	int num_t;//number of threads
	int k=0;//BFS level index

	//whether or not to adjust "k", see comment on "BFS_kernel_multi_blk_inGPU" for more details 
	int * switch_kd;
	( hipMalloc( (void**) &switch_kd, sizeof(int)));
	int * num_td;//number of threads
	( hipMalloc( (void**) &num_td, sizeof(int)));

	//whether to stay within a kernel, used in "BFS_kernel_multi_blk_inGPU"
	bool *stay;
	( hipMalloc( (void**) &stay, sizeof(bool)));
	int switch_k;

	//max number of frontier nodes assigned to a block
	int * max_nodes_per_block_d;
	( hipMalloc( (void**) &max_nodes_per_block_d, sizeof(int)));
	#ifdef DIS_COMP
	int max_nodes_per_block;
	#endif
    int *global_kt_d;
	( hipMalloc( (void**) &global_kt_d, sizeof(int)));
	(hipMemcpy(global_kt_d,&global_kt, sizeof(int),hipMemcpyHostToDevice));
	
	do
	{
		( hipMemcpy( &num_t, tail, sizeof(int), hipMemcpyDeviceToHost) );
		(hipMemcpy(tail,&zero,sizeof(int),hipMemcpyHostToDevice));

		if(num_t == 0){//frontier is empty
			(hipFree(stay));
			(hipFree(switch_kd));
			(hipFree(num_td));
			(hipMemcpy(&global_kt,global_kt_d, sizeof(int),hipMemcpyDeviceToHost));
			return false;
		}

		num_of_blocks = 1;
		num_of_threads_per_block = num_t;
		if(num_of_threads_per_block <NUM_BIN)
			num_of_threads_per_block = NUM_BIN;
		if(num_t>MAX_THREADS_PER_BLOCK)
		{
			num_of_blocks = (int)ceil(num_t/(double)MAX_THREADS_PER_BLOCK); 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
		}
		if(num_of_blocks == 1)//will call "BFS_in_GPU_kernel" 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
		if(num_of_blocks >1 && num_of_blocks <= NUM_SM)// will call "BFS_kernel_multi_blk_inGPU"
			num_of_blocks = NUM_SM;

		//assume "num_of_blocks" can not be very large
	        dim3  grid( num_of_blocks, 1, 1);
		dim3  threads( num_of_threads_per_block, 1, 1);

		if(k%2 == 0){
			if(num_of_blocks == 1){
				BFS_in_GPU_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost,num_t , tail,GRAY0,k );
			}
			else if(num_of_blocks <= NUM_SM){
				#ifdef DIS_COMP
				max_nodes_per_block = ceil(float(num_t)/NUM_SM);
				(hipMemcpy(max_nodes_per_block_d,
					&max_nodes_per_block,sizeof(int), hipMemcpyHostToDevice));
				#endif
				(hipMemcpy(num_td,&num_t,sizeof(int),
					hipMemcpyHostToDevice));
				BFS_kernel_multi_blk_inGPU
				<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_td, tail,GRAY0,k,
				switch_kd, max_nodes_per_block_d, global_kt_d);
				(hipMemcpy(&switch_k,switch_kd, sizeof(int),
				hipMemcpyDeviceToHost));
				if(!switch_k){
					k--;
				}
			}
			else{
				BFS_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_t, tail,GRAY0,k);
			}
		}
		else{
			if(num_of_blocks == 1){
				BFS_in_GPU_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_t, tail,GRAY1,k);
			}
			else if(num_of_blocks <= NUM_SM){
				#ifdef DIS_COMP
				max_nodes_per_block = ceil(float(num_t)/NUM_SM);
				(hipMemcpy(max_nodes_per_block_d,
					&max_nodes_per_block,sizeof(int), hipMemcpyHostToDevice));
				#endif
				(hipMemcpy(num_td,&num_t,sizeof(int),
					hipMemcpyHostToDevice));
				BFS_kernel_multi_blk_inGPU
				<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_td, tail,GRAY1,k,
				 switch_kd, max_nodes_per_block_d, global_kt_d);
				(hipMemcpy(&switch_k,switch_kd, sizeof(int),
				hipMemcpyDeviceToHost));
				if(!switch_k){
					k--;
				}
			}
			else{
				BFS_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
				 d_graph_edges, d_color, d_cost, num_t, tail, GRAY1,k);
			}
		}
			
		// check if kernel execution generated any error
		//CUT_CHECK_ERROR("Kernel execution failed");
		
		k++;
	}
	while(1);
}
///////////////////////////////
//FUNCTION:only run GPU version 
////////////////////////////////////////////
void runGPU( int argc, char** argv) 
{

    struct pb_Parameters *params;
    struct pb_TimerSet timers;

    pb_InitializeTimerSet(&timers);
    params = pb_ReadParameters(&argc, argv);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {
        fprintf(stderr, "Expecting one input filename\n");
        exit(-1);
    }

    pb_SwitchToTimer(&timers, pb_TimerID_IO);
	//printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(params->inpFiles[0],"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}
	int source;

	fscanf(fp,"%d",&no_of_nodes);
	// allocate host memory
	Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
	int *color = (int*) malloc(sizeof(int)*no_of_nodes);
	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].x = start;
		h_graph_nodes[i].y = edgeno;
		color[i]=WHITE;
	}
	//read the source node from the file
	fscanf(fp,"%d",&source);
	fscanf(fp,"%d",&edge_list_size);
	int id,cost;
	Edge* h_graph_edges = (Edge*) malloc(sizeof(Edge)*edge_list_size);
	for(int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i].x = id;
		h_graph_edges[i].y = cost;
	}
	if(fp)
		fclose(fp);    

//	printf("Read File\n");

	// allocate mem for the result on host side
	int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
	for(int i = 0; i < no_of_nodes; i++){
		h_cost[i] = INF;
	}
	h_cost[source] = 0;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

	int * temp = NULL;
	( hipMalloc( (void**) &temp, sizeof(int)*no_of_nodes) );
	( hipFree( temp) );
//	unsigned int copy_timer = 0;
//	cutilCheckError(cutCreateTimer(&copy_timer));
//	cutilCheckError(cutStartTimer(copy_timer));

	//Copy the Node list to device memory
	Node* d_graph_nodes;
	( hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) );
	( hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) );
	//Copy the Edge List to device Memory
	Edge* d_graph_edges;
	( hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) );
	( hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) );


	int* d_color;
	( hipMalloc( (void**) &d_color, sizeof(int)*no_of_nodes) );
	int* d_cost;
	( hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes));
	int * d_q1;
	int * d_q2;
	( hipMalloc( (void**) &d_q1, sizeof(int)*no_of_nodes));
	( hipMalloc( (void**) &d_q2, sizeof(int)*no_of_nodes));
	int * tail;
	( hipMalloc( (void**) &tail, sizeof(int)));
	int *front_cost_d;
	( hipMalloc( (void**) &front_cost_d, sizeof(int)));
	( hipMemcpy( d_color, color, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) );
	( hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) );
	
//	printf("Copied Everything to GPU memory\n");


	//bind the texture memory with global memory
	(hipBindTexture(0,g_graph_node_ref,d_graph_nodes, sizeof(Node)*no_of_nodes));
	(hipBindTexture(0,g_graph_edge_ref,d_graph_edges,sizeof(Edge)*edge_list_size));

	int cur_count = 0;
	
    printf("Starting GPU kernel\n");
	(hipDeviceSynchronize());
    pb_SwitchToTimer(&timers, pb_TimerID_GPU);
	BFS_GPU( d_graph_nodes,d_graph_edges,
		d_color, d_cost, d_q1, d_q2,  tail, source, cur_count);
	(hipDeviceSynchronize());
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    printf("GPU kernel done\n");


	// copy result from device to host
	( hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) );
	( hipMemcpy( color, d_color, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) );
	( hipUnbindTexture(g_graph_node_ref) );
	( hipUnbindTexture(g_graph_edge_ref) );

	(hipFree(d_graph_nodes));
	(hipFree(d_graph_edges));
	(hipFree(d_color));
	(hipFree(d_cost));
	(hipFree(tail));
	(hipFree(front_cost_d));
	//Store the result into a file
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
	FILE *fp = fopen(params->outFile,"w");
    fprintf(fp, "%d\n", no_of_nodes);
	for(int i=0;i<no_of_nodes;i++)
		fprintf(fp,"%d %d\n",i,h_cost[i]);
	fclose(fp);
	//printf("Result stored in %s\n", params->outFile);

	// cleanup memory
	free( h_graph_nodes);
	free( h_graph_edges);
	free( color);
	free( h_cost);
    pb_SwitchToTimer(&timers, pb_TimerID_NONE);
    pb_PrintTimerSet(&timers);
    pb_FreeParameters(params);
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void
helloKernel(){
  if(threadIdx.x & 1 > 0){ //odd thread
    printf("Hello world, from odd thread %d\n", threadIdx.x);
  }else{
    printf("Hello world, from even thread %d\n", threadIdx.x);
  }
}

int
main(int argc, char* argv[]){
  dim3 grid(1,1);
  dim3 block(20,1);
  helloKernel<<<grid, block>>>();
  return 0;
  //cudaDeviceReset(); //needed to flush printf cuda buffer
}



/*  \file TestExceptions.cu
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date Tuesday November 9, 2010
	\brief A CUDA assembly test for unstructured control 
		flow mimicking exceptions.
*/


#include <hip/hip_runtime.h>
#include <cstdlib>

const unsigned int threads = 512;
const unsigned int iterations = 100;

__device__ unsigned int output[threads];
__device__ unsigned int input[threads];

extern "C" __global__ void exception_in_divergent_call(unsigned int id)
{
	goto Try_Label;
	
	Try_Label:
		unsigned int result = 0;
		if(threadIdx.x == id)
		{
			goto Function_1_Begin;
		}
		else
		{
			goto Function_2_Begin;
		}

	Function_1_Begin:
		if(input[threadIdx.x] != id) goto Catch_Label;
		
		result = input[threadIdx.x];
		goto Function_Return;

	Function_2_Begin:
		if(input[threadIdx.x] == id) goto Catch_Label;
		
		result = threadIdx.x;
		goto Function_Return;

	Function_Return:
		for(unsigned int i = 0; i < iterations; ++i)
		{
			result = (result >> 1) ^ threadIdx.x;
		}
		
		output[threadIdx.x] = result;
		return;

	Catch_Label:
		output[threadIdx.x] = (unsigned int)-1; // error occurred
		return;

}

extern "C" __global__ void exception_in_loop(int iterations)
{
	goto Try_Label;
	
	Try_Label:
		unsigned int result = 0;
		goto Function_Begin;

	Function_Begin:
		for(unsigned int i = 0; i < threadIdx.x; ++i)
		{
			if(input[i] > blockDim.x) goto Catch_Label;
			result += input[i];
		}
		
		for(unsigned int i = 0; i < iterations; ++i)
		{
			result = (result >> 1) ^ threadIdx.x;
		}
		
		goto Function_Return;



	Function_Return:	
		output[threadIdx.x] = result;
		return;

	Catch_Label:
		output[threadIdx.x] = (unsigned int)-1; // error occurred
		return;
}

extern "C" __global__ void exception_in_conditional()
{
	goto Try_Label;
	
	Try_Label:
		unsigned int result = 0;
		goto Function_Begin;

	Function_Begin:
		if(input[threadIdx.x] > 0)
		{
			if(input[threadIdx.x] > blockDim.x) goto Catch_Label;
			result = input[threadIdx.x];
		}
		
		for(unsigned int i = 0; i < iterations; ++i)
		{
			result = (result >> 1) + threadIdx.x;
		}
		
		goto Function_Return;

	Function_Return:	
		output[threadIdx.x] = result;
		return;

	Catch_Label:
		output[threadIdx.x] = (unsigned int)-1; // error occurred
		return;
}

int main(int argc, char** argv)
{
	unsigned int* in;

	srand(0);

	hipGetSymbolAddress((void**)&in, HIP_SYMBOL("input"));

	unsigned int refIn[threads];

	for(unsigned int i = 0; i < threads; i++)
	{
		refIn[i] = i;
	}
	
	hipMemcpy(in, refIn, threads * sizeof(unsigned int),
		hipMemcpyHostToDevice);
	
	exception_in_divergent_call<<<1, threads>>>(0);
	
	for(unsigned int i = 0; i < threads; i++)
	{
		refIn[i] = std::rand() % threads;
	}
	
	hipMemcpy(in, refIn, threads * sizeof(unsigned int),
		hipMemcpyHostToDevice);
	
	exception_in_loop<<<1, threads>>>(iterations);
	
	for(unsigned int i = 0; i < threads; i++)
	{
		refIn[i] = std::rand() & 1;
	}
	
	hipMemcpy(in, refIn, threads * sizeof(unsigned int),
		hipMemcpyHostToDevice);
	
	exception_in_conditional<<<1, threads>>>();
}



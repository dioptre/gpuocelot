
/* Ocelot 0.4.72  issue.
   Steve Worley  Oct 27 2009
   sw@worley.com

   Ocelot fails when running kernels using dynamic shared memory, in 32 bit only.



   32 bit (ONLY!)  Ubuntu 9.04.
   CUDA 2.3
   
   Compile with:

nvcc ocbug.cu -lOcelotExecutive -lOcelotTrace -lOcelotIr -lOcelotParser -lhydrazine -lcudart

run, and you get the error output:

a.out: ocelot/executive/implementation/CooperativeThreadArray.cpp:1093: ir::PTXU32 executive::CooperativeThreadArray::operandAsU32(int, const ir::PTXOperand&): Assertion `0 == "invalid address mode of operand"' failed.

Likely it has to do with the dynamic shared memory. Static variables work fine.


*/


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(int *source)
{  
  extern __shared__ int s[];

  s[threadIdx.x]=source[threadIdx.x];
}


int main()
{
  int *src;
  int host[10000]={0};
    
  hipSetDevice(0);
  hipMalloc((void**)&src, 10000*sizeof(int));

  hipMemcpy(src, host, 10000*sizeof(int), hipMemcpyHostToDevice);
  
  kernel<<<128, 128, 15000>>>(src);
  printf("TEST PASSED\n");
  return 0;
}

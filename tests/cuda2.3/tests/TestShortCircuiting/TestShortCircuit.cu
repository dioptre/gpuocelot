
#include <hip/hip_runtime.h>
/*  \file TestShortCircuit.cu
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date Tuesday November 9, 2010
	\brief A CUDA assembly test for short-circuiting control flow.
*/

const unsigned int threads = 512;

__device__ bool out[threads];

__global__ void short_circuit()
{
	unsigned int id = threadIdx.x;
	
	bool b0 = (id >> 0) & 0x1;
	bool b1 = (id >> 1) & 0x1;
	bool b2 = (id >> 2) & 0x1;
	bool b3 = (id >> 3) & 0x1;
	bool b4 = (id >> 4) & 0x1;
	bool b5 = (id >> 5) & 0x1;
	bool b6 = (id >> 6) & 0x1;
	bool b7 = (id >> 7) & 0x1;
	bool b8 = (id >> 8) & 0x1;
	
	
	if(((b0 && (b1 || b2)) || (b3 || (b4 && b5))) && (b6 || (b7 && b8)))
	{
		out[id] = true;
	}
	else
	{
		out[id] = false;
	}
}

int main(int argc, char** argv)
{
	short_circuit<<<1, threads>>>();
}



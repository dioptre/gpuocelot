#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 *
 *
 * This sample illustrates a 3D stencil computation over a uniform grid, a
 * computation common in finite difference codes.  The kernel advances 2D
 * threadblocks along the slowest-varying dimension of the 3D data set.
 * Data is kept in registers and shared memory for each computation, thus
 * effectively streaming the input.  Data ends up being read twice, due to 
 * the halos (16x16 output region for each threadblock, 4 halo regions, each
 * 16x4).  For more details please refer to:
 *  P. Micikevicius, 3D finite difference computation on GPUs using CUDA. In 
 *  Proceedings of 2nd Workshop on General Purpose Processing on Graphics 
 *  Processing Units (Washington, D.C., March 08 - 08, 2009). GPGPU-2, 
 *  vol. 383. ACM, New York, NY, 79-84.
 *
 *  CUDA Optimization slides, Supercomputing 08 CUDA totorial
 *  http://gpgpu.org/static/sc2008/M02-04_Optimization.pdf
 *
*/

#include <stdio.h>
#include "reference.h"

#define BLOCK_DIMX	16
#define BLOCK_DIMY	16
#define RADIUS		4

__constant__ float c_coeff[RADIUS+1];

__global__ void stencil_3D_16x16_order8(float *g_output, float *g_input, const int dimx, const int dimy, const int dimz)
{
    __shared__ float s_data[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS];

    int ix  = blockIdx.x*blockDim.x + threadIdx.x;
    int iy  = blockIdx.y*blockDim.y + threadIdx.y;
    int in_idx  = iy*dimx + ix;
    int out_idx = 0;
    int stride  = dimx*dimy;

    float infront1, infront2, infront3, infront4;
    float behind1, behind2, behind3, behind4;
    float current;

    int tx = threadIdx.x + RADIUS;
    int ty = threadIdx.y + RADIUS;

    // fill the "in-front" and "behind" data
    behind3  = g_input[in_idx];	in_idx += stride;
    behind2  = g_input[in_idx];	in_idx += stride;
    behind1  = g_input[in_idx];	in_idx += stride;

    current   = g_input[in_idx];	out_idx = in_idx;	in_idx += stride;

    infront1 = g_input[in_idx];	in_idx += stride;
    infront2 = g_input[in_idx];	in_idx += stride;
    infront3 = g_input[in_idx];	in_idx += stride;
    infront4 = g_input[in_idx];	in_idx += stride;

    for(int i=RADIUS; i<dimz-RADIUS; i++)
    {
        //////////////////////////////////////////
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = g_input[in_idx];

        in_idx  += stride;
        out_idx += stride;
        __syncthreads();

        /////////////////////////////////////////
        // update the data slice in smem

        if(threadIdx.y<RADIUS)	// halo above/below
        {
            s_data[threadIdx.y][tx]					  = g_input[out_idx-RADIUS*dimx];
            s_data[threadIdx.y+BLOCK_DIMY+RADIUS][tx] = g_input[out_idx+BLOCK_DIMY*dimx];
        }

        if(threadIdx.x<RADIUS)	// halo left/right
        {
            s_data[ty][threadIdx.x]					  = g_input[out_idx-RADIUS];
            s_data[ty][threadIdx.x+BLOCK_DIMX+RADIUS] = g_input[out_idx+BLOCK_DIMX];
        }

        // update the slice in smem
        s_data[ty][tx] = current;
        __syncthreads();

        /////////////////////////////////////////
        // compute the output value
        float value  = c_coeff[0] * current;
        value += c_coeff[1]*( infront1 + behind1 + s_data[ty-1][tx]+ s_data[ty+1][tx]+ s_data[ty][tx-1]+ s_data[ty][tx+1] );
        value += c_coeff[2]*( infront2 + behind2 + s_data[ty-2][tx]+ s_data[ty+2][tx]+ s_data[ty][tx-2]+ s_data[ty][tx+2] );
        value += c_coeff[3]*( infront3 + behind3 + s_data[ty-3][tx]+ s_data[ty+3][tx]+ s_data[ty][tx-3]+ s_data[ty][tx+3] );
        value += c_coeff[4]*( infront4 + behind4 + s_data[ty-4][tx]+ s_data[ty+4][tx]+ s_data[ty][tx-4]+ s_data[ty][tx+4] );
        g_output[out_idx] = value;
    }
}


int main(int argc, char *argv[])
{
    /////////////////////////////////////////////
    // choose the GPU for execution
    //
    int device = 0;
    hipSetDevice(device);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    printf("3DFD running on: %s\n", properties.name);
    if (properties.totalGlobalMem >= 1024*1024*1024) {
	    printf("Total GPU Memory: %.4f GB\n", properties.totalGlobalMem/(1024.f*1024.f*1024.f) );
	} else {
	    printf("Total GPU Memory: %.4f MB\n", properties.totalGlobalMem/(1024.f*1024.f) );
	}

    /////////////////////////////////////////////
    // process command-line arguments,
    // set execution parameters
    //
    int pad  = 0;
    int dimx = 48+pad;
    int dimy = 48;
    int dimz = 40;
    int nreps = 1;              // number of time-steps, over which performance is averaged
    int check_correctness = 1;  // 1=check correcness, 0-don't.  Note that CPU code is very
                                //   naive and not optimized, so many steps will take a 
                                //   long time on CPU

    if( argc >= 4 )
    {
        dimx  = atoi(argv[1]);
        dimy  = atoi(argv[2]);
        dimz  = atoi(argv[3]);
    
    }
    if( argc >= 5)
        nreps = atoi(argv[4]);
    if( argc >= 6)
        check_correctness = atoi(argv[5]);

    printf("%dx%dx%d\n", dimx, dimy, dimz);


    /////////////////////////////////////////////
    // setup data
    //

    // allocate CPU and GPU memory
    float *d_input=0, *d_output=0;
    int nbytes = dimx*dimy*dimz*sizeof(float);
    hipMalloc( (void**)&d_input, nbytes);
    hipMalloc( (void**)&d_output, nbytes);
    if( 0==d_input || 0==d_output )
    {
        printf("Unable to allocate %.4f Mbytes of GPU memory\n", (float)nbytes/(1024.0f*1024.0f) );
        printf("  TEST PASSED!\n");
		exit(EXIT_SUCCESS);
//        exit(1);
    }
    printf("allocated %.1f MB on device\n", (2.f*nbytes)/(1024.f*1024.f));

    // initialize data
    float *h_data=0, *h_reference=0;
    h_data = (float*)malloc(nbytes);
    h_reference = (float*)malloc(nbytes);
    if( 0==h_data || 0==h_reference )
    {
        printf("couldn't allocate CPU memory\n");
        printf("  TEST PASSED!\n");
		exit(EXIT_SUCCESS);
//        exit(-1);
    }
    random_data( h_data, dimx,dimy,dimz, 1, 5 );
    hipMemcpy( d_input, h_data, nbytes, hipMemcpyHostToDevice );
    if( hipGetLastError() != hipSuccess )
    {
        printf("data upload to GPU failed\n");
        printf("  TEST FAILED!\n");
        exit(-2);
    }

    // setup coefficients
    float h_coeff_symmetric[RADIUS+1] = {1.f, 1.f, 1.f, 1.f, 1.f};
    hipMemcpyToSymbol(HIP_SYMBOL( c_coeff), h_coeff_symmetric, (RADIUS+1)*sizeof(float) );
    if( hipGetLastError() != hipSuccess )
    {
        printf("coefficient upload to GPU failed\n");
        printf("  TEST FAILED!\n");
        exit(-3);
    }

    // kernel launch configuration
    dim3 block(BLOCK_DIMX,BLOCK_DIMY);
    dim3 grid( dimx/block.x, dimy/block.y );
    printf("(%d,%d)x(%d,%d) grid\n", grid.x,grid.y, block.x,block.y);

    // variables for measuring performance
    float elapsed_time_ms=0.0f, throughput_mpoints=0.0f;
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop  );


    /////////////////////////////////////////////
    // kernel execution
    //
    hipEventRecord( start, 0 );
    for(int i=0; i<nreps; i++)
        stencil_3D_16x16_order8<<<grid,block>>>(d_output, d_input, dimx, dimy, dimz);
    hipEventRecord( stop, 0 );

    hipDeviceSynchronize();
    hipEventElapsedTime( &elapsed_time_ms, start, stop );
    elapsed_time_ms /= nreps;
    throughput_mpoints = (dimx*dimy*(dimz-2*RADIUS))/(elapsed_time_ms*1e3f);

    printf("-------------------------------\n");
    printf("time:       %8.2f ms\n", elapsed_time_ms );
    printf("throughput: %8.2f MPoints/s\n", throughput_mpoints );
    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()) );


    /////////////////////////////////////////////
    // check the correctness
    //
    if( check_correctness)
    {
        printf("-------------------------------\n");
        printf("comparing to CPU result...\n");
        reference_3D( h_reference, h_data, h_coeff_symmetric, dimx,dimy,dimz, RADIUS );
        hipMemcpy( h_data, d_output, nbytes, hipMemcpyDeviceToHost );
        if( within_epsilon( h_data, h_reference, dimx,dimy,dimz, RADIUS*nreps, 0.0001f ) ) {
            printf("  Result within epsilon\n");
            printf("  TEST PASSED!\n");
        } else {
            printf("  Incorrect result\n");	
            printf("  TEST FAILED!\n");
        }
    }


    /////////////////////////////////////////////
    // release the resources
    //
    hipEventDestroy( start );
    hipEventDestroy( stop );
    if( d_input )
        hipFree( d_input );
    if( d_output )
        hipFree( d_output );
    if( h_data )
        free( h_data );
    if( h_reference )
        free( h_reference );

    hipDeviceReset();

	exit(EXIT_SUCCESS);
    return 0;
}



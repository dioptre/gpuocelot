#ifdef _GLIBCXX_USE_INT128
#undef _GLIBCXX_USE_INT128
#endif 

#ifdef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_ATOMIC_BUILTINS
#endif

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <hip/hip_runtime.h>

int main(void)
{
    size_t N = 10;

    // raw pointer to device memory
    int * raw_ptr;
    hipMalloc((void **) &raw_ptr, N * sizeof(int));

    // wrap raw pointer with a device_ptr 
    thrust::device_ptr<int> dev_ptr(raw_ptr);

    // use device_ptr in thrust algorithms
    thrust::fill(dev_ptr, dev_ptr + N, (int) 0);

    // access device memory through device_ptr
    dev_ptr[0] = 1;

    // free memory
    hipFree(raw_ptr);

 	std::cout << "TEST PASSED\n";

   return 0;
}

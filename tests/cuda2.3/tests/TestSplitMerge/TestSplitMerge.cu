
#include <hip/hip_runtime.h>
/*  \file TestDivergentRecursion.cu
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date Tuesday November 9, 2010
	\brief A CUDA assembly test for short-circuiting control flow.
*/

const unsigned int threads    = 512;
const unsigned int iterations = 100;

__device__ bool out[threads];

__device__ unsigned int divergent_function(unsigned int id)
{
	unsigned int count               = id;
	unsigned int functionReturnPoint = 0;

	if(threadIdx.x & 0x1)
	{
		functionReturnPoint = 0;
		goto FunctionEntryPoint;
		FunctionReturnPoint0:
		count |= 0x1;
	}
	else if(threadIdx.x != 0)
	{
		functionReturnPoint = 1;
		goto FunctionEntryPoint;
		FunctionReturnPoint1:
		count |= 0x2;
	}
	else
	{
		count = 0;
	}

	return count;

	FunctionEntryPoint:
		for(unsigned int i = 0; i < iterations; ++i) count = (count >> 1) + i;		
		if(functionReturnPoint == 0)
			goto FunctionReturnPoint0;
		else
			goto FunctionReturnPoint1;
}

__global__ void divergent_recusion()
{
	out[threadIdx.x] = divergent_function(threadIdx.x);
}

int main(int argc, char** argv)
{
	divergent_recusion<<<1, threads>>>();
}


